#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include "merge.h"
#include "zone_map.h"


struct MurmurHash64D
{

    const void* key;
    unsigned long long* output;
    const int* len;
    const unsigned int* seed;
    const unsigned int* off;
    const unsigned int* off_count;


    MurmurHash64D(const void* _key, unsigned long long* _output, const int* _len, const unsigned int* _seed,
                  const unsigned int* _off, const unsigned int* _off_count):
        key(_key), output(_output), len(_len), seed(_seed), off(_off), off_count(_off_count) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {


        const uint64_t m = 0xc6a4a7935bd1e995;
        const int r = 47;
        uint64_t h = *seed ^ (*len * m);

        const uint64_t* data = (const uint64_t *)((char*)key + i*(*len));
        const uint64_t* end = data + (*len/8);

        while(data != end)
        {
            uint64_t k = *data++;

            k *= m;
            k ^= k >> r;
            k *= m;

            h ^= k;
            h *= m;
        }

        const unsigned char * data2 = (const unsigned char*)data;

        switch(*len & 7)
        {
        case 7:
            h ^= uint64_t(data2[6]) << 48;
        case 6:
            h ^= uint64_t(data2[5]) << 40;
        case 5:
            h ^= uint64_t(data2[4]) << 32;
        case 4:
            h ^= uint64_t(data2[3]) << 24;
        case 3:
            h ^= uint64_t(data2[2]) << 16;
        case 2:
            h ^= uint64_t(data2[1]) << 8;
        case 1:
            h ^= uint64_t(data2[0]);
            h *= m;
        };

        h ^= h >> r;
        h *= m;
        h ^= h >> r;

        //printf("WRITE TO OFFSET %d %d %lld \n", i ,  i*(*off) + (*off_count), h);
        output[i*(*off) + (*off_count)] = h;
    }
};

struct MurmurHash64D_F
{

    const void* key;
    unsigned long long* output;
    const int* len;
    const unsigned int* seed;


    MurmurHash64D_F(const void* _key, unsigned long long* _output, const int* _len, const unsigned int* _seed):
        key(_key), output(_output), len(_len), seed(_seed) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {


        const uint64_t m = 0xc6a4a7935bd1e995;
        const int r = 47;
        uint64_t h = *seed ^ (*len * m);

        const uint64_t* data = (const uint64_t *)((char*)key + i*(*len));
        const uint64_t* end = data + (*len/8);

        while(data != end)
        {
            uint64_t k = *data++;

            k *= m;
            k ^= k >> r;
            k *= m;

            h ^= k;
            h *= m;
        }

        const unsigned char * data2 = (const unsigned char*)data;

        switch(*len & 7)
        {
        case 7:
            h ^= uint64_t(data2[6]) << 48;
        case 6:
            h ^= uint64_t(data2[5]) << 40;
        case 5:
            h ^= uint64_t(data2[4]) << 32;
        case 4:
            h ^= uint64_t(data2[3]) << 24;
        case 3:
            h ^= uint64_t(data2[2]) << 16;
        case 2:
            h ^= uint64_t(data2[1]) << 8;
        case 1:
            h ^= uint64_t(data2[0]);
            h *= m;
        };

        h ^= h >> r;
        h *= m;
        h ^= h >> r;

        output[i] = h;
    }
};

struct float_avg
{
    __host__  float_type operator()(const float_type &lhs, const int_type &rhs) const {
        return lhs/rhs;
    }
};

struct float_avg1
{
    __host__  float_type operator()(const int_type &lhs, const int_type &rhs) const {
        return ((float_type)lhs)/rhs;
    }
};


/*struct float_avg  : public binary_function<float_type,int_type,float_type>
{
  __host__ __device__ float_type operator()(const float_type &lhs, const int_type &rhs) const {return lhs/(float_type)rhs;}
}; // end not_equal_to
*/





typedef thrust::device_vector<int_type>::iterator    IntIterator;
typedef thrust::tuple<IntIterator,IntIterator> IteratorTuple;
typedef thrust::zip_iterator<IteratorTuple> ZipIterator;
unsigned int hash_seed = 100;
thrust::host_vector<unsigned long long int> h_merge;

using namespace std;
using namespace thrust::placeholders;


void create_c(CudaSet* c, CudaSet* b)
{
    map<string,unsigned int>::iterator it;
    c->not_compressed = 1;
    c->segCount = 1;

    for (  it=b->columnNames.begin() ; it != b->columnNames.end(); ++it ) {
        c->columnNames[(*it).first] = (*it).second;
    };
	

    c->grp_type = new unsigned int[c->mColumnCount];
	h_merge.clear();

    for(unsigned int i=0; i < b->mColumnCount; i++) {
	
        c->cols[i] = b->cols[i];
        c->type[i] = b->type[i];
        c->grp_type[i] = b->grp_type[i];

        if (b->type[i] == 0) {
            c->h_columns_int.push_back(thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >());
            c->d_columns_int.push_back(thrust::device_vector<int_type>());
            c->type_index[i] = c->h_columns_int.size()-1;
        }
        else if (b->type[i] == 1) {
            c->h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >());
            c->d_columns_float.push_back(thrust::device_vector<float_type>());
            c->type_index[i] = c->h_columns_float.size()-1;
        }
        else {
            c->h_columns_char.push_back(NULL);
            c->d_columns_char.push_back(NULL);
            c->char_size.push_back(b->char_size[b->type_index[i]]);
            c->type_index[i] = c->h_columns_char.size()-1;
        };
    };
}

void add(CudaSet* c, CudaSet* b, queue<string> op_v3, map<string,string> aliases,
         vector<thrust::device_vector<int_type> >& distinct_tmp, vector<thrust::device_vector<int_type> >& distinct_val,
         vector<thrust::device_vector<int_type> >& distinct_hash, CudaSet* a)
{		
    if (c->columnNames.empty()) {
        // create d_columns and h_columns
        create_c(c,b);
    }

    size_t cycle_sz = op_v3.size();	
	
    vector<unsigned int> opv;
    queue<string> ss;
    for(unsigned int z = 0; z < cycle_sz; z++) {
        opv.push_back(b->columnNames[aliases[op_v3.front()]]);
        ss.push(aliases[op_v3.front()]);
        op_v3.pop();
    };

    // create hashes of groupby columns
    thrust::device_vector<unsigned long long int> hashes(b->mRecCount);

    unsigned int idx;
    thrust::device_vector<unsigned long long int> sum(cycle_sz*b->mRecCount);
    thrust::device_vector<unsigned int> seed(1);
    seed[0] = hash_seed;
    thrust::device_vector<int> len(1);
    thrust::device_vector<unsigned int> off(1);
    thrust::device_vector<unsigned int> off_count(1);

    thrust::counting_iterator<unsigned int> begin(0);
    for(unsigned int z = 0; z < cycle_sz; z++) {
        idx = opv[z];

        if(b->type[idx] == 0) {  //int
            len[0] = 8;
            off[0] = cycle_sz;
            off_count[0] = z;
            MurmurHash64D ff((void*)(thrust::raw_pointer_cast(b->d_columns_int[b->type_index[idx]].data())),
                             thrust::raw_pointer_cast(sum.data()),
                             thrust::raw_pointer_cast(len.data()), thrust::raw_pointer_cast(seed.data()),
                             thrust::raw_pointer_cast(off.data()), thrust::raw_pointer_cast(off_count.data()));
            thrust::for_each(begin, begin + b->mRecCount, ff);
        }
        else if(b->type[idx] == 2) {  //string
            len[0] = b->char_size[b->type_index[idx]];
            off[0] = cycle_sz;
            off_count[0] = z;
            MurmurHash64D ff((void*)b->d_columns_char[b->type_index[idx]],
                             thrust::raw_pointer_cast(sum.data()),
                             thrust::raw_pointer_cast(len.data()), thrust::raw_pointer_cast(seed.data()),
                             thrust::raw_pointer_cast(off.data()), thrust::raw_pointer_cast(off_count.data()));
            thrust::for_each(begin, begin + b->mRecCount, ff);
        }
        else {  //float
            cout << "No group by on float/decimal columns " << endl;
            exit(0);
        };
    };

    //for(int i = 0; i < cycle_sz*b->mRecCount;i++)
    //cout << "SUM " << sum[0] << endl;
	

    len[0] = 8*cycle_sz;
    MurmurHash64D_F ff(thrust::raw_pointer_cast(sum.data()),
                       thrust::raw_pointer_cast(hashes.data()),
                       thrust::raw_pointer_cast(len.data()), thrust::raw_pointer_cast(seed.data()));
    thrust::for_each(begin, begin + b->mRecCount, ff);

    //for(int i = 0; i < b->mRecCount;i++)
    //cout << "DEV HASH " << hashes[0] << endl;

    // sort the results by hash
    thrust::device_ptr<unsigned int> v = thrust::device_malloc<unsigned int>(b->mRecCount);
    thrust::sequence(v, v + b->mRecCount, 0, 1);

    size_t max_c	= max_char(b);
    if(max_c < 8) {
        max_c = 8;
    };
    void* d;
    CUDA_SAFE_CALL(hipMalloc((void **) &d, b->mRecCount*max_c));
    thrust::sort_by_key(hashes.begin(), hashes.end(), v);
	

    for(unsigned int i = 0; i < b->mColumnCount; i++) {

        if(b->type[i] == 0) {
            thrust::device_ptr<int_type> d_tmp((int_type*)d);
            thrust::gather(v, v+b->mRecCount, b->d_columns_int[b->type_index[i]].begin(), d_tmp);
            thrust::copy(d_tmp, d_tmp + b->mRecCount, b->d_columns_int[b->type_index[i]].begin());
        }
        else if(b->type[i] == 1) {
            thrust::device_ptr<float_type> d_tmp((float_type*)d);
            thrust::gather(v, v+b->mRecCount, b->d_columns_float[b->type_index[i]].begin(), d_tmp);
            thrust::copy(d_tmp, d_tmp + b->mRecCount, b->d_columns_float[b->type_index[i]].begin());
        }
        else {
            thrust::device_ptr<char> d_tmp((char*)d);
            str_gather(thrust::raw_pointer_cast(v), b->mRecCount, (void*)b->d_columns_char[b->type_index[i]], (void*) thrust::raw_pointer_cast(d_tmp), b->char_size[b->type_index[i]]);
            hipMemcpy( (void*)b->d_columns_char[b->type_index[i]], (void*) thrust::raw_pointer_cast(d_tmp), b->mRecCount*b->char_size[b->type_index[i]], hipMemcpyDeviceToDevice);
        };
    };
    hipFree(d);
    thrust::device_free(v);


    b->CopyToHost(0, b->mRecCount);
    thrust::host_vector<unsigned long long int> hh = hashes;
    char* tmp = new char[max_c*(c->mRecCount + b->mRecCount)];
    c->resize(b->mRecCount);
    //lets merge every column
	
    for(unsigned int i = 0; i < b->mColumnCount; i++) {
	

        if(b->type[i] == 0) {
            thrust::merge_by_key(h_merge.begin(), h_merge.end(),
                                 hh.begin(), hh.end(),
                                 c->h_columns_int[c->type_index[i]].begin(), b->h_columns_int[b->type_index[i]].begin(),
                                 thrust::make_discard_iterator(), (int_type*)tmp);
            thrust::copy((int_type*)tmp, (int_type*)tmp + h_merge.size() + b->mRecCount, c->h_columns_int[c->type_index[i]].begin());
        }
        else if(b->type[i] == 1) {
            thrust::merge_by_key(h_merge.begin(), h_merge.end(),
                                 hh.begin(), hh.end(),
                                 c->h_columns_float[c->type_index[i]].begin(), b->h_columns_float[b->type_index[i]].begin(),
                                 thrust::make_discard_iterator(), (float_type*)tmp);										 
            thrust::copy((float_type*)tmp, (float_type*)tmp + h_merge.size() + b->mRecCount, c->h_columns_float[c->type_index[i]].begin());			
			
        }
        else {
            str_merge_by_key(h_merge, hh, c->h_columns_char[c->type_index[i]], b->h_columns_char[b->type_index[i]], b->char_size[b->type_index[i]], tmp);
            thrust::copy(tmp, tmp + (h_merge.size() + b->mRecCount)*b->char_size[b->type_index[i]],	c->h_columns_char[c->type_index[i]]);
        };
    };
	

    //merge the keys
    thrust::merge(h_merge.begin(), h_merge.end(),
                  hh.begin(), hh.end(), (unsigned long long int*)tmp);

    size_t cpy_sz = h_merge.size() + b->mRecCount;
    h_merge.resize(h_merge.size() + b->mRecCount);
    thrust::copy((unsigned long long int*)tmp, (unsigned long long int*)tmp + cpy_sz, h_merge.begin());
    delete [] tmp;
	

    //cout << endl << "end b and c " << b->mRecCount << " " << c->mRecCount << endl;
    //for(int i = 0; i < h_merge.size();i++)
    //cout << "H " << h_merge[i] << endl;

    /*   bool dis_exists = 0;
        for(unsigned int j=0; j < c->mColumnCount; j++) {
            if (c->grp_type[j] == 6)
                dis_exists = 1;
        };

        if (dis_exists) {
            bool grp_scanned = 0;
            thrust::device_ptr<bool> d_di(a->grp);
            thrust::device_ptr<unsigned int> d_dii = thrust::device_malloc<unsigned int>(a->mRecCount);
            thrust::identity<bool> op;
            thrust::transform(d_di, d_di+a->mRecCount, d_dii, op);

            thrust::device_ptr<int_type> tmp = thrust::device_malloc<int_type>(a->mRecCount);

            unsigned int dist_count = 0;

            for(unsigned int j=0; j < c->mColumnCount; j++) {

                if (c->grp_type[j] == 6) {

                    if(!grp_scanned) {

                        d_dii[a->mRecCount-1] = 0;
                        thrust::inclusive_scan(d_dii, d_dii + a->mRecCount, d_dii);
                        thrust::gather(d_dii, d_dii + a->mRecCount, hashes.begin(), tmp);	// now hashes are in tmp
                        grp_scanned = 1;
                    };
                    unsigned int offset = distinct_val[dist_count].size();

                    distinct_val[dist_count].resize(distinct_val[dist_count].size() + a->mRecCount);
                    distinct_hash[dist_count].resize(distinct_hash[dist_count].size() + a->mRecCount);

                    thrust::copy(distinct_tmp[dist_count].begin(), distinct_tmp[dist_count].begin() + a->mRecCount, distinct_val[dist_count].begin() + offset);
                    thrust::copy(tmp, tmp + a->mRecCount, distinct_hash[dist_count].begin() + offset);

                    thrust::stable_sort_by_key(distinct_val[dist_count].begin(), distinct_val[dist_count].end(), distinct_hash[dist_count].begin());
                    thrust::stable_sort_by_key(distinct_hash[dist_count].begin(), distinct_hash[dist_count].end(), distinct_val[dist_count].begin());

                    ZipIterator new_last = thrust::unique(thrust::make_zip_iterator(thrust::make_tuple(distinct_hash[dist_count].begin(), distinct_val[dist_count].begin())),
                                                          thrust::make_zip_iterator(thrust::make_tuple(distinct_hash[dist_count].end(), distinct_val[dist_count].end())));

                    IteratorTuple t = new_last.get_iterator_tuple();
                    distinct_val[dist_count].resize(thrust::get<0>(t) - distinct_hash[dist_count].begin());
                    distinct_hash[dist_count].resize(thrust::get<0>(t) - distinct_hash[dist_count].begin());

                    dist_count++;

                };
            };
            thrust::device_free(tmp);
            thrust::device_free(d_dii);
        };
    	*/


}


void count_simple(CudaSet* c)
{
    int_type count;

    for(unsigned int i = 0; i < c->mColumnCount; i++) {
        if(c->grp_type[i] == 0) { // COUNT
            count = thrust::reduce(c->h_columns_int[c->type_index[i]].begin(), c->h_columns_int[c->type_index[i]].begin() + c->mRecCount);
            c->h_columns_int[c->type_index[i]][0] = count;
        };
    };


    if (c->mRecCount != 0) {

        for(unsigned int k = 0; k < c->mColumnCount; k++)	{
            if(c->grp_type[k] == 1) {   // AVG
                if(c->type[k] == 0) {
                    int_type sum  = thrust::reduce(c->h_columns_int[c->type_index[k]].begin(), c->h_columns_int[c->type_index[k]].begin() + c->mRecCount);
                    c->h_columns_int[c->type_index[k]][0] = sum/count;
                }
                if(c->type[k] == 1) {
                    float_type sum  = thrust::reduce(c->h_columns_float[c->type_index[k]].begin(), c->h_columns_float[c->type_index[k]].begin() + c->mRecCount);
                    c->h_columns_float[c->type_index[k]][0] = sum/count;
                };
            }
            else if(c->grp_type[k] == 2) {   // SUM
                if(c->type[k] == 0) {
                    int_type sum  = thrust::reduce(c->h_columns_int[c->type_index[k]].begin(), c->h_columns_int[c->type_index[k]].begin() + c->mRecCount);
                    c->h_columns_int[c->type_index[k]][0] = sum;
                }
                if(c->type[k] == 1) {
                    float_type sum  = thrust::reduce(c->h_columns_float[c->type_index[k]].begin(), c->h_columns_float[c->type_index[k]].begin() + c->mRecCount);
                    c->h_columns_float[c->type_index[k]][0] = sum;
                };

            }
        };
    }
    c->mRecCount = 1;
};


void count_avg(CudaSet* c,  vector<thrust::device_vector<int_type> >& distinct_hash)
{
    int countIndex;

    for(unsigned int i = 0; i < c->mColumnCount; i++) {
        if(c->grp_type[i] == 0) { // COUNT
            countIndex = i;
            break;
        };
    };

    thrust::host_vector<bool> grp;
    size_t res_count;

    if(h_merge.size()) {
        grp.resize(h_merge.size());
        thrust::adjacent_difference(h_merge.begin(), h_merge.end(), grp.begin());
        res_count = h_merge.size() - thrust::count(grp.begin(), grp.end(), 0);
    };


    if (c->mRecCount != 0) {

        //unsigned int dis_count = 0;
        if (h_merge.size()) {
            for(unsigned int k = 0; k < c->mColumnCount; k++)	{

                if(c->grp_type[k] <= 2) { //sum || avg || count
                    if (c->type[k] == 0 ) { // int

                        int_type* tmp =  new int_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->type_index[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_int[c->type_index[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->type_index[k]].begin());
                        delete [] tmp;
                    }
                    else if (c->type[k] == 1 ) { // float
                        float_type* tmp =  new float_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->type_index[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_float[c->type_index[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_float[c->type_index[k]].begin());
                        delete [] tmp;
                    };
                }
                if(c->grp_type[k] == 4) { //min
                    if (c->type[k] == 0 ) { // int
                        int_type* tmp =  new int_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->type_index[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_int[c->type_index[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->type_index[k]].begin());
                        delete [] tmp;
                    }
                    else if (c->type[k] == 1 ) { // float
                        float_type* tmp =  new float_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->type_index[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_float[c->type_index[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_float[c->type_index[k]].begin());
                        delete [] tmp;
                    };
                }
                if(c->grp_type[k] == 5) { //max
                    if (c->type[k] == 0 ) { // int
                        int_type* tmp =  new int_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->type_index[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_int[c->type_index[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->type_index[k]].begin());
                        delete [] tmp;
                    }
                    else if (c->type[k] == 1 ) { // float
                        float_type* tmp =  new float_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->type_index[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_float[c->type_index[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_float[c->type_index[k]].begin());
                        delete [] tmp;
                    };
                }
                else if(c->grp_type[k] == 3) { //no group function
                    if (c->type[k] == 0 ) { // int
                        int_type* tmp =  new int_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_int[c->type_index[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_int[c->type_index[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_int[c->type_index[k]].begin());
                        delete [] tmp;
                    }
                    else if (c->type[k] == 1 ) { // float
                        float_type* tmp =  new float_type[res_count];
                        thrust::reduce_by_key(h_merge.begin(), h_merge.end(), c->h_columns_float[c->type_index[k]].begin(),
                                              thrust::make_discard_iterator(), tmp);
                        c->h_columns_float[c->type_index[k]].resize(res_count);
                        thrust::copy(tmp, tmp + res_count, c->h_columns_float[c->type_index[k]].begin());
                        delete [] tmp;
                    }
                    else { //char
                        char* tmp = new char[res_count*c->char_size[c->type_index[k]]];
                        str_copy_if_host(c->h_columns_char[c->type_index[k]], c->mRecCount, tmp, grp, c->char_size[c->type_index[k]]);
                        thrust::copy(tmp, tmp + c->char_size[c->type_index[k]]*res_count, c->h_columns_char[c->type_index[k]]);
                        delete [] tmp;
                    };
                };
            };
            c->mRecCount = res_count;
        };

        for(unsigned int k = 0; k < c->mColumnCount; k++)	{
            if(c->grp_type[k] == 1) {   // AVG

                if (c->type[k] == 0 ) { // int
                    //create a float column k
                    c->h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >(c->mRecCount));
                    size_t idx = c->h_columns_float.size()-1;

                    thrust::transform(c->h_columns_int[c->type_index[k]].begin(), c->h_columns_int[c->type_index[k]].begin() + c->mRecCount,
                                      c->h_columns_int[c->type_index[countIndex]].begin(), c->h_columns_float[idx].begin(), float_avg1());
                    c->type[k] = 1;
                    c->h_columns_int[c->type_index[k]].resize(0);
                    c->h_columns_int[c->type_index[k]].shrink_to_fit();
                    c->type_index[k] = idx;
                    c->grp_type[k] = 3;
                }
                else {              // float
                    thrust::transform(c->h_columns_float[c->type_index[k]].begin(), c->h_columns_float[c->type_index[k]].begin() + c->mRecCount,
                                      c->h_columns_int[c->type_index[countIndex]].begin(), c->h_columns_float[c->type_index[k]].begin(), float_avg());
                };
            }
            else if(c->grp_type[k] == 6) {
                /*   unsigned int res_count = 0;

                   thrust::host_vector<int_type> h_hash = distinct_hash[dis_count];
                   int_type curr_val = h_hash[0];
                   unsigned int cycle_sz = h_hash.size();

                   for(unsigned int i = 0; i < cycle_sz; i++) {
                       if (h_hash[i] == curr_val) {
                           res_count++;
                           if(i == cycle_sz-1) {
                               c->h_columns_int[c->type_index[k]][mymap[h_hash[i]]] = res_count;
                           };
                       }
                       else {
                           unsigned int idx = mymap[h_hash[i-1]];
                           c->h_columns_int[c->type_index[k]][idx] = res_count;
                           curr_val = h_hash[i];
                           res_count = 1;
                       };
                   };
                   dis_count++;*/
            }
            else if(c->grp_type[k] == 2) {

            };
        };

    };

    c->segCount = 1;
    c->maxRecs = c->mRecCount;
};

