#include "hip/hip_runtime.h"
/*
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */


// PFOR and PFOR-DELTA Compression and decompression routines

#include <stdio.h>
#include <fstream>
#include <iomanip>
#include <exception>
#include <thrust/device_vector.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/extrema.h>
#include "cm.h"

using namespace std;

unsigned long long int* raw_decomp = NULL;
unsigned int raw_decomp_length = 0;

std::map<string, unsigned int> cnt_counts;
string curr_file;


struct bool_to_int
{
    __host__ __device__
    unsigned int operator()(const bool x)
    {
        return (unsigned int)x;
    }
};

struct ui_to_ll
{
    __host__ __device__
    long long int operator()(const unsigned int x)
    {
        return (long long int)x;
    }
};


struct compress_functor_int
{

    const int_type * source;
    unsigned long long int * dest;
    const long long int * start_val;
    const unsigned int * vals;


    compress_functor_int(const int_type * _source, unsigned long long int  * _dest,
                         const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
        long long int val = source[i] - start_val[0];
        unsigned int shifted = vals[2] - vals[0] - (i%vals[1])*vals[0];
        dest[i] = val << shifted;
    }
};

struct compress_functor_float
{

    const long long int * source;
    unsigned long long int * dest;
    const long long int * start_val;
    const unsigned int * vals;


    compress_functor_float(const long long int * _source, unsigned long long int  * _dest,
                           const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {
        long long int val;

        unsigned int bits = vals[0];
        unsigned int fit_count = vals[1];
        unsigned int int_sz = vals[2];

        val = source[i] - start_val[0];
        unsigned int z = i%fit_count;

        unsigned int shifted = int_sz - bits - z*bits;
        dest[i] = val << shifted;
    }
};



struct decompress_functor_int
{

    const unsigned long long int * source;
    int_type * dest;
    const long long int * start_val;
    const unsigned int * vals;


    decompress_functor_int(const unsigned long long int * _source, int_type * _dest,
                           const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned long long int tmp = source[i/vals[1]]  >> (vals[2] - vals[0] - (i%vals[1])*vals[0]);
        // set  the rest of bits to 0
        tmp	= tmp << (vals[2] - vals[0]);
        tmp	= tmp >> (vals[2] - vals[0]);
        dest[i] = tmp + start_val[0];

    }
};


struct decompress_functor_float
{

    const unsigned long long int * source;
    long long int * dest;
    const long long int * start_val;
    const unsigned int * vals;


    decompress_functor_float(const unsigned long long int * _source, long long int * _dest,
                             const long long int * _start_val, const unsigned int * _vals):
        source(_source), dest(_dest), start_val(_start_val), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned long long int tmp = source[i/vals[1]]  >> (vals[2] - vals[0] - (i%vals[1])*vals[0]);
        // set  the rest of bits to 0
        tmp	= tmp << (vals[2] - vals[0]);
        tmp	= tmp >> (vals[2] - vals[0]);

        dest[i] = tmp + start_val[0];

    }
};


struct decompress_functor_str
{

    const unsigned long long  * source;
    unsigned int * dest;
    const unsigned int * vals;


    decompress_functor_str(const unsigned long long int * _source, unsigned int * _dest,
                           const unsigned int * _vals):
        source(_source), dest(_dest), vals(_vals) {}

    template <typename IndexType>
    __host__ __device__
    void operator()(const IndexType & i) {

        unsigned int bits = vals[0];
        unsigned int fit_count = vals[1];
        unsigned int int_sz = 64;

        //find the source index
        unsigned int src_idx = i/fit_count;
        // find the exact location
        unsigned int src_loc = i%fit_count;
        //right shift the values
        unsigned int shifted = ((fit_count-src_loc)-1)*bits;
        unsigned long long int tmp = source[src_idx]  >> shifted;
        // set  the rest of bits to 0
        tmp	= tmp << (int_sz - bits);
        tmp	= tmp >> (int_sz - bits);


        dest[i] = tmp;

    }
};




unsigned int pfor_decompress(void* destination, void* host, void* d_v, void* s_v)
{

    unsigned int bits, cnt, fit_count, orig_recCount;
    long long int  orig_lower_val;
    unsigned int bit_count = 64;
    unsigned int comp_type;
    long long int start_val;

    cnt = ((unsigned int*)host)[0];
    orig_recCount = ((unsigned int*)host + cnt*2)[7];
    bits = ((unsigned int*)host + cnt*2)[8];
    orig_lower_val = ((long long int*)((unsigned int*)host + cnt*2 + 9))[0];
    fit_count = ((unsigned int*)host + cnt*2)[11];
    start_val = ((long long int*)((unsigned int*)host + cnt*2 + 12))[0];
    comp_type = ((unsigned int*)host + cnt*2)[14];

    //*mRecCount = orig_recCount;

    //cout << "Decomp Header " <<  orig_recCount << " " << bits << " " << orig_lower_val << " " << cnt << " " << fit_count << " " << comp_type << endl;


    if(raw_decomp_length < cnt*8) {
        if(raw_decomp != NULL) {
            hipFree(raw_decomp);
        };
        hipMalloc((void **) &raw_decomp, cnt*8);
        raw_decomp_length = cnt*8;
    };

    hipMemcpy( (void*)raw_decomp, (void*)((unsigned int*)host + 5), cnt*8, hipMemcpyHostToDevice);
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
    thrust::device_ptr<long long int> dd_sv((long long int*)s_v);

    dd_sv[0] = orig_lower_val;
    dd_v[0] = bits;
    dd_v[1] = fit_count;
    dd_v[2] = bit_count;

    thrust::counting_iterator<unsigned int> begin(0);
    decompress_functor_int ff1(raw_decomp,(int_type*)destination, (long long int*)s_v, (unsigned int*)d_v);
    thrust::for_each(begin, begin + orig_recCount, ff1);
    if(comp_type == 1) {
        thrust::device_ptr<int_type> d_int((int_type*)destination);
        d_int[0] = start_val;
        thrust::inclusive_scan(d_int, d_int + orig_recCount, d_int);
    };
    return orig_recCount;
}


template< typename T>
unsigned long long int pfor_delta_compress(void* source, size_t source_len, string file_name, thrust::host_vector<T, pinned_allocator<T> >& host, bool tp, unsigned long long int sz)
{
    long long int orig_lower_val, orig_upper_val, start_val, real_lower, real_upper;
    unsigned int  bits, recCount;
    unsigned int bit_count = 8*8;
    unsigned int fit_count;
    unsigned int comp_type = 1; // FOR-DELTA

    if(tp == 0)
        recCount = source_len/int_size;
    else
        recCount = source_len/float_size;

    void* ss;
    CUDA_SAFE_CALL(hipMalloc((void **) &ss, recCount*float_size));


    if (tp == 0) {
        thrust::device_ptr<int_type> s((int_type*)source);
        thrust::device_ptr<int_type> d_ss((int_type*)ss);
        thrust::adjacent_difference(s, s+recCount, d_ss);

        start_val = d_ss[0];
        if(recCount > 1)
            d_ss[0] = d_ss[1];

        orig_lower_val = *(thrust::min_element(d_ss, d_ss + recCount));
        orig_upper_val = *(thrust::max_element(d_ss, d_ss + recCount));

        real_lower = s[0];
        real_upper = s[recCount-1];
        //cout << "orig " << orig_upper_val << " " <<  orig_lower_val << endl;
        //cout << "We need for delta " << (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1))) << " bits to encode " <<  orig_upper_val-orig_lower_val << " values " << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1)));
        if (bits == 0)
            bits = 1;

    }
    else {
        thrust::device_ptr<long long int> s((long long int*)source);
        thrust::device_ptr<long long int> d_ss((long long int*)ss);
        thrust::adjacent_difference(s, s+recCount, d_ss);
        start_val = d_ss[0];
        if(recCount > 1)
            d_ss[0] = d_ss[1];

        orig_lower_val = *(thrust::min_element(d_ss, d_ss + recCount));
        orig_upper_val = *(thrust::max_element(d_ss, d_ss + recCount));
        real_lower = s[0];
        real_upper = s[recCount-1];

        //cout << "orig " << orig_upper_val << " " <<  orig_lower_val << endl;
        //cout << "We need for delta " << (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1))) << " bits to encode " << orig_upper_val-orig_lower_val << " values" << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val-orig_lower_val)+1)));
        if (bits == 0)
            bits = 1;
    };

    thrust::counting_iterator<unsigned int> begin(0);

    fit_count = bit_count/bits;
    void* d_v1;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_v1, 12));
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v1);

    void* s_v1;
    CUDA_SAFE_CALL(hipMalloc((void **) &s_v1, 8));
    thrust::device_ptr<long long int> dd_sv((long long int*)s_v1);

    dd_sv[0] = orig_lower_val;
    dd_v[0] = bits;
    dd_v[1] = fit_count;
    dd_v[2] = bit_count;


    //void* d;
    //CUDA_SAFE_CALL(hipMalloc((void **) &d, recCount*float_size));

    thrust::device_ptr<char> dd((char*)source);
    thrust::fill(dd, dd+source_len,0);

    //cout << "FF " << orig_lower_val << " " << bits << " " << fit_count << " " << bit_count << endl;

    if (tp == 0) {
        compress_functor_int ff((int_type*)ss,(unsigned long long int*)source, (long long int*)s_v1, (unsigned int*)d_v1);
        thrust::for_each(begin, begin + recCount, ff);
    }
    else {
        compress_functor_float ff((long long int*)ss,(unsigned long long int*)source, (long long int*)s_v1, (unsigned int*)d_v1);
        thrust::for_each(begin, begin + recCount, ff);
    };


    thrust::device_ptr<unsigned long long int> s_copy1((unsigned long long int*)source);

    // make an addition  sequence

    thrust::device_ptr<unsigned long long int> add_seq((unsigned long long int*)ss);
    thrust::constant_iterator<unsigned long long int> iter(fit_count);
    thrust::sequence(add_seq, add_seq + recCount, 0, 1);
    thrust::transform(add_seq, add_seq + recCount, iter, add_seq, thrust::divides<unsigned long long int>());

    unsigned int cnt = (recCount)/fit_count;
    if (recCount%fit_count > 0)
        cnt++;

    thrust::device_ptr<unsigned long long int> fin_seq = thrust::device_malloc<unsigned long long int>(cnt);

    thrust::reduce_by_key(add_seq, add_seq+recCount,s_copy1,thrust::make_discard_iterator(),
                          fin_seq);

    //for(int i = 0; i < 10;i++)
    //  cout << "FIN " << fin_seq[i] << endl;

    // copy fin_seq to host
    unsigned long long int * raw_src = thrust::raw_pointer_cast(fin_seq);

    if(!file_name.empty()) {
        hipMemcpy( host.data(), (void *)raw_src, cnt*8, hipMemcpyDeviceToHost);
        fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::app);
        binary_file.write((char *)&cnt, 4);
        binary_file.write((char *)&real_lower, 8);
        binary_file.write((char *)&real_upper, 8);
        binary_file.write((char *)host.data(),cnt*8);
        binary_file.write((char *)&comp_type, 4);
        binary_file.write((char *)&cnt, 4);
        binary_file.write((char *)&recCount, 4);
        binary_file.write((char *)&bits, 4);
        binary_file.write((char *)&orig_lower_val, 8);
        binary_file.write((char *)&fit_count, 4);
        binary_file.write((char *)&start_val, 8);
        binary_file.write((char *)&comp_type, 4);
        binary_file.write((char *)&comp_type, 4); //filler
        binary_file.close();
        if(cnt_counts[curr_file] < cnt)
            cnt_counts[curr_file] = cnt;
    }
    else {
        char* hh;
        //resize_compressed(host, sz, cnt*8 + 15*4, 0);
        host.resize(sz+cnt+8);
        hh = (char*)(host.data() + sz);
        ((unsigned int*)hh)[0] = cnt;
        ((long long int*)(hh+4))[0] = real_lower;
        ((long long int*)(hh+12))[0] = real_upper;
        hipMemcpy( hh + 20, (void *)raw_src, cnt*8, hipMemcpyDeviceToHost);
        ((unsigned int*)hh)[5+cnt*2] = comp_type;
        ((unsigned int*)hh)[6+cnt*2] = cnt;
        ((unsigned int*)hh)[7+cnt*2] = recCount;
        ((unsigned int*)hh)[8+cnt*2] = bits;
        ((long long int*)((char*)hh+36+cnt*8))[0] = orig_lower_val;
        ((unsigned int*)hh)[11+cnt*2] = fit_count;
        ((long long int*)((char*)hh+48+cnt*8))[0] = start_val;
        ((unsigned int*)hh)[14+cnt*2] = comp_type;
    };

    thrust::device_free(fin_seq);
    hipFree(ss);
    hipFree(d_v1);
    hipFree(s_v1);
    return sz + cnt + 8;
}


template< typename T>
unsigned long long int pfor_compress(void* source, size_t source_len, string file_name, thrust::host_vector<T, pinned_allocator<T> >& host,  bool tp, unsigned long long int sz)
{
    unsigned int recCount;
    long long int orig_lower_val;
    long long int orig_upper_val;
    unsigned int  bits;
    unsigned int bit_count = 8*8;
    unsigned int fit_count;
    unsigned int comp_type = 0; // FOR
    long long int start_val = 0;
    bool sorted = 0;

    // check if sorted

    if (tp == 0) {
        recCount = source_len/int_size;
        thrust::device_ptr<int_type> s((int_type*)source);
        sorted = thrust::is_sorted(s, s+recCount-1);
    }
    else {
        recCount = source_len/float_size;
        thrust::device_ptr<long long int> s((long long int*)source);
        sorted = thrust::is_sorted(s, s+recCount);
    };
    //cout << "file " << file_name << " is sorted " << sorted << endl;

    if(sorted)
        return pfor_delta_compress(source, source_len, file_name, host, tp, sz);

// sort the sequence

    if (tp == 0) {
        thrust::device_ptr<int_type> s((int_type*)source);

        orig_lower_val = *(thrust::min_element(s, s + recCount));
        orig_upper_val = *(thrust::max_element(s, s + recCount));

        //cout << "We need " << (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1))) << " bits to encode original range of " << orig_lower_val << " to " << orig_upper_val << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1)));
    }
    else {

        thrust::device_ptr<long long int> s((long long int*)source);

        orig_lower_val = *(thrust::min_element(s, s + recCount));
        orig_upper_val = *(thrust::max_element(s, s + recCount));

        //cout << "We need " << (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1))) << " bits to encode original range of " << orig_lower_val << " to " << orig_upper_val << endl;
        bits = (unsigned int)ceil(log2((double)((orig_upper_val - orig_lower_val) + 1)));
    };

    thrust::counting_iterator<unsigned int> begin(0);

    fit_count = bit_count/bits;
    void* d_v1;
    CUDA_SAFE_CALL(hipMalloc((void **) &d_v1, 12));
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v1);

    void* s_v1;
    CUDA_SAFE_CALL(hipMalloc((void **) &s_v1, 8));
    thrust::device_ptr<long long int> dd_sv((long long int*)s_v1);

    dd_sv[0] = orig_lower_val;
    dd_v[0] = bits;
    dd_v[1] = fit_count;
    dd_v[2] = bit_count;

    void* d;
    CUDA_SAFE_CALL(hipMalloc((void **) &d, recCount*float_size));
    thrust::device_ptr<char> dd((char*)d);
    thrust::fill(dd, dd+source_len,0);

    if (tp == 0) {
        compress_functor_int ff((int_type*)source,(unsigned long long int*)d, (long long int*)s_v1, (unsigned int*)d_v1);
        thrust::for_each(begin, begin + recCount, ff);
    }
    else {
        compress_functor_float ff((long long int*)source,(unsigned long long int*)d, (long long int*)s_v1, (unsigned int*)d_v1);
        thrust::for_each(begin, begin + recCount, ff);
    };


    thrust::device_ptr<unsigned long long int> s_copy1((unsigned long long int*)d);

    // make an addition  sequence
    thrust::device_ptr<unsigned int> add_seq = thrust::device_malloc<unsigned int>(recCount);
    thrust::constant_iterator<unsigned int> iter(fit_count);
    thrust::sequence(add_seq, add_seq + recCount, 0, 1);
    thrust::transform(add_seq, add_seq + recCount, iter, add_seq, thrust::divides<unsigned int>());

    unsigned int cnt = (recCount)/fit_count;
    if(cnt == 0)
        cnt = 1; // need at least 1

    if (recCount%fit_count > 0)
        cnt++;

    //thrust::device_ptr<unsigned long long int> fin_seq = thrust::device_malloc<unsigned long long int>(cnt);
    thrust::device_ptr<unsigned long long int> fin_seq((unsigned long long int*)source);

    thrust::reduce_by_key(add_seq, add_seq+recCount,s_copy1,thrust::make_discard_iterator(),
                          fin_seq);

    // copy fin_seq to host
    unsigned long long int * raw_src = thrust::raw_pointer_cast(fin_seq);

    //cout << file_name << " CNT  " << cnt << endl;

    if(!file_name.empty()) {
        hipMemcpy( host.data(), (void *)raw_src, cnt*8, hipMemcpyDeviceToHost);
        fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::app);
        binary_file.write((char *)&cnt, 4);
        binary_file.write((char *)&orig_lower_val, 8);
        binary_file.write((char *)&orig_upper_val, 8);
        binary_file.write((char *)host.data(),cnt*8);
        binary_file.write((char *)&comp_type, 4);
        binary_file.write((char *)&cnt, 4);
        binary_file.write((char *)&recCount, 4);
        binary_file.write((char *)&bits, 4);
        binary_file.write((char *)&orig_lower_val, 8);
        binary_file.write((char *)&fit_count, 4);
        binary_file.write((char *)&start_val, 8);
        binary_file.write((char *)&comp_type, 4);
        binary_file.write((char *)&comp_type, 4); //filler
        binary_file.close();
        if(cnt_counts[curr_file] < cnt)
            cnt_counts[curr_file] = cnt;
    }
    else {
        char* hh;
        // resize host to sz + cnt*8 + 15
        host.resize(sz+cnt+8);
        hh = (char*)(host.data() + sz);
        ((unsigned int*)hh)[0] = cnt;
        ((long long int*)(hh+4))[0] = orig_lower_val;
        ((long long int*)(hh+12))[0] = orig_upper_val;
        hipMemcpy( hh + 20, (void *)raw_src, cnt*8, hipMemcpyDeviceToHost);
        ((unsigned int*)hh)[5+cnt*2] = comp_type;
        ((unsigned int*)hh)[6+cnt*2] = cnt;
        ((unsigned int*)hh)[7+cnt*2] = recCount;
        ((unsigned int*)hh)[8+cnt*2] = bits;
        ((long long int*)(hh+36+cnt*8))[0] = orig_lower_val;
        ((unsigned int*)hh)[11+cnt*2] = fit_count;
        ((long long int*)(hh+48+cnt*8))[0] = start_val;
        ((unsigned int*)hh)[14+cnt*2] = comp_type;
    };

    thrust::device_free(add_seq);
    hipFree(d);
    hipFree(d_v1);
    hipFree(s_v1);
    return sz + cnt + 8;
}