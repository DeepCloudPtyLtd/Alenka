#include "hip/hip_runtime.h"

/* A Bison parser, made by GNU Bison 2.4.1.  */

/* Skeleton implementation for Bison's Yacc-like parsers in C
   
      Copyright (C) 1984, 1989, 1990, 2000, 2001, 2002, 2003, 2004, 2005, 2006
   Free Software Foundation, Inc.
   
   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.  */

/* As a special exception, you may create a larger work that contains
   part or all of the Bison parser skeleton and distribute that work
   under terms of your choice, so long as that work isn't itself a
   parser generator using the skeleton or a modified version thereof
   as a parser skeleton.  Alternatively, if you modify or redistribute
   the parser skeleton itself, you may (at your option) remove this
   special exception, which will cause the skeleton and the resulting
   Bison output files to be licensed under the GNU General Public
   License without this special exception.
   
   This special exception was added by the Free Software Foundation in
   version 2.2 of Bison.  */

/* C LALR(1) parser skeleton written by Richard Stallman, by
   simplifying the original so-called "semantic" parser.  */

/* All symbols defined below should begin with yy or YY, to avoid
   infringing on user name space.  This should be done even for local
   variables, as they might otherwise be expanded by user macros.
   There are some unavoidable exceptions within include files to
   define necessary library symbols; they are noted "INFRINGES ON
   USER NAME SPACE" below.  */

/* Identify Bison output.  */
#define YYBISON 1

/* Bison version.  */
#define YYBISON_VERSION "2.4.1"

/* Skeleton name.  */
#define YYSKELETON_NAME "yacc.c"

/* Pure parsers.  */
#define YYPURE 0

/* Push parsers.  */
#define YYPUSH 0

/* Pull parsers.  */
#define YYPULL 1

/* Using locations.  */
#define YYLSP_NEEDED 0



/* Copy the first part of user declarations.  */

/* Line 189 of yacc.c  */
#line 15 "bison.y"



#include "lex.yy.c"
#include "cm.h"
#include "operators.h"




/* Line 189 of yacc.c  */
#line 84 "bison.cu"

/* Enabling traces.  */
#ifndef YYDEBUG
# define YYDEBUG 0
#endif

/* Enabling verbose error messages.  */
#ifdef YYERROR_VERBOSE
# undef YYERROR_VERBOSE
# define YYERROR_VERBOSE 1
#else
# define YYERROR_VERBOSE 0
#endif

/* Enabling the token table.  */
#ifndef YYTOKEN_TABLE
# define YYTOKEN_TABLE 0
#endif


/* Tokens.  */
#ifndef YYTOKENTYPE
# define YYTOKENTYPE
   /* Put the tokens into the symbol table, so that GDB and other debuggers
      know about them.  */
   enum yytokentype {
     FILENAME = 258,
     NAME = 259,
     STRING = 260,
     INTNUM = 261,
     DECIMAL1 = 262,
     BOOL1 = 263,
     APPROXNUM = 264,
     USERVAR = 265,
     ASSIGN = 266,
     EQUAL = 267,
     NONEQUAL = 268,
     OR = 269,
     XOR = 270,
     AND = 271,
     DISTINCT = 272,
     YEAR = 273,
     REGEXP = 274,
     LIKE = 275,
     IS = 276,
     IN = 277,
     NOT = 278,
     BETWEEN = 279,
     COMPARISON = 280,
     SHIFT = 281,
     MOD = 282,
     FROM = 283,
     DELETE = 284,
     LOAD = 285,
     FILTER = 286,
     BY = 287,
     JOIN = 288,
     STORE = 289,
     INTO = 290,
     GROUP = 291,
     SELECT = 292,
     AS = 293,
     ORDER = 294,
     ASC = 295,
     DESC = 296,
     COUNT = 297,
     USING = 298,
     SUM = 299,
     AVG = 300,
     MIN = 301,
     MAX = 302,
     LIMIT = 303,
     ON = 304,
     BINARY = 305,
     LEFT = 306,
     RIGHT = 307,
     OUTER = 308,
     SEMI = 309,
     ANTI = 310,
     SORT = 311,
     SEGMENTS = 312,
     PRESORTED = 313,
     PARTITION = 314,
     INSERT = 315,
     WHERE = 316,
     DISPLAY = 317,
     CASE = 318,
     WHEN = 319,
     THEN = 320,
     ELSE = 321,
     END = 322,
     SHOW = 323,
     TABLES = 324,
     TABLE = 325,
     DESCRIBE = 326,
     DROP = 327,
     CREATE = 328,
     INDEX = 329,
     INTERVAL = 330,
     APPEND = 331
   };
#endif



#if ! defined YYSTYPE && ! defined YYSTYPE_IS_DECLARED
typedef union YYSTYPE
{

/* Line 214 of yacc.c  */
#line 25 "bison.y"

    long long int intval;
    double floatval;
    char *strval;
    int subtok;



/* Line 214 of yacc.c  */
#line 205 "bison.cu"
} YYSTYPE;
# define YYSTYPE_IS_TRIVIAL 1
# define yystype YYSTYPE /* obsolescent; will be withdrawn */
# define YYSTYPE_IS_DECLARED 1
#endif


/* Copy the second part of user declarations.  */


/* Line 264 of yacc.c  */
#line 217 "bison.cu"

#ifdef short
# undef short
#endif

#ifdef YYTYPE_UINT8
typedef YYTYPE_UINT8 yytype_uint8;
#else
typedef unsigned char yytype_uint8;
#endif

#ifdef YYTYPE_INT8
typedef YYTYPE_INT8 yytype_int8;
#elif (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
typedef signed char yytype_int8;
#else
typedef short int yytype_int8;
#endif

#ifdef YYTYPE_UINT16
typedef YYTYPE_UINT16 yytype_uint16;
#else
typedef unsigned short int yytype_uint16;
#endif

#ifdef YYTYPE_INT16
typedef YYTYPE_INT16 yytype_int16;
#else
typedef short int yytype_int16;
#endif

#ifndef YYSIZE_T
# ifdef __SIZE_TYPE__
#  define YYSIZE_T __SIZE_TYPE__
# elif defined size_t
#  define YYSIZE_T size_t
# elif ! defined YYSIZE_T && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#  include <stddef.h> /* INFRINGES ON USER NAME SPACE */
#  define YYSIZE_T size_t
# else
#  define YYSIZE_T unsigned int
# endif
#endif

#define YYSIZE_MAXIMUM ((YYSIZE_T) -1)

#ifndef YY_
# if YYENABLE_NLS
#  if ENABLE_NLS
#   include <libintl.h> /* INFRINGES ON USER NAME SPACE */
#   define YY_(msgid) dgettext ("bison-runtime", msgid)
#  endif
# endif
# ifndef YY_
#  define YY_(msgid) msgid
# endif
#endif

/* Suppress unused-variable warnings by "using" E.  */
#if ! defined lint || defined __GNUC__
# define YYUSE(e) ((void) (e))
#else
# define YYUSE(e) /* empty */
#endif

/* Identity function, used to suppress warnings about constant conditions.  */
#ifndef lint
# define YYID(n) (n)
#else
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static int
YYID (int yyi)
#else
static int
YYID (yyi)
    int yyi;
#endif
{
  return yyi;
}
#endif

#if ! defined yyoverflow || YYERROR_VERBOSE

/* The parser invokes alloca or malloc; define the necessary symbols.  */

# ifdef YYSTACK_USE_ALLOCA
#  if YYSTACK_USE_ALLOCA
#   ifdef __GNUC__
#    define YYSTACK_ALLOC __builtin_alloca
#   elif defined __BUILTIN_VA_ARG_INCR
#    include <alloca.h> /* INFRINGES ON USER NAME SPACE */
#   elif defined _AIX
#    define YYSTACK_ALLOC __alloca
#   elif defined _MSC_VER
#    include <malloc.h> /* INFRINGES ON USER NAME SPACE */
#    define alloca _alloca
#   else
#    define YYSTACK_ALLOC alloca
#    if ! defined _ALLOCA_H && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#     include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#     ifndef _STDLIB_H
#      define _STDLIB_H 1
#     endif
#    endif
#   endif
#  endif
# endif

# ifdef YYSTACK_ALLOC
   /* Pacify GCC's `empty if-body' warning.  */
#  define YYSTACK_FREE(Ptr) do { /* empty */; } while (YYID (0))
#  ifndef YYSTACK_ALLOC_MAXIMUM
    /* The OS might guarantee only one guard page at the bottom of the stack,
       and a page size can be as small as 4096 bytes.  So we cannot safely
       invoke alloca (N) if N exceeds 4096.  Use a slightly smaller number
       to allow for a few compiler-allocated temporary stack slots.  */
#   define YYSTACK_ALLOC_MAXIMUM 4032 /* reasonable circa 2006 */
#  endif
# else
#  define YYSTACK_ALLOC YYMALLOC
#  define YYSTACK_FREE YYFREE
#  ifndef YYSTACK_ALLOC_MAXIMUM
#   define YYSTACK_ALLOC_MAXIMUM YYSIZE_MAXIMUM
#  endif
#  if (defined __cplusplus && ! defined _STDLIB_H \
       && ! ((defined YYMALLOC || defined malloc) \
	     && (defined YYFREE || defined free)))
#   include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#   ifndef _STDLIB_H
#    define _STDLIB_H 1
#   endif
#  endif
#  ifndef YYMALLOC
#   define YYMALLOC malloc
#   if ! defined malloc && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void *malloc (YYSIZE_T); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
#  ifndef YYFREE
#   define YYFREE free
#   if ! defined free && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void free (void *); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
# endif
#endif /* ! defined yyoverflow || YYERROR_VERBOSE */


#if (! defined yyoverflow \
     && (! defined __cplusplus \
	 || (defined YYSTYPE_IS_TRIVIAL && YYSTYPE_IS_TRIVIAL)))

/* A type that is properly aligned for any stack member.  */
union yyalloc
{
  yytype_int16 yyss_alloc;
  YYSTYPE yyvs_alloc;
};

/* The size of the maximum gap between one aligned stack and the next.  */
# define YYSTACK_GAP_MAXIMUM (sizeof (union yyalloc) - 1)

/* The size of an array large to enough to hold all stacks, each with
   N elements.  */
# define YYSTACK_BYTES(N) \
     ((N) * (sizeof (yytype_int16) + sizeof (YYSTYPE)) \
      + YYSTACK_GAP_MAXIMUM)

/* Copy COUNT objects from FROM to TO.  The source and destination do
   not overlap.  */
# ifndef YYCOPY
#  if defined __GNUC__ && 1 < __GNUC__
#   define YYCOPY(To, From, Count) \
      __builtin_memcpy (To, From, (Count) * sizeof (*(From)))
#  else
#   define YYCOPY(To, From, Count)		\
      do					\
	{					\
	  YYSIZE_T yyi;				\
	  for (yyi = 0; yyi < (Count); yyi++)	\
	    (To)[yyi] = (From)[yyi];		\
	}					\
      while (YYID (0))
#  endif
# endif

/* Relocate STACK from its old location to the new one.  The
   local variables YYSIZE and YYSTACKSIZE give the old and new number of
   elements in the stack, and YYPTR gives the new location of the
   stack.  Advance YYPTR to a properly aligned location for the next
   stack.  */
# define YYSTACK_RELOCATE(Stack_alloc, Stack)				\
    do									\
      {									\
	YYSIZE_T yynewbytes;						\
	YYCOPY (&yyptr->Stack_alloc, Stack, yysize);			\
	Stack = &yyptr->Stack_alloc;					\
	yynewbytes = yystacksize * sizeof (*Stack) + YYSTACK_GAP_MAXIMUM; \
	yyptr += yynewbytes / sizeof (*yyptr);				\
      }									\
    while (YYID (0))

#endif

/* YYFINAL -- State number of the termination state.  */
#define YYFINAL  23
/* YYLAST -- Last index in YYTABLE.  */
#define YYLAST   835

/* YYNTOKENS -- Number of terminals.  */
#define YYNTOKENS  94
/* YYNNTS -- Number of nonterminals.  */
#define YYNNTS  14
/* YYNRULES -- Number of rules.  */
#define YYNRULES  96
/* YYNRULES -- Number of states.  */
#define YYNSTATES  286

/* YYTRANSLATE(YYLEX) -- Bison symbol number corresponding to YYLEX.  */
#define YYUNDEFTOK  2
#define YYMAXUTOK   331

#define YYTRANSLATE(YYX)						\
  ((unsigned int) (YYX) <= YYMAXUTOK ? yytranslate[YYX] : YYUNDEFTOK)

/* YYTRANSLATE[YYLEX] -- Bison symbol number corresponding to YYLEX.  */
static const yytype_uint8 yytranslate[] =
{
       0,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    23,     2,     2,     2,    34,    28,     2,
      87,    88,    32,    30,    90,    31,    89,    33,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,    93,    86,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,    36,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    91,    27,    92,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     1,     2,     3,     4,
       5,     6,     7,     8,     9,    10,    11,    12,    13,    14,
      15,    16,    17,    18,    19,    20,    21,    22,    24,    25,
      26,    29,    35,    37,    38,    39,    40,    41,    42,    43,
      44,    45,    46,    47,    48,    49,    50,    51,    52,    53,
      54,    55,    56,    57,    58,    59,    60,    61,    62,    63,
      64,    65,    66,    67,    68,    69,    70,    71,    72,    73,
      74,    75,    76,    77,    78,    79,    80,    81,    82,    83,
      84,    85
};

#if YYDEBUG
/* YYPRHS[YYN] -- Index of the first RHS symbol of rule number YYN in
   YYRHS.  */
static const yytype_uint16 yyprhs[] =
{
       0,     0,     3,     6,    10,    12,    20,    33,    39,    46,
      55,    65,    73,    82,    85,    93,    99,   107,   110,   114,
     137,   146,   157,   159,   163,   165,   167,   169,   171,   173,
     175,   187,   197,   204,   207,   210,   215,   220,   225,   230,
     235,   238,   243,   247,   251,   255,   259,   263,   267,   271,
     275,   279,   283,   287,   291,   294,   297,   301,   305,   311,
     315,   324,   328,   333,   334,   338,   342,   348,   350,   352,
     356,   358,   362,   363,   365,   368,   373,   380,   387,   394,
     400,   406,   413,   419,   425,   433,   441,   448,   456,   463,
     471,   478,   479,   482,   483,   488,   496
};

/* YYRHS -- A `-1'-separated list of the rules' RHS.  */
static const yytype_int8 yyrhs[] =
{
      95,     0,    -1,    96,    86,    -1,    95,    96,    86,    -1,
      97,    -1,     4,    11,    46,   100,    37,     4,    99,    -1,
       4,    11,    39,     3,    52,    87,     3,    88,    47,    87,
     101,    88,    -1,     4,    11,    40,     4,   104,    -1,     4,
      11,    48,     4,    41,   103,    -1,     4,    11,    46,   100,
      37,     4,   105,    99,    -1,    43,     4,    44,     3,    52,
      87,     3,    88,   106,    -1,    43,     4,    44,     3,   106,
      59,   107,    -1,    43,     4,    44,     3,    85,   106,    59,
     107,    -1,    80,     4,    -1,    69,    44,     4,    46,   100,
      37,     4,    -1,    38,    37,     4,    70,    98,    -1,    71,
       4,    52,    87,     3,    88,   106,    -1,    77,    78,    -1,
      81,    79,     4,    -1,    82,    83,     4,    58,     4,    87,
       4,    89,     4,    88,    37,     4,    90,     4,    70,     4,
      89,     4,    12,     4,    89,     4,    -1,    82,    83,     4,
      58,     4,    87,     4,    88,    -1,    82,    84,     4,    58,
       4,    87,     4,    90,     4,    88,    -1,     4,    -1,     4,
      89,     4,    -1,    10,    -1,     5,    -1,     6,    -1,     7,
      -1,     9,    -1,     8,    -1,     4,    91,     6,    92,    93,
       4,    87,     6,    90,     6,    88,    -1,     4,    91,     6,
      92,    93,     4,    87,     6,    88,    -1,     4,    91,     6,
      92,    93,     4,    -1,     4,    49,    -1,     4,    50,    -1,
      51,    87,    98,    88,    -1,    53,    87,    98,    88,    -1,
      54,    87,    98,    88,    -1,    55,    87,    98,    88,    -1,
      56,    87,    98,    88,    -1,    17,    98,    -1,    18,    87,
      98,    88,    -1,    98,    30,    98,    -1,    98,    31,    98,
      -1,    98,    32,    98,    -1,    98,    33,    98,    -1,    98,
      34,    98,    -1,    98,    35,    98,    -1,    98,    16,    98,
      -1,    98,    12,    98,    -1,    98,    13,    98,    -1,    98,
      14,    98,    -1,    98,    15,    98,    -1,    98,    29,    98,
      -1,    24,    98,    -1,    23,    98,    -1,    98,    26,    98,
      -1,    98,    20,    98,    -1,    98,    26,    87,    97,    88,
      -1,    87,    98,    88,    -1,    72,    73,    98,    74,    98,
      75,    98,    76,    -1,    98,    21,     8,    -1,    98,    21,
      24,     8,    -1,    -1,    45,    41,   102,    -1,    98,    47,
       4,    -1,   100,    90,    98,    47,     4,    -1,    32,    -1,
      98,    -1,   101,    90,    98,    -1,    98,    -1,    98,    90,
     102,    -1,    -1,   102,    -1,    41,    98,    -1,    42,     4,
      58,    98,    -1,    60,    64,    42,     4,    58,    98,    -1,
      61,    64,    42,     4,    58,    98,    -1,    60,    63,    42,
       4,    58,    98,    -1,    60,    42,     4,    58,    98,    -1,
      61,    42,     4,    58,    98,    -1,    61,    63,    42,     4,
      58,    98,    -1,    62,    42,     4,    58,    98,    -1,    42,
       4,    58,    98,   105,    -1,    60,    64,    42,     4,    58,
      98,   105,    -1,    61,    64,    42,     4,    58,    98,   105,
      -1,    60,    42,     4,    58,    98,   105,    -1,    60,    63,
      42,     4,    58,    98,   105,    -1,    61,    42,     4,    58,
      98,   105,    -1,    61,    63,    42,     4,    58,    98,   105,
      -1,    62,    42,     4,    58,    98,   105,    -1,    -1,    57,
       6,    -1,    -1,    65,    66,    41,     4,    -1,    65,    66,
      41,     4,    68,    41,     6,    -1,    67,    41,     4,    -1
};

/* YYRLINE[YYN] -- source line where rule number YYN was defined.  */
static const yytype_uint16 yyrline[] =
{
       0,   124,   124,   125,   129,   132,   134,   136,   138,   140,
     142,   144,   146,   148,   150,   152,   154,   156,   158,   160,
     162,   164,   170,   171,   172,   173,   174,   175,   176,   177,
     178,   179,   180,   181,   182,   183,   184,   185,   186,   187,
     188,   189,   193,   194,   195,   196,   197,   198,   199,   200,
     201,   202,   203,   204,   205,   206,   207,   208,   210,   211,
     212,   216,   217,   220,   223,   227,   228,   229,   233,   234,
     238,   239,   242,   244,   247,   251,   252,   253,   254,   255,
     256,   257,   258,   259,   260,   261,   262,   263,   264,   265,
     266,   268,   271,   273,   276,   277,   278
};
#endif

#if YYDEBUG || YYERROR_VERBOSE || YYTOKEN_TABLE
/* YYTNAME[SYMBOL-NUM] -- String name of the symbol SYMBOL-NUM.
   First, the terminals, then, starting at YYNTOKENS, nonterminals.  */
static const char *const yytname[] =
{
  "$end", "error", "$undefined", "FILENAME", "NAME", "STRING", "INTNUM",
  "DECIMAL1", "BOOL1", "APPROXNUM", "USERVAR", "ASSIGN", "EQUAL",
  "NONEQUAL", "OR", "XOR", "AND", "DISTINCT", "YEAR", "REGEXP", "LIKE",
  "IS", "IN", "'!'", "NOT", "BETWEEN", "COMPARISON", "'|'", "'&'", "SHIFT",
  "'+'", "'-'", "'*'", "'/'", "'%'", "MOD", "'^'", "FROM", "DELETE",
  "LOAD", "FILTER", "BY", "JOIN", "STORE", "INTO", "GROUP", "SELECT", "AS",
  "ORDER", "ASC", "DESC", "COUNT", "USING", "SUM", "AVG", "MIN", "MAX",
  "LIMIT", "ON", "BINARY", "LEFT", "RIGHT", "OUTER", "SEMI", "ANTI",
  "SORT", "SEGMENTS", "PRESORTED", "PARTITION", "INSERT", "WHERE",
  "DISPLAY", "CASE", "WHEN", "THEN", "ELSE", "END", "SHOW", "TABLES",
  "TABLE", "DESCRIBE", "DROP", "CREATE", "INDEX", "INTERVAL", "APPEND",
  "';'", "'('", "')'", "'.'", "','", "'{'", "'}'", "':'", "$accept",
  "stmt_list", "stmt", "select_stmt", "expr", "opt_group_list",
  "expr_list", "load_list", "val_list", "opt_val_list", "opt_where",
  "join_list", "opt_limit", "sort_def", 0
};
#endif

# ifdef YYPRINT
/* YYTOKNUM[YYLEX-NUM] -- Internal token number corresponding to
   token YYLEX-NUM.  */
static const yytype_uint16 yytoknum[] =
{
       0,   256,   257,   258,   259,   260,   261,   262,   263,   264,
     265,   266,   267,   268,   269,   270,   271,   272,   273,   274,
     275,   276,   277,    33,   278,   279,   280,   124,    38,   281,
      43,    45,    42,    47,    37,   282,    94,   283,   284,   285,
     286,   287,   288,   289,   290,   291,   292,   293,   294,   295,
     296,   297,   298,   299,   300,   301,   302,   303,   304,   305,
     306,   307,   308,   309,   310,   311,   312,   313,   314,   315,
     316,   317,   318,   319,   320,   321,   322,   323,   324,   325,
     326,   327,   328,   329,   330,   331,    59,    40,    41,    46,
      44,   123,   125,    58
};
# endif

/* YYR1[YYN] -- Symbol number of symbol that rule YYN derives.  */
static const yytype_uint8 yyr1[] =
{
       0,    94,    95,    95,    96,    97,    97,    97,    97,    97,
      97,    97,    97,    97,    97,    97,    97,    97,    97,    97,
      97,    97,    98,    98,    98,    98,    98,    98,    98,    98,
      98,    98,    98,    98,    98,    98,    98,    98,    98,    98,
      98,    98,    98,    98,    98,    98,    98,    98,    98,    98,
      98,    98,    98,    98,    98,    98,    98,    98,    98,    98,
      98,    98,    98,    99,    99,   100,   100,   100,   101,   101,
     102,   102,   103,   103,   104,   105,   105,   105,   105,   105,
     105,   105,   105,   105,   105,   105,   105,   105,   105,   105,
     105,   106,   106,   107,   107,   107,   107
};

/* YYR2[YYN] -- Number of symbols composing right hand side of rule YYN.  */
static const yytype_uint8 yyr2[] =
{
       0,     2,     2,     3,     1,     7,    12,     5,     6,     8,
       9,     7,     8,     2,     7,     5,     7,     2,     3,    22,
       8,    10,     1,     3,     1,     1,     1,     1,     1,     1,
      11,     9,     6,     2,     2,     4,     4,     4,     4,     4,
       2,     4,     3,     3,     3,     3,     3,     3,     3,     3,
       3,     3,     3,     3,     2,     2,     3,     3,     5,     3,
       8,     3,     4,     0,     3,     3,     5,     1,     1,     3,
       1,     3,     0,     1,     2,     4,     6,     6,     6,     5,
       5,     6,     5,     5,     7,     7,     6,     7,     6,     7,
       6,     0,     2,     0,     4,     7,     3
};

/* YYDEFACT[STATE-NAME] -- Default rule to reduce with in state
   STATE-NUM when YYTABLE doesn't specify something else to do.  Zero
   means the default is an error.  */
static const yytype_uint8 yydefact[] =
{
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     4,     0,     0,     0,     0,     0,    17,    13,
       0,     0,     0,     1,     0,     2,     0,     0,     0,     0,
       0,     0,     0,     0,    18,     0,     0,     3,     0,     0,
      22,    25,    26,    27,    29,    28,    24,     0,     0,     0,
       0,    67,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,    91,     0,     0,     0,     0,     0,     0,
       7,    33,    34,     0,     0,    40,     0,    55,    54,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,    72,    15,     0,     0,    91,     0,
       0,     0,     0,     0,     0,    74,    23,     0,     0,     0,
       0,     0,     0,     0,     0,    59,    49,    50,    51,    52,
      48,    57,    61,     0,     0,    56,    53,    42,    43,    44,
      45,    46,    47,    65,    63,     0,    70,    73,     8,     0,
      92,     0,    93,     0,    91,     0,     0,     0,     0,    41,
      35,    36,    37,    38,    39,     0,    62,    22,     0,     0,
       0,     0,     0,     0,     5,    63,     0,     0,     0,    93,
       0,     0,    11,    14,    16,     0,     0,     0,     0,     0,
      58,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       9,    66,    71,    91,    12,     0,     0,    20,     0,     0,
       0,    32,     0,     0,    64,     0,     0,     0,     0,     0,
       0,     0,    10,     0,    96,     0,     0,     0,     0,     0,
      75,     0,     0,     0,     0,     0,     0,     0,    94,     0,
      21,    68,     0,     0,    60,    83,    79,     0,     0,    80,
       0,     0,    82,     0,     0,     6,     0,    31,     0,    86,
      78,    76,    88,    81,    77,    90,     0,     0,    69,     0,
      87,    84,    89,    85,    95,     0,    30,     0,     0,     0,
       0,     0,     0,     0,     0,    19
};

/* YYDEFGOTO[NTERM-NUM].  */
static const yytype_int16 yydefgoto[] =
{
      -1,    10,    11,    12,   146,   174,    60,   242,   147,   148,
      70,   175,   109,   182
};

/* YYPACT[STATE-NUM] -- Index in YYTABLE of the portion describing
   STATE-NUM.  */
#define YYPACT_NINF -164
static const yytype_int16 yypact[] =
{
     273,    -5,    -9,     5,   -32,    36,    -2,    73,     7,   -34,
     251,     2,  -164,    68,    85,    46,    90,    41,  -164,  -164,
      91,    95,    96,  -164,    25,  -164,   110,   122,   159,   138,
      77,   141,   102,    62,  -164,    92,    94,  -164,   105,   112,
     -47,  -164,  -164,  -164,  -164,  -164,  -164,   230,    71,   230,
     230,  -164,    75,    83,    84,    86,    87,    99,   230,   656,
     -33,   119,   230,   -42,   159,   177,   182,   183,   107,   230,
    -164,  -164,  -164,   184,   189,   784,   230,   266,   266,   230,
     230,   230,   230,   230,   230,   344,   230,   230,   230,   230,
     230,   230,     0,   252,   230,   230,   230,   230,   230,   230,
     230,   192,   193,   230,   230,   704,   111,   194,   144,   143,
     -26,   116,   118,   120,   205,   704,  -164,   124,   368,   392,
     428,   452,   476,   512,   589,  -164,   704,   727,   748,   768,
     784,   800,  -164,   203,    74,   404,   101,   106,   106,  -164,
    -164,  -164,  -164,  -164,   -29,   680,    89,  -164,  -164,   214,
    -164,   162,   -30,   220,   144,   221,   222,   130,   134,  -164,
    -164,  -164,  -164,  -164,  -164,   230,  -164,    -4,   142,   225,
     191,   -37,   -25,   199,  -164,   188,   238,   230,   155,   -30,
     178,   204,  -164,  -164,  -164,   -15,   160,   202,   248,   560,
    -164,   206,   230,   259,   223,   224,   263,   226,   229,   268,
    -164,  -164,  -164,   144,  -164,   232,   270,  -164,   274,   275,
     195,   200,   230,   230,  -164,   233,   276,   284,   235,   286,
     300,   254,  -164,   305,  -164,   227,   231,   230,   304,   536,
     613,   230,   255,   256,   230,   260,   265,   230,   253,   288,
    -164,   704,   -70,   -65,  -164,  -164,   613,   230,   230,   613,
     230,   230,   613,   285,   323,  -164,   230,  -164,   324,  -164,
     613,   613,  -164,   613,   613,  -164,   328,   239,   704,   247,
    -164,  -164,  -164,  -164,  -164,   332,  -164,   267,   334,   257,
     336,   329,   339,   258,   341,  -164
};

/* YYPGOTO[NTERM-NUM].  */
static const yytype_int16 yypgoto[] =
{
    -164,  -164,   338,   215,   -28,   176,   297,  -164,  -163,  -164,
    -164,   -71,  -107,   173
};

/* YYTABLE[YYPACT[STATE-NUM]].  What to do in state STATE-NUM.  If
   positive, shift that token.  If negative, reduce the rule which
   number is the opposite.  If zero, do what YYDEFACT says.
   If YYTABLE_NINF, syntax error.  */
#define YYTABLE_NINF -1
static const yytype_int16 yytable[] =
{
      59,   151,    71,    72,   102,   193,    13,    13,   132,    15,
     106,   153,    16,   169,   202,   107,   170,   196,   255,    75,
     256,    77,    78,   257,   133,   258,   194,   195,    14,   214,
      85,   171,   172,   173,   105,   180,    59,   181,   197,   198,
      17,   115,    73,   108,    74,    71,    72,   184,   118,    21,
      22,   119,   120,   121,   122,   123,   124,   103,   126,   127,
     128,   129,   130,   131,   103,   135,   136,   137,   138,   139,
     140,   141,   142,   207,   208,   145,    18,    19,   167,    41,
      42,    43,    44,    45,    46,    73,    20,    74,    25,    30,
      31,    47,    48,    33,    32,    34,   222,    49,    50,    35,
      36,    86,    87,    88,    89,    90,    85,    26,    27,    91,
      92,    37,     2,    38,    28,    93,    29,     3,    94,    95,
      96,    97,    98,    99,   100,    52,    39,    53,    54,    55,
      56,    95,    96,    97,    98,    99,   100,   189,    97,    98,
      99,   100,    61,     4,    63,     5,    57,    62,    64,    65,
      66,     6,    67,    69,     7,     8,     9,    68,    76,   245,
     104,    58,    79,    40,    41,    42,    43,    44,    45,    46,
      80,    81,    84,    82,    83,   259,    47,    48,   262,   177,
     111,   265,    49,    50,   229,   230,   112,   113,   116,   270,
     271,    51,   272,   273,   114,   117,   143,   144,   149,   241,
     150,   107,   152,   246,   154,   155,   249,   156,   157,   252,
      52,   166,    53,    54,    55,    56,   158,   178,   187,   260,
     261,   179,   263,   264,   183,   185,   186,   188,   268,   191,
     190,    57,   192,   170,    40,    41,    42,    43,    44,    45,
      46,   199,   201,   203,   205,   206,    58,    47,    48,   210,
     209,    23,   211,    49,    50,     1,    40,    41,    42,    43,
      44,    45,    46,   215,   213,   216,   217,   218,   219,    47,
      48,   220,   221,   223,   224,    49,    50,     1,   225,   226,
     232,    52,   227,    53,    54,    55,    56,   228,   233,     2,
     235,   231,    93,   234,     3,    94,    95,    96,    97,    98,
      99,   100,    57,    52,   236,    53,    54,    55,    56,   238,
     243,     2,   237,   247,   248,   239,     3,    58,   250,   240,
       4,   253,     5,   251,    57,   254,   266,   267,     6,   275,
     269,     7,     8,     9,   274,   276,   277,   278,   279,   134,
     281,   282,     4,   283,     5,   285,   280,   284,    24,   168,
       6,   200,   204,     7,     8,     9,    86,    87,    88,    89,
      90,   110,     0,     0,    91,    92,     0,     0,     0,     0,
      93,     0,     0,    94,    95,    96,    97,    98,    99,   100,
      86,    87,    88,    89,    90,     0,     0,     0,    91,    92,
       0,     0,     0,     0,    93,     0,     0,    94,    95,    96,
      97,    98,    99,   100,    86,    87,    88,    89,    90,     0,
       0,     0,    91,    92,     0,     0,     0,     0,    93,     0,
       0,    94,    95,    96,    97,    98,    99,   100,     0,     0,
       0,     0,   125,    94,    95,    96,    97,    98,    99,   100,
      86,    87,    88,    89,    90,     0,     0,     0,    91,    92,
       0,     0,     0,     0,    93,     0,   159,    94,    95,    96,
      97,    98,    99,   100,    86,    87,    88,    89,    90,     0,
       0,     0,    91,    92,     0,     0,     0,     0,    93,     0,
     160,    94,    95,    96,    97,    98,    99,   100,    86,    87,
      88,    89,    90,     0,     0,     0,    91,    92,     0,     0,
       0,     0,    93,     0,     0,    94,    95,    96,    97,    98,
      99,   100,     0,     0,     0,     0,   161,     0,     0,     0,
       0,     0,     0,     0,    86,    87,    88,    89,    90,     0,
       0,     0,    91,    92,     0,     0,     0,     0,    93,     0,
     162,    94,    95,    96,    97,    98,    99,   100,    86,    87,
      88,    89,    90,     0,     0,     0,    91,    92,     0,     0,
       0,     0,    93,     0,   163,    94,    95,    96,    97,    98,
      99,   100,    86,    87,    88,    89,    90,     0,     0,     0,
      91,    92,     0,     0,     0,     0,    93,     0,     0,    94,
      95,    96,    97,    98,    99,   100,     0,     0,     0,     0,
     164,    86,    87,    88,    89,    90,     0,     0,     0,    91,
      92,     0,   244,     0,     0,    93,     0,     0,    94,    95,
      96,    97,    98,    99,   100,    86,    87,    88,    89,    90,
       0,     0,     0,    91,    92,   212,     0,     0,     0,    93,
       0,     0,    94,    95,    96,    97,    98,    99,   100,     0,
       0,     0,     0,     0,     0,   169,     0,     0,     0,     0,
       0,     0,     0,   165,     0,     0,     0,     0,    86,    87,
      88,    89,    90,   171,   172,   173,    91,    92,     0,     0,
       0,     0,    93,     0,     0,    94,    95,    96,    97,    98,
      99,   100,    86,    87,    88,    89,    90,     0,     0,     0,
      91,    92,     0,   101,     0,     0,    93,     0,     0,    94,
      95,    96,    97,    98,    99,   100,    86,    87,    88,    89,
      90,     0,     0,     0,    91,    92,     0,   176,     0,     0,
      93,     0,     0,    94,    95,    96,    97,    98,    99,   100,
      87,    88,    89,    90,     0,     0,     0,    91,    92,     0,
       0,     0,     0,    93,     0,     0,    94,    95,    96,    97,
      98,    99,   100,    89,    90,     0,     0,     0,    91,    92,
       0,     0,     0,     0,    93,     0,     0,    94,    95,    96,
      97,    98,    99,   100,    90,     0,     0,     0,    91,    92,
       0,     0,     0,     0,    93,     0,     0,    94,    95,    96,
      97,    98,    99,   100,    91,    92,     0,     0,     0,     0,
      93,     0,     0,    94,    95,    96,    97,    98,    99,   100,
      -1,    -1,     0,     0,     0,     0,    93,     0,     0,    94,
      95,    96,    97,    98,    99,   100
};

static const yytype_int16 yycheck[] =
{
      28,   108,    49,    50,    37,    42,    11,    11,     8,     4,
      52,    37,    44,    42,   177,    57,    45,    42,    88,    47,
      90,    49,    50,    88,    24,    90,    63,    64,    37,   192,
      58,    60,    61,    62,    62,    65,    64,    67,    63,    64,
       4,    69,    89,    85,    91,    49,    50,   154,    76,    83,
      84,    79,    80,    81,    82,    83,    84,    90,    86,    87,
      88,    89,    90,    91,    90,    93,    94,    95,    96,    97,
      98,    99,   100,    88,    89,   103,    78,     4,     4,     5,
       6,     7,     8,     9,    10,    89,    79,    91,    86,     4,
      44,    17,    18,    52,     4,     4,   203,    23,    24,     4,
       4,    12,    13,    14,    15,    16,   134,    39,    40,    20,
      21,    86,    38,     3,    46,    26,    48,    43,    29,    30,
      31,    32,    33,    34,    35,    51,     4,    53,    54,    55,
      56,    30,    31,    32,    33,    34,    35,   165,    32,    33,
      34,    35,     4,    69,     3,    71,    72,    70,    46,    87,
      58,    77,    58,    41,    80,    81,    82,    52,    87,   230,
      41,    87,    87,     4,     5,     6,     7,     8,     9,    10,
      87,    87,    73,    87,    87,   246,    17,    18,   249,    90,
       3,   252,    23,    24,   212,   213,     4,     4,     4,   260,
     261,    32,   263,   264,    87,     6,     4,     4,    87,   227,
       6,    57,    59,   231,    88,    87,   234,    87,     3,   237,
      51,     8,    53,    54,    55,    56,    92,     3,    88,   247,
     248,    59,   250,   251,     4,     4,     4,    93,   256,     4,
      88,    72,    41,    45,     4,     5,     6,     7,     8,     9,
      10,    42,     4,    88,    66,    41,    87,    17,    18,    47,
      90,     0,     4,    23,    24,     4,     4,     5,     6,     7,
       8,     9,    10,     4,    58,    42,    42,     4,    42,    17,
      18,    42,     4,    41,     4,    23,    24,     4,     4,     4,
       4,    51,    87,    53,    54,    55,    56,    87,     4,    38,
       4,    58,    26,    58,    43,    29,    30,    31,    32,    33,
      34,    35,    72,    51,     4,    53,    54,    55,    56,     4,
       6,    38,    58,    58,    58,    88,    43,    87,    58,    88,
      69,    68,    71,    58,    72,    37,    41,     4,    77,    90,
       6,    80,    81,    82,     6,    88,     4,    70,     4,    87,
       4,    12,    69,     4,    71,     4,    89,    89,    10,   134,
      77,   175,   179,    80,    81,    82,    12,    13,    14,    15,
      16,    64,    -1,    -1,    20,    21,    -1,    -1,    -1,    -1,
      26,    -1,    -1,    29,    30,    31,    32,    33,    34,    35,
      12,    13,    14,    15,    16,    -1,    -1,    -1,    20,    21,
      -1,    -1,    -1,    -1,    26,    -1,    -1,    29,    30,    31,
      32,    33,    34,    35,    12,    13,    14,    15,    16,    -1,
      -1,    -1,    20,    21,    -1,    -1,    -1,    -1,    26,    -1,
      -1,    29,    30,    31,    32,    33,    34,    35,    -1,    -1,
      -1,    -1,    88,    29,    30,    31,    32,    33,    34,    35,
      12,    13,    14,    15,    16,    -1,    -1,    -1,    20,    21,
      -1,    -1,    -1,    -1,    26,    -1,    88,    29,    30,    31,
      32,    33,    34,    35,    12,    13,    14,    15,    16,    -1,
      -1,    -1,    20,    21,    -1,    -1,    -1,    -1,    26,    -1,
      88,    29,    30,    31,    32,    33,    34,    35,    12,    13,
      14,    15,    16,    -1,    -1,    -1,    20,    21,    -1,    -1,
      -1,    -1,    26,    -1,    -1,    29,    30,    31,    32,    33,
      34,    35,    -1,    -1,    -1,    -1,    88,    -1,    -1,    -1,
      -1,    -1,    -1,    -1,    12,    13,    14,    15,    16,    -1,
      -1,    -1,    20,    21,    -1,    -1,    -1,    -1,    26,    -1,
      88,    29,    30,    31,    32,    33,    34,    35,    12,    13,
      14,    15,    16,    -1,    -1,    -1,    20,    21,    -1,    -1,
      -1,    -1,    26,    -1,    88,    29,    30,    31,    32,    33,
      34,    35,    12,    13,    14,    15,    16,    -1,    -1,    -1,
      20,    21,    -1,    -1,    -1,    -1,    26,    -1,    -1,    29,
      30,    31,    32,    33,    34,    35,    -1,    -1,    -1,    -1,
      88,    12,    13,    14,    15,    16,    -1,    -1,    -1,    20,
      21,    -1,    76,    -1,    -1,    26,    -1,    -1,    29,    30,
      31,    32,    33,    34,    35,    12,    13,    14,    15,    16,
      -1,    -1,    -1,    20,    21,    75,    -1,    -1,    -1,    26,
      -1,    -1,    29,    30,    31,    32,    33,    34,    35,    -1,
      -1,    -1,    -1,    -1,    -1,    42,    -1,    -1,    -1,    -1,
      -1,    -1,    -1,    74,    -1,    -1,    -1,    -1,    12,    13,
      14,    15,    16,    60,    61,    62,    20,    21,    -1,    -1,
      -1,    -1,    26,    -1,    -1,    29,    30,    31,    32,    33,
      34,    35,    12,    13,    14,    15,    16,    -1,    -1,    -1,
      20,    21,    -1,    47,    -1,    -1,    26,    -1,    -1,    29,
      30,    31,    32,    33,    34,    35,    12,    13,    14,    15,
      16,    -1,    -1,    -1,    20,    21,    -1,    47,    -1,    -1,
      26,    -1,    -1,    29,    30,    31,    32,    33,    34,    35,
      13,    14,    15,    16,    -1,    -1,    -1,    20,    21,    -1,
      -1,    -1,    -1,    26,    -1,    -1,    29,    30,    31,    32,
      33,    34,    35,    15,    16,    -1,    -1,    -1,    20,    21,
      -1,    -1,    -1,    -1,    26,    -1,    -1,    29,    30,    31,
      32,    33,    34,    35,    16,    -1,    -1,    -1,    20,    21,
      -1,    -1,    -1,    -1,    26,    -1,    -1,    29,    30,    31,
      32,    33,    34,    35,    20,    21,    -1,    -1,    -1,    -1,
      26,    -1,    -1,    29,    30,    31,    32,    33,    34,    35,
      20,    21,    -1,    -1,    -1,    -1,    26,    -1,    -1,    29,
      30,    31,    32,    33,    34,    35
};

/* YYSTOS[STATE-NUM] -- The (internal number of the) accessing
   symbol of state STATE-NUM.  */
static const yytype_uint8 yystos[] =
{
       0,     4,    38,    43,    69,    71,    77,    80,    81,    82,
      95,    96,    97,    11,    37,     4,    44,     4,    78,     4,
      79,    83,    84,     0,    96,    86,    39,    40,    46,    48,
       4,    44,     4,    52,     4,     4,     4,    86,     3,     4,
       4,     5,     6,     7,     8,     9,    10,    17,    18,    23,
      24,    32,    51,    53,    54,    55,    56,    72,    87,    98,
     100,     4,    70,     3,    46,    87,    58,    58,    52,    41,
     104,    49,    50,    89,    91,    98,    87,    98,    98,    87,
      87,    87,    87,    87,    73,    98,    12,    13,    14,    15,
      16,    20,    21,    26,    29,    30,    31,    32,    33,    34,
      35,    47,    37,    90,    41,    98,    52,    57,    85,   106,
     100,     3,     4,     4,    87,    98,     4,     6,    98,    98,
      98,    98,    98,    98,    98,    88,    98,    98,    98,    98,
      98,    98,     8,    24,    87,    98,    98,    98,    98,    98,
      98,    98,    98,     4,     4,    98,    98,   102,   103,    87,
       6,   106,    59,    37,    88,    87,    87,     3,    92,    88,
      88,    88,    88,    88,    88,    74,     8,     4,    97,    42,
      45,    60,    61,    62,    99,   105,    47,    90,     3,    59,
      65,    67,   107,     4,   106,     4,     4,    88,    93,    98,
      88,     4,    41,    42,    63,    64,    42,    63,    64,    42,
      99,     4,   102,    88,   107,    66,    41,    88,    89,    90,
      47,     4,    75,    58,   102,     4,    42,    42,     4,    42,
      42,     4,   106,    41,     4,     4,     4,    87,    87,    98,
      98,    58,     4,     4,    58,     4,     4,    58,     4,    88,
      88,    98,   101,     6,    76,   105,    98,    58,    58,    98,
      58,    58,    98,    68,    37,    88,    90,    88,    90,   105,
      98,    98,   105,    98,    98,   105,    41,     4,    98,     6,
     105,   105,   105,   105,     6,    90,    88,     4,    70,     4,
      89,     4,    12,     4,    89,     4
};

#define yyerrok		(yyerrstatus = 0)
#define yyclearin	(yychar = YYEMPTY)
#define YYEMPTY		(-2)
#define YYEOF		0

#define YYACCEPT	goto yyacceptlab
#define YYABORT		goto yyabortlab
#define YYERROR		goto yyerrorlab


/* Like YYERROR except do call yyerror.  This remains here temporarily
   to ease the transition to the new meaning of YYERROR, for GCC.
   Once GCC version 2 has supplanted version 1, this can go.  */

#define YYFAIL		goto yyerrlab

#define YYRECOVERING()  (!!yyerrstatus)

#define YYBACKUP(Token, Value)					\
do								\
  if (yychar == YYEMPTY && yylen == 1)				\
    {								\
      yychar = (Token);						\
      yylval = (Value);						\
      yytoken = YYTRANSLATE (yychar);				\
      YYPOPSTACK (1);						\
      goto yybackup;						\
    }								\
  else								\
    {								\
      yyerror (YY_("syntax error: cannot back up")); \
      YYERROR;							\
    }								\
while (YYID (0))


#define YYTERROR	1
#define YYERRCODE	256


/* YYLLOC_DEFAULT -- Set CURRENT to span from RHS[1] to RHS[N].
   If N is 0, then set CURRENT to the empty location which ends
   the previous symbol: RHS[0] (always defined).  */

#define YYRHSLOC(Rhs, K) ((Rhs)[K])
#ifndef YYLLOC_DEFAULT
# define YYLLOC_DEFAULT(Current, Rhs, N)				\
    do									\
      if (YYID (N))                                                    \
	{								\
	  (Current).first_line   = YYRHSLOC (Rhs, 1).first_line;	\
	  (Current).first_column = YYRHSLOC (Rhs, 1).first_column;	\
	  (Current).last_line    = YYRHSLOC (Rhs, N).last_line;		\
	  (Current).last_column  = YYRHSLOC (Rhs, N).last_column;	\
	}								\
      else								\
	{								\
	  (Current).first_line   = (Current).last_line   =		\
	    YYRHSLOC (Rhs, 0).last_line;				\
	  (Current).first_column = (Current).last_column =		\
	    YYRHSLOC (Rhs, 0).last_column;				\
	}								\
    while (YYID (0))
#endif


/* YY_LOCATION_PRINT -- Print the location on the stream.
   This macro was not mandated originally: define only if we know
   we won't break user code: when these are the locations we know.  */

#ifndef YY_LOCATION_PRINT
# if YYLTYPE_IS_TRIVIAL
#  define YY_LOCATION_PRINT(File, Loc)			\
     fprintf (File, "%d.%d-%d.%d",			\
	      (Loc).first_line, (Loc).first_column,	\
	      (Loc).last_line,  (Loc).last_column)
# else
#  define YY_LOCATION_PRINT(File, Loc) ((void) 0)
# endif
#endif


/* YYLEX -- calling `yylex' with the right arguments.  */

#ifdef YYLEX_PARAM
# define YYLEX yylex (YYLEX_PARAM)
#else
# define YYLEX yylex ()
#endif

/* Enable debugging if requested.  */
#if YYDEBUG

# ifndef YYFPRINTF
#  include <stdio.h> /* INFRINGES ON USER NAME SPACE */
#  define YYFPRINTF fprintf
# endif

# define YYDPRINTF(Args)			\
do {						\
  if (yydebug)					\
    YYFPRINTF Args;				\
} while (YYID (0))

# define YY_SYMBOL_PRINT(Title, Type, Value, Location)			  \
do {									  \
  if (yydebug)								  \
    {									  \
      YYFPRINTF (stderr, "%s ", Title);					  \
      yy_symbol_print (stderr,						  \
		  Type, Value); \
      YYFPRINTF (stderr, "\n");						  \
    }									  \
} while (YYID (0))


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_value_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_value_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (!yyvaluep)
    return;
# ifdef YYPRINT
  if (yytype < YYNTOKENS)
    YYPRINT (yyoutput, yytoknum[yytype], *yyvaluep);
# else
  YYUSE (yyoutput);
# endif
  switch (yytype)
    {
      default:
	break;
    }
}


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (yytype < YYNTOKENS)
    YYFPRINTF (yyoutput, "token %s (", yytname[yytype]);
  else
    YYFPRINTF (yyoutput, "nterm %s (", yytname[yytype]);

  yy_symbol_value_print (yyoutput, yytype, yyvaluep);
  YYFPRINTF (yyoutput, ")");
}

/*------------------------------------------------------------------.
| yy_stack_print -- Print the state stack from its BOTTOM up to its |
| TOP (included).                                                   |
`------------------------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_stack_print (yytype_int16 *yybottom, yytype_int16 *yytop)
#else
static void
yy_stack_print (yybottom, yytop)
    yytype_int16 *yybottom;
    yytype_int16 *yytop;
#endif
{
  YYFPRINTF (stderr, "Stack now");
  for (; yybottom <= yytop; yybottom++)
    {
      int yybot = *yybottom;
      YYFPRINTF (stderr, " %d", yybot);
    }
  YYFPRINTF (stderr, "\n");
}

# define YY_STACK_PRINT(Bottom, Top)				\
do {								\
  if (yydebug)							\
    yy_stack_print ((Bottom), (Top));				\
} while (YYID (0))


/*------------------------------------------------.
| Report that the YYRULE is going to be reduced.  |
`------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_reduce_print (YYSTYPE *yyvsp, int yyrule)
#else
static void
yy_reduce_print (yyvsp, yyrule)
    YYSTYPE *yyvsp;
    int yyrule;
#endif
{
  int yynrhs = yyr2[yyrule];
  int yyi;
  unsigned long int yylno = yyrline[yyrule];
  YYFPRINTF (stderr, "Reducing stack by rule %d (line %lu):\n",
	     yyrule - 1, yylno);
  /* The symbols being reduced.  */
  for (yyi = 0; yyi < yynrhs; yyi++)
    {
      YYFPRINTF (stderr, "   $%d = ", yyi + 1);
      yy_symbol_print (stderr, yyrhs[yyprhs[yyrule] + yyi],
		       &(yyvsp[(yyi + 1) - (yynrhs)])
		       		       );
      YYFPRINTF (stderr, "\n");
    }
}

# define YY_REDUCE_PRINT(Rule)		\
do {					\
  if (yydebug)				\
    yy_reduce_print (yyvsp, Rule); \
} while (YYID (0))

/* Nonzero means print parse trace.  It is left uninitialized so that
   multiple parsers can coexist.  */
int yydebug;
#else /* !YYDEBUG */
# define YYDPRINTF(Args)
# define YY_SYMBOL_PRINT(Title, Type, Value, Location)
# define YY_STACK_PRINT(Bottom, Top)
# define YY_REDUCE_PRINT(Rule)
#endif /* !YYDEBUG */


/* YYINITDEPTH -- initial size of the parser's stacks.  */
#ifndef	YYINITDEPTH
# define YYINITDEPTH 200
#endif

/* YYMAXDEPTH -- maximum size the stacks can grow to (effective only
   if the built-in stack extension method is used).

   Do not make this value too large; the results are undefined if
   YYSTACK_ALLOC_MAXIMUM < YYSTACK_BYTES (YYMAXDEPTH)
   evaluated with infinite-precision integer arithmetic.  */

#ifndef YYMAXDEPTH
# define YYMAXDEPTH 10000
#endif



#if YYERROR_VERBOSE

# ifndef yystrlen
#  if defined __GLIBC__ && defined _STRING_H
#   define yystrlen strlen
#  else
/* Return the length of YYSTR.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static YYSIZE_T
yystrlen (const char *yystr)
#else
static YYSIZE_T
yystrlen (yystr)
    const char *yystr;
#endif
{
  YYSIZE_T yylen;
  for (yylen = 0; yystr[yylen]; yylen++)
    continue;
  return yylen;
}
#  endif
# endif

# ifndef yystpcpy
#  if defined __GLIBC__ && defined _STRING_H && defined _GNU_SOURCE
#   define yystpcpy stpcpy
#  else
/* Copy YYSRC to YYDEST, returning the address of the terminating '\0' in
   YYDEST.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static char *
yystpcpy (char *yydest, const char *yysrc)
#else
static char *
yystpcpy (yydest, yysrc)
    char *yydest;
    const char *yysrc;
#endif
{
  char *yyd = yydest;
  const char *yys = yysrc;

  while ((*yyd++ = *yys++) != '\0')
    continue;

  return yyd - 1;
}
#  endif
# endif

# ifndef yytnamerr
/* Copy to YYRES the contents of YYSTR after stripping away unnecessary
   quotes and backslashes, so that it's suitable for yyerror.  The
   heuristic is that double-quoting is unnecessary unless the string
   contains an apostrophe, a comma, or backslash (other than
   backslash-backslash).  YYSTR is taken from yytname.  If YYRES is
   null, do not copy; instead, return the length of what the result
   would have been.  */
static YYSIZE_T
yytnamerr (char *yyres, const char *yystr)
{
  if (*yystr == '"')
    {
      YYSIZE_T yyn = 0;
      char const *yyp = yystr;

      for (;;)
	switch (*++yyp)
	  {
	  case '\'':
	  case ',':
	    goto do_not_strip_quotes;

	  case '\\':
	    if (*++yyp != '\\')
	      goto do_not_strip_quotes;
	    /* Fall through.  */
	  default:
	    if (yyres)
	      yyres[yyn] = *yyp;
	    yyn++;
	    break;

	  case '"':
	    if (yyres)
	      yyres[yyn] = '\0';
	    return yyn;
	  }
    do_not_strip_quotes: ;
    }

  if (! yyres)
    return yystrlen (yystr);

  return yystpcpy (yyres, yystr) - yyres;
}
# endif

/* Copy into YYRESULT an error message about the unexpected token
   YYCHAR while in state YYSTATE.  Return the number of bytes copied,
   including the terminating null byte.  If YYRESULT is null, do not
   copy anything; just return the number of bytes that would be
   copied.  As a special case, return 0 if an ordinary "syntax error"
   message will do.  Return YYSIZE_MAXIMUM if overflow occurs during
   size calculation.  */
static YYSIZE_T
yysyntax_error (char *yyresult, int yystate, int yychar)
{
  int yyn = yypact[yystate];

  if (! (YYPACT_NINF < yyn && yyn <= YYLAST))
    return 0;
  else
    {
      int yytype = YYTRANSLATE (yychar);
      YYSIZE_T yysize0 = yytnamerr (0, yytname[yytype]);
      YYSIZE_T yysize = yysize0;
      YYSIZE_T yysize1;
      int yysize_overflow = 0;
      enum { YYERROR_VERBOSE_ARGS_MAXIMUM = 5 };
      char const *yyarg[YYERROR_VERBOSE_ARGS_MAXIMUM];
      int yyx;

# if 0
      /* This is so xgettext sees the translatable formats that are
	 constructed on the fly.  */
      YY_("syntax error, unexpected %s");
      YY_("syntax error, unexpected %s, expecting %s");
      YY_("syntax error, unexpected %s, expecting %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s or %s");
# endif
      char *yyfmt;
      char const *yyf;
      static char const yyunexpected[] = "syntax error, unexpected %s";
      static char const yyexpecting[] = ", expecting %s";
      static char const yyor[] = " or %s";
      char yyformat[sizeof yyunexpected
		    + sizeof yyexpecting - 1
		    + ((YYERROR_VERBOSE_ARGS_MAXIMUM - 2)
		       * (sizeof yyor - 1))];
      char const *yyprefix = yyexpecting;

      /* Start YYX at -YYN if negative to avoid negative indexes in
	 YYCHECK.  */
      int yyxbegin = yyn < 0 ? -yyn : 0;

      /* Stay within bounds of both yycheck and yytname.  */
      int yychecklim = YYLAST - yyn + 1;
      int yyxend = yychecklim < YYNTOKENS ? yychecklim : YYNTOKENS;
      int yycount = 1;

      yyarg[0] = yytname[yytype];
      yyfmt = yystpcpy (yyformat, yyunexpected);

      for (yyx = yyxbegin; yyx < yyxend; ++yyx)
	if (yycheck[yyx + yyn] == yyx && yyx != YYTERROR)
	  {
	    if (yycount == YYERROR_VERBOSE_ARGS_MAXIMUM)
	      {
		yycount = 1;
		yysize = yysize0;
		yyformat[sizeof yyunexpected - 1] = '\0';
		break;
	      }
	    yyarg[yycount++] = yytname[yyx];
	    yysize1 = yysize + yytnamerr (0, yytname[yyx]);
	    yysize_overflow |= (yysize1 < yysize);
	    yysize = yysize1;
	    yyfmt = yystpcpy (yyfmt, yyprefix);
	    yyprefix = yyor;
	  }

      yyf = YY_(yyformat);
      yysize1 = yysize + yystrlen (yyf);
      yysize_overflow |= (yysize1 < yysize);
      yysize = yysize1;

      if (yysize_overflow)
	return YYSIZE_MAXIMUM;

      if (yyresult)
	{
	  /* Avoid sprintf, as that infringes on the user's name space.
	     Don't have undefined behavior even if the translation
	     produced a string with the wrong number of "%s"s.  */
	  char *yyp = yyresult;
	  int yyi = 0;
	  while ((*yyp = *yyf) != '\0')
	    {
	      if (*yyp == '%' && yyf[1] == 's' && yyi < yycount)
		{
		  yyp += yytnamerr (yyp, yyarg[yyi++]);
		  yyf += 2;
		}
	      else
		{
		  yyp++;
		  yyf++;
		}
	    }
	}
      return yysize;
    }
}
#endif /* YYERROR_VERBOSE */


/*-----------------------------------------------.
| Release the memory associated to this symbol.  |
`-----------------------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yydestruct (const char *yymsg, int yytype, YYSTYPE *yyvaluep)
#else
static void
yydestruct (yymsg, yytype, yyvaluep)
    const char *yymsg;
    int yytype;
    YYSTYPE *yyvaluep;
#endif
{
  YYUSE (yyvaluep);

  if (!yymsg)
    yymsg = "Deleting";
  YY_SYMBOL_PRINT (yymsg, yytype, yyvaluep, yylocationp);

  switch (yytype)
    {

      default:
	break;
    }
}

/* Prevent warnings from -Wmissing-prototypes.  */
#ifdef YYPARSE_PARAM
#if defined __STDC__ || defined __cplusplus
int yyparse (void *YYPARSE_PARAM);
#else
int yyparse ();
#endif
#else /* ! YYPARSE_PARAM */
#if defined __STDC__ || defined __cplusplus
int yyparse (void);
#else
int yyparse ();
#endif
#endif /* ! YYPARSE_PARAM */


/* The lookahead symbol.  */
int yychar;

/* The semantic value of the lookahead symbol.  */
YYSTYPE yylval;

/* Number of syntax errors so far.  */
int yynerrs;



/*-------------------------.
| yyparse or yypush_parse.  |
`-------------------------*/

#ifdef YYPARSE_PARAM
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void *YYPARSE_PARAM)
#else
int
yyparse (YYPARSE_PARAM)
    void *YYPARSE_PARAM;
#endif
#else /* ! YYPARSE_PARAM */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void)
#else
int
yyparse ()

#endif
#endif
{


    int yystate;
    /* Number of tokens to shift before error messages enabled.  */
    int yyerrstatus;

    /* The stacks and their tools:
       `yyss': related to states.
       `yyvs': related to semantic values.

       Refer to the stacks thru separate pointers, to allow yyoverflow
       to reallocate them elsewhere.  */

    /* The state stack.  */
    yytype_int16 yyssa[YYINITDEPTH];
    yytype_int16 *yyss;
    yytype_int16 *yyssp;

    /* The semantic value stack.  */
    YYSTYPE yyvsa[YYINITDEPTH];
    YYSTYPE *yyvs;
    YYSTYPE *yyvsp;

    YYSIZE_T yystacksize;

  int yyn;
  int yyresult;
  /* Lookahead token as an internal (translated) token number.  */
  int yytoken;
  /* The variables used to return semantic value and location from the
     action routines.  */
  YYSTYPE yyval;

#if YYERROR_VERBOSE
  /* Buffer for error messages, and its allocated size.  */
  char yymsgbuf[128];
  char *yymsg = yymsgbuf;
  YYSIZE_T yymsg_alloc = sizeof yymsgbuf;
#endif

#define YYPOPSTACK(N)   (yyvsp -= (N), yyssp -= (N))

  /* The number of symbols on the RHS of the reduced rule.
     Keep to zero when no symbol should be popped.  */
  int yylen = 0;

  yytoken = 0;
  yyss = yyssa;
  yyvs = yyvsa;
  yystacksize = YYINITDEPTH;

  YYDPRINTF ((stderr, "Starting parse\n"));

  yystate = 0;
  yyerrstatus = 0;
  yynerrs = 0;
  yychar = YYEMPTY; /* Cause a token to be read.  */

  /* Initialize stack pointers.
     Waste one element of value and location stack
     so that they stay on the same level as the state stack.
     The wasted elements are never initialized.  */
  yyssp = yyss;
  yyvsp = yyvs;

  goto yysetstate;

/*------------------------------------------------------------.
| yynewstate -- Push a new state, which is found in yystate.  |
`------------------------------------------------------------*/
 yynewstate:
  /* In all cases, when you get here, the value and location stacks
     have just been pushed.  So pushing a state here evens the stacks.  */
  yyssp++;

 yysetstate:
  *yyssp = yystate;

  if (yyss + yystacksize - 1 <= yyssp)
    {
      /* Get the current used size of the three stacks, in elements.  */
      YYSIZE_T yysize = yyssp - yyss + 1;

#ifdef yyoverflow
      {
	/* Give user a chance to reallocate the stack.  Use copies of
	   these so that the &'s don't force the real ones into
	   memory.  */
	YYSTYPE *yyvs1 = yyvs;
	yytype_int16 *yyss1 = yyss;

	/* Each stack pointer address is followed by the size of the
	   data in use in that stack, in bytes.  This used to be a
	   conditional around just the two extra args, but that might
	   be undefined if yyoverflow is a macro.  */
	yyoverflow (YY_("memory exhausted"),
		    &yyss1, yysize * sizeof (*yyssp),
		    &yyvs1, yysize * sizeof (*yyvsp),
		    &yystacksize);

	yyss = yyss1;
	yyvs = yyvs1;
      }
#else /* no yyoverflow */
# ifndef YYSTACK_RELOCATE
      goto yyexhaustedlab;
# else
      /* Extend the stack our own way.  */
      if (YYMAXDEPTH <= yystacksize)
	goto yyexhaustedlab;
      yystacksize *= 2;
      if (YYMAXDEPTH < yystacksize)
	yystacksize = YYMAXDEPTH;

      {
	yytype_int16 *yyss1 = yyss;
	union yyalloc *yyptr =
	  (union yyalloc *) YYSTACK_ALLOC (YYSTACK_BYTES (yystacksize));
	if (! yyptr)
	  goto yyexhaustedlab;
	YYSTACK_RELOCATE (yyss_alloc, yyss);
	YYSTACK_RELOCATE (yyvs_alloc, yyvs);
#  undef YYSTACK_RELOCATE
	if (yyss1 != yyssa)
	  YYSTACK_FREE (yyss1);
      }
# endif
#endif /* no yyoverflow */

      yyssp = yyss + yysize - 1;
      yyvsp = yyvs + yysize - 1;

      YYDPRINTF ((stderr, "Stack size increased to %lu\n",
		  (unsigned long int) yystacksize));

      if (yyss + yystacksize - 1 <= yyssp)
	YYABORT;
    }

  YYDPRINTF ((stderr, "Entering state %d\n", yystate));

  if (yystate == YYFINAL)
    YYACCEPT;

  goto yybackup;

/*-----------.
| yybackup.  |
`-----------*/
yybackup:

  /* Do appropriate processing given the current state.  Read a
     lookahead token if we need one and don't already have one.  */

  /* First try to decide what to do without reference to lookahead token.  */
  yyn = yypact[yystate];
  if (yyn == YYPACT_NINF)
    goto yydefault;

  /* Not known => get a lookahead token if don't already have one.  */

  /* YYCHAR is either YYEMPTY or YYEOF or a valid lookahead symbol.  */
  if (yychar == YYEMPTY)
    {
      YYDPRINTF ((stderr, "Reading a token: "));
      yychar = YYLEX;
    }

  if (yychar <= YYEOF)
    {
      yychar = yytoken = YYEOF;
      YYDPRINTF ((stderr, "Now at end of input.\n"));
    }
  else
    {
      yytoken = YYTRANSLATE (yychar);
      YY_SYMBOL_PRINT ("Next token is", yytoken, &yylval, &yylloc);
    }

  /* If the proper action on seeing token YYTOKEN is to reduce or to
     detect an error, take that action.  */
  yyn += yytoken;
  if (yyn < 0 || YYLAST < yyn || yycheck[yyn] != yytoken)
    goto yydefault;
  yyn = yytable[yyn];
  if (yyn <= 0)
    {
      if (yyn == 0 || yyn == YYTABLE_NINF)
	goto yyerrlab;
      yyn = -yyn;
      goto yyreduce;
    }

  /* Count tokens shifted since error; after three, turn off error
     status.  */
  if (yyerrstatus)
    yyerrstatus--;

  /* Shift the lookahead token.  */
  YY_SYMBOL_PRINT ("Shifting", yytoken, &yylval, &yylloc);

  /* Discard the shifted token.  */
  yychar = YYEMPTY;

  yystate = yyn;
  *++yyvsp = yylval;

  goto yynewstate;


/*-----------------------------------------------------------.
| yydefault -- do the default action for the current state.  |
`-----------------------------------------------------------*/
yydefault:
  yyn = yydefact[yystate];
  if (yyn == 0)
    goto yyerrlab;
  goto yyreduce;


/*-----------------------------.
| yyreduce -- Do a reduction.  |
`-----------------------------*/
yyreduce:
  /* yyn is the number of a rule to reduce with.  */
  yylen = yyr2[yyn];

  /* If YYLEN is nonzero, implement the default value of the action:
     `$$ = $1'.

     Otherwise, the following line sets YYVAL to garbage.
     This behavior is undocumented and Bison
     users should not rely upon it.  Assigning to YYVAL
     unconditionally makes the parser a bit smaller, and it avoids a
     GCC warning that YYVAL may be used uninitialized.  */
  yyval = yyvsp[1-yylen];


  YY_REDUCE_PRINT (yyn);
  switch (yyn)
    {
        case 4:

/* Line 1455 of yacc.c  */
#line 129 "bison.y"
    { emit("STMT"); ;}
    break;

  case 5:

/* Line 1455 of yacc.c  */
#line 133 "bison.y"
    { emit_select((yyvsp[(1) - (7)].strval), (yyvsp[(6) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 6:

/* Line 1455 of yacc.c  */
#line 135 "bison.y"
    {  emit_load((yyvsp[(1) - (12)].strval), (yyvsp[(4) - (12)].strval), (yyvsp[(11) - (12)].intval), (yyvsp[(7) - (12)].strval)); ;}
    break;

  case 7:

/* Line 1455 of yacc.c  */
#line 137 "bison.y"
    {  emit_filter((yyvsp[(1) - (5)].strval), (yyvsp[(4) - (5)].strval));;}
    break;

  case 8:

/* Line 1455 of yacc.c  */
#line 139 "bison.y"
    {  emit_order((yyvsp[(1) - (6)].strval), (yyvsp[(4) - (6)].strval), (yyvsp[(6) - (6)].intval));;}
    break;

  case 9:

/* Line 1455 of yacc.c  */
#line 141 "bison.y"
    {  emit_join((yyvsp[(1) - (8)].strval),(yyvsp[(6) - (8)].strval),(yyvsp[(7) - (8)].intval),0,-1); ;}
    break;

  case 10:

/* Line 1455 of yacc.c  */
#line 143 "bison.y"
    {  emit_store((yyvsp[(2) - (9)].strval),(yyvsp[(4) - (9)].strval),(yyvsp[(7) - (9)].strval)); ;}
    break;

  case 11:

/* Line 1455 of yacc.c  */
#line 145 "bison.y"
    {  emit_store_binary((yyvsp[(2) - (7)].strval),(yyvsp[(4) - (7)].strval),0); ;}
    break;

  case 12:

/* Line 1455 of yacc.c  */
#line 147 "bison.y"
    {  emit_store_binary((yyvsp[(2) - (8)].strval),(yyvsp[(4) - (8)].strval),1); ;}
    break;

  case 13:

/* Line 1455 of yacc.c  */
#line 149 "bison.y"
    {  emit_describe_table((yyvsp[(2) - (2)].strval));;}
    break;

  case 14:

/* Line 1455 of yacc.c  */
#line 151 "bison.y"
    {  emit_insert((yyvsp[(3) - (7)].strval), (yyvsp[(7) - (7)].strval));;}
    break;

  case 15:

/* Line 1455 of yacc.c  */
#line 153 "bison.y"
    {  emit_delete((yyvsp[(3) - (5)].strval));;}
    break;

  case 16:

/* Line 1455 of yacc.c  */
#line 155 "bison.y"
    {  emit_display((yyvsp[(2) - (7)].strval), (yyvsp[(5) - (7)].strval));;}
    break;

  case 17:

/* Line 1455 of yacc.c  */
#line 157 "bison.y"
    {  emit_show_tables();;}
    break;

  case 18:

/* Line 1455 of yacc.c  */
#line 159 "bison.y"
    {  emit_drop_table((yyvsp[(3) - (3)].strval));;}
    break;

  case 19:

/* Line 1455 of yacc.c  */
#line 161 "bison.y"
    {  emit_create_bitmap_index((yyvsp[(3) - (22)].strval), (yyvsp[(5) - (22)].strval), (yyvsp[(7) - (22)].strval), (yyvsp[(9) - (22)].strval), (yyvsp[(18) - (22)].strval), (yyvsp[(22) - (22)].strval));;}
    break;

  case 20:

/* Line 1455 of yacc.c  */
#line 163 "bison.y"
    {  emit_create_index((yyvsp[(3) - (8)].strval), (yyvsp[(5) - (8)].strval), (yyvsp[(7) - (8)].strval));;}
    break;

  case 21:

/* Line 1455 of yacc.c  */
#line 165 "bison.y"
    {  emit_create_interval((yyvsp[(3) - (10)].strval), (yyvsp[(5) - (10)].strval), (yyvsp[(7) - (10)].strval), (yyvsp[(9) - (10)].strval));;}
    break;

  case 22:

/* Line 1455 of yacc.c  */
#line 170 "bison.y"
    { emit_name((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 23:

/* Line 1455 of yacc.c  */
#line 171 "bison.y"
    { emit_fieldname((yyvsp[(1) - (3)].strval), (yyvsp[(3) - (3)].strval)); ;}
    break;

  case 24:

/* Line 1455 of yacc.c  */
#line 172 "bison.y"
    { emit("USERVAR %s", (yyvsp[(1) - (1)].strval)); ;}
    break;

  case 25:

/* Line 1455 of yacc.c  */
#line 173 "bison.y"
    { emit_string((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 26:

/* Line 1455 of yacc.c  */
#line 174 "bison.y"
    { emit_number((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 27:

/* Line 1455 of yacc.c  */
#line 175 "bison.y"
    { emit_decimal((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 28:

/* Line 1455 of yacc.c  */
#line 176 "bison.y"
    { emit_float((yyvsp[(1) - (1)].floatval)); ;}
    break;

  case 29:

/* Line 1455 of yacc.c  */
#line 177 "bison.y"
    { emit("BOOL %d", (yyvsp[(1) - (1)].intval)); ;}
    break;

  case 30:

/* Line 1455 of yacc.c  */
#line 178 "bison.y"
    { emit_vardecimal((yyvsp[(1) - (11)].strval), (yyvsp[(3) - (11)].intval), (yyvsp[(6) - (11)].strval),  (yyvsp[(8) - (11)].intval), (yyvsp[(10) - (11)].intval));;}
    break;

  case 31:

/* Line 1455 of yacc.c  */
#line 179 "bison.y"
    { emit_varchar((yyvsp[(1) - (9)].strval), (yyvsp[(3) - (9)].intval), (yyvsp[(6) - (9)].strval), (yyvsp[(8) - (9)].intval), "", "");;}
    break;

  case 32:

/* Line 1455 of yacc.c  */
#line 180 "bison.y"
    { emit_var((yyvsp[(1) - (6)].strval), (yyvsp[(3) - (6)].intval), (yyvsp[(6) - (6)].strval), "", "");;}
    break;

  case 33:

/* Line 1455 of yacc.c  */
#line 181 "bison.y"
    { emit_var_asc((yyvsp[(1) - (2)].strval));;}
    break;

  case 34:

/* Line 1455 of yacc.c  */
#line 182 "bison.y"
    { emit_var_desc((yyvsp[(1) - (2)].strval));;}
    break;

  case 35:

/* Line 1455 of yacc.c  */
#line 183 "bison.y"
    { emit_count(); ;}
    break;

  case 36:

/* Line 1455 of yacc.c  */
#line 184 "bison.y"
    { emit_sum(); ;}
    break;

  case 37:

/* Line 1455 of yacc.c  */
#line 185 "bison.y"
    { emit_average(); ;}
    break;

  case 38:

/* Line 1455 of yacc.c  */
#line 186 "bison.y"
    { emit_min(); ;}
    break;

  case 39:

/* Line 1455 of yacc.c  */
#line 187 "bison.y"
    { emit_max(); ;}
    break;

  case 40:

/* Line 1455 of yacc.c  */
#line 188 "bison.y"
    { emit_distinct(); ;}
    break;

  case 41:

/* Line 1455 of yacc.c  */
#line 189 "bison.y"
    { emit_year(); ;}
    break;

  case 42:

/* Line 1455 of yacc.c  */
#line 193 "bison.y"
    { emit_add(); ;}
    break;

  case 43:

/* Line 1455 of yacc.c  */
#line 194 "bison.y"
    { emit_minus(); ;}
    break;

  case 44:

/* Line 1455 of yacc.c  */
#line 195 "bison.y"
    { emit_mul(); ;}
    break;

  case 45:

/* Line 1455 of yacc.c  */
#line 196 "bison.y"
    { emit_div(); ;}
    break;

  case 46:

/* Line 1455 of yacc.c  */
#line 197 "bison.y"
    { emit("MOD"); ;}
    break;

  case 47:

/* Line 1455 of yacc.c  */
#line 198 "bison.y"
    { emit("MOD"); ;}
    break;

  case 48:

/* Line 1455 of yacc.c  */
#line 199 "bison.y"
    { emit_and(); ;}
    break;

  case 49:

/* Line 1455 of yacc.c  */
#line 200 "bison.y"
    { emit_eq(); ;}
    break;

  case 50:

/* Line 1455 of yacc.c  */
#line 201 "bison.y"
    { emit_neq(); ;}
    break;

  case 51:

/* Line 1455 of yacc.c  */
#line 202 "bison.y"
    { emit_or(); ;}
    break;

  case 52:

/* Line 1455 of yacc.c  */
#line 203 "bison.y"
    { emit("XOR"); ;}
    break;

  case 53:

/* Line 1455 of yacc.c  */
#line 204 "bison.y"
    { emit("SHIFT %s", (yyvsp[(2) - (3)].subtok)==1?"left":"right"); ;}
    break;

  case 54:

/* Line 1455 of yacc.c  */
#line 205 "bison.y"
    { emit("NOT"); ;}
    break;

  case 55:

/* Line 1455 of yacc.c  */
#line 206 "bison.y"
    { emit("NOT"); ;}
    break;

  case 56:

/* Line 1455 of yacc.c  */
#line 207 "bison.y"
    { emit_cmp((yyvsp[(2) - (3)].subtok)); ;}
    break;

  case 57:

/* Line 1455 of yacc.c  */
#line 208 "bison.y"
    { emit_cmp(7); ;}
    break;

  case 58:

/* Line 1455 of yacc.c  */
#line 210 "bison.y"
    { emit("CMPSELECT %d", (yyvsp[(2) - (5)].subtok)); ;}
    break;

  case 59:

/* Line 1455 of yacc.c  */
#line 211 "bison.y"
    {emit("EXPR");;}
    break;

  case 60:

/* Line 1455 of yacc.c  */
#line 212 "bison.y"
    { emit_case(); ;}
    break;

  case 61:

/* Line 1455 of yacc.c  */
#line 216 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(3) - (3)].intval)); ;}
    break;

  case 62:

/* Line 1455 of yacc.c  */
#line 217 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(4) - (4)].intval)); emit("NOT"); ;}
    break;

  case 63:

/* Line 1455 of yacc.c  */
#line 220 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 64:

/* Line 1455 of yacc.c  */
#line 223 "bison.y"
    { (yyval.intval) = (yyvsp[(3) - (3)].intval);;}
    break;

  case 65:

/* Line 1455 of yacc.c  */
#line 227 "bison.y"
    { (yyval.intval) = 1; emit_sel_name((yyvsp[(3) - (3)].strval));;}
    break;

  case 66:

/* Line 1455 of yacc.c  */
#line 228 "bison.y"
    { (yyval.intval) = (yyvsp[(1) - (5)].intval) + 1; emit_sel_name((yyvsp[(5) - (5)].strval));;}
    break;

  case 67:

/* Line 1455 of yacc.c  */
#line 229 "bison.y"
    { emit_sel_name("*");;}
    break;

  case 68:

/* Line 1455 of yacc.c  */
#line 233 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 69:

/* Line 1455 of yacc.c  */
#line 234 "bison.y"
    {(yyval.intval) = (yyvsp[(1) - (3)].intval) + 1; ;}
    break;

  case 70:

/* Line 1455 of yacc.c  */
#line 238 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 71:

/* Line 1455 of yacc.c  */
#line 239 "bison.y"
    { (yyval.intval) = 1 + (yyvsp[(3) - (3)].intval); ;}
    break;

  case 72:

/* Line 1455 of yacc.c  */
#line 242 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 74:

/* Line 1455 of yacc.c  */
#line 247 "bison.y"
    { emit("FILTER BY"); ;}
    break;

  case 75:

/* Line 1455 of yacc.c  */
#line 251 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (4)].strval), 'I');;}
    break;

  case 76:

/* Line 1455 of yacc.c  */
#line 252 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (6)].strval), '3');;}
    break;

  case 77:

/* Line 1455 of yacc.c  */
#line 253 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (6)].strval), '4');;}
    break;

  case 78:

/* Line 1455 of yacc.c  */
#line 254 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (6)].strval), '1');;}
    break;

  case 79:

/* Line 1455 of yacc.c  */
#line 255 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'S');;}
    break;

  case 80:

/* Line 1455 of yacc.c  */
#line 256 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'R');;}
    break;

  case 81:

/* Line 1455 of yacc.c  */
#line 257 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (6)].strval), '2');;}
    break;

  case 82:

/* Line 1455 of yacc.c  */
#line 258 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'O');;}
    break;

  case 83:

/* Line 1455 of yacc.c  */
#line 259 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (5)].strval), 'I'); ;}
    break;

  case 84:

/* Line 1455 of yacc.c  */
#line 260 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (7)].strval), '3'); ;}
    break;

  case 85:

/* Line 1455 of yacc.c  */
#line 261 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (7)].strval), '4'); ;}
    break;

  case 86:

/* Line 1455 of yacc.c  */
#line 262 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'L'); ;}
    break;

  case 87:

/* Line 1455 of yacc.c  */
#line 263 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (7)].strval), '1'); ;}
    break;

  case 88:

/* Line 1455 of yacc.c  */
#line 264 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'R'); ;}
    break;

  case 89:

/* Line 1455 of yacc.c  */
#line 265 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(4) - (7)].strval), 'R'); ;}
    break;

  case 90:

/* Line 1455 of yacc.c  */
#line 266 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'O'); ;}
    break;

  case 91:

/* Line 1455 of yacc.c  */
#line 268 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 92:

/* Line 1455 of yacc.c  */
#line 271 "bison.y"
    { emit_limit((yyvsp[(2) - (2)].intval)); ;}
    break;

  case 93:

/* Line 1455 of yacc.c  */
#line 273 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 94:

/* Line 1455 of yacc.c  */
#line 276 "bison.y"
    { emit_sort((yyvsp[(4) - (4)].strval), 0); ;}
    break;

  case 95:

/* Line 1455 of yacc.c  */
#line 277 "bison.y"
    { emit_sort((yyvsp[(4) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 96:

/* Line 1455 of yacc.c  */
#line 278 "bison.y"
    { emit_presort((yyvsp[(3) - (3)].strval)); ;}
    break;



/* Line 1455 of yacc.c  */
#line 2413 "bison.cu"
      default: break;
    }
  YY_SYMBOL_PRINT ("-> $$ =", yyr1[yyn], &yyval, &yyloc);

  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);

  *++yyvsp = yyval;

  /* Now `shift' the result of the reduction.  Determine what state
     that goes to, based on the state we popped back to and the rule
     number reduced by.  */

  yyn = yyr1[yyn];

  yystate = yypgoto[yyn - YYNTOKENS] + *yyssp;
  if (0 <= yystate && yystate <= YYLAST && yycheck[yystate] == *yyssp)
    yystate = yytable[yystate];
  else
    yystate = yydefgoto[yyn - YYNTOKENS];

  goto yynewstate;


/*------------------------------------.
| yyerrlab -- here on detecting error |
`------------------------------------*/
yyerrlab:
  /* If not already recovering from an error, report this error.  */
  if (!yyerrstatus)
    {
      ++yynerrs;
#if ! YYERROR_VERBOSE
      yyerror (YY_("syntax error"));
#else
      {
	YYSIZE_T yysize = yysyntax_error (0, yystate, yychar);
	if (yymsg_alloc < yysize && yymsg_alloc < YYSTACK_ALLOC_MAXIMUM)
	  {
	    YYSIZE_T yyalloc = 2 * yysize;
	    if (! (yysize <= yyalloc && yyalloc <= YYSTACK_ALLOC_MAXIMUM))
	      yyalloc = YYSTACK_ALLOC_MAXIMUM;
	    if (yymsg != yymsgbuf)
	      YYSTACK_FREE (yymsg);
	    yymsg = (char *) YYSTACK_ALLOC (yyalloc);
	    if (yymsg)
	      yymsg_alloc = yyalloc;
	    else
	      {
		yymsg = yymsgbuf;
		yymsg_alloc = sizeof yymsgbuf;
	      }
	  }

	if (0 < yysize && yysize <= yymsg_alloc)
	  {
	    (void) yysyntax_error (yymsg, yystate, yychar);
	    yyerror (yymsg);
	  }
	else
	  {
	    yyerror (YY_("syntax error"));
	    if (yysize != 0)
	      goto yyexhaustedlab;
	  }
      }
#endif
    }



  if (yyerrstatus == 3)
    {
      /* If just tried and failed to reuse lookahead token after an
	 error, discard it.  */

      if (yychar <= YYEOF)
	{
	  /* Return failure if at end of input.  */
	  if (yychar == YYEOF)
	    YYABORT;
	}
      else
	{
	  yydestruct ("Error: discarding",
		      yytoken, &yylval);
	  yychar = YYEMPTY;
	}
    }

  /* Else will try to reuse lookahead token after shifting the error
     token.  */
  goto yyerrlab1;


/*---------------------------------------------------.
| yyerrorlab -- error raised explicitly by YYERROR.  |
`---------------------------------------------------*/
yyerrorlab:

  /* Pacify compilers like GCC when the user code never invokes
     YYERROR and the label yyerrorlab therefore never appears in user
     code.  */
  if (/*CONSTCOND*/ 0)
     goto yyerrorlab;

  /* Do not reclaim the symbols of the rule which action triggered
     this YYERROR.  */
  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);
  yystate = *yyssp;
  goto yyerrlab1;


/*-------------------------------------------------------------.
| yyerrlab1 -- common code for both syntax error and YYERROR.  |
`-------------------------------------------------------------*/
yyerrlab1:
  yyerrstatus = 3;	/* Each real token shifted decrements this.  */

  for (;;)
    {
      yyn = yypact[yystate];
      if (yyn != YYPACT_NINF)
	{
	  yyn += YYTERROR;
	  if (0 <= yyn && yyn <= YYLAST && yycheck[yyn] == YYTERROR)
	    {
	      yyn = yytable[yyn];
	      if (0 < yyn)
		break;
	    }
	}

      /* Pop the current state because it cannot handle the error token.  */
      if (yyssp == yyss)
	YYABORT;


      yydestruct ("Error: popping",
		  yystos[yystate], yyvsp);
      YYPOPSTACK (1);
      yystate = *yyssp;
      YY_STACK_PRINT (yyss, yyssp);
    }

  *++yyvsp = yylval;


  /* Shift the error token.  */
  YY_SYMBOL_PRINT ("Shifting", yystos[yyn], yyvsp, yylsp);

  yystate = yyn;
  goto yynewstate;


/*-------------------------------------.
| yyacceptlab -- YYACCEPT comes here.  |
`-------------------------------------*/
yyacceptlab:
  yyresult = 0;
  goto yyreturn;

/*-----------------------------------.
| yyabortlab -- YYABORT comes here.  |
`-----------------------------------*/
yyabortlab:
  yyresult = 1;
  goto yyreturn;

#if !defined(yyoverflow) || YYERROR_VERBOSE
/*-------------------------------------------------.
| yyexhaustedlab -- memory exhaustion comes here.  |
`-------------------------------------------------*/
yyexhaustedlab:
  yyerror (YY_("memory exhausted"));
  yyresult = 2;
  /* Fall through.  */
#endif

yyreturn:
  if (yychar != YYEMPTY)
     yydestruct ("Cleanup: discarding lookahead",
		 yytoken, &yylval);
  /* Do not reclaim the symbols of the rule which action triggered
     this YYABORT or YYACCEPT.  */
  YYPOPSTACK (yylen);
  YY_STACK_PRINT (yyss, yyssp);
  while (yyssp != yyss)
    {
      yydestruct ("Cleanup: popping",
		  yystos[*yyssp], yyvsp);
      YYPOPSTACK (1);
    }
#ifndef yyoverflow
  if (yyss != yyssa)
    YYSTACK_FREE (yyss);
#endif
#if YYERROR_VERBOSE
  if (yymsg != yymsgbuf)
    YYSTACK_FREE (yymsg);
#endif
  /* Make sure YYID is used.  */
  return YYID (yyresult);
}



/* Line 1675 of yacc.c  */
#line 280 "bison.y"


bool scan_state;
unsigned int statement_count;

int execute_file(int ac, char **av)
{
    bool just_once  = 0;
    string script;
    process_count = 1000000000; //1GB by default
    verbose = 0;
	ssd = 0;
	delta = 0;
    total_buffer_size = 0;
	hash_seed = 100;

    for (int i = 1; i < ac; i++) {
        if(strcmp(av[i],"-l") == 0) {
            process_count = 1000000*atoff(av[i+1]);
        }
        else if(strcmp(av[i],"-v") == 0) {
            verbose = 1;
        }
        else if(strcmp(av[i],"-delta") == 0) {
            delta = 1;
        }		
        else if(strcmp(av[i],"-ssd") == 0) {
            ssd = 1;
        }		
        else if(strcmp(av[i],"-i") == 0) {
            interactive = 1;
            break;
        }
        else if(strcmp(av[i],"-s") == 0) {
            just_once = 1;
            interactive = 1;
            script = av[i+1];
        };
    };

    load_col_data(data_dict, "data.dictionary");
	tot_disk = 0;

    if (!interactive) {
        if((yyin = fopen(av[ac-1], "r")) == nullptr) {
            perror(av[ac-1]);
            exit(1);
        };

        if(yyparse()) {
            printf("SQL scan parse failed\n");
            exit(1);
        };

        scan_state = 1;
        std::clock_t start1 = std::clock();

        load_vars();
		
        statement_count = 0;
        clean_queues();
		filter_var.clear();

        yyin = fopen(av[ac-1], "r");
        PROC_FLUSH_BUF ( yyin );
        statement_count = 0;

        extern FILE *yyin;
        context = CreateCudaDevice(0, nullptr, verbose);

        if(!yyparse()) {
            if(verbose)
                cout << "SQL scan parse worked " << endl;
        }
        else
            cout << "SQL scan parse failed" << endl;

        fclose(yyin);
        for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
            (*it).second->free();
        };

        if(verbose) {
            cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
			cout<< "disk time " << ( tot_disk / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
        };
    }
    else {
        context = CreateCudaDevice(0, nullptr, verbose);        
        if(!just_once)
            getline(cin, script);

        while (script != "exit" && script != "EXIT") {

            used_vars.clear();
            yy_scan_string(script.c_str());
            scan_state = 0;
            statement_count = 0;
            clean_queues();
            if(yyparse()) {
                printf("SQL scan parse failed \n");
                getline(cin, script);
                continue;
            };

            scan_state = 1;

            load_vars();

            statement_count = 0;
            clean_queues();
			filter_var.clear();
            yy_scan_string(script.c_str());
            std::clock_t start1 = std::clock();

            if(!yyparse()) {
                if(verbose)
                    cout << "SQL scan parse worked " <<  endl;
            };
            for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
                (*it).second->free();
            };
            varNames.clear();

            if(verbose) {
                cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << endl;
            };
            if(!just_once)
                getline(cin, script);
            else
                script = "exit";
        };

        while(!buffer_names.empty()) {
            //delete [] buffers[buffer_names.front()];
			hipHostFree(buffers[buffer_names.front()]);
            buffer_sizes.erase(buffer_names.front());
            buffers.erase(buffer_names.front());
            buffer_names.pop();
        };
		for(auto it = index_buffers.begin(); it != index_buffers.end();it++) {
			hipHostFree(it->second);
        };

    };
    if(save_dict) {
        save_col_data(data_dict,"data.dictionary");
	};	

    if(alloced_sz) {
        hipFree(alloced_tmp);
        alloced_sz = 0;
    };
	if(scratch.size()) {
		scratch.resize(0);
		scratch.shrink_to_fit();
	};	
	if(ranj.size()) {
		ranj.resize(0);
		ranj.shrink_to_fit();
	};	
    return 0;
}



//external c global to report errors
//char alenka_err[4048];


int alenkaExecute(char *s)
{
    YY_BUFFER_STATE bp;

    total_buffer_size = 0;
    scan_state = 0;
    load_col_data(data_dict, "data.dictionary");
    std::clock_t start;

    if(verbose)
        start = std::clock();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    int ret = yyparse();
    //printf("execute: returned [%d]\n", ret);
    if(!ret) {
        if(verbose)
            cout << "SQL scan parse worked" << endl;
    }

    scan_state = 1;
    load_vars();
    statement_count = 0;
    clean_queues();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    if(!yyparse()) {
        if(verbose)
            cout << "SQL scan parse worked " << endl;
    }
    else
        cout << "SQL scan parse failed" << endl;

    yy_delete_buffer(bp);

    // Clear Vars
    for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
        (*it).second->free();
    };
    varNames.clear();

    if(verbose)
        cout<< "statement time " <<  ( ( std::clock() - start ) / (double)CLOCKS_PER_SEC ) << endl;
    if(save_dict)
        save_col_data(data_dict,"data.dictionary");
    return ret;
}



