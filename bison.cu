#include "hip/hip_runtime.h"

/* A Bison parser, made by GNU Bison 2.4.1.  */

/* Skeleton implementation for Bison's Yacc-like parsers in C
   
      Copyright (C) 1984, 1989, 1990, 2000, 2001, 2002, 2003, 2004, 2005, 2006
   Free Software Foundation, Inc.
   
   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.  */

/* As a special exception, you may create a larger work that contains
   part or all of the Bison parser skeleton and distribute that work
   under terms of your choice, so long as that work isn't itself a
   parser generator using the skeleton or a modified version thereof
   as a parser skeleton.  Alternatively, if you modify or redistribute
   the parser skeleton itself, you may (at your option) remove this
   special exception, which will cause the skeleton and the resulting
   Bison output files to be licensed under the GNU General Public
   License without this special exception.
   
   This special exception was added by the Free Software Foundation in
   version 2.2 of Bison.  */

/* C LALR(1) parser skeleton written by Richard Stallman, by
   simplifying the original so-called "semantic" parser.  */

/* All symbols defined below should begin with yy or YY, to avoid
   infringing on user name space.  This should be done even for local
   variables, as they might otherwise be expanded by user macros.
   There are some unavoidable exceptions within include files to
   define necessary library symbols; they are noted "INFRINGES ON
   USER NAME SPACE" below.  */

/* Identify Bison output.  */
#define YYBISON 1

/* Bison version.  */
#define YYBISON_VERSION "2.4.1"

/* Skeleton name.  */
#define YYSKELETON_NAME "yacc.c"

/* Pure parsers.  */
#define YYPURE 0

/* Push parsers.  */
#define YYPUSH 0

/* Pull parsers.  */
#define YYPULL 1

/* Using locations.  */
#define YYLSP_NEEDED 0



/* Copy the first part of user declarations.  */

/* Line 189 of yacc.c  */
#line 17 "bison.y"


#include "lex.yy.c"
#include "cm.h"

    void clean_queues();
    void order_inplace(CudaSet* a, stack<string> exe_type);
    void yyerror(char *s, ...);
    void emit(char *s, ...);
    void emit_mul();
    void emit_add();
    void emit_minus();
    void emit_distinct();
    void emit_div();
    void emit_and();
    void emit_eq();
    void emit_or();
    void emit_cmp(int val);
    void emit_var(char *s, int c, char *f);
    void emit_var_asc(char *s);
    void emit_var_desc(char *s);
    void emit_name(char *name);
    void emit_count();
    void emit_sum();
    void emit_average();
    void emit_min();
    void emit_max();
    void emit_string(char *str);
    void emit_number(int_type val);
    void emit_float(float_type val);
    void emit_decimal(float_type val);
    void emit_sel_name(char* name);
    void emit_limit(int val);
    void emit_union(char *s, char *f1, char *f2);
    void emit_varchar(char *s, int c, char *f, int d);
    void emit_load(char *s, char *f, int d, char* sep);
    void emit_load_binary(char *s, char *f, int d);
    void emit_store(char *s, char *f, char* sep);
    void emit_store_binary(char *s, char *f, char* sep);
    void emit_store_binary(char *s, char *f);
    void emit_filter(char *s, char *f, int e);
    void emit_order(char *s, char *f, int e, int ll = 0);
    void emit_group(char *s, char *f, int e);
    void emit_select(char *s, char *f, int ll);
    void emit_join(char *s, char *j1, int grp);
    void emit_join_tab(char *s, char tp);
    void emit_distinct();
	void emit_join();
	void emit_sort(char* s);



/* Line 189 of yacc.c  */
#line 126 "bison.cu"

/* Enabling traces.  */
#ifndef YYDEBUG
# define YYDEBUG 0
#endif

/* Enabling verbose error messages.  */
#ifdef YYERROR_VERBOSE
# undef YYERROR_VERBOSE
# define YYERROR_VERBOSE 1
#else
# define YYERROR_VERBOSE 0
#endif

/* Enabling the token table.  */
#ifndef YYTOKEN_TABLE
# define YYTOKEN_TABLE 0
#endif


/* Tokens.  */
#ifndef YYTOKENTYPE
# define YYTOKENTYPE
   /* Put the tokens into the symbol table, so that GDB and other debuggers
      know about them.  */
   enum yytokentype {
     FILENAME = 258,
     NAME = 259,
     STRING = 260,
     INTNUM = 261,
     DECIMAL1 = 262,
     BOOL1 = 263,
     APPROXNUM = 264,
     USERVAR = 265,
     ASSIGN = 266,
     EQUAL = 267,
     OR = 268,
     XOR = 269,
     AND = 270,
     DISTINCT = 271,
     REGEXP = 272,
     LIKE = 273,
     IS = 274,
     IN = 275,
     NOT = 276,
     BETWEEN = 277,
     COMPARISON = 278,
     SHIFT = 279,
     MOD = 280,
     UMINUS = 281,
     LOAD = 282,
     STREAM = 283,
     FILTER = 284,
     BY = 285,
     JOIN = 286,
     STORE = 287,
     INTO = 288,
     GROUP = 289,
     FROM = 290,
     SELECT = 291,
     AS = 292,
     ORDER = 293,
     ASC = 294,
     DESC = 295,
     COUNT = 296,
     USING = 297,
     SUM = 298,
     AVG = 299,
     MIN = 300,
     MAX = 301,
     LIMIT = 302,
     ON = 303,
     BINARY = 304,
     LEFT = 305,
     RIGHT = 306,
     OUTER = 307,
     SORT = 308,
     SEGMENTS = 309
   };
#endif



#if ! defined YYSTYPE && ! defined YYSTYPE_IS_DECLARED
typedef union YYSTYPE
{

/* Line 214 of yacc.c  */
#line 69 "bison.y"

    int intval;
    float floatval;
    char *strval;
    int subtok;



/* Line 214 of yacc.c  */
#line 225 "bison.cu"
} YYSTYPE;
# define YYSTYPE_IS_TRIVIAL 1
# define yystype YYSTYPE /* obsolescent; will be withdrawn */
# define YYSTYPE_IS_DECLARED 1
#endif


/* Copy the second part of user declarations.  */


/* Line 264 of yacc.c  */
#line 237 "bison.cu"

#ifdef short
# undef short
#endif

#ifdef YYTYPE_UINT8
typedef YYTYPE_UINT8 yytype_uint8;
#else
typedef unsigned char yytype_uint8;
#endif

#ifdef YYTYPE_INT8
typedef YYTYPE_INT8 yytype_int8;
#elif (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
typedef signed char yytype_int8;
#else
typedef short int yytype_int8;
#endif

#ifdef YYTYPE_UINT16
typedef YYTYPE_UINT16 yytype_uint16;
#else
typedef unsigned short int yytype_uint16;
#endif

#ifdef YYTYPE_INT16
typedef YYTYPE_INT16 yytype_int16;
#else
typedef short int yytype_int16;
#endif

#ifndef YYSIZE_T
# ifdef __SIZE_TYPE__
#  define YYSIZE_T __SIZE_TYPE__
# elif defined size_t
#  define YYSIZE_T size_t
# elif ! defined YYSIZE_T && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#  include <stddef.h> /* INFRINGES ON USER NAME SPACE */
#  define YYSIZE_T size_t
# else
#  define YYSIZE_T unsigned int
# endif
#endif

#define YYSIZE_MAXIMUM ((YYSIZE_T) -1)

#ifndef YY_
# if YYENABLE_NLS
#  if ENABLE_NLS
#   include <libintl.h> /* INFRINGES ON USER NAME SPACE */
#   define YY_(msgid) dgettext ("bison-runtime", msgid)
#  endif
# endif
# ifndef YY_
#  define YY_(msgid) msgid
# endif
#endif

/* Suppress unused-variable warnings by "using" E.  */
#if ! defined lint || defined __GNUC__
# define YYUSE(e) ((void) (e))
#else
# define YYUSE(e) /* empty */
#endif

/* Identity function, used to suppress warnings about constant conditions.  */
#ifndef lint
# define YYID(n) (n)
#else
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static int
YYID (int yyi)
#else
static int
YYID (yyi)
    int yyi;
#endif
{
  return yyi;
}
#endif

#if ! defined yyoverflow || YYERROR_VERBOSE

/* The parser invokes alloca or malloc; define the necessary symbols.  */

# ifdef YYSTACK_USE_ALLOCA
#  if YYSTACK_USE_ALLOCA
#   ifdef __GNUC__
#    define YYSTACK_ALLOC __builtin_alloca
#   elif defined __BUILTIN_VA_ARG_INCR
#    include <alloca.h> /* INFRINGES ON USER NAME SPACE */
#   elif defined _AIX
#    define YYSTACK_ALLOC __alloca
#   elif defined _MSC_VER
#    include <malloc.h> /* INFRINGES ON USER NAME SPACE */
#    define alloca _alloca
#   else
#    define YYSTACK_ALLOC alloca
#    if ! defined _ALLOCA_H && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#     include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#     ifndef _STDLIB_H
#      define _STDLIB_H 1
#     endif
#    endif
#   endif
#  endif
# endif

# ifdef YYSTACK_ALLOC
   /* Pacify GCC's `empty if-body' warning.  */
#  define YYSTACK_FREE(Ptr) do { /* empty */; } while (YYID (0))
#  ifndef YYSTACK_ALLOC_MAXIMUM
    /* The OS might guarantee only one guard page at the bottom of the stack,
       and a page size can be as small as 4096 bytes.  So we cannot safely
       invoke alloca (N) if N exceeds 4096.  Use a slightly smaller number
       to allow for a few compiler-allocated temporary stack slots.  */
#   define YYSTACK_ALLOC_MAXIMUM 4032 /* reasonable circa 2006 */
#  endif
# else
#  define YYSTACK_ALLOC YYMALLOC
#  define YYSTACK_FREE YYFREE
#  ifndef YYSTACK_ALLOC_MAXIMUM
#   define YYSTACK_ALLOC_MAXIMUM YYSIZE_MAXIMUM
#  endif
#  if (defined __cplusplus && ! defined _STDLIB_H \
       && ! ((defined YYMALLOC || defined malloc) \
	     && (defined YYFREE || defined free)))
#   include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#   ifndef _STDLIB_H
#    define _STDLIB_H 1
#   endif
#  endif
#  ifndef YYMALLOC
#   define YYMALLOC malloc
#   if ! defined malloc && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void *malloc (YYSIZE_T); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
#  ifndef YYFREE
#   define YYFREE free
#   if ! defined free && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void free (void *); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
# endif
#endif /* ! defined yyoverflow || YYERROR_VERBOSE */


#if (! defined yyoverflow \
     && (! defined __cplusplus \
	 || (defined YYSTYPE_IS_TRIVIAL && YYSTYPE_IS_TRIVIAL)))

/* A type that is properly aligned for any stack member.  */
union yyalloc
{
  yytype_int16 yyss_alloc;
  YYSTYPE yyvs_alloc;
};

/* The size of the maximum gap between one aligned stack and the next.  */
# define YYSTACK_GAP_MAXIMUM (sizeof (union yyalloc) - 1)

/* The size of an array large to enough to hold all stacks, each with
   N elements.  */
# define YYSTACK_BYTES(N) \
     ((N) * (sizeof (yytype_int16) + sizeof (YYSTYPE)) \
      + YYSTACK_GAP_MAXIMUM)

/* Copy COUNT objects from FROM to TO.  The source and destination do
   not overlap.  */
# ifndef YYCOPY
#  if defined __GNUC__ && 1 < __GNUC__
#   define YYCOPY(To, From, Count) \
      __builtin_memcpy (To, From, (Count) * sizeof (*(From)))
#  else
#   define YYCOPY(To, From, Count)		\
      do					\
	{					\
	  YYSIZE_T yyi;				\
	  for (yyi = 0; yyi < (Count); yyi++)	\
	    (To)[yyi] = (From)[yyi];		\
	}					\
      while (YYID (0))
#  endif
# endif

/* Relocate STACK from its old location to the new one.  The
   local variables YYSIZE and YYSTACKSIZE give the old and new number of
   elements in the stack, and YYPTR gives the new location of the
   stack.  Advance YYPTR to a properly aligned location for the next
   stack.  */
# define YYSTACK_RELOCATE(Stack_alloc, Stack)				\
    do									\
      {									\
	YYSIZE_T yynewbytes;						\
	YYCOPY (&yyptr->Stack_alloc, Stack, yysize);			\
	Stack = &yyptr->Stack_alloc;					\
	yynewbytes = yystacksize * sizeof (*Stack) + YYSTACK_GAP_MAXIMUM; \
	yyptr += yynewbytes / sizeof (*yyptr);				\
      }									\
    while (YYID (0))

#endif

/* YYFINAL -- State number of the termination state.  */
#define YYFINAL  8
/* YYLAST -- Last index in YYTABLE.  */
#define YYLAST   478

/* YYNTOKENS -- Number of terminals.  */
#define YYNTOKENS  72
/* YYNNTS -- Number of nonterminals.  */
#define YYNNTS  14
/* YYNRULES -- Number of rules.  */
#define YYNRULES  72
/* YYNRULES -- Number of states.  */
#define YYNSTATES  180

/* YYTRANSLATE(YYLEX) -- Bison symbol number corresponding to YYLEX.  */
#define YYUNDEFTOK  2
#define YYMAXUTOK   309

#define YYTRANSLATE(YYX)						\
  ((unsigned int) (YYX) <= YYMAXUTOK ? yytranslate[YYX] : YYUNDEFTOK)

/* YYTRANSLATE[YYLEX] -- Bison symbol number corresponding to YYLEX.  */
static const yytype_uint8 yytranslate[] =
{
       0,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    21,     2,     2,     2,    32,    26,     2,
      65,    66,    30,    28,    71,    29,    67,    31,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,    70,    64,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,    34,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    68,    25,    69,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     1,     2,     3,     4,
       5,     6,     7,     8,     9,    10,    11,    12,    13,    14,
      15,    16,    17,    18,    19,    20,    22,    23,    24,    27,
      33,    35,    36,    37,    38,    39,    40,    41,    42,    43,
      44,    45,    46,    47,    48,    49,    50,    51,    52,    53,
      54,    55,    56,    57,    58,    59,    60,    61,    62,    63
};

#if YYDEBUG
/* YYPRHS[YYN] -- Index of the first RHS symbol of rule number YYN in
   YYRHS.  */
static const yytype_uint16 yyprhs[] =
{
       0,     0,     3,     6,    10,    12,    20,    33,    43,    49,
      56,    65,    75,    83,    85,    89,    91,    93,    95,    97,
      99,   101,   111,   118,   121,   124,   129,   134,   139,   144,
     149,   152,   154,   158,   162,   166,   170,   174,   178,   182,
     186,   190,   194,   198,   201,   204,   208,   214,   218,   222,
     227,   228,   232,   236,   242,   244,   248,   250,   254,   255,
     257,   260,   265,   271,   277,   283,   289,   296,   303,   310,
     311,   314,   315
};

/* YYRHS -- A `-1'-separated list of the rules' RHS.  */
static const yytype_int8 yyrhs[] =
{
      73,     0,    -1,    74,    64,    -1,    73,    74,    64,    -1,
      75,    -1,     4,    11,    45,    78,    44,     4,    77,    -1,
       4,    11,    36,     3,    51,    65,     3,    66,    46,    65,
      79,    66,    -1,     4,    11,    36,     3,    58,    46,    65,
      79,    66,    -1,     4,    11,    38,     4,    82,    -1,     4,
      11,    47,     4,    39,    81,    -1,     4,    11,    45,    78,
      44,     4,    83,    77,    -1,    41,     4,    42,     3,    51,
      65,     3,    66,    84,    -1,    41,     4,    42,     3,    84,
      58,    85,    -1,     4,    -1,     4,    67,     4,    -1,    10,
      -1,     5,    -1,     6,    -1,     9,    -1,     7,    -1,     8,
      -1,     4,    68,     6,    69,    70,     4,    65,     6,    66,
      -1,     4,    68,     6,    69,    70,     4,    -1,     4,    48,
      -1,     4,    49,    -1,    50,    65,    76,    66,    -1,    52,
      65,    76,    66,    -1,    53,    65,    76,    66,    -1,    54,
      65,    76,    66,    -1,    55,    65,    76,    66,    -1,    16,
      76,    -1,    40,    -1,    76,    28,    76,    -1,    76,    29,
      76,    -1,    76,    30,    76,    -1,    76,    31,    76,    -1,
      76,    32,    76,    -1,    76,    33,    76,    -1,    76,    15,
      76,    -1,    76,    12,    76,    -1,    76,    13,    76,    -1,
      76,    14,    76,    -1,    76,    27,    76,    -1,    22,    76,
      -1,    21,    76,    -1,    76,    24,    76,    -1,    76,    24,
      65,    75,    66,    -1,    65,    76,    66,    -1,    76,    19,
       8,    -1,    76,    19,    22,     8,    -1,    -1,    43,    39,
      80,    -1,    76,    46,     4,    -1,    78,    71,    76,    46,
       4,    -1,    76,    -1,    79,    71,    76,    -1,    76,    -1,
      76,    71,    80,    -1,    -1,    80,    -1,    39,    76,    -1,
      40,     4,    57,    76,    -1,    59,    40,     4,    57,    76,
      -1,    60,    40,     4,    57,    76,    -1,    61,    40,     4,
      57,    76,    -1,    40,     4,    57,    76,    83,    -1,    59,
      40,     4,    57,    76,    83,    -1,    60,    40,     4,    57,
      76,    83,    -1,    61,    40,     4,    57,    76,    83,    -1,
      -1,    56,     6,    -1,    -1,    62,    63,    39,     4,    -1
};

/* YYRLINE[YYN] -- source line where rule number YYN was defined.  */
static const yytype_uint16 yyrline[] =
{
       0,   145,   145,   146,   150,   153,   155,   157,   159,   161,
     163,   165,   167,   172,   173,   174,   175,   176,   177,   178,
     179,   180,   181,   182,   183,   184,   185,   186,   187,   188,
     189,   190,   194,   195,   196,   197,   198,   199,   201,   202,
     203,   204,   205,   206,   207,   208,   210,   211,   215,   216,
     219,   222,   226,   227,   231,   232,   236,   237,   240,   242,
     245,   248,   249,   250,   251,   252,   253,   254,   255,   257,
     260,   262,   265
};
#endif

#if YYDEBUG || YYERROR_VERBOSE || YYTOKEN_TABLE
/* YYTNAME[SYMBOL-NUM] -- String name of the symbol SYMBOL-NUM.
   First, the terminals, then, starting at YYNTOKENS, nonterminals.  */
static const char *const yytname[] =
{
  "$end", "error", "$undefined", "FILENAME", "NAME", "STRING", "INTNUM",
  "DECIMAL1", "BOOL1", "APPROXNUM", "USERVAR", "ASSIGN", "EQUAL", "OR",
  "XOR", "AND", "DISTINCT", "REGEXP", "LIKE", "IS", "IN", "'!'", "NOT",
  "BETWEEN", "COMPARISON", "'|'", "'&'", "SHIFT", "'+'", "'-'", "'*'",
  "'/'", "'%'", "MOD", "'^'", "UMINUS", "LOAD", "STREAM", "FILTER", "BY",
  "JOIN", "STORE", "INTO", "GROUP", "FROM", "SELECT", "AS", "ORDER", "ASC",
  "DESC", "COUNT", "USING", "SUM", "AVG", "MIN", "MAX", "LIMIT", "ON",
  "BINARY", "LEFT", "RIGHT", "OUTER", "SORT", "SEGMENTS", "';'", "'('",
  "')'", "'.'", "'{'", "'}'", "':'", "','", "$accept", "stmt_list", "stmt",
  "select_stmt", "expr", "opt_group_list", "expr_list", "load_list",
  "val_list", "opt_val_list", "opt_where", "join_list", "opt_limit",
  "sort_def", 0
};
#endif

# ifdef YYPRINT
/* YYTOKNUM[YYLEX-NUM] -- Internal token number corresponding to
   token YYLEX-NUM.  */
static const yytype_uint16 yytoknum[] =
{
       0,   256,   257,   258,   259,   260,   261,   262,   263,   264,
     265,   266,   267,   268,   269,   270,   271,   272,   273,   274,
     275,    33,   276,   277,   278,   124,    38,   279,    43,    45,
      42,    47,    37,   280,    94,   281,   282,   283,   284,   285,
     286,   287,   288,   289,   290,   291,   292,   293,   294,   295,
     296,   297,   298,   299,   300,   301,   302,   303,   304,   305,
     306,   307,   308,   309,    59,    40,    41,    46,   123,   125,
      58,    44
};
# endif

/* YYR1[YYN] -- Symbol number of symbol that rule YYN derives.  */
static const yytype_uint8 yyr1[] =
{
       0,    72,    73,    73,    74,    75,    75,    75,    75,    75,
      75,    75,    75,    76,    76,    76,    76,    76,    76,    76,
      76,    76,    76,    76,    76,    76,    76,    76,    76,    76,
      76,    76,    76,    76,    76,    76,    76,    76,    76,    76,
      76,    76,    76,    76,    76,    76,    76,    76,    76,    76,
      77,    77,    78,    78,    79,    79,    80,    80,    81,    81,
      82,    83,    83,    83,    83,    83,    83,    83,    83,    84,
      84,    85,    85
};

/* YYR2[YYN] -- Number of symbols composing right hand side of rule YYN.  */
static const yytype_uint8 yyr2[] =
{
       0,     2,     2,     3,     1,     7,    12,     9,     5,     6,
       8,     9,     7,     1,     3,     1,     1,     1,     1,     1,
       1,     9,     6,     2,     2,     4,     4,     4,     4,     4,
       2,     1,     3,     3,     3,     3,     3,     3,     3,     3,
       3,     3,     3,     2,     2,     3,     5,     3,     3,     4,
       0,     3,     3,     5,     1,     3,     1,     3,     0,     1,
       2,     4,     5,     5,     5,     5,     6,     6,     6,     0,
       2,     0,     4
};

/* YYDEFACT[STATE-NAME] -- Default rule to reduce with in state
   STATE-NUM when YYTABLE doesn't specify something else to do.  Zero
   means the default is an error.  */
static const yytype_uint8 yydefact[] =
{
       0,     0,     0,     0,     0,     4,     0,     0,     1,     0,
       2,     0,     0,     0,     0,     0,     3,     0,     0,    13,
      16,    17,    19,    20,    18,    15,     0,     0,     0,    31,
       0,     0,     0,     0,     0,     0,     0,     0,     0,    69,
       0,     0,     0,     8,    23,    24,     0,     0,    30,    44,
      43,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,    58,     0,     0,     0,     0,     0,    60,
      14,     0,     0,     0,     0,     0,     0,    47,    39,    40,
      41,    38,    48,     0,     0,    45,    42,    32,    33,    34,
      35,    36,    37,    52,    50,     0,    56,    59,     9,     0,
      70,    71,     0,     0,     0,    25,    26,    27,    28,    29,
      49,    13,     0,     0,     0,     0,     0,     0,     5,    50,
       0,     0,     0,     0,    12,     0,    54,     0,     0,    46,
       0,     0,     0,     0,     0,    10,    53,    57,    69,     0,
       0,     7,     0,    22,     0,    51,     0,     0,     0,    11,
       0,     0,    55,     0,    61,     0,     0,     0,    72,     0,
       0,    65,    62,    63,    64,     6,    21,    66,    67,    68
};

/* YYDEFGOTO[NTERM-NUM].  */
static const yytype_int16 yydefgoto[] =
{
      -1,     3,     4,     5,   106,   128,    37,   137,   107,   108,
      43,   129,    76,   134
};

/* YYPACT[STATE-NUM] -- Index in YYTABLE of the portion describing
   STATE-NUM.  */
#define YYPACT_NINF -122
static const yytype_int16 yypact[] =
{
       7,   -10,     8,     2,   -37,  -122,   122,    15,  -122,    -6,
    -122,    65,    66,    82,    69,    77,  -122,    16,    30,    34,
    -122,  -122,  -122,  -122,  -122,  -122,    82,    82,    82,  -122,
      40,    42,    47,    55,    58,    82,   317,   -41,    60,   -47,
      59,    80,    82,  -122,  -122,  -122,   123,   127,   422,   432,
     432,    82,    82,    82,    82,    82,   163,    82,    82,    82,
      82,    -1,   109,    82,    82,    82,    82,    82,    82,    82,
     125,   134,    82,    82,    75,   140,    92,   148,    90,   368,
    -122,    87,   185,   207,   229,   251,   273,  -122,   368,   388,
     407,   422,  -122,   149,    56,   439,   445,   112,   112,  -122,
    -122,  -122,  -122,  -122,   -35,   345,     4,  -122,  -122,   165,
    -122,   108,   105,    82,   102,  -122,  -122,  -122,  -122,  -122,
    -122,   117,   107,   175,   141,   143,   146,   161,  -122,   138,
     184,    82,   136,   126,  -122,   157,   368,     5,   201,  -122,
     150,    82,   202,   204,   206,  -122,  -122,  -122,   155,   186,
     158,  -122,    82,   159,    82,  -122,   170,   171,   173,  -122,
     228,    82,   368,   227,   295,    82,    82,    82,  -122,    13,
     179,  -122,   295,   295,   295,  -122,  -122,  -122,  -122,  -122
};

/* YYPGOTO[NTERM-NUM].  */
static const yytype_int16 yypgoto[] =
{
    -122,  -122,   243,   153,   -13,   120,  -122,    89,  -121,  -122,
    -122,   -79,   104,  -122
};

/* YYTABLE[YYPACT[STATE-NUM]].  What to do in state STATE-NUM.  If
   positive, shift that token.  If negative, reduce the rule which
   number is the opposite.  If zero, do what YYDEFACT says.
   If YYTABLE_NINF, syntax error.  */
#define YYTABLE_NINF -1
static const yytype_uint8 yytable[] =
{
      36,     6,     8,    71,    74,   123,     1,    92,   124,    75,
     147,     1,     7,    48,    49,    50,    57,    58,    59,    60,
     155,    93,    56,    61,   125,   126,   127,    10,    62,    79,
      72,    63,    64,    65,    66,    67,    68,    69,    82,    83,
      84,    85,    86,     2,    88,    89,    90,    91,     2,    95,
      96,    97,    98,    99,   100,   101,   102,    15,    16,   105,
     121,    20,    21,    22,    23,    24,    25,    40,    17,    42,
      18,   151,    26,    38,    41,   131,   152,    27,    28,   175,
      39,    56,    44,    45,   152,   171,    19,    20,    21,    22,
      23,    24,    25,   177,   178,   179,    29,     2,    26,    73,
     136,    46,    47,    27,    28,    51,    30,    52,    31,    32,
      33,    34,    53,    19,    20,    21,    22,    23,    24,    25,
      54,    35,    29,    55,    77,    26,    78,    80,     6,   103,
      27,    28,    30,    81,    31,    32,    33,    34,   104,   162,
     109,   164,    66,    67,    68,    69,   110,    35,   136,    29,
     111,   112,   172,   173,   174,   113,   114,   120,    11,    30,
      12,    31,    32,    33,    34,    44,    45,    13,   132,    14,
     133,   135,   138,   139,    94,    57,    58,    59,    60,   140,
     141,   124,    61,   142,    46,    47,   143,    62,   146,   149,
      63,    64,    65,    66,    67,    68,    69,    57,    58,    59,
      60,   144,   148,   150,    61,   153,   156,   154,   157,    62,
     158,    75,    63,    64,    65,    66,    67,    68,    69,    57,
      58,    59,    60,   161,   163,   160,    61,   165,   166,    87,
     167,    62,   168,   170,    63,    64,    65,    66,    67,    68,
      69,    57,    58,    59,    60,   176,     9,   122,    61,   145,
     169,   115,   159,    62,     0,     0,    63,    64,    65,    66,
      67,    68,    69,    57,    58,    59,    60,     0,     0,     0,
      61,     0,     0,   116,     0,    62,     0,     0,    63,    64,
      65,    66,    67,    68,    69,    57,    58,    59,    60,     0,
       0,     0,    61,     0,     0,   117,     0,    62,     0,     0,
      63,    64,    65,    66,    67,    68,    69,    57,    58,    59,
      60,     0,     0,     0,    61,     0,     0,   118,     0,    62,
       0,     0,    63,    64,    65,    66,    67,    68,    69,    57,
      58,    59,    60,     0,     0,   123,    61,     0,     0,   119,
       0,    62,     0,     0,    63,    64,    65,    66,    67,    68,
      69,     0,     0,     0,   125,   126,   127,    57,    58,    59,
      60,     0,     0,    70,    61,     0,     0,     0,     0,    62,
       0,     0,    63,    64,    65,    66,    67,    68,    69,     0,
      57,    58,    59,    60,     0,     0,     0,    61,     0,     0,
       0,   130,    62,     0,     0,    63,    64,    65,    66,    67,
      68,    69,    59,    60,     0,     0,     0,    61,     0,     0,
       0,     0,    62,     0,     0,    63,    64,    65,    66,    67,
      68,    69,    60,     0,     0,     0,    61,     0,     0,     0,
       0,    62,     0,     0,    63,    64,    65,    66,    67,    68,
      69,    61,     0,     0,     0,     0,    62,     0,     0,    63,
      64,    65,    66,    67,    68,    69,    62,     0,     0,    63,
      64,    65,    66,    67,    68,    69,    63,    64,    65,    66,
      67,    68,    69,    64,    65,    66,    67,    68,    69
};

static const yytype_int16 yycheck[] =
{
      13,    11,     0,    44,    51,    40,     4,     8,    43,    56,
     131,     4,     4,    26,    27,    28,    12,    13,    14,    15,
     141,    22,    35,    19,    59,    60,    61,    64,    24,    42,
      71,    27,    28,    29,    30,    31,    32,    33,    51,    52,
      53,    54,    55,    41,    57,    58,    59,    60,    41,    62,
      63,    64,    65,    66,    67,    68,    69,    42,    64,    72,
       4,     5,     6,     7,     8,     9,    10,    51,     3,    39,
       4,    66,    16,     4,    58,    71,    71,    21,    22,    66,
       3,    94,    48,    49,    71,   164,     4,     5,     6,     7,
       8,     9,    10,   172,   173,   174,    40,    41,    16,    39,
     113,    67,    68,    21,    22,    65,    50,    65,    52,    53,
      54,    55,    65,     4,     5,     6,     7,     8,     9,    10,
      65,    65,    40,    65,    65,    16,    46,     4,    11,     4,
      21,    22,    50,     6,    52,    53,    54,    55,     4,   152,
      65,   154,    30,    31,    32,    33,     6,    65,   161,    40,
      58,     3,   165,   166,   167,    65,    69,     8,    36,    50,
      38,    52,    53,    54,    55,    48,    49,    45,     3,    47,
      62,    66,    70,    66,    65,    12,    13,    14,    15,     4,
      39,    43,    19,    40,    67,    68,    40,    24,     4,    63,
      27,    28,    29,    30,    31,    32,    33,    12,    13,    14,
      15,    40,    66,    46,    19,     4,     4,    57,     4,    24,
       4,    56,    27,    28,    29,    30,    31,    32,    33,    12,
      13,    14,    15,    65,    65,    39,    19,    57,    57,    66,
      57,    24,     4,     6,    27,    28,    29,    30,    31,    32,
      33,    12,    13,    14,    15,    66,     3,    94,    19,   129,
     161,    66,   148,    24,    -1,    -1,    27,    28,    29,    30,
      31,    32,    33,    12,    13,    14,    15,    -1,    -1,    -1,
      19,    -1,    -1,    66,    -1,    24,    -1,    -1,    27,    28,
      29,    30,    31,    32,    33,    12,    13,    14,    15,    -1,
      -1,    -1,    19,    -1,    -1,    66,    -1,    24,    -1,    -1,
      27,    28,    29,    30,    31,    32,    33,    12,    13,    14,
      15,    -1,    -1,    -1,    19,    -1,    -1,    66,    -1,    24,
      -1,    -1,    27,    28,    29,    30,    31,    32,    33,    12,
      13,    14,    15,    -1,    -1,    40,    19,    -1,    -1,    66,
      -1,    24,    -1,    -1,    27,    28,    29,    30,    31,    32,
      33,    -1,    -1,    -1,    59,    60,    61,    12,    13,    14,
      15,    -1,    -1,    46,    19,    -1,    -1,    -1,    -1,    24,
      -1,    -1,    27,    28,    29,    30,    31,    32,    33,    -1,
      12,    13,    14,    15,    -1,    -1,    -1,    19,    -1,    -1,
      -1,    46,    24,    -1,    -1,    27,    28,    29,    30,    31,
      32,    33,    14,    15,    -1,    -1,    -1,    19,    -1,    -1,
      -1,    -1,    24,    -1,    -1,    27,    28,    29,    30,    31,
      32,    33,    15,    -1,    -1,    -1,    19,    -1,    -1,    -1,
      -1,    24,    -1,    -1,    27,    28,    29,    30,    31,    32,
      33,    19,    -1,    -1,    -1,    -1,    24,    -1,    -1,    27,
      28,    29,    30,    31,    32,    33,    24,    -1,    -1,    27,
      28,    29,    30,    31,    32,    33,    27,    28,    29,    30,
      31,    32,    33,    28,    29,    30,    31,    32,    33
};

/* YYSTOS[STATE-NUM] -- The (internal number of the) accessing
   symbol of state STATE-NUM.  */
static const yytype_uint8 yystos[] =
{
       0,     4,    41,    73,    74,    75,    11,     4,     0,    74,
      64,    36,    38,    45,    47,    42,    64,     3,     4,     4,
       5,     6,     7,     8,     9,    10,    16,    21,    22,    40,
      50,    52,    53,    54,    55,    65,    76,    78,     4,     3,
      51,    58,    39,    82,    48,    49,    67,    68,    76,    76,
      76,    65,    65,    65,    65,    65,    76,    12,    13,    14,
      15,    19,    24,    27,    28,    29,    30,    31,    32,    33,
      46,    44,    71,    39,    51,    56,    84,    65,    46,    76,
       4,     6,    76,    76,    76,    76,    76,    66,    76,    76,
      76,    76,     8,    22,    65,    76,    76,    76,    76,    76,
      76,    76,    76,     4,     4,    76,    76,    80,    81,    65,
       6,    58,     3,    65,    69,    66,    66,    66,    66,    66,
       8,     4,    75,    40,    43,    59,    60,    61,    77,    83,
      46,    71,     3,    62,    85,    66,    76,    79,    70,    66,
       4,    39,    40,    40,    40,    77,     4,    80,    66,    63,
      46,    66,    71,     4,    57,    80,     4,     4,     4,    84,
      39,    65,    76,    65,    76,    57,    57,    57,     4,    79,
       6,    83,    76,    76,    76,    66,    66,    83,    83,    83
};

#define yyerrok		(yyerrstatus = 0)
#define yyclearin	(yychar = YYEMPTY)
#define YYEMPTY		(-2)
#define YYEOF		0

#define YYACCEPT	goto yyacceptlab
#define YYABORT		goto yyabortlab
#define YYERROR		goto yyerrorlab


/* Like YYERROR except do call yyerror.  This remains here temporarily
   to ease the transition to the new meaning of YYERROR, for GCC.
   Once GCC version 2 has supplanted version 1, this can go.  */

#define YYFAIL		goto yyerrlab

#define YYRECOVERING()  (!!yyerrstatus)

#define YYBACKUP(Token, Value)					\
do								\
  if (yychar == YYEMPTY && yylen == 1)				\
    {								\
      yychar = (Token);						\
      yylval = (Value);						\
      yytoken = YYTRANSLATE (yychar);				\
      YYPOPSTACK (1);						\
      goto yybackup;						\
    }								\
  else								\
    {								\
      yyerror (YY_("syntax error: cannot back up")); \
      YYERROR;							\
    }								\
while (YYID (0))


#define YYTERROR	1
#define YYERRCODE	256


/* YYLLOC_DEFAULT -- Set CURRENT to span from RHS[1] to RHS[N].
   If N is 0, then set CURRENT to the empty location which ends
   the previous symbol: RHS[0] (always defined).  */

#define YYRHSLOC(Rhs, K) ((Rhs)[K])
#ifndef YYLLOC_DEFAULT
# define YYLLOC_DEFAULT(Current, Rhs, N)				\
    do									\
      if (YYID (N))                                                    \
	{								\
	  (Current).first_line   = YYRHSLOC (Rhs, 1).first_line;	\
	  (Current).first_column = YYRHSLOC (Rhs, 1).first_column;	\
	  (Current).last_line    = YYRHSLOC (Rhs, N).last_line;		\
	  (Current).last_column  = YYRHSLOC (Rhs, N).last_column;	\
	}								\
      else								\
	{								\
	  (Current).first_line   = (Current).last_line   =		\
	    YYRHSLOC (Rhs, 0).last_line;				\
	  (Current).first_column = (Current).last_column =		\
	    YYRHSLOC (Rhs, 0).last_column;				\
	}								\
    while (YYID (0))
#endif


/* YY_LOCATION_PRINT -- Print the location on the stream.
   This macro was not mandated originally: define only if we know
   we won't break user code: when these are the locations we know.  */

#ifndef YY_LOCATION_PRINT
# if YYLTYPE_IS_TRIVIAL
#  define YY_LOCATION_PRINT(File, Loc)			\
     fprintf (File, "%d.%d-%d.%d",			\
	      (Loc).first_line, (Loc).first_column,	\
	      (Loc).last_line,  (Loc).last_column)
# else
#  define YY_LOCATION_PRINT(File, Loc) ((void) 0)
# endif
#endif


/* YYLEX -- calling `yylex' with the right arguments.  */

#ifdef YYLEX_PARAM
# define YYLEX yylex (YYLEX_PARAM)
#else
# define YYLEX yylex ()
#endif

/* Enable debugging if requested.  */
#if YYDEBUG

# ifndef YYFPRINTF
#  include <stdio.h> /* INFRINGES ON USER NAME SPACE */
#  define YYFPRINTF fprintf
# endif

# define YYDPRINTF(Args)			\
do {						\
  if (yydebug)					\
    YYFPRINTF Args;				\
} while (YYID (0))

# define YY_SYMBOL_PRINT(Title, Type, Value, Location)			  \
do {									  \
  if (yydebug)								  \
    {									  \
      YYFPRINTF (stderr, "%s ", Title);					  \
      yy_symbol_print (stderr,						  \
		  Type, Value); \
      YYFPRINTF (stderr, "\n");						  \
    }									  \
} while (YYID (0))


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_value_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_value_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (!yyvaluep)
    return;
# ifdef YYPRINT
  if (yytype < YYNTOKENS)
    YYPRINT (yyoutput, yytoknum[yytype], *yyvaluep);
# else
  YYUSE (yyoutput);
# endif
  switch (yytype)
    {
      default:
	break;
    }
}


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (yytype < YYNTOKENS)
    YYFPRINTF (yyoutput, "token %s (", yytname[yytype]);
  else
    YYFPRINTF (yyoutput, "nterm %s (", yytname[yytype]);

  yy_symbol_value_print (yyoutput, yytype, yyvaluep);
  YYFPRINTF (yyoutput, ")");
}

/*------------------------------------------------------------------.
| yy_stack_print -- Print the state stack from its BOTTOM up to its |
| TOP (included).                                                   |
`------------------------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_stack_print (yytype_int16 *yybottom, yytype_int16 *yytop)
#else
static void
yy_stack_print (yybottom, yytop)
    yytype_int16 *yybottom;
    yytype_int16 *yytop;
#endif
{
  YYFPRINTF (stderr, "Stack now");
  for (; yybottom <= yytop; yybottom++)
    {
      int yybot = *yybottom;
      YYFPRINTF (stderr, " %d", yybot);
    }
  YYFPRINTF (stderr, "\n");
}

# define YY_STACK_PRINT(Bottom, Top)				\
do {								\
  if (yydebug)							\
    yy_stack_print ((Bottom), (Top));				\
} while (YYID (0))


/*------------------------------------------------.
| Report that the YYRULE is going to be reduced.  |
`------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_reduce_print (YYSTYPE *yyvsp, int yyrule)
#else
static void
yy_reduce_print (yyvsp, yyrule)
    YYSTYPE *yyvsp;
    int yyrule;
#endif
{
  int yynrhs = yyr2[yyrule];
  int yyi;
  unsigned long int yylno = yyrline[yyrule];
  YYFPRINTF (stderr, "Reducing stack by rule %d (line %lu):\n",
	     yyrule - 1, yylno);
  /* The symbols being reduced.  */
  for (yyi = 0; yyi < yynrhs; yyi++)
    {
      YYFPRINTF (stderr, "   $%d = ", yyi + 1);
      yy_symbol_print (stderr, yyrhs[yyprhs[yyrule] + yyi],
		       &(yyvsp[(yyi + 1) - (yynrhs)])
		       		       );
      YYFPRINTF (stderr, "\n");
    }
}

# define YY_REDUCE_PRINT(Rule)		\
do {					\
  if (yydebug)				\
    yy_reduce_print (yyvsp, Rule); \
} while (YYID (0))

/* Nonzero means print parse trace.  It is left uninitialized so that
   multiple parsers can coexist.  */
int yydebug;
#else /* !YYDEBUG */
# define YYDPRINTF(Args)
# define YY_SYMBOL_PRINT(Title, Type, Value, Location)
# define YY_STACK_PRINT(Bottom, Top)
# define YY_REDUCE_PRINT(Rule)
#endif /* !YYDEBUG */


/* YYINITDEPTH -- initial size of the parser's stacks.  */
#ifndef	YYINITDEPTH
# define YYINITDEPTH 200
#endif

/* YYMAXDEPTH -- maximum size the stacks can grow to (effective only
   if the built-in stack extension method is used).

   Do not make this value too large; the results are undefined if
   YYSTACK_ALLOC_MAXIMUM < YYSTACK_BYTES (YYMAXDEPTH)
   evaluated with infinite-precision integer arithmetic.  */

#ifndef YYMAXDEPTH
# define YYMAXDEPTH 10000
#endif



#if YYERROR_VERBOSE

# ifndef yystrlen
#  if defined __GLIBC__ && defined _STRING_H
#   define yystrlen strlen
#  else
/* Return the length of YYSTR.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static YYSIZE_T
yystrlen (const char *yystr)
#else
static YYSIZE_T
yystrlen (yystr)
    const char *yystr;
#endif
{
  YYSIZE_T yylen;
  for (yylen = 0; yystr[yylen]; yylen++)
    continue;
  return yylen;
}
#  endif
# endif

# ifndef yystpcpy
#  if defined __GLIBC__ && defined _STRING_H && defined _GNU_SOURCE
#   define yystpcpy stpcpy
#  else
/* Copy YYSRC to YYDEST, returning the address of the terminating '\0' in
   YYDEST.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static char *
yystpcpy (char *yydest, const char *yysrc)
#else
static char *
yystpcpy (yydest, yysrc)
    char *yydest;
    const char *yysrc;
#endif
{
  char *yyd = yydest;
  const char *yys = yysrc;

  while ((*yyd++ = *yys++) != '\0')
    continue;

  return yyd - 1;
}
#  endif
# endif

# ifndef yytnamerr
/* Copy to YYRES the contents of YYSTR after stripping away unnecessary
   quotes and backslashes, so that it's suitable for yyerror.  The
   heuristic is that double-quoting is unnecessary unless the string
   contains an apostrophe, a comma, or backslash (other than
   backslash-backslash).  YYSTR is taken from yytname.  If YYRES is
   null, do not copy; instead, return the length of what the result
   would have been.  */
static YYSIZE_T
yytnamerr (char *yyres, const char *yystr)
{
  if (*yystr == '"')
    {
      YYSIZE_T yyn = 0;
      char const *yyp = yystr;

      for (;;)
	switch (*++yyp)
	  {
	  case '\'':
	  case ',':
	    goto do_not_strip_quotes;

	  case '\\':
	    if (*++yyp != '\\')
	      goto do_not_strip_quotes;
	    /* Fall through.  */
	  default:
	    if (yyres)
	      yyres[yyn] = *yyp;
	    yyn++;
	    break;

	  case '"':
	    if (yyres)
	      yyres[yyn] = '\0';
	    return yyn;
	  }
    do_not_strip_quotes: ;
    }

  if (! yyres)
    return yystrlen (yystr);

  return yystpcpy (yyres, yystr) - yyres;
}
# endif

/* Copy into YYRESULT an error message about the unexpected token
   YYCHAR while in state YYSTATE.  Return the number of bytes copied,
   including the terminating null byte.  If YYRESULT is null, do not
   copy anything; just return the number of bytes that would be
   copied.  As a special case, return 0 if an ordinary "syntax error"
   message will do.  Return YYSIZE_MAXIMUM if overflow occurs during
   size calculation.  */
static YYSIZE_T
yysyntax_error (char *yyresult, int yystate, int yychar)
{
  int yyn = yypact[yystate];

  if (! (YYPACT_NINF < yyn && yyn <= YYLAST))
    return 0;
  else
    {
      int yytype = YYTRANSLATE (yychar);
      YYSIZE_T yysize0 = yytnamerr (0, yytname[yytype]);
      YYSIZE_T yysize = yysize0;
      YYSIZE_T yysize1;
      int yysize_overflow = 0;
      enum { YYERROR_VERBOSE_ARGS_MAXIMUM = 5 };
      char const *yyarg[YYERROR_VERBOSE_ARGS_MAXIMUM];
      int yyx;

# if 0
      /* This is so xgettext sees the translatable formats that are
	 constructed on the fly.  */
      YY_("syntax error, unexpected %s");
      YY_("syntax error, unexpected %s, expecting %s");
      YY_("syntax error, unexpected %s, expecting %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s or %s");
# endif
      char *yyfmt;
      char const *yyf;
      static char const yyunexpected[] = "syntax error, unexpected %s";
      static char const yyexpecting[] = ", expecting %s";
      static char const yyor[] = " or %s";
      char yyformat[sizeof yyunexpected
		    + sizeof yyexpecting - 1
		    + ((YYERROR_VERBOSE_ARGS_MAXIMUM - 2)
		       * (sizeof yyor - 1))];
      char const *yyprefix = yyexpecting;

      /* Start YYX at -YYN if negative to avoid negative indexes in
	 YYCHECK.  */
      int yyxbegin = yyn < 0 ? -yyn : 0;

      /* Stay within bounds of both yycheck and yytname.  */
      int yychecklim = YYLAST - yyn + 1;
      int yyxend = yychecklim < YYNTOKENS ? yychecklim : YYNTOKENS;
      int yycount = 1;

      yyarg[0] = yytname[yytype];
      yyfmt = yystpcpy (yyformat, yyunexpected);

      for (yyx = yyxbegin; yyx < yyxend; ++yyx)
	if (yycheck[yyx + yyn] == yyx && yyx != YYTERROR)
	  {
	    if (yycount == YYERROR_VERBOSE_ARGS_MAXIMUM)
	      {
		yycount = 1;
		yysize = yysize0;
		yyformat[sizeof yyunexpected - 1] = '\0';
		break;
	      }
	    yyarg[yycount++] = yytname[yyx];
	    yysize1 = yysize + yytnamerr (0, yytname[yyx]);
	    yysize_overflow |= (yysize1 < yysize);
	    yysize = yysize1;
	    yyfmt = yystpcpy (yyfmt, yyprefix);
	    yyprefix = yyor;
	  }

      yyf = YY_(yyformat);
      yysize1 = yysize + yystrlen (yyf);
      yysize_overflow |= (yysize1 < yysize);
      yysize = yysize1;

      if (yysize_overflow)
	return YYSIZE_MAXIMUM;

      if (yyresult)
	{
	  /* Avoid sprintf, as that infringes on the user's name space.
	     Don't have undefined behavior even if the translation
	     produced a string with the wrong number of "%s"s.  */
	  char *yyp = yyresult;
	  int yyi = 0;
	  while ((*yyp = *yyf) != '\0')
	    {
	      if (*yyp == '%' && yyf[1] == 's' && yyi < yycount)
		{
		  yyp += yytnamerr (yyp, yyarg[yyi++]);
		  yyf += 2;
		}
	      else
		{
		  yyp++;
		  yyf++;
		}
	    }
	}
      return yysize;
    }
}
#endif /* YYERROR_VERBOSE */


/*-----------------------------------------------.
| Release the memory associated to this symbol.  |
`-----------------------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yydestruct (const char *yymsg, int yytype, YYSTYPE *yyvaluep)
#else
static void
yydestruct (yymsg, yytype, yyvaluep)
    const char *yymsg;
    int yytype;
    YYSTYPE *yyvaluep;
#endif
{
  YYUSE (yyvaluep);

  if (!yymsg)
    yymsg = "Deleting";
  YY_SYMBOL_PRINT (yymsg, yytype, yyvaluep, yylocationp);

  switch (yytype)
    {

      default:
	break;
    }
}

/* Prevent warnings from -Wmissing-prototypes.  */
#ifdef YYPARSE_PARAM
#if defined __STDC__ || defined __cplusplus
int yyparse (void *YYPARSE_PARAM);
#else
int yyparse ();
#endif
#else /* ! YYPARSE_PARAM */
#if defined __STDC__ || defined __cplusplus
int yyparse (void);
#else
int yyparse ();
#endif
#endif /* ! YYPARSE_PARAM */


/* The lookahead symbol.  */
int yychar;

/* The semantic value of the lookahead symbol.  */
YYSTYPE yylval;

/* Number of syntax errors so far.  */
int yynerrs;



/*-------------------------.
| yyparse or yypush_parse.  |
`-------------------------*/

#ifdef YYPARSE_PARAM
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void *YYPARSE_PARAM)
#else
int
yyparse (YYPARSE_PARAM)
    void *YYPARSE_PARAM;
#endif
#else /* ! YYPARSE_PARAM */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void)
#else
int
yyparse ()

#endif
#endif
{


    int yystate;
    /* Number of tokens to shift before error messages enabled.  */
    int yyerrstatus;

    /* The stacks and their tools:
       `yyss': related to states.
       `yyvs': related to semantic values.

       Refer to the stacks thru separate pointers, to allow yyoverflow
       to reallocate them elsewhere.  */

    /* The state stack.  */
    yytype_int16 yyssa[YYINITDEPTH];
    yytype_int16 *yyss;
    yytype_int16 *yyssp;

    /* The semantic value stack.  */
    YYSTYPE yyvsa[YYINITDEPTH];
    YYSTYPE *yyvs;
    YYSTYPE *yyvsp;

    YYSIZE_T yystacksize;

  int yyn;
  int yyresult;
  /* Lookahead token as an internal (translated) token number.  */
  int yytoken;
  /* The variables used to return semantic value and location from the
     action routines.  */
  YYSTYPE yyval;

#if YYERROR_VERBOSE
  /* Buffer for error messages, and its allocated size.  */
  char yymsgbuf[128];
  char *yymsg = yymsgbuf;
  YYSIZE_T yymsg_alloc = sizeof yymsgbuf;
#endif

#define YYPOPSTACK(N)   (yyvsp -= (N), yyssp -= (N))

  /* The number of symbols on the RHS of the reduced rule.
     Keep to zero when no symbol should be popped.  */
  int yylen = 0;

  yytoken = 0;
  yyss = yyssa;
  yyvs = yyvsa;
  yystacksize = YYINITDEPTH;

  YYDPRINTF ((stderr, "Starting parse\n"));

  yystate = 0;
  yyerrstatus = 0;
  yynerrs = 0;
  yychar = YYEMPTY; /* Cause a token to be read.  */

  /* Initialize stack pointers.
     Waste one element of value and location stack
     so that they stay on the same level as the state stack.
     The wasted elements are never initialized.  */
  yyssp = yyss;
  yyvsp = yyvs;

  goto yysetstate;

/*------------------------------------------------------------.
| yynewstate -- Push a new state, which is found in yystate.  |
`------------------------------------------------------------*/
 yynewstate:
  /* In all cases, when you get here, the value and location stacks
     have just been pushed.  So pushing a state here evens the stacks.  */
  yyssp++;

 yysetstate:
  *yyssp = yystate;

  if (yyss + yystacksize - 1 <= yyssp)
    {
      /* Get the current used size of the three stacks, in elements.  */
      YYSIZE_T yysize = yyssp - yyss + 1;

#ifdef yyoverflow
      {
	/* Give user a chance to reallocate the stack.  Use copies of
	   these so that the &'s don't force the real ones into
	   memory.  */
	YYSTYPE *yyvs1 = yyvs;
	yytype_int16 *yyss1 = yyss;

	/* Each stack pointer address is followed by the size of the
	   data in use in that stack, in bytes.  This used to be a
	   conditional around just the two extra args, but that might
	   be undefined if yyoverflow is a macro.  */
	yyoverflow (YY_("memory exhausted"),
		    &yyss1, yysize * sizeof (*yyssp),
		    &yyvs1, yysize * sizeof (*yyvsp),
		    &yystacksize);

	yyss = yyss1;
	yyvs = yyvs1;
      }
#else /* no yyoverflow */
# ifndef YYSTACK_RELOCATE
      goto yyexhaustedlab;
# else
      /* Extend the stack our own way.  */
      if (YYMAXDEPTH <= yystacksize)
	goto yyexhaustedlab;
      yystacksize *= 2;
      if (YYMAXDEPTH < yystacksize)
	yystacksize = YYMAXDEPTH;

      {
	yytype_int16 *yyss1 = yyss;
	union yyalloc *yyptr =
	  (union yyalloc *) YYSTACK_ALLOC (YYSTACK_BYTES (yystacksize));
	if (! yyptr)
	  goto yyexhaustedlab;
	YYSTACK_RELOCATE (yyss_alloc, yyss);
	YYSTACK_RELOCATE (yyvs_alloc, yyvs);
#  undef YYSTACK_RELOCATE
	if (yyss1 != yyssa)
	  YYSTACK_FREE (yyss1);
      }
# endif
#endif /* no yyoverflow */

      yyssp = yyss + yysize - 1;
      yyvsp = yyvs + yysize - 1;

      YYDPRINTF ((stderr, "Stack size increased to %lu\n",
		  (unsigned long int) yystacksize));

      if (yyss + yystacksize - 1 <= yyssp)
	YYABORT;
    }

  YYDPRINTF ((stderr, "Entering state %d\n", yystate));

  if (yystate == YYFINAL)
    YYACCEPT;

  goto yybackup;

/*-----------.
| yybackup.  |
`-----------*/
yybackup:

  /* Do appropriate processing given the current state.  Read a
     lookahead token if we need one and don't already have one.  */

  /* First try to decide what to do without reference to lookahead token.  */
  yyn = yypact[yystate];
  if (yyn == YYPACT_NINF)
    goto yydefault;

  /* Not known => get a lookahead token if don't already have one.  */

  /* YYCHAR is either YYEMPTY or YYEOF or a valid lookahead symbol.  */
  if (yychar == YYEMPTY)
    {
      YYDPRINTF ((stderr, "Reading a token: "));
      yychar = YYLEX;
    }

  if (yychar <= YYEOF)
    {
      yychar = yytoken = YYEOF;
      YYDPRINTF ((stderr, "Now at end of input.\n"));
    }
  else
    {
      yytoken = YYTRANSLATE (yychar);
      YY_SYMBOL_PRINT ("Next token is", yytoken, &yylval, &yylloc);
    }

  /* If the proper action on seeing token YYTOKEN is to reduce or to
     detect an error, take that action.  */
  yyn += yytoken;
  if (yyn < 0 || YYLAST < yyn || yycheck[yyn] != yytoken)
    goto yydefault;
  yyn = yytable[yyn];
  if (yyn <= 0)
    {
      if (yyn == 0 || yyn == YYTABLE_NINF)
	goto yyerrlab;
      yyn = -yyn;
      goto yyreduce;
    }

  /* Count tokens shifted since error; after three, turn off error
     status.  */
  if (yyerrstatus)
    yyerrstatus--;

  /* Shift the lookahead token.  */
  YY_SYMBOL_PRINT ("Shifting", yytoken, &yylval, &yylloc);

  /* Discard the shifted token.  */
  yychar = YYEMPTY;

  yystate = yyn;
  *++yyvsp = yylval;

  goto yynewstate;


/*-----------------------------------------------------------.
| yydefault -- do the default action for the current state.  |
`-----------------------------------------------------------*/
yydefault:
  yyn = yydefact[yystate];
  if (yyn == 0)
    goto yyerrlab;
  goto yyreduce;


/*-----------------------------.
| yyreduce -- Do a reduction.  |
`-----------------------------*/
yyreduce:
  /* yyn is the number of a rule to reduce with.  */
  yylen = yyr2[yyn];

  /* If YYLEN is nonzero, implement the default value of the action:
     `$$ = $1'.

     Otherwise, the following line sets YYVAL to garbage.
     This behavior is undocumented and Bison
     users should not rely upon it.  Assigning to YYVAL
     unconditionally makes the parser a bit smaller, and it avoids a
     GCC warning that YYVAL may be used uninitialized.  */
  yyval = yyvsp[1-yylen];


  YY_REDUCE_PRINT (yyn);
  switch (yyn)
    {
        case 4:

/* Line 1455 of yacc.c  */
#line 150 "bison.y"
    { emit("STMT"); ;}
    break;

  case 5:

/* Line 1455 of yacc.c  */
#line 154 "bison.y"
    { emit_select((yyvsp[(1) - (7)].strval), (yyvsp[(6) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 6:

/* Line 1455 of yacc.c  */
#line 156 "bison.y"
    {  emit_load((yyvsp[(1) - (12)].strval), (yyvsp[(4) - (12)].strval), (yyvsp[(11) - (12)].intval), (yyvsp[(7) - (12)].strval)); ;}
    break;

  case 7:

/* Line 1455 of yacc.c  */
#line 158 "bison.y"
    {  emit_load_binary((yyvsp[(1) - (9)].strval), (yyvsp[(4) - (9)].strval), (yyvsp[(8) - (9)].intval)); ;}
    break;

  case 8:

/* Line 1455 of yacc.c  */
#line 160 "bison.y"
    {  emit_filter((yyvsp[(1) - (5)].strval), (yyvsp[(4) - (5)].strval), (yyvsp[(5) - (5)].intval));;}
    break;

  case 9:

/* Line 1455 of yacc.c  */
#line 162 "bison.y"
    {  emit_order((yyvsp[(1) - (6)].strval), (yyvsp[(4) - (6)].strval), (yyvsp[(6) - (6)].intval));;}
    break;

  case 10:

/* Line 1455 of yacc.c  */
#line 164 "bison.y"
    { emit_join((yyvsp[(1) - (8)].strval),(yyvsp[(6) - (8)].strval),(yyvsp[(7) - (8)].intval)); ;}
    break;

  case 11:

/* Line 1455 of yacc.c  */
#line 166 "bison.y"
    { emit_store((yyvsp[(2) - (9)].strval),(yyvsp[(4) - (9)].strval),(yyvsp[(7) - (9)].strval)); ;}
    break;

  case 12:

/* Line 1455 of yacc.c  */
#line 168 "bison.y"
    { emit_store_binary((yyvsp[(2) - (7)].strval),(yyvsp[(4) - (7)].strval)); ;}
    break;

  case 13:

/* Line 1455 of yacc.c  */
#line 172 "bison.y"
    { emit_name((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 14:

/* Line 1455 of yacc.c  */
#line 173 "bison.y"
    { emit("FIELDNAME %s.%s", (yyvsp[(1) - (3)].strval), (yyvsp[(3) - (3)].strval)); ;}
    break;

  case 15:

/* Line 1455 of yacc.c  */
#line 174 "bison.y"
    { emit("USERVAR %s", (yyvsp[(1) - (1)].strval)); ;}
    break;

  case 16:

/* Line 1455 of yacc.c  */
#line 175 "bison.y"
    { emit_string((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 17:

/* Line 1455 of yacc.c  */
#line 176 "bison.y"
    { emit_number((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 18:

/* Line 1455 of yacc.c  */
#line 177 "bison.y"
    { emit_float((yyvsp[(1) - (1)].floatval)); ;}
    break;

  case 19:

/* Line 1455 of yacc.c  */
#line 178 "bison.y"
    { emit_decimal((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 20:

/* Line 1455 of yacc.c  */
#line 179 "bison.y"
    { emit("BOOL %d", (yyvsp[(1) - (1)].intval)); ;}
    break;

  case 21:

/* Line 1455 of yacc.c  */
#line 180 "bison.y"
    { emit_varchar((yyvsp[(1) - (9)].strval), (yyvsp[(3) - (9)].intval), (yyvsp[(6) - (9)].strval), (yyvsp[(8) - (9)].intval));;}
    break;

  case 22:

/* Line 1455 of yacc.c  */
#line 181 "bison.y"
    { emit_var((yyvsp[(1) - (6)].strval), (yyvsp[(3) - (6)].intval), (yyvsp[(6) - (6)].strval));;}
    break;

  case 23:

/* Line 1455 of yacc.c  */
#line 182 "bison.y"
    { emit_var_asc((yyvsp[(1) - (2)].strval));;}
    break;

  case 24:

/* Line 1455 of yacc.c  */
#line 183 "bison.y"
    { emit_var_desc((yyvsp[(1) - (2)].strval));;}
    break;

  case 25:

/* Line 1455 of yacc.c  */
#line 184 "bison.y"
    { emit_count(); ;}
    break;

  case 26:

/* Line 1455 of yacc.c  */
#line 185 "bison.y"
    { emit_sum(); ;}
    break;

  case 27:

/* Line 1455 of yacc.c  */
#line 186 "bison.y"
    { emit_average(); ;}
    break;

  case 28:

/* Line 1455 of yacc.c  */
#line 187 "bison.y"
    { emit_min(); ;}
    break;

  case 29:

/* Line 1455 of yacc.c  */
#line 188 "bison.y"
    { emit_max(); ;}
    break;

  case 30:

/* Line 1455 of yacc.c  */
#line 189 "bison.y"
    { emit_distinct(); ;}
    break;

  case 31:

/* Line 1455 of yacc.c  */
#line 190 "bison.y"
    { emit_join(); ;}
    break;

  case 32:

/* Line 1455 of yacc.c  */
#line 194 "bison.y"
    { emit_add(); ;}
    break;

  case 33:

/* Line 1455 of yacc.c  */
#line 195 "bison.y"
    { emit_minus(); ;}
    break;

  case 34:

/* Line 1455 of yacc.c  */
#line 196 "bison.y"
    { emit_mul(); ;}
    break;

  case 35:

/* Line 1455 of yacc.c  */
#line 197 "bison.y"
    { emit_div(); ;}
    break;

  case 36:

/* Line 1455 of yacc.c  */
#line 198 "bison.y"
    { emit("MOD"); ;}
    break;

  case 37:

/* Line 1455 of yacc.c  */
#line 199 "bison.y"
    { emit("MOD"); ;}
    break;

  case 38:

/* Line 1455 of yacc.c  */
#line 201 "bison.y"
    { emit_and(); ;}
    break;

  case 39:

/* Line 1455 of yacc.c  */
#line 202 "bison.y"
    { emit_eq(); ;}
    break;

  case 40:

/* Line 1455 of yacc.c  */
#line 203 "bison.y"
    { emit_or(); ;}
    break;

  case 41:

/* Line 1455 of yacc.c  */
#line 204 "bison.y"
    { emit("XOR"); ;}
    break;

  case 42:

/* Line 1455 of yacc.c  */
#line 205 "bison.y"
    { emit("SHIFT %s", (yyvsp[(2) - (3)].subtok)==1?"left":"right"); ;}
    break;

  case 43:

/* Line 1455 of yacc.c  */
#line 206 "bison.y"
    { emit("NOT"); ;}
    break;

  case 44:

/* Line 1455 of yacc.c  */
#line 207 "bison.y"
    { emit("NOT"); ;}
    break;

  case 45:

/* Line 1455 of yacc.c  */
#line 208 "bison.y"
    { emit_cmp((yyvsp[(2) - (3)].subtok)); ;}
    break;

  case 46:

/* Line 1455 of yacc.c  */
#line 210 "bison.y"
    { emit("CMPSELECT %d", (yyvsp[(2) - (5)].subtok)); ;}
    break;

  case 47:

/* Line 1455 of yacc.c  */
#line 211 "bison.y"
    {emit("EXPR");;}
    break;

  case 48:

/* Line 1455 of yacc.c  */
#line 215 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(3) - (3)].intval)); ;}
    break;

  case 49:

/* Line 1455 of yacc.c  */
#line 216 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(4) - (4)].intval)); emit("NOT"); ;}
    break;

  case 50:

/* Line 1455 of yacc.c  */
#line 219 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 51:

/* Line 1455 of yacc.c  */
#line 222 "bison.y"
    { (yyval.intval) = (yyvsp[(3) - (3)].intval);}
    break;

  case 52:

/* Line 1455 of yacc.c  */
#line 226 "bison.y"
    { (yyval.intval) = 1; emit_sel_name((yyvsp[(3) - (3)].strval));;}
    break;

  case 53:

/* Line 1455 of yacc.c  */
#line 227 "bison.y"
    { (yyval.intval) = (yyvsp[(1) - (5)].intval) + 1; emit_sel_name((yyvsp[(5) - (5)].strval));;}
    break;

  case 54:

/* Line 1455 of yacc.c  */
#line 231 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 55:

/* Line 1455 of yacc.c  */
#line 232 "bison.y"
    {(yyval.intval) = (yyvsp[(1) - (3)].intval) + 1; ;}
    break;

  case 56:

/* Line 1455 of yacc.c  */
#line 236 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 57:

/* Line 1455 of yacc.c  */
#line 237 "bison.y"
    { (yyval.intval) = 1 + (yyvsp[(3) - (3)].intval); ;}
    break;

  case 58:

/* Line 1455 of yacc.c  */
#line 240 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 60:

/* Line 1455 of yacc.c  */
#line 245 "bison.y"
    { emit("FILTER BY"); ;}
    break;

  case 61:

/* Line 1455 of yacc.c  */
#line 248 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (4)].strval), 'I');;}
    break;

  case 62:

/* Line 1455 of yacc.c  */
#line 249 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'L');;}
    break;

  case 63:

/* Line 1455 of yacc.c  */
#line 250 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'R');;}
    break;

  case 64:

/* Line 1455 of yacc.c  */
#line 251 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'O');;}
    break;

  case 65:

/* Line 1455 of yacc.c  */
#line 252 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (5)].strval), 'I'); ;}
    break;

  case 66:

/* Line 1455 of yacc.c  */
#line 253 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'L'); ;}
    break;

  case 67:

/* Line 1455 of yacc.c  */
#line 254 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'R'); ;}
    break;

  case 68:

/* Line 1455 of yacc.c  */
#line 255 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'O'); ;}
    break;

  case 69:

/* Line 1455 of yacc.c  */
#line 257 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 70:

/* Line 1455 of yacc.c  */
#line 260 "bison.y"
    { emit_limit((yyvsp[(2) - (2)].intval)); ;}
    break;

  case 71:

/* Line 1455 of yacc.c  */
#line 262 "bison.y"
    { /* nil */
    (yyval.intval) = 0
;}
    break;

  case 72:

/* Line 1455 of yacc.c  */
#line 265 "bison.y"
    { emit_sort((yyvsp[(4) - (4)].strval)); ;}
    break;



/* Line 1455 of yacc.c  */
#line 2126 "bison.cu"
      default: break;
    }
  YY_SYMBOL_PRINT ("-> $$ =", yyr1[yyn], &yyval, &yyloc);

  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);

  *++yyvsp = yyval;

  /* Now `shift' the result of the reduction.  Determine what state
     that goes to, based on the state we popped back to and the rule
     number reduced by.  */

  yyn = yyr1[yyn];

  yystate = yypgoto[yyn - YYNTOKENS] + *yyssp;
  if (0 <= yystate && yystate <= YYLAST && yycheck[yystate] == *yyssp)
    yystate = yytable[yystate];
  else
    yystate = yydefgoto[yyn - YYNTOKENS];

  goto yynewstate;


/*------------------------------------.
| yyerrlab -- here on detecting error |
`------------------------------------*/
yyerrlab:
  /* If not already recovering from an error, report this error.  */
  if (!yyerrstatus)
    {
      ++yynerrs;
#if ! YYERROR_VERBOSE
      yyerror (YY_("syntax error"));
#else
      {
	YYSIZE_T yysize = yysyntax_error (0, yystate, yychar);
	if (yymsg_alloc < yysize && yymsg_alloc < YYSTACK_ALLOC_MAXIMUM)
	  {
	    YYSIZE_T yyalloc = 2 * yysize;
	    if (! (yysize <= yyalloc && yyalloc <= YYSTACK_ALLOC_MAXIMUM))
	      yyalloc = YYSTACK_ALLOC_MAXIMUM;
	    if (yymsg != yymsgbuf)
	      YYSTACK_FREE (yymsg);
	    yymsg = (char *) YYSTACK_ALLOC (yyalloc);
	    if (yymsg)
	      yymsg_alloc = yyalloc;
	    else
	      {
		yymsg = yymsgbuf;
		yymsg_alloc = sizeof yymsgbuf;
	      }
	  }

	if (0 < yysize && yysize <= yymsg_alloc)
	  {
	    (void) yysyntax_error (yymsg, yystate, yychar);
	    yyerror (yymsg);
	  }
	else
	  {
	    yyerror (YY_("syntax error"));
	    if (yysize != 0)
	      goto yyexhaustedlab;
	  }
      }
#endif
    }



  if (yyerrstatus == 3)
    {
      /* If just tried and failed to reuse lookahead token after an
	 error, discard it.  */

      if (yychar <= YYEOF)
	{
	  /* Return failure if at end of input.  */
	  if (yychar == YYEOF)
	    YYABORT;
	}
      else
	{
	  yydestruct ("Error: discarding",
		      yytoken, &yylval);
	  yychar = YYEMPTY;
	}
    }

  /* Else will try to reuse lookahead token after shifting the error
     token.  */
  goto yyerrlab1;


/*---------------------------------------------------.
| yyerrorlab -- error raised explicitly by YYERROR.  |
`---------------------------------------------------*/
yyerrorlab:

  /* Pacify compilers like GCC when the user code never invokes
     YYERROR and the label yyerrorlab therefore never appears in user
     code.  */
  if (/*CONSTCOND*/ 0)
     goto yyerrorlab;

  /* Do not reclaim the symbols of the rule which action triggered
     this YYERROR.  */
  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);
  yystate = *yyssp;
  goto yyerrlab1;


/*-------------------------------------------------------------.
| yyerrlab1 -- common code for both syntax error and YYERROR.  |
`-------------------------------------------------------------*/
yyerrlab1:
  yyerrstatus = 3;	/* Each real token shifted decrements this.  */

  for (;;)
    {
      yyn = yypact[yystate];
      if (yyn != YYPACT_NINF)
	{
	  yyn += YYTERROR;
	  if (0 <= yyn && yyn <= YYLAST && yycheck[yyn] == YYTERROR)
	    {
	      yyn = yytable[yyn];
	      if (0 < yyn)
		break;
	    }
	}

      /* Pop the current state because it cannot handle the error token.  */
      if (yyssp == yyss)
	YYABORT;


      yydestruct ("Error: popping",
		  yystos[yystate], yyvsp);
      YYPOPSTACK (1);
      yystate = *yyssp;
      YY_STACK_PRINT (yyss, yyssp);
    }

  *++yyvsp = yylval;


  /* Shift the error token.  */
  YY_SYMBOL_PRINT ("Shifting", yystos[yyn], yyvsp, yylsp);

  yystate = yyn;
  goto yynewstate;


/*-------------------------------------.
| yyacceptlab -- YYACCEPT comes here.  |
`-------------------------------------*/
yyacceptlab:
  yyresult = 0;
  goto yyreturn;

/*-----------------------------------.
| yyabortlab -- YYABORT comes here.  |
`-----------------------------------*/
yyabortlab:
  yyresult = 1;
  goto yyreturn;

#if !defined(yyoverflow) || YYERROR_VERBOSE
/*-------------------------------------------------.
| yyexhaustedlab -- memory exhaustion comes here.  |
`-------------------------------------------------*/
yyexhaustedlab:
  yyerror (YY_("memory exhausted"));
  yyresult = 2;
  /* Fall through.  */
#endif

yyreturn:
  if (yychar != YYEMPTY)
     yydestruct ("Cleanup: discarding lookahead",
		 yytoken, &yylval);
  /* Do not reclaim the symbols of the rule which action triggered
     this YYABORT or YYACCEPT.  */
  YYPOPSTACK (yylen);
  YY_STACK_PRINT (yyss, yyssp);
  while (yyssp != yyss)
    {
      yydestruct ("Cleanup: popping",
		  yystos[*yyssp], yyvsp);
      YYPOPSTACK (1);
    }
#ifndef yyoverflow
  if (yyss != yyssa)
    YYSTACK_FREE (yyss);
#endif
#if YYERROR_VERBOSE
  if (yymsg != yymsgbuf)
    YYSTACK_FREE (yymsg);
#endif
  /* Make sure YYID is used.  */
  return YYID (yyresult);
}



/* Line 1675 of yacc.c  */
#line 267 "bison.y"


#include "filter.h"
#include "select.h"
#include "merge.h"
#include "zone_map.h"
#include "atof.h"
#include "cudpp_src_2.0/include/cudpp_hash.h"
#include "moderngpu-master/include/kernels/join.cuh"
#include "moderngpu-master/include/util/mgpucontext.h"
#include "sstream"
#include "sorts.cu"

string to_string1(long long int i) {
	stringstream res;
	res << i;
	return res.str();
}

using namespace mgpu;

size_t int_size = sizeof(int_type);
size_t float_size = sizeof(float_type);

FILE *file_pointer;
queue<string> namevars;
queue<string> typevars;
queue<int> sizevars;
queue<int> cols;

queue<unsigned int> j_col_count;
unsigned int sel_count = 0;
unsigned int join_cnt = 0;
unsigned int distinct_cnt = 0;
unsigned int join_col_cnt = 0;
unsigned int join_tab_cnt = 0;
unsigned int tab_cnt = 0;
queue<string> op_join;
queue<char> join_type;


unsigned int statement_count = 0;
map<string,unsigned int> stat;
map<unsigned int, unsigned int> join_and_cnt;
bool scan_state = 0;
string separator, f_file;
unsigned int int_col_count;
CUDPPHandle theCudpp;
ContextPtr context;

void emit_multijoin(string s, string j1, string j2, unsigned int tab, char* res_name);

using namespace thrust::placeholders;


void emit_name(char *name)
{
    op_type.push("NAME");
    op_value.push(name);
}

void emit_limit(int val)
{
    op_nums.push(val);
}


void emit_string(char *str)
{   // remove the float_type quotes
    string sss(str,1, strlen(str)-2);
    op_type.push("STRING");
    op_value.push(sss);
}


void emit_number(int_type val)
{
    op_type.push("NUMBER");
    op_nums.push(val);
}

void emit_float(float_type val)
{
    op_type.push("FLOAT");
    op_nums_f.push(val);
}

void emit_decimal(float_type val)
{
    op_type.push("DECIMAL");
    op_nums_f.push(val);
}



void emit_mul()
{
    op_type.push("MUL");
}

void emit_add()
{
    op_type.push("ADD");
}

void emit_div()
{
    op_type.push("DIV");
}

unsigned int misses = 0;

void emit_and()
{
    op_type.push("AND");
    join_col_cnt++;
	//cout << "AND "  << endl;	
}

void emit_eq()
{    
    op_type.push("JOIN");
	if(misses == 0) {
		join_and_cnt[tab_cnt] = join_col_cnt;	
		//cout << "ASSIGN " << tab_cnt << " " << join_and_cnt[tab_cnt] << endl;
		misses = join_col_cnt;
		join_col_cnt = 0;		
		tab_cnt++;
	}
	else {
		misses--;
	}
	//cout << "eq " << endl;
}

void emit_distinct()
{
    op_type.push("DISTINCT");
    distinct_cnt++;
}

void emit_join()
{
   cout << "emit join " << endl;
}


void emit_or()
{
    op_type.push("OR");
}


void emit_minus()
{
    op_type.push("MINUS");
}

void emit_cmp(int val)
{
    op_type.push("CMP");
    op_nums.push(val);
}

void emit(char *s, ...)
{


}

void emit_var(char *s, int c, char *f)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(0);
    cols.push(c);
}

void emit_var_asc(char *s)
{
    op_type.push(s);
    op_value.push("ASC");
}

void emit_var_desc(char *s)
{
    op_type.push(s);
    op_value.push("DESC");
}

void emit_sort(char *s)
{
	op_sort.push(s); 
}



void emit_varchar(char *s, int c, char *f, int d)
{
    namevars.push(s);
    typevars.push(f);
    sizevars.push(d);
    cols.push(c);
}

void emit_sel_name(char *s)
{
    op_type.push("emit sel_name");
    op_value.push(s);
    sel_count++;
}

void emit_count()
{
    op_type.push("COUNT");
}

void emit_sum()
{
    op_type.push("SUM");
}


void emit_average()
{
    op_type.push("AVG");
}

void emit_min()
{
    op_type.push("MIN");
}

void emit_max()
{
    op_type.push("MAX");
}

void emit_join_tab(char *s, char tp)
{
    op_join.push(s);
	join_tab_cnt++;
    join_type.push(tp);
	//cout << "join tab " << join_tab_cnt << endl;
};


void order_inplace(CudaSet* a, stack<string> exe_type, set<string> field_names)
{
    //std::clock_t start1 = std::clock();
    unsigned int sz = a->mRecCount;
    thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(sz);
    thrust::sequence(permutation, permutation+sz,0,1);

    unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);
    void* temp;
    // find the largest mRecSize of all data sources exe_type.top()
    unsigned int maxSize = 0;
    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        CudaSet *t = varNames[setMap[*it]];
        if(t->mRecCount > maxSize)
            maxSize = t->mRecCount;
    };


    unsigned int max_c = max_char(a, field_names);
	//cout << "max_c " << max_c << " " << maxSize << " " << getFreeMem() << endl;

    if(max_c > float_size)
        CUDA_SAFE_CALL(hipMalloc((void **) &temp, maxSize*max_c));
    else
        CUDA_SAFE_CALL(hipMalloc((void **) &temp, maxSize*float_size));

    unsigned int str_count = 0;
	
	
    for(int i=0; !exe_type.empty(); ++i, exe_type.pop()) {
        int colInd = (a->columnNames).find(exe_type.top())->second;
        if (a->type[colInd] == 0)
            update_permutation(a->d_columns_int[a->type_index[colInd]], raw_ptr, sz, "ASC", (int_type*)temp);
        else if (a->type[colInd] == 1)
            update_permutation(a->d_columns_float[a->type_index[colInd]], raw_ptr, sz,"ASC", (float_type*)temp);
        else {
            // use int col int_col_count
	        update_permutation(a->d_columns_int[int_col_count+str_count], raw_ptr, sz, "ASC", (int_type*)temp);
	        str_count++;
        };
    };
	
    str_count = 0;

    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        int i = a->columnNames[*it];
        if (a->type[i] == 0) {
            apply_permutation(a->d_columns_int[a->type_index[i]], raw_ptr, sz, (int_type*)temp);			
		}	
        else if (a->type[i] == 1)
            apply_permutation(a->d_columns_float[a->type_index[i]], raw_ptr, sz, (float_type*)temp);
        else {
            apply_permutation_char(a->d_columns_char[a->type_index[i]], raw_ptr, sz, (char*)temp, a->char_size[a->type_index[i]]);			
            apply_permutation(a->d_columns_int[int_col_count + str_count], raw_ptr, sz, (int_type*)temp);
            str_count++;
        };
    };
	
    hipFree(temp);
    thrust::device_free(permutation);

}

bool check_star_join(string j1)
{
    queue<string> op_vals(op_value);
	queue<string> op_j(op_join);
	CudaSet* fact_table;
	
    for(unsigned int i=0; i < sel_count; i++) {        
        op_vals.pop();
        op_vals.pop();
    };
	
    if(join_tab_cnt > 1) {
	    fact_table = varNames[j1];
		
		while(op_vals.size()) {
			if (fact_table->columnNames.find(op_vals.front()) != fact_table->columnNames.end()) {
				op_vals.pop();
				op_vals.pop();
			}
            else {
				return 0;
			};	
		};
		return 1;
		
	}
	else
		return 0;


}

std::ostream &operator<<(std::ostream &os, const uint2 &x)
{
  os << x.x << ", " << x.y;
  return os;
}

void star_join(char *s, string j1)
{
   //need to copy to gpu all dimension keys, sort the dimension tables and
   //build an array of hash tables for the dimension tables
    CUDPPResult result;
	map<string,bool> already_copied;
   
    //cout << j1 << endl;
	CudaSet* left = varNames.find(j1)->second;
	
    queue<string> op_sel;
    queue<string> op_sel_as;
    for(int i=0; i < sel_count; i++) {
        op_sel.push(op_value.front());
        op_value.pop();
        op_sel_as.push(op_value.front());
        op_value.pop();
    };	
	queue<string> op_sel_s(op_sel);
	queue<string> op_sel_s_as(op_sel_as);
	queue<string> op_g(op_value);
	
	CudaSet* c = new CudaSet(op_sel_s, op_sel_s_as);

	
    CUDPPHandle* hash_table_handle = new CUDPPHandle[join_tab_cnt];
    CUDPPHashTableConfig config;
    config.type = CUDPP_MULTIVALUE_HASH_TABLE;    
    config.space_usage = 1.1f;  
    bool str_join = 0;	
	string f1, f2;
	unsigned int colInd1, tt = 0;
	bool v64bit = 0;
	unsigned int colInd2;
	map<string, unsigned int> tab_map;
	map<string, string> var_map;
	
	for(unsigned int i = 0; i < join_tab_cnt; i++) {

	    f1 = op_g.front();
		op_g.pop();
		f2 = op_g.front();
		op_g.pop();
	
        queue<string> op_jj(op_join);	
		for(unsigned int z = 0; z < (join_tab_cnt-1) - i; z++)
		    op_jj.pop();

		cout << "PROCESSING " << f2 <<   endl;
		
        unsigned int rcount;
        curr_segment = 10000000;
        queue<string> op_vd(op_g);
        queue<string> op_alt(op_sel);
        unsigned int jc = join_col_cnt;
        while(jc) {
            jc--;
            op_vd.pop();
            op_alt.push(op_vd.front());
            op_vd.pop();
        };
		
		//cout << "right is " << op_jj.front() << endl;
		tab_map[op_jj.front()] = i;
		var_map[op_jj.front()] = f1;

		CudaSet* right = varNames.find(op_jj.front())->second;
		colInd2 = right->columnNames[f2];
		
        unsigned int cnt_r = load_queue(op_alt, right, str_join, f2, rcount); // put all used columns into GPU
		
		bool sorted = thrust::is_sorted(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r);


		if(!sorted) {

			queue<string> ss(op_sel);
			thrust::device_vector<unsigned int> v(cnt_r);
			thrust::sequence(v.begin(),v.end(),0,1);

			unsigned int max_c	= max_char(right);
			unsigned int mm;
			if(max_c > 8)
				mm = (max_c/8) + 1;
			else
				mm = 1;

			thrust::device_ptr<int_type> d_tmp = thrust::device_malloc<int_type>(cnt_r*mm);
			thrust::sort_by_key(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r, v.begin());

			unsigned int i;
			while(!ss.empty()) {
				if (right->columnNames.find(ss.front()) != right->columnNames.end()) {
					i = right->columnNames[ss.front()];

					if(i != colInd2) {
						if(right->type[i] == 0) {
							thrust::gather(v.begin(), v.end(), right->d_columns_int[right->type_index[i]].begin(), d_tmp);
							thrust::copy(d_tmp, d_tmp + cnt_r, right->d_columns_int[right->type_index[i]].begin());
						}
						else if(right->type[i] == 1) {
							thrust::gather(v.begin(), v.end(), right->d_columns_float[right->type_index[i]].begin(), d_tmp);
							thrust::copy(d_tmp, d_tmp + cnt_r, right->d_columns_float[right->type_index[i]].begin());
						}
						else {
							str_gather(thrust::raw_pointer_cast(v.data()), cnt_r, (void*)right->d_columns_char[right->type_index[i]], (void*) thrust::raw_pointer_cast(d_tmp), right->char_size[right->type_index[i]]);
							hipMemcpy( (void*)right->d_columns_char[right->type_index[i]], (void*) thrust::raw_pointer_cast(d_tmp), cnt_r*right->char_size[right->type_index[i]], hipMemcpyDeviceToDevice);
						};
					};
				};
				ss.pop();
			};
			thrust::device_free(d_tmp);
		};

		if(right->d_columns_int[right->type_index[colInd2]][cnt_r-1] > std::numeric_limits<unsigned int>::max())
			v64bit = 1;
			
		colInd1 = (left->columnNames).find(f1)->second;			
		if (left->type[colInd1]  == 2) {
			cout << "Joins are not yet supported in star joins" << endl;
			exit(0);
		}
		else {
		    queue<string> cc;
			cc.push(f1);
			allocColumns(left, cc);
		};		
	
	    config.kInputSize = cnt_r;
		//cout << "creating table with " << cnt_r << " " << getFreeMem()  << endl;
		result = cudppHashTable(theCudpp, &hash_table_handle[i], &config);

		if (result == CUDPP_SUCCESS)
			cout << "hash tables created " << getFreeMem() << endl;
		
		
		if(left->maxRecs > rcount)
			tt = left->maxRecs;
		else {
		    if (rcount > tt)
				tt = rcount;
		};	
		thrust::device_vector<unsigned int> d_rr(tt);		
		thrust::device_vector<unsigned int> v(cnt_r);
		thrust::sequence(v.begin(),v.end(),0,1);
	
		thrust::copy(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r,
					 d_rr.begin());				 
		result = cudppHashInsert(hash_table_handle[i], thrust::raw_pointer_cast(d_rr.data()),
								 thrust::raw_pointer_cast(v.data()), cnt_r);

		if (result == CUDPP_SUCCESS)
			cout << "hash table inserted " << getFreeMem() << endl;		
	
	};
	
	thrust::device_ptr<unsigned int> d_r = thrust::device_malloc<unsigned int>(tt);
	thrust::device_vector<unsigned int> d_s(tt);
	
	thrust::device_ptr<uint2> res = thrust::device_malloc<uint2>(left->maxRecs);
	
    thrust::device_vector<unsigned int> d_res1;
    thrust::device_vector<unsigned int> d_res2;
	
	thrust::device_vector<bool> d_star(left->maxRecs);
		
    unsigned int cnt_l, res_count, tot_count = 0, offset = 0, k = 0;
	string ttt;
	queue<string> lc;
		
	
    for (unsigned int i = 0; i < left->segCount; i++) {
	       
        cout << "segment " << i << " " << getFreeMem() <<  '\xd';		
	    thrust::sequence(d_star.begin(), d_star.end(),1,0);	

		   //for every hash table
	    queue<string> op_g1(op_value);
	    for(unsigned int z = 0; z < join_tab_cnt; z++) {			
			
	        cnt_l = 0;
			f1 = op_g1.front();
		    op_g1.pop();
		    f2 = op_g1.front();
		    op_g1.pop();	

			while(lc.size())
				lc.pop();
			lc.push(f1);			
			copyColumns(left, lc, i, cnt_l);
			already_copied[f1] = 1;
		
			if(left->prm.empty()) {
				cnt_l = left->mRecCount;
			}
			else {
				cnt_l = left->prm_count[i];
			};
			

            queue<string> op_jj(op_join);	
		    for(unsigned int j = 0; j < (join_tab_cnt-1) - z; j++) {
				op_jj.pop();
			};	
			
				
			unsigned int idx;	
			if (cnt_l) {								
				
				idx = left->type_index[left->columnNames[lc.front()]];	
				//cout << "left idx " << idx << endl;
				//cout << "right col " << op_jj.front() << endl;
                CudaSet* right = varNames.find(op_jj.front())->second;				
				colInd2 = right->columnNames[f2];

				thrust::copy(left->d_columns_int[idx].begin(), left->d_columns_int[idx].begin() + cnt_l, d_r);

                result = cudppHashRetrieve(hash_table_handle[z], thrust::raw_pointer_cast(d_r),
										   thrust::raw_pointer_cast(res), cnt_l);
				if (result != CUDPP_SUCCESS)
					cout << "Failed retrieve " << endl;

				uint2 rr = thrust::reduce(res, res+cnt_l, make_uint2(0,0), Uint2Sum());
			
	    
				res_count = rr.y;
				d_res1.resize(res_count);
				d_res2.resize(res_count);
				//cout << "res cnt of " << f2 << " = " << res_count << endl;

				if(res_count) {
					thrust::counting_iterator<unsigned int> begin(0);
					uint2_split ff(thrust::raw_pointer_cast(res),thrust::raw_pointer_cast(d_r));
					thrust::for_each(begin, begin + cnt_l, ff);
					
					if(!v64bit) {
						thrust::transform(d_star.begin(), d_star.begin() + cnt_l, d_r, d_star.begin(), thrust::logical_and<bool>());
					};

					thrust::exclusive_scan(d_r, d_r+cnt_l, d_r );  // addresses
					join_functor1 ff1(thrust::raw_pointer_cast(res),
									  thrust::raw_pointer_cast(d_r),
									  thrust::raw_pointer_cast(d_res1.data()),
									  thrust::raw_pointer_cast(d_res2.data()));
					thrust::for_each(begin, begin + cnt_l, ff1);
					
					if(v64bit) {// need to check the upper 32 bits
						thrust::device_ptr<bool> d_add = thrust::device_malloc<bool>(d_res1.size());
						thrust::permutation_iterator<ElementIterator_int,IndexIterator> iter_left(left->d_columns_int[idx].begin(), d_res1.begin());
						thrust::permutation_iterator<ElementIterator_int,IndexIterator> iter_right(right->d_columns_int[right->type_index[colInd2]].begin(), d_res2.begin());						
						thrust::transform(iter_left, iter_left+d_res2.size(), iter_right, d_add, int_upper_equal_to());
						unsigned int new_cnt = thrust::count(d_add, d_add+d_res1.size(), 1);
						if(new_cnt == 0)
							break;
						thrust::stable_partition(d_res1.begin(), d_res1.begin() + d_res2.size(), d_add, thrust::identity<unsigned int>());
						thrust::stable_partition(d_res2.begin(), d_res2.end(), d_add, thrust::identity<unsigned int>());
                        
						thrust::transform(d_star.begin(), d_star.end(), d_add, d_star.begin(), thrust::logical_and<bool>());
						thrust::device_free(d_add);
						d_res2.resize(new_cnt);
						d_res1.resize(new_cnt);
					
					};
                }
                else {
				    thrust::sequence(d_star.begin(), d_star.end(),0,0);	
					break;
				};	
            };			
		};	
        // if our bool vector is not all zeroes then load all left columns and also get indexes and gather values 
 		// from right hash tables	
		unsigned int n_cnt = thrust::count(d_star.begin(), d_star.begin() + cnt_l, 1);
		//cout << "Star join result count " << n_cnt << endl;
		tot_count = tot_count + n_cnt;
		queue<string> cc;
		if(n_cnt) { //gather		
		
			offset = c->mRecCount;
			if(i == 0 && left->segCount != 1) {
				c->reserve(n_cnt*(left->segCount+1));				
			};	
            c->resize_join(n_cnt);
            queue<string> op_sel1(op_sel_s);
            unsigned int colInd, c_colInd;
            
            while(!op_sel1.empty()) {
				
				
                while(!cc.empty())
                    cc.pop();

                cc.push(op_sel1.front());
				if(c->columnNames.find(op_sel1.front()) != c->columnNames.end()) {
                    c_colInd = c->columnNames[op_sel1.front()];						
				};	

                if(left->columnNames.find(op_sel1.front()) !=  left->columnNames.end()) {
                    // copy field's segment to device, gather it and copy to the host
                    colInd = left->columnNames[op_sel1.front()];
					//cout << "gathering left " << op_sel1.front() << endl;  
						
					if(already_copied.count(op_sel1.front()) == 0) {	
						reset_offsets();
						allocColumns(left, cc);
						copyColumns(left, cc, i, k);
					};	
					
                        //gather
                    if(left->type[colInd] == 0) {
						thrust::device_ptr<int_type> d_tmp = thrust::device_malloc<int_type>(n_cnt);
						thrust::copy_if(left->d_columns_int[left->type_index[colInd]].begin(), left->d_columns_int[left->type_index[colInd]].begin() + cnt_l,
						                d_star.begin(), d_tmp, thrust::identity<bool>());
						thrust::copy(d_tmp, d_tmp + n_cnt, c->h_columns_int[c->type_index[c_colInd]].begin() + offset);				
						thrust::device_free(d_tmp);				
                    }
                    else if(left->type[colInd] == 1) {
						thrust::device_ptr<float_type> d_tmp = thrust::device_malloc<float_type>(n_cnt);
						thrust::copy_if(left->d_columns_float[left->type_index[colInd]].begin(), left->d_columns_float[left->type_index[colInd]].begin() + cnt_l,
						                d_star.begin(), d_tmp, thrust::identity<bool>());						
                        thrust::copy(d_tmp, d_tmp + n_cnt, c->h_columns_float[c->type_index[c_colInd]].begin() + offset);
						thrust::device_free(d_tmp);				
                    }
                    else { //strings
                        thrust::device_ptr<char> d_tmp = thrust::device_malloc<char>(n_cnt*left->char_size[left->type_index[colInd]]);
						
						thrust::device_ptr<bool> d_g(thrust::raw_pointer_cast(d_star.data()));
						
                        str_copy_if(left->d_columns_char[left->type_index[colInd]], cnt_l, thrust::raw_pointer_cast(d_tmp),
						             d_g, c->char_size[c->type_index[c_colInd]]);
                        hipMemcpy( (void*)&c->h_columns_char[c->type_index[c_colInd]][offset*c->char_size[c->type_index[c_colInd]]], (void*) thrust::raw_pointer_cast(d_tmp),
                                    c->char_size[c->type_index[c_colInd]] * n_cnt, hipMemcpyDeviceToHost);
                        thrust::device_free(d_tmp);
                    }
                    //left->deAllocColumnOnDevice(colInd);

                }
                else { 
				
				    //cout << "gathering right " << op_sel1.front() << endl;  
                    string right_tab_name;
                    queue<string> op_j(op_join);	
		            while(!op_j.empty()) {
					    if(varNames[op_j.front()]->columnNames.count(op_sel1.front())) {
							right_tab_name = op_j.front();
							break;
						};
						op_j.pop();
					};	   
   
					colInd = left->columnNames[var_map[right_tab_name]];
					//cout << "leftcolind " << colInd << endl;
					
					CudaSet* right = varNames[right_tab_name];				
					unsigned int r_colInd = right->columnNames[op_sel1.front()];
					
					//cout << "rcolind " << r_colInd << endl;
					
	                while(!cc.empty())
						cc.pop();
                    cc.push(var_map[right_tab_name]);
					
					if(c->columnNames.find(op_sel1.front()) != c->columnNames.end()) {
						c_colInd = c->columnNames[op_sel1.front()];						
					};	
					
					if(already_copied.count(var_map[right_tab_name]) == 0) {
						reset_offsets();
						allocColumns(left, cc);
						copyColumns(left, cc, i, k);
					};	
					
					thrust::device_ptr<int_type> d_t = thrust::device_malloc<int_type>(n_cnt);
					thrust::copy_if(left->d_columns_int[left->type_index[colInd]].begin(), left->d_columns_int[left->type_index[colInd]].begin() + cnt_l,
					                d_star.begin(), d_t, thrust::identity<bool>());
									
                    // get the values from hash table
					unsigned int hash_ind = tab_map[right_tab_name];
					
					thrust::copy(d_t, d_t + n_cnt, d_r);
					thrust::device_free(d_t);	
					result = cudppHashRetrieve(hash_table_handle[hash_ind], thrust::raw_pointer_cast(d_r),
											thrust::raw_pointer_cast(res), n_cnt);					
					if (result != CUDPP_SUCCESS)
						cout << "Failed retrieve " << endl;
	
					thrust::counting_iterator<unsigned int> begin(0);
					uint2_split_left ff(thrust::raw_pointer_cast(res),thrust::raw_pointer_cast(d_s.data()));
					thrust::for_each(begin, begin + n_cnt, ff);	

                        //gather
					if(right->type[r_colInd] == 0) {
						thrust::permutation_iterator<ElementIterator_int,IndexIterator> iter(right->d_columns_int[right->type_index[r_colInd]].begin(), d_s.begin());
                        thrust::copy(iter, iter + n_cnt, c->h_columns_int[c->type_index[c_colInd]].begin() + offset);
                    }
                    else if(right->type[r_colInd] == 1) {
                        thrust::permutation_iterator<ElementIterator_float,IndexIterator> iter(right->d_columns_float[right->type_index[r_colInd]].begin(), d_s.begin());
                        thrust::copy(iter, iter + n_cnt, c->h_columns_float[c->type_index[c_colInd]].begin() + offset);
                    }
                    else { //strings
                        thrust::device_ptr<char> d_tmp1 = thrust::device_malloc<char>(n_cnt*right->char_size[right->type_index[r_colInd]]);
                        str_gather(thrust::raw_pointer_cast(d_s.data()), n_cnt, (void*)right->d_columns_char[right->type_index[r_colInd]],
                                   (void*) thrust::raw_pointer_cast(d_tmp1), right->char_size[right->type_index[r_colInd]]);
                        hipMemcpy( (void*)(c->h_columns_char[c->type_index[c_colInd]] + offset*c->char_size[c->type_index[c_colInd]]), (void*) thrust::raw_pointer_cast(d_tmp1),
                                    c->char_size[c->type_index[c_colInd]] * n_cnt, hipMemcpyDeviceToHost);
                        thrust::device_free(d_tmp1);
                    }
					//cout << "right gathered " << endl;
                }		
				
                op_sel1.pop();		
		    };
		};		
		
	};
	
    while(!op_join.empty()) {
		varNames[op_join.front()]->deAllocOnDevice();
		op_join.pop();
	};	   
	left->deAllocOnDevice();	
	
	for(unsigned int i = 0; i < join_tab_cnt; i++) {
		cudppDestroyHashTable(theCudpp, hash_table_handle[i]);
	};	
	delete [] hash_table_handle;
	
    varNames[s] = c;
    c->mRecCount = tot_count;
    c->maxRecs = tot_count;
	cout << endl << "join count " << tot_count << endl;
    for ( map<string,int>::iterator it=c->columnNames.begin() ; it != c->columnNames.end(); ++it ) {
        setMap[(*it).first] = s;			
	};	 
};


void emit_join(char *s, char *j1, int grp)
{

    statement_count++;
    if (scan_state == 0) {
        if (stat.find(j1) == stat.end()) {
            cout << "Join : couldn't find variable " << j1 << endl;
            exit(1);
        };
        if (stat.find(op_join.front()) == stat.end()) {
            cout << "Join : couldn't find variable " << op_join.front() << endl;
            exit(1);
        };
        stat[s] = statement_count;
        stat[j1] = statement_count;
		while(!op_join.empty()) {
            stat[op_join.front()] = statement_count;			
			op_join.pop();
		};		
        return;
    };


	queue<string> op_m(op_value);
      
    if(check_star_join(j1)) {	   
	    cout << "executing star join !! " << endl;
		star_join(s, j1);
    }
	else {
		if(join_tab_cnt > 1) {
			string tab_name;
			for(unsigned int i = 1; i <= join_tab_cnt; i++) {
	  
				if(i == join_tab_cnt)
					tab_name = s;
				else	 
					tab_name = s + to_string1((long long int)i);
			  
				string j, j2;	  
				if(i == 1) {	  		      
					j2 = op_join.front();
					op_join.pop();
					j = op_join.front();
					op_join.pop();
				}
				else {
					if(!op_join.empty()) {
						j = op_join.front();	
						op_join.pop();
					}	
					else
						j = j1;			  					
					j2 = s + to_string1((long long int)i-1);
				};
				emit_multijoin(tab_name, j, j2, i, s);
				op_value = op_m;
			};	
		}
		else {
			string j2 = op_join.front();	
			op_join.pop();
			emit_multijoin(s, j1, j2, 1, s);
		}; 
    };		
	
    clean_queues();
   
    if(stat[s] == statement_count) {
        varNames[s]->free();
        varNames.erase(s);
    };

    /*if(stat[j1] == statement_count) {
        varNames[j1]->free();
        varNames.erase(j1);
    };
	*/
	

	if(op_join.size()) {
		if(stat[op_join.front()] == statement_count && op_join.front().compare(j1) != 0) {
			varNames[op_join.front()]->free();
			varNames.erase(op_join.front());
		};
	};	
   
}


void emit_multijoin(string s, string j1, string j2, unsigned int tab, char* res_name)
{

//	cout << "j2 " << j2 << endl;
//	cout << "j1 " << j1 << endl;
    

    if(varNames.find(j1) == varNames.end() || varNames.find(j2) == varNames.end()) {
        clean_queues();
		if(varNames.find(j1) == varNames.end())
		    cout << "Couldn't find j1 " << j1 << endl;
		if(varNames.find(j2) == varNames.end())
		    cout << "Couldn't find j2 " << j2 << endl;

        return;
    };

    CudaSet* left = varNames.find(j1)->second;
    CudaSet* right = varNames.find(j2)->second;
	

    queue<string> op_sel;
    queue<string> op_sel_as;
    for(int i=0; i < sel_count; i++) {
        op_sel.push(op_value.front());
        op_value.pop();
        op_sel_as.push(op_value.front());
        op_value.pop();
    };
	
	queue<string> op_sel_s(op_sel);
	queue<string> op_sel_s_as(op_sel_as);
	queue<string> op_g(op_value);	
	
	//cout << "join_col_cnt " << join_col_cnt << endl;			 
	if(tab > 0) {			
	    for(unsigned int z = 0; z < join_tab_cnt - tab; z++) {
			for(unsigned int j = 0; j < join_and_cnt[z]*2 + 2; j++) {
				op_sel_s.push(op_g.front());
				op_sel_s_as.push(op_g.front());						
				op_g.pop();	
			};		
		};
	};
	

    string f1 = op_g.front();
    op_g.pop();
    string f2 = op_g.front();
    op_g.pop();

    cout << "JOIN " << s <<  " " <<  f1 << " " << f2 << " " << getFreeMem() <<  endl;
	
    std::clock_t start1 = std::clock();
	//cout << "creating c with " << op_sel.size() << endl;
	if(tab != join_tab_cnt) {
	//	op_sel_s.push(f1);
	//	op_sel_s.push(f2);
	//	op_sel_s_as.push(f1);
	//	op_sel_s_as.push(f2);
	};	
	
		
    CudaSet* c = new CudaSet(right, left, op_sel_s, op_sel_s_as);

    if (left->mRecCount == 0 || right->mRecCount == 0) {
        c = new CudaSet(left, right, op_sel_s, op_sel_s_as);
        varNames[res_name] = c;
        clean_queues();
        cout << "Join result " << res_name << " : " << c->mRecCount << endl; 		
        return;
    };
	
	//cout << "tab = " << tab << " " << join_tab_cnt << endl;
	if(join_tab_cnt > 1 && tab < join_tab_cnt)
	    c->tmp_table = 1;
	else
        c->tmp_table = 0;	

    unsigned int colInd1, colInd2;
    string tmpstr;
    if (left->columnNames.find(f1) != left->columnNames.end()) {
        colInd1 = (left->columnNames).find(f1)->second;
        if (right->columnNames.find(f2) != right->columnNames.end()) {
            colInd2 = (right->columnNames).find(f2)->second;
        }
        else {
            cout << "Couldn't find column " << f2 << endl;
            exit(0);
        };
    }
    else if (right->columnNames.find(f1) != right->columnNames.end()) {
        colInd2 = (right->columnNames).find(f1)->second;
        tmpstr = f1;
        f1 = f2;
        if (left->columnNames.find(f2) != left->columnNames.end()) {
            colInd1 = (left->columnNames).find(f2)->second;
            f2 = tmpstr;
        }
        else {
            cout << "Couldn't find column " << f2 << endl;
            exit(0);
        };
    }
    else {
        cout << "Couldn't find column " << f1 << endl;
        exit(0);
    };


    if (!((left->type[colInd1] == 0 && right->type[colInd2]  == 0) || (left->type[colInd1] == 2 && right->type[colInd2]  == 2)
            || (left->type[colInd1] == 1 && right->type[colInd2]  == 1 && left->decimal[colInd1] && right->decimal[colInd2]))) {
        cout << "Joins on floats are not supported " << endl;
        exit(0);
    };
    bool decimal_join = 0;
    if (left->type[colInd1] == 1 && right->type[colInd2]  == 1)
        decimal_join = 1;

    set<string> field_names;
    stack<string> exe_type;
    exe_type.push(f2);
    field_names.insert(f2);

    bool str_join = 0;
	unsigned int cnt_r;
    //if join is on strings then add integer columns to left and right tables and modify colInd1 and colInd2

    if (right->type[colInd2]  == 2) {
        str_join = 1;
        right->d_columns_int.push_back(thrust::device_vector<int_type>());
        for(unsigned int i = 0; i < right->segCount; i++) {
            right->add_hashed_strings(f2, i, right->d_columns_int.size()-1);
        };
		cnt_r = right->d_columns_int[right->d_columns_int.size()-1].size();
    };

    // need to allocate all right columns
    queue<string> cc;
    unsigned int rcount;
    curr_segment = 10000000;


    queue<string> op_vd(op_g);
    queue<string> op_alt(op_sel);
    unsigned int jc = join_and_cnt[join_tab_cnt - tab];
    while(jc) {
        jc--;
        op_vd.pop();
        op_alt.push(op_vd.front());
        op_vd.pop();
    };

	
	string empty = "";
	if(right->not_compressed) {
	    queue<string> op_alt1;
		op_alt1.push(f2);
		cnt_r = load_queue(op_alt1, right, str_join, empty, rcount);
	}
	else {
		cnt_r = load_queue(op_alt, right, str_join, f2, rcount);
	};	
	
    if(str_join) {
        colInd2 = right->mColumnCount+1;
        right->type_index[colInd2] = right->d_columns_int.size()-1;
    };


    //here we need to make sure that right column is ordered. If not then we order it and keep the permutation
	
	thrust::device_ptr<unsigned long long int> d_col_r((unsigned long long int*)thrust::raw_pointer_cast(right->d_columns_int[right->type_index[colInd2]].data()));					
	
    bool sorted;
	
	if(str_join) {
	    sorted = thrust::is_sorted(d_col_r, d_col_r + cnt_r);
	}	
    else if(!decimal_join) {        
		sorted = thrust::is_sorted(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r);
	}	
    else
        sorted = thrust::is_sorted(right->d_columns_float[right->type_index[colInd2]].begin(), right->d_columns_float[right->type_index[colInd2]].begin() + cnt_r);


			
    if(!sorted) {

	    typedef thrust::device_ptr<unsigned int> IndexIterator2;
        queue<string> ss(op_sel);
		thrust::device_ptr<unsigned int> v = thrust::device_malloc<unsigned int>(cnt_r);
        thrust::sequence(v, v + cnt_r, 0, 1);	

		unsigned int max_c	= max_char(right);
        unsigned int mm;
        if(max_c > 8)
            mm = max_c;
        else
            mm = 8;        
        void* d;
        CUDA_SAFE_CALL(hipMalloc((void **) &d, cnt_r*mm)); 		

		if(str_join) {
			thrust::sort_by_key(d_col_r, d_col_r + cnt_r, v);
		}	
		else {
			thrust::sort_by_key(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r, v);
		};				

        unsigned int i;
        while(!ss.empty()) {
            if (right->columnNames.find(ss.front()) != right->columnNames.end()) {
                i = right->columnNames[ss.front()];

                if(i != colInd2) {
				
					if(right->not_compressed) {		
					    queue<string> op_alt1;
						op_alt1.push(ss.front());	
						cnt_r = load_queue(op_alt1, right, str_join, empty, rcount);
						
					};							

                    if(right->type[i] == 0) {
					    thrust::device_ptr<int_type> d_tmp((int_type*)d);
                        thrust::gather(v, v+cnt_r, right->d_columns_int[right->type_index[i]].begin(), d_tmp);
                        thrust::copy(d_tmp, d_tmp + cnt_r, right->d_columns_int[right->type_index[i]].begin());
                    }
                    else if(right->type[i] == 1) {
					    thrust::device_ptr<float_type> d_tmp((float_type*)d);
                        thrust::gather(v, v+cnt_r, right->d_columns_float[right->type_index[i]].begin(), d_tmp);
                        thrust::copy(d_tmp, d_tmp + cnt_r, right->d_columns_float[right->type_index[i]].begin());
                    }
                    else {
					    thrust::device_ptr<char> d_tmp((char*)d);
                        str_gather(thrust::raw_pointer_cast(v), cnt_r, (void*)right->d_columns_char[right->type_index[i]], (void*) thrust::raw_pointer_cast(d_tmp), right->char_size[right->type_index[i]]);
                        hipMemcpy( (void*)right->d_columns_char[right->type_index[i]], (void*) thrust::raw_pointer_cast(d_tmp), cnt_r*right->char_size[right->type_index[i]], hipMemcpyDeviceToDevice);
                    };
                };
            };
            ss.pop();
        };
		thrust::device_free(v);
		hipFree(d);
    }
    else {
		if(right->not_compressed) {
			queue<string> op_alt1;
			while(!op_alt.empty()) {
				if(f2.compare(op_alt.front())) {
					if (right->columnNames.find(op_alt.front()) != right->columnNames.end()) {
						op_alt1.push(op_alt.front());
					};	
				};	
				op_alt.pop();	
			};
			cnt_r = load_queue(op_alt1, right, str_join, empty, rcount);			
		};
    };	
	
   

    while(!cc.empty())
        cc.pop();

    if (left->type[colInd1]  == 2) {
        left->d_columns_int.push_back(thrust::device_vector<int_type>());
    }
    else {
        cc.push(f1);
        allocColumns(left, cc);
    };
	
	
	left->oldRecCount = left->mRecCount;
	
    unsigned int cnt_l, res_count, tot_count = 0, offset = 0, k = 0;
    queue<string> lc(cc);
    curr_segment = 10000000;	
	thrust::device_vector<int> p_tmp;	
	thrust::device_vector<unsigned int> v_l(left->maxRecs);		
	MGPU_MEM(int) aIndicesDevice, bIndicesDevice;			

    for (unsigned int i = 0; i < left->segCount; i++) {

        cout << "segment " << i <<  '\xd';
		//cout << "segment " << i <<  endl;
				
        cnt_l = 0;
		
        if (left->type[colInd1]  != 2) {
            copyColumns(left, lc, i, cnt_l);		
        }
        else {
		    //left->d_columns_int.resize(0);
            left->add_hashed_strings(f1, i, left->d_columns_int.size()-1);
        };		
		
		
	    if(left->prm.empty()) {
            //copy all records
			if (left->type[colInd1]  != 2) 
				cnt_l = left->mRecCount;
			else
				cnt_l = left->d_columns_int[left->d_columns_int.size()-1].size();
        }
        else {
            cnt_l = left->prm_count[i];
        };
		
	
        if (cnt_l) {
 
            unsigned int idx;
            if(!str_join)
                idx = left->type_index[colInd1];
            else
                idx = left->d_columns_int.size()-1;				
				
			// sort the left index column, save the permutation vector, it might be needed later
			
			thrust::sequence(v_l.begin(), v_l.begin() + cnt_l,0,1);
			
			thrust::device_ptr<unsigned long long int> d_col((unsigned long long int*)thrust::raw_pointer_cast(left->d_columns_int[idx].data()));					
			bool do_sort = 1;
			if(!left->sorted_fields.empty()) {
				if(left->sorted_fields.front() == idx) {
					do_sort = 0;
				};	
			}
			if(do_sort)
				thrust::sort_by_key(d_col, d_col + cnt_l, v_l.begin());			    						
		    //cout << endl << "j1 " << getFreeMem() << endl;
			//cout << "join " << cnt_l << ":" << cnt_r << " " << join_type.front() << endl;
			//cout << "MIN MAX " << left->d_columns_int[idx][0] << " - " << left->d_columns_int[idx][cnt_l-1] << " : " << right->d_columns_int[right->type_index[colInd2]][0] << "-" << right->d_columns_int[right->type_index[colInd2]][cnt_r-1] << endl; 
			
			
			char join_kind = join_type.front();
			join_type.pop();			

			
			if (left->type[colInd1] == 2) {
					res_count = RelationalJoin<MgpuJoinKindInner>(thrust::raw_pointer_cast(d_col), cnt_l,
									thrust::raw_pointer_cast(d_col_r), cnt_r,
									&aIndicesDevice, &bIndicesDevice,
									mgpu::less<unsigned long long int>(), *context);
														
			}
			else {

				if (join_kind == 'I')
					res_count = RelationalJoin<MgpuJoinKindInner>(thrust::raw_pointer_cast(left->d_columns_int[idx].data()), cnt_l,
									thrust::raw_pointer_cast(right->d_columns_int[right->type_index[colInd2]].data()), cnt_r,
									&aIndicesDevice, &bIndicesDevice,
									mgpu::less<int_type>(), *context);
				else if(join_kind == 'L')					
					res_count = RelationalJoin<MgpuJoinKindLeft>(thrust::raw_pointer_cast(left->d_columns_int[idx].data()), cnt_l,
									thrust::raw_pointer_cast(right->d_columns_int[right->type_index[colInd2]].data()), cnt_r,
									&aIndicesDevice, &bIndicesDevice,
									mgpu::less<int_type>(), *context);
				else if(join_kind == 'R')					
					res_count = RelationalJoin<MgpuJoinKindRight>(thrust::raw_pointer_cast(left->d_columns_int[idx].data()), cnt_l,
									thrust::raw_pointer_cast(right->d_columns_int[right->type_index[colInd2]].data()), cnt_r,
									&aIndicesDevice, &bIndicesDevice,
									mgpu::less<int_type>(), *context);
				else if(join_kind == 'O')					
					res_count = RelationalJoin<MgpuJoinKindOuter>(thrust::raw_pointer_cast(left->d_columns_int[idx].data()), cnt_l,
									thrust::raw_pointer_cast(right->d_columns_int[right->type_index[colInd2]].data()), cnt_r,
									&aIndicesDevice, &bIndicesDevice,
									mgpu::less<int_type>(), *context);								
			};	

		
			//cout << "total " << res_count << endl;
			int* r1 = aIndicesDevice->get(); 
            thrust::device_ptr<int> d_res1((int*)r1);
			int* r2 = bIndicesDevice->get(); 
			thrust::device_ptr<int> d_res2((int*)r2);		
		
			if(res_count) {						
				p_tmp.resize(res_count);
				thrust::sequence(p_tmp.begin(), p_tmp.end(),-1);
				thrust::gather_if(d_res1, d_res1+res_count, d_res1, v_l.begin(), p_tmp.begin(), is_positive());		
			};		
			
		
			//std::cout<< endl << "join time " <<  ( ( std::clock() - start3 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;            
	
            // check if the join is a multicolumn join
			unsigned int mul_cnt = join_and_cnt[join_tab_cnt - tab];
			while(mul_cnt) {			
			    		    
                mul_cnt--;
                string f3 = op_g.front();
                op_g.pop();
                string f4 = op_g.front();
                op_g.pop();
				
				//cout << "ADDITIONAL COL JOIN " << f3 << " " << f4 << " " << getFreeMem() << endl;
			
                queue<string> rc;
                rc.push(f3);

                allocColumns(left, rc);
                copyColumns(left, rc, i, cnt_l);
                rc.pop();	
				
		        void* temp;
				CUDA_SAFE_CALL(hipMalloc((void **) &temp, res_count*float_size));
		        void* temp1;
				CUDA_SAFE_CALL(hipMalloc((void **) &temp1, res_count*float_size));
				hipMemset(temp,0,res_count);
				hipMemset(temp1,0,res_count);

                				
                if (res_count) {
				    unsigned int new_cnt;
                    unsigned int colInd3 = (left->columnNames).find(f3)->second;
                    unsigned int colInd4 = (right->columnNames).find(f4)->second;    
					thrust::device_ptr<bool> d_add = thrust::device_malloc<bool>(res_count);
					
                    if (left->type[colInd3] == 1 && right->type[colInd4]  == 1) {
					
                        if(right->d_columns_float[right->type_index[colInd4]].size() == 0)
                            unsigned int cnt_r = load_queue(rc, right, 0, f4, rcount);									
		                
						thrust::device_ptr<float_type> d_tmp((float_type*)temp);	
						thrust::device_ptr<float_type> d_tmp1((float_type*)temp1);	
		                thrust::gather_if(p_tmp.begin(), p_tmp.end(), p_tmp.begin(), left->d_columns_float[left->type_index[colInd3]].begin(), d_tmp, is_positive());						
                        thrust::gather_if(d_res2, d_res2+res_count, d_res2, right->d_columns_float[right->type_index[colInd4]].begin(), d_tmp1, is_positive());																		
						thrust::transform(d_tmp, d_tmp+res_count, d_tmp1, d_add, float_equal_to());
                    }
                    else {
                        if(right->d_columns_int[right->type_index[colInd4]].size() == 0) {
                            unsigned int cnt_r = load_queue(rc, right, 0, f4, rcount);		
			            };                        					
						thrust::device_ptr<int_type> d_tmp((int_type*)temp);	
						thrust::device_ptr<int_type> d_tmp1((int_type*)temp1);	
		                thrust::gather_if(p_tmp.begin(), p_tmp.end(), p_tmp.begin(), left->d_columns_int[left->type_index[colInd3]].begin(), d_tmp, is_positive());						
                        thrust::gather_if(d_res2, d_res2+res_count, d_res2, right->d_columns_int[right->type_index[colInd4]].begin(), d_tmp1, is_positive());												
                        thrust::transform(d_tmp, d_tmp+res_count, d_tmp1, d_add, thrust::equal_to<int_type>());
                    };

					if (join_kind == 'I') {  // result count changes only in case of an inner join
						new_cnt = thrust::count(d_add, d_add+res_count, 1);	
						thrust::stable_partition(d_res2, d_res2 + res_count, d_add, thrust::identity<unsigned int>());
						thrust::stable_partition(p_tmp.begin(), p_tmp.end(), d_add, thrust::identity<unsigned int>());
						thrust::device_free(d_add);						
						res_count = new_cnt;
					}
					else { //otherwise we consider it a valid left join result with non-nulls on the left side and nulls on the right side
						thrust::transform(d_res2, d_res2 + res_count, d_add , d_res2, set_minus());	
					};
                };
				hipFree(temp);
				hipFree(temp1);				
            };			
            
            tot_count = tot_count + res_count;			
		
			
            if(res_count) {			

                offset = c->mRecCount;
                if(i == 0 && left->segCount != 1) {
                    c->reserve(res_count*(left->segCount+1));
				};	
                c->resize_join(res_count);	
				
                queue<string> op_sel1(op_sel_s);
                unsigned int colInd, c_colInd;
				
								
		        void* temp;
				unsigned int max_c = max_char(c);
		
				if(max_c > float_size) {
					CUDA_SAFE_CALL(hipMalloc((void **) &temp, res_count*max_c));
				}	
				else
					CUDA_SAFE_CALL(hipMalloc((void **) &temp, res_count*float_size));
					
               
                while(!op_sel1.empty()) {
				
			
                    while(!cc.empty())
                        cc.pop();

                    cc.push(op_sel1.front());
					if(c->columnNames.find(op_sel1.front()) != c->columnNames.end()) {
                        c_colInd = c->columnNames[op_sel1.front()];						
					};	
					
					if(left->columnNames.find(op_sel1.front()) !=  left->columnNames.end()) {
                        // copy field's segment to device, gather it and copy to the host
                        colInd = left->columnNames[op_sel1.front()];						
                    
                        reset_offsets();	
                        allocColumns(left, cc);
                        copyColumns(left, cc, i, k);//possible that in some cases a join column would be copied to device twice					
						
                        //gather
                        if(left->type[colInd] == 0) {
							thrust::device_ptr<int_type> d_tmp((int_type*)temp);	
							thrust::sequence(d_tmp, d_tmp+res_count,0,0);
                            //thrust::permutation_iterator<ElementIterator_int,IndexIterator> iter(left->d_columns_int[left->type_index[colInd]].begin(), p_tmp.begin());
							thrust::gather_if(p_tmp.begin(), p_tmp.begin() + res_count, p_tmp.begin(), left->d_columns_int[left->type_index[colInd]].begin(), d_tmp, is_positive());							
							thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_int[c->type_index[c_colInd]].begin() + offset);							
                        }
                        else if(left->type[colInd] == 1) {
						    thrust::device_ptr<float_type> d_tmp((float_type*)temp);	
							thrust::sequence(d_tmp, d_tmp+res_count,0,0);
                            //thrust::permutation_iterator<ElementIterator_float,IndexIterator> iter(left->d_columns_float[left->type_index[colInd]].begin(), p_tmp.begin());
							thrust::gather_if(p_tmp.begin(), p_tmp.begin() + res_count, p_tmp.begin(), left->d_columns_float[left->type_index[colInd]].begin(), d_tmp, is_positive());
                            thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_float[c->type_index[c_colInd]].begin() + offset);
                        }
                        else { //strings
                            thrust::device_ptr<char> d_tmp((char*)temp);							
						
							thrust::fill(d_tmp, d_tmp+res_count*left->char_size[left->type_index[colInd]],0);
                            str_gather(thrust::raw_pointer_cast(p_tmp.data()), res_count, (void*)left->d_columns_char[left->type_index[colInd]],
                                       (void*) thrust::raw_pointer_cast(d_tmp), left->char_size[left->type_index[colInd]]);
									   
									   
                            hipMemcpy( (void*)&c->h_columns_char[c->type_index[c_colInd]][offset*c->char_size[c->type_index[c_colInd]]], (void*) thrust::raw_pointer_cast(d_tmp),
                                        c->char_size[c->type_index[c_colInd]] * res_count, hipMemcpyDeviceToHost);
                        };
						if(colInd != colInd1)
							left->deAllocColumnOnDevice(colInd);

                    }
                    else if(right->columnNames.find(op_sel1.front()) !=  right->columnNames.end()) {
                        colInd = right->columnNames[op_sel1.front()];

                        //gather
                        if(right->type[colInd] == 0) {			
							thrust::device_ptr<int_type> d_tmp((int_type*)temp);	
							thrust::sequence(d_tmp, d_tmp+res_count,0,0);
                            //thrust::permutation_iterator<ElementIterator_int,IndexIterator1> iter(right->d_columns_int[right->type_index[colInd]].begin(), d_res2);
							thrust::gather_if(d_res2, d_res2 + res_count, d_res2, right->d_columns_int[right->type_index[colInd]].begin(), d_tmp, is_positive());
                            thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_int[c->type_index[c_colInd]].begin() + offset);
                        }
                        else if(right->type[colInd] == 1) {
   						    thrust::device_ptr<float_type> d_tmp((float_type*)temp);	
							thrust::sequence(d_tmp, d_tmp+res_count,0,0);
                            //thrust::permutation_iterator<ElementIterator_float,IndexIterator1> iter(right->d_columns_float[right->type_index[colInd]].begin(), d_res2);
							thrust::gather_if(d_res2, d_res2 + res_count, d_res2, right->d_columns_float[right->type_index[colInd]].begin(), d_tmp, is_positive());
                            thrust::copy(d_tmp, d_tmp + res_count, c->h_columns_float[c->type_index[c_colInd]].begin() + offset);							
                        }
                        else { //strings						
						
	                        thrust::device_ptr<char> d_tmp((char*)temp);
							thrust::sequence(d_tmp, d_tmp+res_count*right->char_size[right->type_index[colInd]],0,0);					
                            str_gather(thrust::raw_pointer_cast(d_res2), res_count, (void*)right->d_columns_char[right->type_index[colInd]],
                                       (void*) thrust::raw_pointer_cast(d_tmp), right->char_size[right->type_index[colInd]]);																   						   										

							
                            hipMemcpy( (void*)&c->h_columns_char[c->type_index[c_colInd]][offset*c->char_size[c->type_index[c_colInd]]], (void*) thrust::raw_pointer_cast(d_tmp),
                                        c->char_size[c->type_index[c_colInd]] * res_count, hipMemcpyDeviceToHost);		

					
                        };						
                    }
                    else {
                        //cout << "Couldn't find field " << op_sel1.front() << endl;
                        //exit(0);
                    };
                    op_sel1.pop();					
                };
				hipFree(temp);				
            };	
        };		
    };	
	
		
    left->deAllocOnDevice();
    right->deAllocOnDevice();
    c->deAllocOnDevice();	

    unsigned int i = 0;	
    while(!col_aliases.empty() && tab == join_tab_cnt) {
        c->columnNames[col_aliases.front()] = i;
        col_aliases.pop();
        i++;
    };

    varNames[s] = c;
    c->mRecCount = tot_count;
    c->maxRecs = tot_count;
	cout << endl << "join count " << tot_count << endl;
    for ( map<string,int>::iterator it=c->columnNames.begin() ; it != c->columnNames.end(); ++it )
        setMap[(*it).first] = s;
		

    if(right->tmp_table == 1) {
        right->free();
        varNames.erase(j2);
	}
	else {
		if(stat[j2] == statement_count) {
			right->free();
			varNames.erase(j2);
		};
		
	};
	
	if(stat[j1] == statement_count) {
        left->free();
        varNames.erase(j1);
    };	
	
    std::cout<< "join time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;	
}


void order_on_host(CudaSet *a, CudaSet* b, queue<string> names, stack<string> exe_type, stack<string> exe_value)
{
    unsigned int tot = 0;
    if(!a->not_compressed) { //compressed
        allocColumns(a, names);

        unsigned int c = 0;
        if(a->prm_count.size())	{
            for(unsigned int i = 0; i < a->prm.size(); i++)
                c = c + a->prm_count[i];
        }
        else
            c = a->mRecCount;
        a->mRecCount = 0;
        a->resize(c);

        unsigned int cnt = 0;
        for(unsigned int i = 0; i < a->segCount; i++) {
            copyColumns(a, names, (a->segCount - i) - 1, cnt);	//uses segment 1 on a host	to copy data from a file to gpu
            if (a->mRecCount) {
                a->CopyToHost((c - tot) - a->mRecCount, a->mRecCount);
                tot = tot + a->mRecCount;
            };
        };
    }
    else
        tot = a->mRecCount;

    b->resize(tot); //resize host arrays
    a->mRecCount = tot;

    unsigned int* permutation = new unsigned int[a->mRecCount];
    thrust::sequence(permutation, permutation + a->mRecCount);

    unsigned int maxSize =  a->mRecCount;
    char* temp;
    unsigned int max_c = max_char(a);

    if(max_c > float_size)
        temp = new char[maxSize*max_c];
    else
        temp = new char[maxSize*float_size];

    // sort on host

    for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {
        int colInd = (a->columnNames).find(exe_type.top())->second;

        if ((a->type)[colInd] == 0)
            update_permutation_host(a->h_columns_int[a->type_index[colInd]].data(), permutation, a->mRecCount, exe_value.top(), (int_type*)temp);
        else if ((a->type)[colInd] == 1)
            update_permutation_host(a->h_columns_float[a->type_index[colInd]].data(), permutation, a->mRecCount,exe_value.top(), (float_type*)temp);
        else {
            update_permutation_char_host(a->h_columns_char[a->type_index[colInd]], permutation, a->mRecCount, exe_value.top(), b->h_columns_char[b->type_index[colInd]], a->char_size[a->type_index[colInd]]);
        };
    };

    for (unsigned int i = 0; i < a->mColumnCount; i++) {
        if ((a->type)[i] == 0) {
            apply_permutation_host(a->h_columns_int[a->type_index[i]].data(), permutation, a->mRecCount, b->h_columns_int[b->type_index[i]].data());
        }
        else if ((a->type)[i] == 1)
            apply_permutation_host(a->h_columns_float[a->type_index[i]].data(), permutation, a->mRecCount, b->h_columns_float[b->type_index[i]].data());
        else {
            apply_permutation_char_host(a->h_columns_char[a->type_index[i]], permutation, a->mRecCount, b->h_columns_char[b->type_index[i]], a->char_size[a->type_index[i]]);
        };
    };
	
    delete [] temp;
    delete [] permutation;
}



void emit_order(char *s, char *f, int e, int ll)
{
    if(ll == 0)
        statement_count++;

    if (scan_state == 0 && ll == 0) {
        if (stat.find(f) == stat.end()) {
            cout << "Order : couldn't find variable " << f << endl;
            exit(1);
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        return;
    };

    if(varNames.find(f) == varNames.end() ) {
        clean_queues();
        return;
    };

    CudaSet* a = varNames.find(f)->second;


    if (a->mRecCount == 0)	{
        if(varNames.find(s) == varNames.end())
            varNames[s] = new CudaSet(0,1);
        else {
            CudaSet* c = varNames.find(s)->second;
            c->mRecCount = 0;
        };
        return;
    };

    stack<string> exe_type, exe_value;

    cout << "order: " << s << " " << f << endl;


    for(int i=0; !op_type.empty(); ++i, op_type.pop(),op_value.pop()) {
        if ((op_type.front()).compare("NAME") == 0) {
            exe_type.push(op_value.front());
            exe_value.push("ASC");
        }
        else {
            exe_type.push(op_type.front());
            exe_value.push(op_value.front());
        };
    };

    stack<string> tp(exe_type);
    queue<string> op_vx;
    while (!tp.empty()) {
        op_vx.push(tp.top());
        tp.pop();
    };

    queue<string> names;
    for ( map<string,int>::iterator it=a->columnNames.begin() ; it != a->columnNames.end(); ++it )
        names.push((*it).first);

    CudaSet *b = a->copyDeviceStruct();

    //lets find out if our data set fits into a GPU
    size_t mem_available = getFreeMem();
    size_t rec_size = 0;
    for(unsigned int i = 0; i < a->mColumnCount; i++) {
        if(a->type[i] == 0)
            rec_size = rec_size + int_size;
        else if(a->type[i] == 1)
            rec_size = rec_size + float_size;
        else
            rec_size = rec_size + a->char_size[a->type_index[i]];
    };
    bool fits;
    if (rec_size*a->mRecCount > (mem_available/2)) // doesn't fit into a GPU
        fits = 0;
    else fits = 1;

    if(!fits) {
        order_on_host(a, b, names, exe_type, exe_value);
    }
    else {
        // initialize permutation to [0, 1, 2, ... ,N-1]
        thrust::device_ptr<unsigned int> permutation = thrust::device_malloc<unsigned int>(a->mRecCount);
        thrust::sequence(permutation, permutation+(a->mRecCount));

        unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation);

        unsigned int maxSize =  a->mRecCount;
        void* temp;
        unsigned int max_c = max_char(a);

        if(max_c > float_size)
            CUDA_SAFE_CALL(hipMalloc((void **) &temp, maxSize*max_c));
        else
            CUDA_SAFE_CALL(hipMalloc((void **) &temp, maxSize*float_size));

        varNames[setMap[exe_type.top()]]->oldRecCount = varNames[setMap[exe_type.top()]]->mRecCount;


        unsigned int rcount;

        a->mRecCount = load_queue(names, a, 1, op_vx.front(), rcount);

        varNames[setMap[exe_type.top()]]->mRecCount = varNames[setMap[exe_type.top()]]->oldRecCount;
        //unsigned int str_count = 0;

        for(int i=0; !exe_type.empty(); ++i, exe_type.pop(),exe_value.pop()) {
            int colInd = (a->columnNames).find(exe_type.top())->second;
            if ((a->type)[colInd] == 0)
                update_permutation(a->d_columns_int[a->type_index[colInd]], raw_ptr, a->mRecCount, exe_value.top(), (int_type*)temp);
            else if ((a->type)[colInd] == 1)
                update_permutation(a->d_columns_float[a->type_index[colInd]], raw_ptr, a->mRecCount,exe_value.top(), (float_type*)temp);
            else {
                update_permutation_char(a->d_columns_char[a->type_index[colInd]], raw_ptr, a->mRecCount, exe_value.top(), (char*)temp, a->char_size[a->type_index[colInd]]);
                //update_permutation(a->d_columns_int[int_col_count+str_count], raw_ptr, a->mRecCount, exe_value.top(), (int_type*)temp);
                //str_count++;
            };
        };

        b->resize(a->mRecCount); //resize host arrays
        b->mRecCount = a->mRecCount;
        //str_count = 0;

        for (unsigned int i = 0; i < a->mColumnCount; i++) {
            if ((a->type)[i] == 0)
                apply_permutation(a->d_columns_int[a->type_index[i]], raw_ptr, a->mRecCount, (int_type*)temp);
            else if ((a->type)[i] == 1)
                apply_permutation(a->d_columns_float[a->type_index[i]], raw_ptr, a->mRecCount, (float_type*)temp);
            else {				
                apply_permutation_char(a->d_columns_char[a->type_index[i]], raw_ptr, a->mRecCount, (char*)temp, a->char_size[a->type_index[i]]);
				//str_count++;
            };
        };

        for(unsigned int i = 0; i < a->mColumnCount; i++) {
            switch(a->type[i]) {
            case 0 :
                thrust::copy(a->d_columns_int[a->type_index[i]].begin(), a->d_columns_int[a->type_index[i]].begin() + a->mRecCount, b->h_columns_int[b->type_index[i]].begin());
                break;
            case 1 :
                thrust::copy(a->d_columns_float[a->type_index[i]].begin(), a->d_columns_float[a->type_index[i]].begin() + a->mRecCount, b->h_columns_float[b->type_index[i]].begin());
                break;
            default :
                hipMemcpy(b->h_columns_char[b->type_index[i]], a->d_columns_char[a->type_index[i]], a->char_size[a->type_index[i]]*a->mRecCount, hipMemcpyDeviceToHost);
            }
        };

        b->deAllocOnDevice();
        a->deAllocOnDevice();


        thrust::device_free(permutation);
        hipFree(temp);
    };

    varNames[s] = b;
    b->segCount = 1;
    b->not_compressed = 1;

    if(stat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(f);
    };
}


void emit_select(char *s, char *f, int ll)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end()) {
            cout << "Select : couldn't find variable " << f << endl;
            exit(1);
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        return;
    };


    if(varNames.find(f) == varNames.end()) {
        clean_queues();
		cout << "Couldn't find " << f << endl;
        return;
    };



    queue<string> op_v1(op_value);
    while(op_v1.size() > ll)
        op_v1.pop();


    stack<string> op_v2;
    queue<string> op_v3;

    for(int i=0; i < ll; ++i) {
        op_v2.push(op_v1.front());
        op_v3.push(op_v1.front());
        op_v1.pop();
    };

    CudaSet *a;
	if(varNames.find(f) != varNames.end())
		a = varNames.find(f)->second;
	else {
		cout << "Couldn't find " << f  << endl;
		exit(0);
    };	
	
    if(a->mRecCount == 0) {
        CudaSet *c;
        c = new CudaSet(0,1);
        varNames[s] = c;
        clean_queues();
		cout << "SELECT " << s << " count : 0,  Mem " << getFreeMem() << endl;
        return;
    };

    cout << "SELECT " << s << " " << f << " " << getFreeMem() << endl;
    std::clock_t start1 = std::clock();

    // here we need to determine the column count and composition

    queue<string> op_v(op_value);
    queue<string> op_vx;
    set<string> field_names;
    map<string,string> aliases;
    string tt;
	

	//cout << "colsize " << a->columnNames.size() << endl;
	
    while(!op_v.empty()) {
        if(a->columnNames.find(op_v.front()) != a->columnNames.end()) {          
			tt = op_v.front();
			if(!op_v.empty()) {
				op_v.pop();
				if(!op_v.empty()) {
					if(a->columnNames.find(op_v.front()) == a->columnNames.end()) {
						if(aliases.count(tt) == 0) {
							aliases[tt] = op_v.front();				
						};	
					}
					else {
						if (!op_v.empty()) {
							while(a->columnNames.find(op_v.front()) == a->columnNames.end())
								op_v.pop();			
						};		
					}; 									
				};	
			};
		};	
		if(!op_v.empty())
			op_v.pop();
	};	
	
	op_v = op_value;
	while(!op_v.empty()) {
		if(a->columnNames.find(op_v.front()) != a->columnNames.end()) {
			field_names.insert(op_v.front());
		};	
		op_v.pop();
	};
	


    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it)  {
        op_vx.push(*it);
    };
	
    // find out how many columns a new set will have
    queue<string> op_t(op_type);
    int_type col_count = 0;

    for(int i=0; !op_t.empty(); ++i, op_t.pop())
        if((op_t.front()).compare("emit sel_name") == 0)
            col_count++;

    CudaSet *b, *c;

    curr_segment = 10000000;
	if(a->segCount <= 1)
		setSegments(a, op_vx);
    allocColumns(a, op_vx);
	
    unsigned int cycle_count;
    if(!a->prm.empty())
        cycle_count = varNames[setMap[op_value.front()]]->segCount;
    else
        cycle_count = a->segCount;

    unsigned long long int ol_count = a->mRecCount;
	unsigned int cnt;
    //varNames[setMap[op_value.front()]]->oldRecCount = varNames[setMap[op_value.front()]]->mRecCount;
    a->oldRecCount = a->mRecCount;
    b = new CudaSet(0, col_count);
    bool b_set = 0, c_set = 0;

    unsigned int long long tmp_size = a->mRecCount;
    if(a->segCount > 1)
        tmp_size = a->maxRecs;		
    
    vector<thrust::device_vector<int_type> > distinct_val; //keeps array of DISTINCT values for every key
    vector<thrust::device_vector<int_type> > distinct_hash; //keeps array of DISTINCT values for every key
    vector<thrust::device_vector<int_type> > distinct_tmp;

    for(unsigned int i = 0; i < distinct_cnt; i++) {
        distinct_tmp.push_back(thrust::device_vector<int_type>(tmp_size));
        distinct_val.push_back(thrust::device_vector<int_type>());
        distinct_hash.push_back(thrust::device_vector<int_type>());
    };
	

// find out how many string columns we have. Add int_type columns to store string hashes for sort/groupby ops.
    stack<string> op_s = op_v2;
    int_col_count = a->d_columns_int.size();

    while(!op_s.empty()) {
        int colInd = (a->columnNames).find(op_s.top())->second;		
        if (a->type[colInd] == 2) {
            a->d_columns_int.push_back(thrust::device_vector<int_type>());
        };
        op_s.pop();
    };
	

    unsigned int s_cnt;
    bool one_liner;

    for(unsigned int i = 0; i < cycle_count; i++) {          // MAIN CYCLE
        cout << "segment " << i << " select mem " << getFreeMem() << '\xd';
		
        cnt = 0;
        copyColumns(a, op_vx, i, cnt);		
        reset_offsets();
        op_s = op_v2;
        s_cnt = 0;
		

        while(!op_s.empty()) {

            int colInd = (a->columnNames).find(op_s.top())->second;
            if (a->type[colInd] == 2) {
                a->d_columns_int[int_col_count + s_cnt].resize(0);
                a->add_hashed_strings(op_s.top(), i, int_col_count + s_cnt);
                s_cnt++;
            };
            op_s.pop();
        };

        if(a->mRecCount) {
            if (ll != 0) {
                order_inplace(a,op_v2,field_names);
                a->GroupBy(op_v2, int_col_count);
            };			
						
            select(op_type,op_value,op_nums, op_nums_f,a,b, distinct_tmp, one_liner);			
	
            if(!b_set) {
                for ( map<string,int>::iterator it=b->columnNames.begin() ; it != b->columnNames.end(); ++it )
                    setMap[(*it).first] = s;
                b_set = 1;
                unsigned int old_cnt = b->mRecCount;
                b->mRecCount = 0;
                b->resize(varNames[setMap[op_vx.front()]]->maxRecs);
                b->mRecCount = old_cnt;
            };			

            if (!c_set) {
                c = new CudaSet(0, col_count);
                create_c(c,b);
                c_set = 1;
            };

            if (ll != 0 && cycle_count > 1  ) {
                add(c,b,op_v3, aliases, distinct_tmp, distinct_val, distinct_hash, a);
            }
            else {
                //copy b to c
                unsigned int c_offset = c->mRecCount;
                c->resize(b->mRecCount);
                for(unsigned int j=0; j < b->mColumnCount; j++) {
                    if (b->type[j] == 0) {
                        thrust::copy(b->d_columns_int[b->type_index[j]].begin(), b->d_columns_int[b->type_index[j]].begin() + b->mRecCount, c->h_columns_int[c->type_index[j]].begin() + c_offset);
                    }
                    else if (b->type[j] == 1) {
                        thrust::copy(b->d_columns_float[b->type_index[j]].begin(), b->d_columns_float[b->type_index[j]].begin() + b->mRecCount, c->h_columns_float[c->type_index[j]].begin() + c_offset);
                    }
                    else {
                        hipMemcpy((void*)(thrust::raw_pointer_cast(c->h_columns_char[c->type_index[j]] + b->char_size[b->type_index[j]]*c_offset)), (void*)thrust::raw_pointer_cast(b->d_columns_char[b->type_index[j]]),
                                   b->char_size[b->type_index[j]] * b->mRecCount, hipMemcpyDeviceToHost);
                    };
                };

            };
        };
    };

    a->mRecCount = ol_count;
    a->mRecCount = a->oldRecCount;
    a->deAllocOnDevice();
    b->deAllocOnDevice();

    if (ll != 0) {
        count_avg(c, distinct_hash);
    }
    else {
        if(one_liner) {
            count_simple(c);
        };
    };

    reset_offsets();
    c->maxRecs = c->mRecCount;
    c->name = s;
    c->keep = 1;

    for ( map<string,int>::iterator it=c->columnNames.begin() ; it != c->columnNames.end(); ++it ) {
        setMap[(*it).first] = s;
    };

    cout << endl << "final select " << c->mRecCount << endl;
    clean_queues();

    varNames[s] = c;
    b->free();
    varNames[s]->keep = 1;

    if(stat[s] == statement_count) {
        varNames[s]->free();
        varNames.erase(s);
    };

    if(stat[f] == statement_count && a->keep == 0) {
        a->free();
        varNames.erase(f);
    };
    std::cout<< "select time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';
}


void emit_filter(char *s, char *f, int e)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(f) == stat.end()) {
            cout << "Filter : couldn't find variable " << f << endl;
            exit(1);
        };
        stat[s] = statement_count;
        stat[f] = statement_count;
        clean_queues();
        return;
    };

    if(varNames.find(f) == varNames.end()) {
        clean_queues();
        return;
    };

    CudaSet *a, *b;

    a = varNames.find(f)->second;
    a->name = f;
    std::clock_t start1 = std::clock();

    if(a->mRecCount == 0) {
        b = new CudaSet(0,1);
    }
    else {
        cout << "FILTER " << s << " " << f << " " << getFreeMem() << endl;

        b = a->copyDeviceStruct();
        b->name = s;
		b->sorted_fields = a->sorted_fields;

        unsigned int cycle_count = 1, cnt = 0;
        allocColumns(a, op_value);
		
        varNames[setMap[op_value.front()]]->oldRecCount = varNames[setMap[op_value.front()]]->mRecCount;

        if(a->segCount != 1)
            cycle_count = varNames[setMap[op_value.front()]]->segCount;

        oldCount = a->mRecCount;
        thrust::device_vector<unsigned int> p(a->maxRecs);

        for(unsigned int i = 0; i < cycle_count; i++) {
            map_check = zone_map_check(op_type,op_value,op_nums, op_nums_f, a, i);
			cout << "MAP CHECK segment " << i << " " << map_check <<  '\xd';
            reset_offsets();
            if(map_check == 'R') {
                copyColumns(a, op_value, i, cnt);
			    filter(op_type,op_value,op_nums, op_nums_f,a, b, i, p);			
            }
            else  {
                setPrm(a,b,map_check,i);
            };			
        };
        a->mRecCount = oldCount;
        varNames[setMap[op_value.front()]]->mRecCount = varNames[setMap[op_value.front()]]->oldRecCount;        
        a->deAllocOnDevice();
		cout << endl << "filter is finished " << b->mRecCount << " " << getFreeMem()  << endl;
    };

    clean_queues();
	
    if (varNames.count(s) > 0)
        varNames[s]->free();
    varNames[s] = b;
	
    if(stat[s] == statement_count) {
        b->free();
        varNames.erase(s);
    };
    if(stat[f] == statement_count && !a->keep) {
        //a->free();
        //varNames.erase(f);
    };
    std::cout<< "filter time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';
}

void emit_store(char *s, char *f, char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(s) == stat.end()) {
            cout << "Store : couldn't find variable " << s << endl;
            exit(1);
        };
        stat[s] = statement_count;
        return;
    };

    if(varNames.find(s) == varNames.end())
        return;

    CudaSet* a = varNames.find(s)->second;
    cout << "STORE: " << s << " " << f << " " << sep << endl;

    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };

    a->Store(f,sep, limit, 0);

    if(stat[s] == statement_count  && a->keep == 0) {
        a->free();
        varNames.erase(s);
    };
};


void emit_store_binary(char *s, char *f)
{
    statement_count++;
    if (scan_state == 0) {
        if (stat.find(s) == stat.end()) {
            cout << "Store : couldn't find variable " << s << endl;
            exit(1);
        };
        stat[s] = statement_count;
        return;
    };

    if(varNames.find(s) == varNames.end())
        return;

    CudaSet* a = varNames.find(s)->second;

    if(stat[f] == statement_count)
        a->deAllocOnDevice();

    printf("STORE: %s %s \n", s, f);

    int limit = 0;
    if(!op_nums.empty()) {
        limit = op_nums.front();
        op_nums.pop();
    };
    total_count = 0;
    total_segments = 0;
    
	if(fact_file_loaded) {
		a->Store(f,"", limit, 1);	
	}
	else { 
		while(!fact_file_loaded)	{
			cout << "LOADING " << f_file << " mem: " << getFreeMem() << endl;
			if(a->text_source)
				fact_file_loaded = a->LoadBigFile(f_file.c_str(), separator.c_str());
			a->Store(f,"", limit, 1);
		};
	};	

    if(stat[f] == statement_count && !a->keep) {
        a->free();
        varNames.erase(s);
    };

};


void emit_load_binary(char *s, char *f, int d)
{
    statement_count++;
    if (scan_state == 0) {
        stat[s] = statement_count;
        return;
    };

    printf("BINARY LOAD: %s %s \n", s, f);

    CudaSet *a;
    unsigned int segCount, maxRecs;
    char f1[100];
    strcpy(f1, f);
    strcat(f1,".");
    char col_pos[3];
    itoaa(cols.front(),col_pos);
    strcat(f1,col_pos);
    strcat(f1,".header");

    FILE* ff = fopen(f1, "rb");
	if(ff == NULL) {
	    cout << "Couldn't open file " << f1 << endl;
		exit(0);
	};	
    fread((char *)&totalRecs, 8, 1, ff);
    fread((char *)&segCount, 4, 1, ff);
    fread((char *)&maxRecs, 4, 1, ff);
    fclose(ff);

	cout << "Reading " << totalRecs << " records" << endl;
    queue<string> names(namevars);
    while(!names.empty()) {
        setMap[names.front()] = s;
        names.pop();
    };

    a = new CudaSet(namevars, typevars, sizevars, cols,totalRecs, f);
    a->segCount = segCount;
    a->maxRecs = maxRecs;
    a->keep = 1;
    varNames[s] = a;

    if(stat[s] == statement_count )  {
        a->free();
        varNames.erase(s);
    };
}


void emit_load(char *s, char *f, int d, char* sep)
{
    statement_count++;
    if (scan_state == 0) {
        stat[s] = statement_count;
        return;
    };

    printf("LOAD: %s %s %d  %s \n", s, f, d, sep);

    CudaSet *a;

    a = new CudaSet(namevars, typevars, sizevars, cols, process_count);
    a->mRecCount = 0;
    a->resize(process_count);
    a->keep = true;
    a->not_compressed = 1;

    string separator1(sep);
    separator = separator1;
    string ff(f);
    f_file = ff;
    a->maxRecs = a->mRecCount;
    a->segCount = 0;
    varNames[s] = a;
	fact_file_loaded = 0;

    if(stat[s] == statement_count)  {
        a->free();
        varNames.erase(s);
    };
}



void yyerror(char *s, ...)
{
    extern int yylineno;
    va_list ap;
    va_start(ap, s);

    fprintf(stderr, "%d: error: ", yylineno);
    vfprintf(stderr, s, ap);
    fprintf(stderr, "\n");
}

void clean_queues()
{
    while(!op_type.empty()) op_type.pop();
    while(!op_value.empty()) op_value.pop();
    while(!op_join.empty()) op_join.pop();
    while(!op_nums.empty()) op_nums.pop();
    while(!op_nums_f.empty()) op_nums_f.pop();
    while(!j_col_count.empty()) j_col_count.pop();
    while(!namevars.empty()) namevars.pop();
    while(!typevars.empty()) typevars.pop();
    while(!sizevars.empty()) sizevars.pop();
    while(!cols.empty()) cols.pop();
	while(!op_sort.empty()) op_sort.pop();
	
	

    sel_count = 0;
    join_cnt = 0;
    join_col_cnt = 0;
    distinct_cnt = 0;
    reset_offsets();
	join_tab_cnt = 0;
	tab_cnt = 0;
	join_and_cnt.clear();
}



int main(int ac, char **av)
{
    extern FILE *yyin;
    //hipDeviceProp_t deviceProp;

    //hipGetDeviceProperties(&deviceProp, 0);
    //if (!deviceProp.canMapHostMemory)
    //    cout << "Device 0 cannot map host memory" << endl;

    //hipSetDeviceFlags(hipDeviceMapHost);
	//context = CreateCudaDevice(0);
	context = CreateCudaDevice(0, av, true);
    AllocPtr standardAlloc(new CudaAllocSimple(&context->Device()));
    context->SetAllocator(standardAlloc);
	
    cudppCreate(&theCudpp);
	
    /*long long int r30 = RAND_MAX*rand()+rand();
    long long int s30 = RAND_MAX*rand()+rand();
    long long int t4  = rand() & 0xf;

    hash_seed = (r30 << 34) + (s30 << 4) + t4;
	*/
	hash_seed = 100;

    if (ac == 1) {
        cout << "Usage : alenka -l process_count script.sql" << endl;
        exit(1);
    };

    if(strcmp(av[1],"-l") == 0) {
        process_count = atoff(av[2]);
        cout << "Process count = " << process_count << endl;
    }
    else {
        process_count = 6200000;
        cout << "Process count = 6200000 " << endl;
    };

    if((yyin = fopen(av[ac-1], "r")) == NULL) {
        perror(av[ac-1]);
        exit(1);
    };

    if(yyparse()) {
        printf("SQL scan parse failed\n");
        exit(1);
    };

    scan_state = 1;

    std::clock_t start1 = std::clock();
    statement_count = 0;
    clean_queues();

    if(ac > 1 && (yyin = fopen(av[ac-1], "r")) == NULL) {
        perror(av[1]);
        exit(1);
    }

    PROC_FLUSH_BUF ( yyin );
    statement_count = 0;

    if(!yyparse())
        cout << "SQL scan parse worked" << endl;
    else
        cout << "SQL scan parse failed" << endl;
		
	std::cout<< "tot disk time " <<  (( tot ) / (double)CLOCKS_PER_SEC ) <<'\n';


    if(alloced_sz)
        hipFree(alloced_tmp);

    fclose(yyin);
    std::cout<< "cycle time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) <<'\n';
    cudppDestroy(theCudpp);

}



