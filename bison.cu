#include "hip/hip_runtime.h"

/* A Bison parser, made by GNU Bison 2.4.1.  */

/* Skeleton implementation for Bison's Yacc-like parsers in C
   
      Copyright (C) 1984, 1989, 1990, 2000, 2001, 2002, 2003, 2004, 2005, 2006
   Free Software Foundation, Inc.
   
   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.  */

/* As a special exception, you may create a larger work that contains
   part or all of the Bison parser skeleton and distribute that work
   under terms of your choice, so long as that work isn't itself a
   parser generator using the skeleton or a modified version thereof
   as a parser skeleton.  Alternatively, if you modify or redistribute
   the parser skeleton itself, you may (at your option) remove this
   special exception, which will cause the skeleton and the resulting
   Bison output files to be licensed under the GNU General Public
   License without this special exception.
   
   This special exception was added by the Free Software Foundation in
   version 2.2 of Bison.  */

/* C LALR(1) parser skeleton written by Richard Stallman, by
   simplifying the original so-called "semantic" parser.  */

/* All symbols defined below should begin with yy or YY, to avoid
   infringing on user name space.  This should be done even for local
   variables, as they might otherwise be expanded by user macros.
   There are some unavoidable exceptions within include files to
   define necessary library symbols; they are noted "INFRINGES ON
   USER NAME SPACE" below.  */

/* Identify Bison output.  */
#define YYBISON 1

/* Bison version.  */
#define YYBISON_VERSION "2.4.1"

/* Skeleton name.  */
#define YYSKELETON_NAME "yacc.c"

/* Pure parsers.  */
#define YYPURE 0

/* Push parsers.  */
#define YYPUSH 0

/* Pull parsers.  */
#define YYPULL 1

/* Using locations.  */
#define YYLSP_NEEDED 0



/* Copy the first part of user declarations.  */

/* Line 189 of yacc.c  */
#line 15 "bison.y"



#include "lex.yy.c"
#include "cm.h"
#include "operators.h"




/* Line 189 of yacc.c  */
#line 84 "bison.cu"

/* Enabling traces.  */
#ifndef YYDEBUG
# define YYDEBUG 0
#endif

/* Enabling verbose error messages.  */
#ifdef YYERROR_VERBOSE
# undef YYERROR_VERBOSE
# define YYERROR_VERBOSE 1
#else
# define YYERROR_VERBOSE 0
#endif

/* Enabling the token table.  */
#ifndef YYTOKEN_TABLE
# define YYTOKEN_TABLE 0
#endif


/* Tokens.  */
#ifndef YYTOKENTYPE
# define YYTOKENTYPE
   /* Put the tokens into the symbol table, so that GDB and other debuggers
      know about them.  */
   enum yytokentype {
     FILENAME = 258,
     NAME = 259,
     STRING = 260,
     INTNUM = 261,
     DECIMAL1 = 262,
     BOOL1 = 263,
     APPROXNUM = 264,
     USERVAR = 265,
     ASSIGN = 266,
     EQUAL = 267,
     OR = 268,
     XOR = 269,
     AND = 270,
     DISTINCT = 271,
     YEAR = 272,
     REGEXP = 273,
     LIKE = 274,
     IS = 275,
     IN = 276,
     NOT = 277,
     BETWEEN = 278,
     COMPARISON = 279,
     SHIFT = 280,
     MOD = 281,
     FROM = 282,
     MULITE = 283,
     DELETE = 284,
     LOAD = 285,
     FILTER = 286,
     BY = 287,
     JOIN = 288,
     STORE = 289,
     INTO = 290,
     GROUP = 291,
     SELECT = 292,
     AS = 293,
     ORDER = 294,
     ASC = 295,
     DESC = 296,
     COUNT = 297,
     USING = 298,
     SUM = 299,
     AVG = 300,
     MIN = 301,
     MAX = 302,
     LIMIT = 303,
     ON = 304,
     BINARY = 305,
     LEFT = 306,
     RIGHT = 307,
     OUTER = 308,
     SORT = 309,
     SEGMENTS = 310,
     PRESORTED = 311,
     PARTITION = 312,
     INSERT = 313,
     WHERE = 314,
     DISPLAY = 315,
     CASE = 316,
     WHEN = 317,
     THEN = 318,
     ELSE = 319,
     END = 320,
     SHOW = 321,
     TABLES = 322,
     TABLE = 323,
     DESCRIBE = 324,
     DROP = 325,
     CREATE = 326,
     INDEX = 327
   };
#endif



#if ! defined YYSTYPE && ! defined YYSTYPE_IS_DECLARED
typedef union YYSTYPE
{

/* Line 214 of yacc.c  */
#line 25 "bison.y"

    long long int intval;
    double floatval;
    char *strval;
    int subtok;



/* Line 214 of yacc.c  */
#line 201 "bison.cu"
} YYSTYPE;
# define YYSTYPE_IS_TRIVIAL 1
# define yystype YYSTYPE /* obsolescent; will be withdrawn */
# define YYSTYPE_IS_DECLARED 1
#endif


/* Copy the second part of user declarations.  */


/* Line 264 of yacc.c  */
#line 213 "bison.cu"

#ifdef short
# undef short
#endif

#ifdef YYTYPE_UINT8
typedef YYTYPE_UINT8 yytype_uint8;
#else
typedef unsigned char yytype_uint8;
#endif

#ifdef YYTYPE_INT8
typedef YYTYPE_INT8 yytype_int8;
#elif (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
typedef signed char yytype_int8;
#else
typedef short int yytype_int8;
#endif

#ifdef YYTYPE_UINT16
typedef YYTYPE_UINT16 yytype_uint16;
#else
typedef unsigned short int yytype_uint16;
#endif

#ifdef YYTYPE_INT16
typedef YYTYPE_INT16 yytype_int16;
#else
typedef short int yytype_int16;
#endif

#ifndef YYSIZE_T
# ifdef __SIZE_TYPE__
#  define YYSIZE_T __SIZE_TYPE__
# elif defined size_t
#  define YYSIZE_T size_t
# elif ! defined YYSIZE_T && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#  include <stddef.h> /* INFRINGES ON USER NAME SPACE */
#  define YYSIZE_T size_t
# else
#  define YYSIZE_T unsigned int
# endif
#endif

#define YYSIZE_MAXIMUM ((YYSIZE_T) -1)

#ifndef YY_
# if YYENABLE_NLS
#  if ENABLE_NLS
#   include <libintl.h> /* INFRINGES ON USER NAME SPACE */
#   define YY_(msgid) dgettext ("bison-runtime", msgid)
#  endif
# endif
# ifndef YY_
#  define YY_(msgid) msgid
# endif
#endif

/* Suppress unused-variable warnings by "using" E.  */
#if ! defined lint || defined __GNUC__
# define YYUSE(e) ((void) (e))
#else
# define YYUSE(e) /* empty */
#endif

/* Identity function, used to suppress warnings about constant conditions.  */
#ifndef lint
# define YYID(n) (n)
#else
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static int
YYID (int yyi)
#else
static int
YYID (yyi)
    int yyi;
#endif
{
  return yyi;
}
#endif

#if ! defined yyoverflow || YYERROR_VERBOSE

/* The parser invokes alloca or malloc; define the necessary symbols.  */

# ifdef YYSTACK_USE_ALLOCA
#  if YYSTACK_USE_ALLOCA
#   ifdef __GNUC__
#    define YYSTACK_ALLOC __builtin_alloca
#   elif defined __BUILTIN_VA_ARG_INCR
#    include <alloca.h> /* INFRINGES ON USER NAME SPACE */
#   elif defined _AIX
#    define YYSTACK_ALLOC __alloca
#   elif defined _MSC_VER
#    include <malloc.h> /* INFRINGES ON USER NAME SPACE */
#    define alloca _alloca
#   else
#    define YYSTACK_ALLOC alloca
#    if ! defined _ALLOCA_H && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#     include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#     ifndef _STDLIB_H
#      define _STDLIB_H 1
#     endif
#    endif
#   endif
#  endif
# endif

# ifdef YYSTACK_ALLOC
   /* Pacify GCC's `empty if-body' warning.  */
#  define YYSTACK_FREE(Ptr) do { /* empty */; } while (YYID (0))
#  ifndef YYSTACK_ALLOC_MAXIMUM
    /* The OS might guarantee only one guard page at the bottom of the stack,
       and a page size can be as small as 4096 bytes.  So we cannot safely
       invoke alloca (N) if N exceeds 4096.  Use a slightly smaller number
       to allow for a few compiler-allocated temporary stack slots.  */
#   define YYSTACK_ALLOC_MAXIMUM 4032 /* reasonable circa 2006 */
#  endif
# else
#  define YYSTACK_ALLOC YYMALLOC
#  define YYSTACK_FREE YYFREE
#  ifndef YYSTACK_ALLOC_MAXIMUM
#   define YYSTACK_ALLOC_MAXIMUM YYSIZE_MAXIMUM
#  endif
#  if (defined __cplusplus && ! defined _STDLIB_H \
       && ! ((defined YYMALLOC || defined malloc) \
	     && (defined YYFREE || defined free)))
#   include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#   ifndef _STDLIB_H
#    define _STDLIB_H 1
#   endif
#  endif
#  ifndef YYMALLOC
#   define YYMALLOC malloc
#   if ! defined malloc && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void *malloc (YYSIZE_T); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
#  ifndef YYFREE
#   define YYFREE free
#   if ! defined free && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void free (void *); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
# endif
#endif /* ! defined yyoverflow || YYERROR_VERBOSE */


#if (! defined yyoverflow \
     && (! defined __cplusplus \
	 || (defined YYSTYPE_IS_TRIVIAL && YYSTYPE_IS_TRIVIAL)))

/* A type that is properly aligned for any stack member.  */
union yyalloc
{
  yytype_int16 yyss_alloc;
  YYSTYPE yyvs_alloc;
};

/* The size of the maximum gap between one aligned stack and the next.  */
# define YYSTACK_GAP_MAXIMUM (sizeof (union yyalloc) - 1)

/* The size of an array large to enough to hold all stacks, each with
   N elements.  */
# define YYSTACK_BYTES(N) \
     ((N) * (sizeof (yytype_int16) + sizeof (YYSTYPE)) \
      + YYSTACK_GAP_MAXIMUM)

/* Copy COUNT objects from FROM to TO.  The source and destination do
   not overlap.  */
# ifndef YYCOPY
#  if defined __GNUC__ && 1 < __GNUC__
#   define YYCOPY(To, From, Count) \
      __builtin_memcpy (To, From, (Count) * sizeof (*(From)))
#  else
#   define YYCOPY(To, From, Count)		\
      do					\
	{					\
	  YYSIZE_T yyi;				\
	  for (yyi = 0; yyi < (Count); yyi++)	\
	    (To)[yyi] = (From)[yyi];		\
	}					\
      while (YYID (0))
#  endif
# endif

/* Relocate STACK from its old location to the new one.  The
   local variables YYSIZE and YYSTACKSIZE give the old and new number of
   elements in the stack, and YYPTR gives the new location of the
   stack.  Advance YYPTR to a properly aligned location for the next
   stack.  */
# define YYSTACK_RELOCATE(Stack_alloc, Stack)				\
    do									\
      {									\
	YYSIZE_T yynewbytes;						\
	YYCOPY (&yyptr->Stack_alloc, Stack, yysize);			\
	Stack = &yyptr->Stack_alloc;					\
	yynewbytes = yystacksize * sizeof (*Stack) + YYSTACK_GAP_MAXIMUM; \
	yyptr += yynewbytes / sizeof (*yyptr);				\
      }									\
    while (YYID (0))

#endif

/* YYFINAL -- State number of the termination state.  */
#define YYFINAL  22
/* YYLAST -- Last index in YYTABLE.  */
#define YYLAST   755

/* YYNTOKENS -- Number of terminals.  */
#define YYNTOKENS  90
/* YYNNTS -- Number of nonterminals.  */
#define YYNNTS  14
/* YYNRULES -- Number of rules.  */
#define YYNRULES  84
/* YYNRULES -- Number of states.  */
#define YYNSTATES  246

/* YYTRANSLATE(YYLEX) -- Bison symbol number corresponding to YYLEX.  */
#define YYUNDEFTOK  2
#define YYMAXUTOK   327

#define YYTRANSLATE(YYX)						\
  ((unsigned int) (YYX) <= YYMAXUTOK ? yytranslate[YYX] : YYUNDEFTOK)

/* YYTRANSLATE[YYLEX] -- Bison symbol number corresponding to YYLEX.  */
static const yytype_uint8 yytranslate[] =
{
       0,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    22,     2,     2,     2,    33,    27,     2,
      83,    84,    31,    29,    86,    30,    85,    32,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,    89,    82,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,    35,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    87,    26,    88,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     1,     2,     3,     4,
       5,     6,     7,     8,     9,    10,    11,    12,    13,    14,
      15,    16,    17,    18,    19,    20,    21,    23,    24,    25,
      28,    34,    36,    37,    38,    39,    40,    41,    42,    43,
      44,    45,    46,    47,    48,    49,    50,    51,    52,    53,
      54,    55,    56,    57,    58,    59,    60,    61,    62,    63,
      64,    65,    66,    67,    68,    69,    70,    71,    72,    73,
      74,    75,    76,    77,    78,    79,    80,    81
};

#if YYDEBUG
/* YYPRHS[YYN] -- Index of the first RHS symbol of rule number YYN in
   YYRHS.  */
static const yytype_uint16 yyprhs[] =
{
       0,     0,     3,     6,    10,    12,    20,    33,    39,    46,
      55,    65,    73,    76,    84,    90,    98,   101,   105,   128,
     130,   134,   136,   138,   140,   142,   144,   146,   158,   168,
     175,   178,   181,   186,   191,   196,   201,   206,   209,   214,
     218,   222,   226,   230,   234,   238,   242,   246,   250,   254,
     258,   261,   264,   268,   272,   278,   282,   291,   295,   300,
     301,   305,   309,   315,   317,   319,   323,   325,   329,   330,
     332,   335,   340,   346,   352,   358,   364,   371,   378,   385,
     386,   389,   390,   395,   403
};

/* YYRHS -- A `-1'-separated list of the rules' RHS.  */
static const yytype_int8 yyrhs[] =
{
      91,     0,    -1,    92,    82,    -1,    91,    92,    82,    -1,
      93,    -1,     4,    11,    46,    96,    36,     4,    95,    -1,
       4,    11,    39,     3,    52,    83,     3,    84,    47,    83,
      97,    84,    -1,     4,    11,    40,     4,   100,    -1,     4,
      11,    48,     4,    41,    99,    -1,     4,    11,    46,    96,
      36,     4,   101,    95,    -1,    43,     4,    44,     3,    52,
      83,     3,    84,   102,    -1,    43,     4,    44,     3,   102,
      59,   103,    -1,    78,     4,    -1,    67,    44,     4,    46,
      96,    36,     4,    -1,    38,    36,     4,    68,    94,    -1,
      69,     4,    52,    83,     3,    84,   102,    -1,    75,    76,
      -1,    79,    77,     4,    -1,    80,    81,     4,    58,     4,
      83,     4,    85,     4,    84,    36,     4,    86,     4,    68,
       4,    85,     4,    12,     4,    85,     4,    -1,     4,    -1,
       4,    85,     4,    -1,    10,    -1,     5,    -1,     6,    -1,
       7,    -1,     9,    -1,     8,    -1,     4,    87,     6,    88,
      89,     4,    83,     6,    86,     6,    84,    -1,     4,    87,
       6,    88,    89,     4,    83,     6,    84,    -1,     4,    87,
       6,    88,    89,     4,    -1,     4,    49,    -1,     4,    50,
      -1,    51,    83,    94,    84,    -1,    53,    83,    94,    84,
      -1,    54,    83,    94,    84,    -1,    55,    83,    94,    84,
      -1,    56,    83,    94,    84,    -1,    16,    94,    -1,    17,
      83,    94,    84,    -1,    94,    29,    94,    -1,    94,    30,
      94,    -1,    94,    31,    94,    -1,    94,    32,    94,    -1,
      94,    33,    94,    -1,    94,    34,    94,    -1,    94,    15,
      94,    -1,    94,    12,    94,    -1,    94,    13,    94,    -1,
      94,    14,    94,    -1,    94,    28,    94,    -1,    23,    94,
      -1,    22,    94,    -1,    94,    25,    94,    -1,    94,    19,
      94,    -1,    94,    25,    83,    93,    84,    -1,    83,    94,
      84,    -1,    70,    71,    94,    72,    94,    73,    94,    74,
      -1,    94,    20,     8,    -1,    94,    20,    23,     8,    -1,
      -1,    45,    41,    98,    -1,    94,    47,     4,    -1,    96,
      86,    94,    47,     4,    -1,    31,    -1,    94,    -1,    97,
      86,    94,    -1,    94,    -1,    94,    86,    98,    -1,    -1,
      98,    -1,    41,    94,    -1,    42,     4,    58,    94,    -1,
      60,    42,     4,    58,    94,    -1,    61,    42,     4,    58,
      94,    -1,    62,    42,     4,    58,    94,    -1,    42,     4,
      58,    94,   101,    -1,    60,    42,     4,    58,    94,   101,
      -1,    61,    42,     4,    58,    94,   101,    -1,    62,    42,
       4,    58,    94,   101,    -1,    -1,    57,     6,    -1,    -1,
      63,    64,    41,     4,    -1,    63,    64,    41,     4,    66,
      41,     6,    -1,    65,    41,     4,    -1
};

/* YYRLINE[YYN] -- source line where rule number YYN was defined.  */
static const yytype_uint16 yyrline[] =
{
       0,   120,   120,   121,   125,   128,   130,   132,   134,   136,
     138,   140,   142,   144,   146,   148,   150,   152,   154,   159,
     160,   161,   162,   163,   164,   165,   166,   167,   168,   169,
     170,   171,   172,   173,   174,   175,   176,   177,   178,   182,
     183,   184,   185,   186,   187,   189,   190,   191,   192,   193,
     194,   195,   196,   197,   199,   200,   201,   205,   206,   209,
     212,   216,   217,   218,   222,   223,   227,   228,   231,   233,
     236,   240,   241,   242,   243,   244,   245,   246,   247,   249,
     252,   254,   257,   258,   259
};
#endif

#if YYDEBUG || YYERROR_VERBOSE || YYTOKEN_TABLE
/* YYTNAME[SYMBOL-NUM] -- String name of the symbol SYMBOL-NUM.
   First, the terminals, then, starting at YYNTOKENS, nonterminals.  */
static const char *const yytname[] =
{
  "$end", "error", "$undefined", "FILENAME", "NAME", "STRING", "INTNUM",
  "DECIMAL1", "BOOL1", "APPROXNUM", "USERVAR", "ASSIGN", "EQUAL", "OR",
  "XOR", "AND", "DISTINCT", "YEAR", "REGEXP", "LIKE", "IS", "IN", "'!'",
  "NOT", "BETWEEN", "COMPARISON", "'|'", "'&'", "SHIFT", "'+'", "'-'",
  "'*'", "'/'", "'%'", "MOD", "'^'", "FROM", "MULITE", "DELETE", "LOAD",
  "FILTER", "BY", "JOIN", "STORE", "INTO", "GROUP", "SELECT", "AS",
  "ORDER", "ASC", "DESC", "COUNT", "USING", "SUM", "AVG", "MIN", "MAX",
  "LIMIT", "ON", "BINARY", "LEFT", "RIGHT", "OUTER", "SORT", "SEGMENTS",
  "PRESORTED", "PARTITION", "INSERT", "WHERE", "DISPLAY", "CASE", "WHEN",
  "THEN", "ELSE", "END", "SHOW", "TABLES", "TABLE", "DESCRIBE", "DROP",
  "CREATE", "INDEX", "';'", "'('", "')'", "'.'", "','", "'{'", "'}'",
  "':'", "$accept", "stmt_list", "stmt", "select_stmt", "expr",
  "opt_group_list", "expr_list", "load_list", "val_list", "opt_val_list",
  "opt_where", "join_list", "opt_limit", "sort_def", 0
};
#endif

# ifdef YYPRINT
/* YYTOKNUM[YYLEX-NUM] -- Internal token number corresponding to
   token YYLEX-NUM.  */
static const yytype_uint16 yytoknum[] =
{
       0,   256,   257,   258,   259,   260,   261,   262,   263,   264,
     265,   266,   267,   268,   269,   270,   271,   272,   273,   274,
     275,   276,    33,   277,   278,   279,   124,    38,   280,    43,
      45,    42,    47,    37,   281,    94,   282,   283,   284,   285,
     286,   287,   288,   289,   290,   291,   292,   293,   294,   295,
     296,   297,   298,   299,   300,   301,   302,   303,   304,   305,
     306,   307,   308,   309,   310,   311,   312,   313,   314,   315,
     316,   317,   318,   319,   320,   321,   322,   323,   324,   325,
     326,   327,    59,    40,    41,    46,    44,   123,   125,    58
};
# endif

/* YYR1[YYN] -- Symbol number of symbol that rule YYN derives.  */
static const yytype_uint8 yyr1[] =
{
       0,    90,    91,    91,    92,    93,    93,    93,    93,    93,
      93,    93,    93,    93,    93,    93,    93,    93,    93,    94,
      94,    94,    94,    94,    94,    94,    94,    94,    94,    94,
      94,    94,    94,    94,    94,    94,    94,    94,    94,    94,
      94,    94,    94,    94,    94,    94,    94,    94,    94,    94,
      94,    94,    94,    94,    94,    94,    94,    94,    94,    95,
      95,    96,    96,    96,    97,    97,    98,    98,    99,    99,
     100,   101,   101,   101,   101,   101,   101,   101,   101,   102,
     102,   103,   103,   103,   103
};

/* YYR2[YYN] -- Number of symbols composing right hand side of rule YYN.  */
static const yytype_uint8 yyr2[] =
{
       0,     2,     2,     3,     1,     7,    12,     5,     6,     8,
       9,     7,     2,     7,     5,     7,     2,     3,    22,     1,
       3,     1,     1,     1,     1,     1,     1,    11,     9,     6,
       2,     2,     4,     4,     4,     4,     4,     2,     4,     3,
       3,     3,     3,     3,     3,     3,     3,     3,     3,     3,
       2,     2,     3,     3,     5,     3,     8,     3,     4,     0,
       3,     3,     5,     1,     1,     3,     1,     3,     0,     1,
       2,     4,     5,     5,     5,     5,     6,     6,     6,     0,
       2,     0,     4,     7,     3
};

/* YYDEFACT[STATE-NAME] -- Default rule to reduce with in state
   STATE-NUM when YYTABLE doesn't specify something else to do.  Zero
   means the default is an error.  */
static const yytype_uint8 yydefact[] =
{
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     4,     0,     0,     0,     0,     0,    16,    12,
       0,     0,     1,     0,     2,     0,     0,     0,     0,     0,
       0,     0,     0,    17,     0,     3,     0,     0,    19,    22,
      23,    24,    26,    25,    21,     0,     0,     0,     0,    63,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,    79,     0,     0,     0,     0,     0,     7,    30,    31,
       0,     0,    37,     0,    51,    50,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
      68,    14,     0,     0,     0,     0,     0,     0,     0,    70,
      20,     0,     0,     0,     0,     0,     0,     0,     0,    55,
      46,    47,    48,    45,    53,    57,     0,     0,    52,    49,
      39,    40,    41,    42,    43,    44,    61,    59,     0,    66,
      69,     8,     0,    80,    81,     0,    79,     0,     0,     0,
      38,    32,    33,    34,    35,    36,     0,    58,    19,     0,
       0,     0,     0,     0,     0,     5,    59,     0,     0,     0,
       0,     0,    11,    13,    15,     0,     0,     0,     0,    54,
       0,     0,     0,     0,     0,     9,    62,    67,    79,     0,
       0,     0,     0,    29,     0,     0,    60,     0,     0,     0,
      10,     0,    84,     0,     0,     0,     0,    71,     0,     0,
       0,    82,     0,    64,     0,     0,    56,    75,    72,    73,
      74,     0,     0,     6,     0,    28,     0,    76,    77,    78,
       0,     0,    65,     0,    83,     0,    27,     0,     0,     0,
       0,     0,     0,     0,     0,    18
};

/* YYDEFGOTO[NTERM-NUM].  */
static const yytype_int16 yydefgoto[] =
{
      -1,    10,    11,    12,   139,   165,    58,   214,   140,   141,
      67,   166,   104,   172
};

/* YYPACT[STATE-NUM] -- Index in YYTABLE of the portion describing
   STATE-NUM.  */
#define YYPACT_NINF -205
static const yytype_int16 yypact[] =
{
      32,     2,   -17,    23,   -13,    33,   -38,    40,   -32,   -33,
       4,    -8,  -205,    69,    73,    52,    98,    53,  -205,  -205,
     102,   109,  -205,    39,  -205,   111,   119,   182,   121,    58,
     124,    82,    47,  -205,    75,  -205,    79,    99,    31,  -205,
    -205,  -205,  -205,  -205,  -205,   202,    56,   202,   202,  -205,
      66,    70,    71,    72,    74,    87,   202,   615,   -31,   122,
     202,   -40,   182,   149,   158,    83,   202,  -205,  -205,  -205,
     161,   170,   285,   202,   113,   113,   202,   202,   202,   202,
     202,   202,   316,   202,   202,   202,   202,   202,    -1,   270,
     202,   202,   202,   202,   202,   202,   202,   174,   175,   202,
     202,   664,    97,   178,   126,   -25,   110,   112,   190,   664,
    -205,   108,   344,   373,   401,   430,   458,   487,   567,  -205,
     664,   685,   705,   285,   721,  -205,   192,    81,   141,   197,
      61,    61,  -205,  -205,  -205,  -205,  -205,   -36,   641,   269,
    -205,  -205,   198,  -205,   -35,   199,   145,   210,   131,   127,
    -205,  -205,  -205,  -205,  -205,  -205,   202,  -205,    -9,   133,
     216,   180,   181,   200,   201,  -205,   177,   228,   202,   150,
     176,   203,  -205,  -205,  -205,   154,   194,   241,   544,  -205,
     188,   202,   243,   244,   245,  -205,  -205,  -205,   145,   209,
     247,   250,   179,   183,   202,   202,  -205,   205,   206,   211,
    -205,   255,  -205,   184,   202,   254,   515,   590,   202,   202,
     202,   195,   231,   664,   -52,    36,  -205,  -205,   590,   590,
     590,   229,   267,  -205,   202,  -205,   284,  -205,  -205,  -205,
     289,   187,   664,   207,  -205,   292,  -205,   238,   303,   223,
     305,   299,   308,   235,   318,  -205
};

/* YYPGOTO[NTERM-NUM].  */
static const yytype_int16 yypgoto[] =
{
    -205,  -205,   317,   212,   -27,   166,   271,  -205,  -158,  -205,
    -205,  -204,  -145,  -205
};

/* YYTABLE[YYPACT[STATE-NUM]].  What to do in state STATE-NUM.  If
   positive, shift that token.  If negative, reduce the rule which
   number is the opposite.  If zero, do what YYDEFACT says.
   If YYTABLE_NINF, syntax error.  */
#define YYTABLE_NINF -1
static const yytype_int16 yytable[] =
{
      57,   174,    13,   217,    22,    98,   160,   125,     1,   161,
     187,   145,   102,    13,   227,   228,   229,   103,    72,    14,
      74,    75,   126,   196,   162,   163,   164,    15,   170,    82,
     171,    16,   223,   101,   224,    57,     1,    17,    18,   109,
      68,    69,     2,   200,    19,    20,   112,     3,    21,   113,
     114,   115,   116,   117,   118,    99,   120,   121,   122,   123,
     124,    99,   128,   129,   130,   131,   132,   133,   134,   135,
       2,     4,   138,     5,    24,     3,    70,    29,    71,     6,
      68,    69,     7,     8,     9,   158,    39,    40,    41,    42,
      43,    44,    93,    94,    95,    96,    30,    45,    46,     4,
      82,     5,    31,    47,    48,    32,    33,     6,    25,    26,
       7,     8,     9,    34,    36,    27,    70,    28,    71,     2,
     225,    35,   226,    37,     3,    59,    60,    61,    62,   178,
      63,    65,    50,    64,    51,    52,    53,    54,    89,    73,
      66,    90,    91,    92,    93,    94,    95,    96,     4,    76,
       5,    55,   106,    77,    78,    79,     6,    80,    81,     7,
       8,     9,   107,   100,    56,   110,   108,   206,   207,    90,
      91,    92,    93,    94,    95,    96,   111,   213,   136,   137,
     142,   218,   219,   220,   143,   144,    38,    39,    40,    41,
      42,    43,    44,   148,   146,   147,   149,   232,    45,    46,
     157,   169,   103,   173,    47,    48,    38,    39,    40,    41,
      42,    43,    44,    49,   175,   176,   177,   179,    45,    46,
     180,   181,   161,   182,    47,    48,    91,    92,    93,    94,
      95,    96,   186,    50,   188,    51,    52,    53,    54,   191,
     189,   192,   183,   184,   190,   193,   195,   197,   198,   199,
     201,   202,    55,    50,   203,    51,    52,    53,    54,   211,
     215,   221,   204,   208,   209,    56,   205,   222,   212,   210,
     230,   231,    55,   235,    38,    39,    40,    41,    42,    43,
      44,    83,    84,    85,    86,    56,    45,    46,    87,    88,
     233,   236,    47,    48,    89,   234,   237,    90,    91,    92,
      93,    94,    95,    96,    87,    88,   238,   239,   240,   241,
      89,   242,   243,    90,    91,    92,    93,    94,    95,    96,
     244,    50,   245,    51,    52,    53,    54,    23,    83,    84,
      85,    86,   185,   105,     0,    87,    88,     0,     0,   159,
      55,    89,     0,     0,    90,    91,    92,    93,    94,    95,
      96,     0,     0,   127,     0,   168,    83,    84,    85,    86,
       0,     0,     0,    87,    88,     0,     0,     0,     0,    89,
       0,     0,    90,    91,    92,    93,    94,    95,    96,     0,
       0,     0,     0,     0,     0,    83,    84,    85,    86,     0,
       0,     0,    87,    88,     0,     0,     0,     0,    89,     0,
     119,    90,    91,    92,    93,    94,    95,    96,     0,     0,
       0,     0,     0,    83,    84,    85,    86,     0,     0,     0,
      87,    88,     0,     0,     0,     0,    89,     0,   150,    90,
      91,    92,    93,    94,    95,    96,     0,     0,     0,     0,
       0,     0,    83,    84,    85,    86,     0,     0,     0,    87,
      88,     0,     0,     0,     0,    89,     0,   151,    90,    91,
      92,    93,    94,    95,    96,     0,     0,     0,     0,     0,
      83,    84,    85,    86,     0,     0,     0,    87,    88,     0,
       0,     0,     0,    89,     0,   152,    90,    91,    92,    93,
      94,    95,    96,     0,     0,     0,     0,     0,     0,    83,
      84,    85,    86,     0,     0,     0,    87,    88,     0,     0,
       0,     0,    89,     0,   153,    90,    91,    92,    93,    94,
      95,    96,     0,     0,     0,     0,     0,    83,    84,    85,
      86,     0,     0,     0,    87,    88,     0,     0,     0,     0,
      89,     0,   154,    90,    91,    92,    93,    94,    95,    96,
       0,     0,     0,     0,     0,     0,    83,    84,    85,    86,
       0,     0,     0,    87,    88,     0,     0,     0,     0,    89,
       0,   155,    90,    91,    92,    93,    94,    95,    96,    83,
      84,    85,    86,     0,     0,     0,    87,    88,     0,   216,
       0,     0,    89,     0,     0,    90,    91,    92,    93,    94,
      95,    96,    83,    84,    85,    86,     0,     0,     0,    87,
      88,     0,     0,     0,     0,    89,     0,   194,    90,    91,
      92,    93,    94,    95,    96,     0,     0,    83,    84,    85,
      86,     0,   160,     0,    87,    88,     0,     0,     0,   156,
      89,     0,     0,    90,    91,    92,    93,    94,    95,    96,
     162,   163,   164,    83,    84,    85,    86,     0,     0,     0,
      87,    88,    97,     0,     0,     0,    89,     0,     0,    90,
      91,    92,    93,    94,    95,    96,    83,    84,    85,    86,
       0,     0,     0,    87,    88,     0,     0,     0,   167,    89,
       0,     0,    90,    91,    92,    93,    94,    95,    96,    85,
      86,     0,     0,     0,    87,    88,     0,     0,     0,     0,
      89,     0,     0,    90,    91,    92,    93,    94,    95,    96,
      86,     0,     0,     0,    87,    88,     0,     0,     0,     0,
      89,     0,     0,    90,    91,    92,    93,    94,    95,    96,
      -1,    -1,     0,     0,     0,     0,    89,     0,     0,    90,
      91,    92,    93,    94,    95,    96
};

static const yytype_int16 yycheck[] =
{
      27,   146,    11,   207,     0,    36,    42,     8,     4,    45,
     168,    36,    52,    11,   218,   219,   220,    57,    45,    36,
      47,    48,    23,   181,    60,    61,    62,     4,    63,    56,
      65,    44,    84,    60,    86,    62,     4,     4,    76,    66,
      49,    50,    38,   188,     4,    77,    73,    43,    81,    76,
      77,    78,    79,    80,    81,    86,    83,    84,    85,    86,
      87,    86,    89,    90,    91,    92,    93,    94,    95,    96,
      38,    67,    99,    69,    82,    43,    85,     4,    87,    75,
      49,    50,    78,    79,    80,     4,     5,     6,     7,     8,
       9,    10,    31,    32,    33,    34,    44,    16,    17,    67,
     127,    69,     4,    22,    23,    52,     4,    75,    39,    40,
      78,    79,    80,     4,     3,    46,    85,    48,    87,    38,
      84,    82,    86,     4,    43,     4,    68,     3,    46,   156,
      83,    52,    51,    58,    53,    54,    55,    56,    25,    83,
      41,    28,    29,    30,    31,    32,    33,    34,    67,    83,
      69,    70,     3,    83,    83,    83,    75,    83,    71,    78,
      79,    80,     4,    41,    83,     4,    83,   194,   195,    28,
      29,    30,    31,    32,    33,    34,     6,   204,     4,     4,
      83,   208,   209,   210,     6,    59,     4,     5,     6,     7,
       8,     9,    10,     3,    84,    83,    88,   224,    16,    17,
       8,     3,    57,     4,    22,    23,     4,     5,     6,     7,
       8,     9,    10,    31,     4,    84,    89,    84,    16,    17,
       4,    41,    45,    42,    22,    23,    29,    30,    31,    32,
      33,    34,     4,    51,    84,    53,    54,    55,    56,    85,
      64,    47,    42,    42,    41,     4,    58,     4,     4,     4,
      41,     4,    70,    51,     4,    53,    54,    55,    56,     4,
       6,    66,    83,    58,    58,    83,    83,    36,    84,    58,
      41,     4,    70,    86,     4,     5,     6,     7,     8,     9,
      10,    12,    13,    14,    15,    83,    16,    17,    19,    20,
       6,    84,    22,    23,    25,     6,     4,    28,    29,    30,
      31,    32,    33,    34,    19,    20,    68,     4,    85,     4,
      25,    12,     4,    28,    29,    30,    31,    32,    33,    34,
      85,    51,     4,    53,    54,    55,    56,    10,    12,    13,
      14,    15,   166,    62,    -1,    19,    20,    -1,    -1,   127,
      70,    25,    -1,    -1,    28,    29,    30,    31,    32,    33,
      34,    -1,    -1,    83,    -1,    86,    12,    13,    14,    15,
      -1,    -1,    -1,    19,    20,    -1,    -1,    -1,    -1,    25,
      -1,    -1,    28,    29,    30,    31,    32,    33,    34,    -1,
      -1,    -1,    -1,    -1,    -1,    12,    13,    14,    15,    -1,
      -1,    -1,    19,    20,    -1,    -1,    -1,    -1,    25,    -1,
      84,    28,    29,    30,    31,    32,    33,    34,    -1,    -1,
      -1,    -1,    -1,    12,    13,    14,    15,    -1,    -1,    -1,
      19,    20,    -1,    -1,    -1,    -1,    25,    -1,    84,    28,
      29,    30,    31,    32,    33,    34,    -1,    -1,    -1,    -1,
      -1,    -1,    12,    13,    14,    15,    -1,    -1,    -1,    19,
      20,    -1,    -1,    -1,    -1,    25,    -1,    84,    28,    29,
      30,    31,    32,    33,    34,    -1,    -1,    -1,    -1,    -1,
      12,    13,    14,    15,    -1,    -1,    -1,    19,    20,    -1,
      -1,    -1,    -1,    25,    -1,    84,    28,    29,    30,    31,
      32,    33,    34,    -1,    -1,    -1,    -1,    -1,    -1,    12,
      13,    14,    15,    -1,    -1,    -1,    19,    20,    -1,    -1,
      -1,    -1,    25,    -1,    84,    28,    29,    30,    31,    32,
      33,    34,    -1,    -1,    -1,    -1,    -1,    12,    13,    14,
      15,    -1,    -1,    -1,    19,    20,    -1,    -1,    -1,    -1,
      25,    -1,    84,    28,    29,    30,    31,    32,    33,    34,
      -1,    -1,    -1,    -1,    -1,    -1,    12,    13,    14,    15,
      -1,    -1,    -1,    19,    20,    -1,    -1,    -1,    -1,    25,
      -1,    84,    28,    29,    30,    31,    32,    33,    34,    12,
      13,    14,    15,    -1,    -1,    -1,    19,    20,    -1,    74,
      -1,    -1,    25,    -1,    -1,    28,    29,    30,    31,    32,
      33,    34,    12,    13,    14,    15,    -1,    -1,    -1,    19,
      20,    -1,    -1,    -1,    -1,    25,    -1,    73,    28,    29,
      30,    31,    32,    33,    34,    -1,    -1,    12,    13,    14,
      15,    -1,    42,    -1,    19,    20,    -1,    -1,    -1,    72,
      25,    -1,    -1,    28,    29,    30,    31,    32,    33,    34,
      60,    61,    62,    12,    13,    14,    15,    -1,    -1,    -1,
      19,    20,    47,    -1,    -1,    -1,    25,    -1,    -1,    28,
      29,    30,    31,    32,    33,    34,    12,    13,    14,    15,
      -1,    -1,    -1,    19,    20,    -1,    -1,    -1,    47,    25,
      -1,    -1,    28,    29,    30,    31,    32,    33,    34,    14,
      15,    -1,    -1,    -1,    19,    20,    -1,    -1,    -1,    -1,
      25,    -1,    -1,    28,    29,    30,    31,    32,    33,    34,
      15,    -1,    -1,    -1,    19,    20,    -1,    -1,    -1,    -1,
      25,    -1,    -1,    28,    29,    30,    31,    32,    33,    34,
      19,    20,    -1,    -1,    -1,    -1,    25,    -1,    -1,    28,
      29,    30,    31,    32,    33,    34
};

/* YYSTOS[STATE-NUM] -- The (internal number of the) accessing
   symbol of state STATE-NUM.  */
static const yytype_uint8 yystos[] =
{
       0,     4,    38,    43,    67,    69,    75,    78,    79,    80,
      91,    92,    93,    11,    36,     4,    44,     4,    76,     4,
      77,    81,     0,    92,    82,    39,    40,    46,    48,     4,
      44,     4,    52,     4,     4,    82,     3,     4,     4,     5,
       6,     7,     8,     9,    10,    16,    17,    22,    23,    31,
      51,    53,    54,    55,    56,    70,    83,    94,    96,     4,
      68,     3,    46,    83,    58,    52,    41,   100,    49,    50,
      85,    87,    94,    83,    94,    94,    83,    83,    83,    83,
      83,    71,    94,    12,    13,    14,    15,    19,    20,    25,
      28,    29,    30,    31,    32,    33,    34,    47,    36,    86,
      41,    94,    52,    57,   102,    96,     3,     4,    83,    94,
       4,     6,    94,    94,    94,    94,    94,    94,    94,    84,
      94,    94,    94,    94,    94,     8,    23,    83,    94,    94,
      94,    94,    94,    94,    94,    94,     4,     4,    94,    94,
      98,    99,    83,     6,    59,    36,    84,    83,     3,    88,
      84,    84,    84,    84,    84,    84,    72,     8,     4,    93,
      42,    45,    60,    61,    62,    95,   101,    47,    86,     3,
      63,    65,   103,     4,   102,     4,    84,    89,    94,    84,
       4,    41,    42,    42,    42,    95,     4,    98,    84,    64,
      41,    85,    47,     4,    73,    58,    98,     4,     4,     4,
     102,    41,     4,     4,    83,    83,    94,    94,    58,    58,
      58,     4,    84,    94,    97,     6,    74,   101,    94,    94,
      94,    66,    36,    84,    86,    84,    86,   101,   101,   101,
      41,     4,    94,     6,     6,    86,    84,     4,    68,     4,
      85,     4,    12,     4,    85,     4
};

#define yyerrok		(yyerrstatus = 0)
#define yyclearin	(yychar = YYEMPTY)
#define YYEMPTY		(-2)
#define YYEOF		0

#define YYACCEPT	goto yyacceptlab
#define YYABORT		goto yyabortlab
#define YYERROR		goto yyerrorlab


/* Like YYERROR except do call yyerror.  This remains here temporarily
   to ease the transition to the new meaning of YYERROR, for GCC.
   Once GCC version 2 has supplanted version 1, this can go.  */

#define YYFAIL		goto yyerrlab

#define YYRECOVERING()  (!!yyerrstatus)

#define YYBACKUP(Token, Value)					\
do								\
  if (yychar == YYEMPTY && yylen == 1)				\
    {								\
      yychar = (Token);						\
      yylval = (Value);						\
      yytoken = YYTRANSLATE (yychar);				\
      YYPOPSTACK (1);						\
      goto yybackup;						\
    }								\
  else								\
    {								\
      yyerror (YY_("syntax error: cannot back up")); \
      YYERROR;							\
    }								\
while (YYID (0))


#define YYTERROR	1
#define YYERRCODE	256


/* YYLLOC_DEFAULT -- Set CURRENT to span from RHS[1] to RHS[N].
   If N is 0, then set CURRENT to the empty location which ends
   the previous symbol: RHS[0] (always defined).  */

#define YYRHSLOC(Rhs, K) ((Rhs)[K])
#ifndef YYLLOC_DEFAULT
# define YYLLOC_DEFAULT(Current, Rhs, N)				\
    do									\
      if (YYID (N))                                                    \
	{								\
	  (Current).first_line   = YYRHSLOC (Rhs, 1).first_line;	\
	  (Current).first_column = YYRHSLOC (Rhs, 1).first_column;	\
	  (Current).last_line    = YYRHSLOC (Rhs, N).last_line;		\
	  (Current).last_column  = YYRHSLOC (Rhs, N).last_column;	\
	}								\
      else								\
	{								\
	  (Current).first_line   = (Current).last_line   =		\
	    YYRHSLOC (Rhs, 0).last_line;				\
	  (Current).first_column = (Current).last_column =		\
	    YYRHSLOC (Rhs, 0).last_column;				\
	}								\
    while (YYID (0))
#endif


/* YY_LOCATION_PRINT -- Print the location on the stream.
   This macro was not mandated originally: define only if we know
   we won't break user code: when these are the locations we know.  */

#ifndef YY_LOCATION_PRINT
# if YYLTYPE_IS_TRIVIAL
#  define YY_LOCATION_PRINT(File, Loc)			\
     fprintf (File, "%d.%d-%d.%d",			\
	      (Loc).first_line, (Loc).first_column,	\
	      (Loc).last_line,  (Loc).last_column)
# else
#  define YY_LOCATION_PRINT(File, Loc) ((void) 0)
# endif
#endif


/* YYLEX -- calling `yylex' with the right arguments.  */

#ifdef YYLEX_PARAM
# define YYLEX yylex (YYLEX_PARAM)
#else
# define YYLEX yylex ()
#endif

/* Enable debugging if requested.  */
#if YYDEBUG

# ifndef YYFPRINTF
#  include <stdio.h> /* INFRINGES ON USER NAME SPACE */
#  define YYFPRINTF fprintf
# endif

# define YYDPRINTF(Args)			\
do {						\
  if (yydebug)					\
    YYFPRINTF Args;				\
} while (YYID (0))

# define YY_SYMBOL_PRINT(Title, Type, Value, Location)			  \
do {									  \
  if (yydebug)								  \
    {									  \
      YYFPRINTF (stderr, "%s ", Title);					  \
      yy_symbol_print (stderr,						  \
		  Type, Value); \
      YYFPRINTF (stderr, "\n");						  \
    }									  \
} while (YYID (0))


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_value_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_value_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (!yyvaluep)
    return;
# ifdef YYPRINT
  if (yytype < YYNTOKENS)
    YYPRINT (yyoutput, yytoknum[yytype], *yyvaluep);
# else
  YYUSE (yyoutput);
# endif
  switch (yytype)
    {
      default:
	break;
    }
}


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (yytype < YYNTOKENS)
    YYFPRINTF (yyoutput, "token %s (", yytname[yytype]);
  else
    YYFPRINTF (yyoutput, "nterm %s (", yytname[yytype]);

  yy_symbol_value_print (yyoutput, yytype, yyvaluep);
  YYFPRINTF (yyoutput, ")");
}

/*------------------------------------------------------------------.
| yy_stack_print -- Print the state stack from its BOTTOM up to its |
| TOP (included).                                                   |
`------------------------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_stack_print (yytype_int16 *yybottom, yytype_int16 *yytop)
#else
static void
yy_stack_print (yybottom, yytop)
    yytype_int16 *yybottom;
    yytype_int16 *yytop;
#endif
{
  YYFPRINTF (stderr, "Stack now");
  for (; yybottom <= yytop; yybottom++)
    {
      int yybot = *yybottom;
      YYFPRINTF (stderr, " %d", yybot);
    }
  YYFPRINTF (stderr, "\n");
}

# define YY_STACK_PRINT(Bottom, Top)				\
do {								\
  if (yydebug)							\
    yy_stack_print ((Bottom), (Top));				\
} while (YYID (0))


/*------------------------------------------------.
| Report that the YYRULE is going to be reduced.  |
`------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_reduce_print (YYSTYPE *yyvsp, int yyrule)
#else
static void
yy_reduce_print (yyvsp, yyrule)
    YYSTYPE *yyvsp;
    int yyrule;
#endif
{
  int yynrhs = yyr2[yyrule];
  int yyi;
  unsigned long int yylno = yyrline[yyrule];
  YYFPRINTF (stderr, "Reducing stack by rule %d (line %lu):\n",
	     yyrule - 1, yylno);
  /* The symbols being reduced.  */
  for (yyi = 0; yyi < yynrhs; yyi++)
    {
      YYFPRINTF (stderr, "   $%d = ", yyi + 1);
      yy_symbol_print (stderr, yyrhs[yyprhs[yyrule] + yyi],
		       &(yyvsp[(yyi + 1) - (yynrhs)])
		       		       );
      YYFPRINTF (stderr, "\n");
    }
}

# define YY_REDUCE_PRINT(Rule)		\
do {					\
  if (yydebug)				\
    yy_reduce_print (yyvsp, Rule); \
} while (YYID (0))

/* Nonzero means print parse trace.  It is left uninitialized so that
   multiple parsers can coexist.  */
int yydebug;
#else /* !YYDEBUG */
# define YYDPRINTF(Args)
# define YY_SYMBOL_PRINT(Title, Type, Value, Location)
# define YY_STACK_PRINT(Bottom, Top)
# define YY_REDUCE_PRINT(Rule)
#endif /* !YYDEBUG */


/* YYINITDEPTH -- initial size of the parser's stacks.  */
#ifndef	YYINITDEPTH
# define YYINITDEPTH 200
#endif

/* YYMAXDEPTH -- maximum size the stacks can grow to (effective only
   if the built-in stack extension method is used).

   Do not make this value too large; the results are undefined if
   YYSTACK_ALLOC_MAXIMUM < YYSTACK_BYTES (YYMAXDEPTH)
   evaluated with infinite-precision integer arithmetic.  */

#ifndef YYMAXDEPTH
# define YYMAXDEPTH 10000
#endif



#if YYERROR_VERBOSE

# ifndef yystrlen
#  if defined __GLIBC__ && defined _STRING_H
#   define yystrlen strlen
#  else
/* Return the length of YYSTR.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static YYSIZE_T
yystrlen (const char *yystr)
#else
static YYSIZE_T
yystrlen (yystr)
    const char *yystr;
#endif
{
  YYSIZE_T yylen;
  for (yylen = 0; yystr[yylen]; yylen++)
    continue;
  return yylen;
}
#  endif
# endif

# ifndef yystpcpy
#  if defined __GLIBC__ && defined _STRING_H && defined _GNU_SOURCE
#   define yystpcpy stpcpy
#  else
/* Copy YYSRC to YYDEST, returning the address of the terminating '\0' in
   YYDEST.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static char *
yystpcpy (char *yydest, const char *yysrc)
#else
static char *
yystpcpy (yydest, yysrc)
    char *yydest;
    const char *yysrc;
#endif
{
  char *yyd = yydest;
  const char *yys = yysrc;

  while ((*yyd++ = *yys++) != '\0')
    continue;

  return yyd - 1;
}
#  endif
# endif

# ifndef yytnamerr
/* Copy to YYRES the contents of YYSTR after stripping away unnecessary
   quotes and backslashes, so that it's suitable for yyerror.  The
   heuristic is that double-quoting is unnecessary unless the string
   contains an apostrophe, a comma, or backslash (other than
   backslash-backslash).  YYSTR is taken from yytname.  If YYRES is
   null, do not copy; instead, return the length of what the result
   would have been.  */
static YYSIZE_T
yytnamerr (char *yyres, const char *yystr)
{
  if (*yystr == '"')
    {
      YYSIZE_T yyn = 0;
      char const *yyp = yystr;

      for (;;)
	switch (*++yyp)
	  {
	  case '\'':
	  case ',':
	    goto do_not_strip_quotes;

	  case '\\':
	    if (*++yyp != '\\')
	      goto do_not_strip_quotes;
	    /* Fall through.  */
	  default:
	    if (yyres)
	      yyres[yyn] = *yyp;
	    yyn++;
	    break;

	  case '"':
	    if (yyres)
	      yyres[yyn] = '\0';
	    return yyn;
	  }
    do_not_strip_quotes: ;
    }

  if (! yyres)
    return yystrlen (yystr);

  return yystpcpy (yyres, yystr) - yyres;
}
# endif

/* Copy into YYRESULT an error message about the unexpected token
   YYCHAR while in state YYSTATE.  Return the number of bytes copied,
   including the terminating null byte.  If YYRESULT is null, do not
   copy anything; just return the number of bytes that would be
   copied.  As a special case, return 0 if an ordinary "syntax error"
   message will do.  Return YYSIZE_MAXIMUM if overflow occurs during
   size calculation.  */
static YYSIZE_T
yysyntax_error (char *yyresult, int yystate, int yychar)
{
  int yyn = yypact[yystate];

  if (! (YYPACT_NINF < yyn && yyn <= YYLAST))
    return 0;
  else
    {
      int yytype = YYTRANSLATE (yychar);
      YYSIZE_T yysize0 = yytnamerr (0, yytname[yytype]);
      YYSIZE_T yysize = yysize0;
      YYSIZE_T yysize1;
      int yysize_overflow = 0;
      enum { YYERROR_VERBOSE_ARGS_MAXIMUM = 5 };
      char const *yyarg[YYERROR_VERBOSE_ARGS_MAXIMUM];
      int yyx;

# if 0
      /* This is so xgettext sees the translatable formats that are
	 constructed on the fly.  */
      YY_("syntax error, unexpected %s");
      YY_("syntax error, unexpected %s, expecting %s");
      YY_("syntax error, unexpected %s, expecting %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s or %s");
# endif
      char *yyfmt;
      char const *yyf;
      static char const yyunexpected[] = "syntax error, unexpected %s";
      static char const yyexpecting[] = ", expecting %s";
      static char const yyor[] = " or %s";
      char yyformat[sizeof yyunexpected
		    + sizeof yyexpecting - 1
		    + ((YYERROR_VERBOSE_ARGS_MAXIMUM - 2)
		       * (sizeof yyor - 1))];
      char const *yyprefix = yyexpecting;

      /* Start YYX at -YYN if negative to avoid negative indexes in
	 YYCHECK.  */
      int yyxbegin = yyn < 0 ? -yyn : 0;

      /* Stay within bounds of both yycheck and yytname.  */
      int yychecklim = YYLAST - yyn + 1;
      int yyxend = yychecklim < YYNTOKENS ? yychecklim : YYNTOKENS;
      int yycount = 1;

      yyarg[0] = yytname[yytype];
      yyfmt = yystpcpy (yyformat, yyunexpected);

      for (yyx = yyxbegin; yyx < yyxend; ++yyx)
	if (yycheck[yyx + yyn] == yyx && yyx != YYTERROR)
	  {
	    if (yycount == YYERROR_VERBOSE_ARGS_MAXIMUM)
	      {
		yycount = 1;
		yysize = yysize0;
		yyformat[sizeof yyunexpected - 1] = '\0';
		break;
	      }
	    yyarg[yycount++] = yytname[yyx];
	    yysize1 = yysize + yytnamerr (0, yytname[yyx]);
	    yysize_overflow |= (yysize1 < yysize);
	    yysize = yysize1;
	    yyfmt = yystpcpy (yyfmt, yyprefix);
	    yyprefix = yyor;
	  }

      yyf = YY_(yyformat);
      yysize1 = yysize + yystrlen (yyf);
      yysize_overflow |= (yysize1 < yysize);
      yysize = yysize1;

      if (yysize_overflow)
	return YYSIZE_MAXIMUM;

      if (yyresult)
	{
	  /* Avoid sprintf, as that infringes on the user's name space.
	     Don't have undefined behavior even if the translation
	     produced a string with the wrong number of "%s"s.  */
	  char *yyp = yyresult;
	  int yyi = 0;
	  while ((*yyp = *yyf) != '\0')
	    {
	      if (*yyp == '%' && yyf[1] == 's' && yyi < yycount)
		{
		  yyp += yytnamerr (yyp, yyarg[yyi++]);
		  yyf += 2;
		}
	      else
		{
		  yyp++;
		  yyf++;
		}
	    }
	}
      return yysize;
    }
}
#endif /* YYERROR_VERBOSE */


/*-----------------------------------------------.
| Release the memory associated to this symbol.  |
`-----------------------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yydestruct (const char *yymsg, int yytype, YYSTYPE *yyvaluep)
#else
static void
yydestruct (yymsg, yytype, yyvaluep)
    const char *yymsg;
    int yytype;
    YYSTYPE *yyvaluep;
#endif
{
  YYUSE (yyvaluep);

  if (!yymsg)
    yymsg = "Deleting";
  YY_SYMBOL_PRINT (yymsg, yytype, yyvaluep, yylocationp);

  switch (yytype)
    {

      default:
	break;
    }
}

/* Prevent warnings from -Wmissing-prototypes.  */
#ifdef YYPARSE_PARAM
#if defined __STDC__ || defined __cplusplus
int yyparse (void *YYPARSE_PARAM);
#else
int yyparse ();
#endif
#else /* ! YYPARSE_PARAM */
#if defined __STDC__ || defined __cplusplus
int yyparse (void);
#else
int yyparse ();
#endif
#endif /* ! YYPARSE_PARAM */


/* The lookahead symbol.  */
int yychar;

/* The semantic value of the lookahead symbol.  */
YYSTYPE yylval;

/* Number of syntax errors so far.  */
int yynerrs;



/*-------------------------.
| yyparse or yypush_parse.  |
`-------------------------*/

#ifdef YYPARSE_PARAM
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void *YYPARSE_PARAM)
#else
int
yyparse (YYPARSE_PARAM)
    void *YYPARSE_PARAM;
#endif
#else /* ! YYPARSE_PARAM */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void)
#else
int
yyparse ()

#endif
#endif
{


    int yystate;
    /* Number of tokens to shift before error messages enabled.  */
    int yyerrstatus;

    /* The stacks and their tools:
       `yyss': related to states.
       `yyvs': related to semantic values.

       Refer to the stacks thru separate pointers, to allow yyoverflow
       to reallocate them elsewhere.  */

    /* The state stack.  */
    yytype_int16 yyssa[YYINITDEPTH];
    yytype_int16 *yyss;
    yytype_int16 *yyssp;

    /* The semantic value stack.  */
    YYSTYPE yyvsa[YYINITDEPTH];
    YYSTYPE *yyvs;
    YYSTYPE *yyvsp;

    YYSIZE_T yystacksize;

  int yyn;
  int yyresult;
  /* Lookahead token as an internal (translated) token number.  */
  int yytoken;
  /* The variables used to return semantic value and location from the
     action routines.  */
  YYSTYPE yyval;

#if YYERROR_VERBOSE
  /* Buffer for error messages, and its allocated size.  */
  char yymsgbuf[128];
  char *yymsg = yymsgbuf;
  YYSIZE_T yymsg_alloc = sizeof yymsgbuf;
#endif

#define YYPOPSTACK(N)   (yyvsp -= (N), yyssp -= (N))

  /* The number of symbols on the RHS of the reduced rule.
     Keep to zero when no symbol should be popped.  */
  int yylen = 0;

  yytoken = 0;
  yyss = yyssa;
  yyvs = yyvsa;
  yystacksize = YYINITDEPTH;

  YYDPRINTF ((stderr, "Starting parse\n"));

  yystate = 0;
  yyerrstatus = 0;
  yynerrs = 0;
  yychar = YYEMPTY; /* Cause a token to be read.  */

  /* Initialize stack pointers.
     Waste one element of value and location stack
     so that they stay on the same level as the state stack.
     The wasted elements are never initialized.  */
  yyssp = yyss;
  yyvsp = yyvs;

  goto yysetstate;

/*------------------------------------------------------------.
| yynewstate -- Push a new state, which is found in yystate.  |
`------------------------------------------------------------*/
 yynewstate:
  /* In all cases, when you get here, the value and location stacks
     have just been pushed.  So pushing a state here evens the stacks.  */
  yyssp++;

 yysetstate:
  *yyssp = yystate;

  if (yyss + yystacksize - 1 <= yyssp)
    {
      /* Get the current used size of the three stacks, in elements.  */
      YYSIZE_T yysize = yyssp - yyss + 1;

#ifdef yyoverflow
      {
	/* Give user a chance to reallocate the stack.  Use copies of
	   these so that the &'s don't force the real ones into
	   memory.  */
	YYSTYPE *yyvs1 = yyvs;
	yytype_int16 *yyss1 = yyss;

	/* Each stack pointer address is followed by the size of the
	   data in use in that stack, in bytes.  This used to be a
	   conditional around just the two extra args, but that might
	   be undefined if yyoverflow is a macro.  */
	yyoverflow (YY_("memory exhausted"),
		    &yyss1, yysize * sizeof (*yyssp),
		    &yyvs1, yysize * sizeof (*yyvsp),
		    &yystacksize);

	yyss = yyss1;
	yyvs = yyvs1;
      }
#else /* no yyoverflow */
# ifndef YYSTACK_RELOCATE
      goto yyexhaustedlab;
# else
      /* Extend the stack our own way.  */
      if (YYMAXDEPTH <= yystacksize)
	goto yyexhaustedlab;
      yystacksize *= 2;
      if (YYMAXDEPTH < yystacksize)
	yystacksize = YYMAXDEPTH;

      {
	yytype_int16 *yyss1 = yyss;
	union yyalloc *yyptr =
	  (union yyalloc *) YYSTACK_ALLOC (YYSTACK_BYTES (yystacksize));
	if (! yyptr)
	  goto yyexhaustedlab;
	YYSTACK_RELOCATE (yyss_alloc, yyss);
	YYSTACK_RELOCATE (yyvs_alloc, yyvs);
#  undef YYSTACK_RELOCATE
	if (yyss1 != yyssa)
	  YYSTACK_FREE (yyss1);
      }
# endif
#endif /* no yyoverflow */

      yyssp = yyss + yysize - 1;
      yyvsp = yyvs + yysize - 1;

      YYDPRINTF ((stderr, "Stack size increased to %lu\n",
		  (unsigned long int) yystacksize));

      if (yyss + yystacksize - 1 <= yyssp)
	YYABORT;
    }

  YYDPRINTF ((stderr, "Entering state %d\n", yystate));

  if (yystate == YYFINAL)
    YYACCEPT;

  goto yybackup;

/*-----------.
| yybackup.  |
`-----------*/
yybackup:

  /* Do appropriate processing given the current state.  Read a
     lookahead token if we need one and don't already have one.  */

  /* First try to decide what to do without reference to lookahead token.  */
  yyn = yypact[yystate];
  if (yyn == YYPACT_NINF)
    goto yydefault;

  /* Not known => get a lookahead token if don't already have one.  */

  /* YYCHAR is either YYEMPTY or YYEOF or a valid lookahead symbol.  */
  if (yychar == YYEMPTY)
    {
      YYDPRINTF ((stderr, "Reading a token: "));
      yychar = YYLEX;
    }

  if (yychar <= YYEOF)
    {
      yychar = yytoken = YYEOF;
      YYDPRINTF ((stderr, "Now at end of input.\n"));
    }
  else
    {
      yytoken = YYTRANSLATE (yychar);
      YY_SYMBOL_PRINT ("Next token is", yytoken, &yylval, &yylloc);
    }

  /* If the proper action on seeing token YYTOKEN is to reduce or to
     detect an error, take that action.  */
  yyn += yytoken;
  if (yyn < 0 || YYLAST < yyn || yycheck[yyn] != yytoken)
    goto yydefault;
  yyn = yytable[yyn];
  if (yyn <= 0)
    {
      if (yyn == 0 || yyn == YYTABLE_NINF)
	goto yyerrlab;
      yyn = -yyn;
      goto yyreduce;
    }

  /* Count tokens shifted since error; after three, turn off error
     status.  */
  if (yyerrstatus)
    yyerrstatus--;

  /* Shift the lookahead token.  */
  YY_SYMBOL_PRINT ("Shifting", yytoken, &yylval, &yylloc);

  /* Discard the shifted token.  */
  yychar = YYEMPTY;

  yystate = yyn;
  *++yyvsp = yylval;

  goto yynewstate;


/*-----------------------------------------------------------.
| yydefault -- do the default action for the current state.  |
`-----------------------------------------------------------*/
yydefault:
  yyn = yydefact[yystate];
  if (yyn == 0)
    goto yyerrlab;
  goto yyreduce;


/*-----------------------------.
| yyreduce -- Do a reduction.  |
`-----------------------------*/
yyreduce:
  /* yyn is the number of a rule to reduce with.  */
  yylen = yyr2[yyn];

  /* If YYLEN is nonzero, implement the default value of the action:
     `$$ = $1'.

     Otherwise, the following line sets YYVAL to garbage.
     This behavior is undocumented and Bison
     users should not rely upon it.  Assigning to YYVAL
     unconditionally makes the parser a bit smaller, and it avoids a
     GCC warning that YYVAL may be used uninitialized.  */
  yyval = yyvsp[1-yylen];


  YY_REDUCE_PRINT (yyn);
  switch (yyn)
    {
        case 4:

/* Line 1455 of yacc.c  */
#line 125 "bison.y"
    { emit("STMT"); ;}
    break;

  case 5:

/* Line 1455 of yacc.c  */
#line 129 "bison.y"
    { emit_select((yyvsp[(1) - (7)].strval), (yyvsp[(6) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 6:

/* Line 1455 of yacc.c  */
#line 131 "bison.y"
    {  emit_load((yyvsp[(1) - (12)].strval), (yyvsp[(4) - (12)].strval), (yyvsp[(11) - (12)].intval), (yyvsp[(7) - (12)].strval)); ;}
    break;

  case 7:

/* Line 1455 of yacc.c  */
#line 133 "bison.y"
    {  emit_filter((yyvsp[(1) - (5)].strval), (yyvsp[(4) - (5)].strval));;}
    break;

  case 8:

/* Line 1455 of yacc.c  */
#line 135 "bison.y"
    {  emit_order((yyvsp[(1) - (6)].strval), (yyvsp[(4) - (6)].strval), (yyvsp[(6) - (6)].intval));;}
    break;

  case 9:

/* Line 1455 of yacc.c  */
#line 137 "bison.y"
    {  emit_join((yyvsp[(1) - (8)].strval),(yyvsp[(6) - (8)].strval),(yyvsp[(7) - (8)].intval),0,-1); ;}
    break;

  case 10:

/* Line 1455 of yacc.c  */
#line 139 "bison.y"
    {  emit_store((yyvsp[(2) - (9)].strval),(yyvsp[(4) - (9)].strval),(yyvsp[(7) - (9)].strval)); ;}
    break;

  case 11:

/* Line 1455 of yacc.c  */
#line 141 "bison.y"
    {  emit_store_binary((yyvsp[(2) - (7)].strval),(yyvsp[(4) - (7)].strval)); ;}
    break;

  case 12:

/* Line 1455 of yacc.c  */
#line 143 "bison.y"
    {  emit_describe_table((yyvsp[(2) - (2)].strval));;}
    break;

  case 13:

/* Line 1455 of yacc.c  */
#line 145 "bison.y"
    {  emit_insert((yyvsp[(3) - (7)].strval), (yyvsp[(7) - (7)].strval));;}
    break;

  case 14:

/* Line 1455 of yacc.c  */
#line 147 "bison.y"
    {  emit_delete((yyvsp[(3) - (5)].strval));;}
    break;

  case 15:

/* Line 1455 of yacc.c  */
#line 149 "bison.y"
    {  emit_display((yyvsp[(2) - (7)].strval), (yyvsp[(5) - (7)].strval));;}
    break;

  case 16:

/* Line 1455 of yacc.c  */
#line 151 "bison.y"
    {  emit_show_tables();;}
    break;

  case 17:

/* Line 1455 of yacc.c  */
#line 153 "bison.y"
    {  emit_drop_table((yyvsp[(3) - (3)].strval));;}
    break;

  case 18:

/* Line 1455 of yacc.c  */
#line 155 "bison.y"
    {  emit_create_bitmap_index((yyvsp[(3) - (22)].strval), (yyvsp[(5) - (22)].strval), (yyvsp[(7) - (22)].strval), (yyvsp[(9) - (22)].strval), (yyvsp[(18) - (22)].strval), (yyvsp[(22) - (22)].strval));;}
    break;

  case 19:

/* Line 1455 of yacc.c  */
#line 159 "bison.y"
    { emit_name((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 20:

/* Line 1455 of yacc.c  */
#line 160 "bison.y"
    { emit_fieldname((yyvsp[(1) - (3)].strval), (yyvsp[(3) - (3)].strval)); ;}
    break;

  case 21:

/* Line 1455 of yacc.c  */
#line 161 "bison.y"
    { emit("USERVAR %s", (yyvsp[(1) - (1)].strval)); ;}
    break;

  case 22:

/* Line 1455 of yacc.c  */
#line 162 "bison.y"
    { emit_string((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 23:

/* Line 1455 of yacc.c  */
#line 163 "bison.y"
    { emit_number((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 24:

/* Line 1455 of yacc.c  */
#line 164 "bison.y"
    { emit_decimal((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 25:

/* Line 1455 of yacc.c  */
#line 165 "bison.y"
    { emit_float((yyvsp[(1) - (1)].floatval)); ;}
    break;

  case 26:

/* Line 1455 of yacc.c  */
#line 166 "bison.y"
    { emit("BOOL %d", (yyvsp[(1) - (1)].intval)); ;}
    break;

  case 27:

/* Line 1455 of yacc.c  */
#line 167 "bison.y"
    { emit_vardecimal((yyvsp[(1) - (11)].strval), (yyvsp[(3) - (11)].intval), (yyvsp[(6) - (11)].strval),  (yyvsp[(8) - (11)].intval), (yyvsp[(10) - (11)].intval));;}
    break;

  case 28:

/* Line 1455 of yacc.c  */
#line 168 "bison.y"
    { emit_varchar((yyvsp[(1) - (9)].strval), (yyvsp[(3) - (9)].intval), (yyvsp[(6) - (9)].strval), (yyvsp[(8) - (9)].intval), "", "");;}
    break;

  case 29:

/* Line 1455 of yacc.c  */
#line 169 "bison.y"
    { emit_var((yyvsp[(1) - (6)].strval), (yyvsp[(3) - (6)].intval), (yyvsp[(6) - (6)].strval), "", "");;}
    break;

  case 30:

/* Line 1455 of yacc.c  */
#line 170 "bison.y"
    { emit_var_asc((yyvsp[(1) - (2)].strval));;}
    break;

  case 31:

/* Line 1455 of yacc.c  */
#line 171 "bison.y"
    { emit_var_desc((yyvsp[(1) - (2)].strval));;}
    break;

  case 32:

/* Line 1455 of yacc.c  */
#line 172 "bison.y"
    { emit_count(); ;}
    break;

  case 33:

/* Line 1455 of yacc.c  */
#line 173 "bison.y"
    { emit_sum(); ;}
    break;

  case 34:

/* Line 1455 of yacc.c  */
#line 174 "bison.y"
    { emit_average(); ;}
    break;

  case 35:

/* Line 1455 of yacc.c  */
#line 175 "bison.y"
    { emit_min(); ;}
    break;

  case 36:

/* Line 1455 of yacc.c  */
#line 176 "bison.y"
    { emit_max(); ;}
    break;

  case 37:

/* Line 1455 of yacc.c  */
#line 177 "bison.y"
    { emit_distinct(); ;}
    break;

  case 38:

/* Line 1455 of yacc.c  */
#line 178 "bison.y"
    { emit_year(); ;}
    break;

  case 39:

/* Line 1455 of yacc.c  */
#line 182 "bison.y"
    { emit_add(); ;}
    break;

  case 40:

/* Line 1455 of yacc.c  */
#line 183 "bison.y"
    { emit_minus(); ;}
    break;

  case 41:

/* Line 1455 of yacc.c  */
#line 184 "bison.y"
    { emit_mul(); ;}
    break;

  case 42:

/* Line 1455 of yacc.c  */
#line 185 "bison.y"
    { emit_div(); ;}
    break;

  case 43:

/* Line 1455 of yacc.c  */
#line 186 "bison.y"
    { emit("MOD"); ;}
    break;

  case 44:

/* Line 1455 of yacc.c  */
#line 187 "bison.y"
    { emit("MOD"); ;}
    break;

  case 45:

/* Line 1455 of yacc.c  */
#line 189 "bison.y"
    { emit_and(); ;}
    break;

  case 46:

/* Line 1455 of yacc.c  */
#line 190 "bison.y"
    { emit_eq(); ;}
    break;

  case 47:

/* Line 1455 of yacc.c  */
#line 191 "bison.y"
    { emit_or(); ;}
    break;

  case 48:

/* Line 1455 of yacc.c  */
#line 192 "bison.y"
    { emit("XOR"); ;}
    break;

  case 49:

/* Line 1455 of yacc.c  */
#line 193 "bison.y"
    { emit("SHIFT %s", (yyvsp[(2) - (3)].subtok)==1?"left":"right"); ;}
    break;

  case 50:

/* Line 1455 of yacc.c  */
#line 194 "bison.y"
    { emit("NOT"); ;}
    break;

  case 51:

/* Line 1455 of yacc.c  */
#line 195 "bison.y"
    { emit("NOT"); ;}
    break;

  case 52:

/* Line 1455 of yacc.c  */
#line 196 "bison.y"
    { emit_cmp((yyvsp[(2) - (3)].subtok)); ;}
    break;

  case 53:

/* Line 1455 of yacc.c  */
#line 197 "bison.y"
    { emit_cmp(7); ;}
    break;

  case 54:

/* Line 1455 of yacc.c  */
#line 199 "bison.y"
    { emit("CMPSELECT %d", (yyvsp[(2) - (5)].subtok)); ;}
    break;

  case 55:

/* Line 1455 of yacc.c  */
#line 200 "bison.y"
    {emit("EXPR");;}
    break;

  case 56:

/* Line 1455 of yacc.c  */
#line 201 "bison.y"
    { emit_case(); ;}
    break;

  case 57:

/* Line 1455 of yacc.c  */
#line 205 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(3) - (3)].intval)); ;}
    break;

  case 58:

/* Line 1455 of yacc.c  */
#line 206 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(4) - (4)].intval)); emit("NOT"); ;}
    break;

  case 59:

/* Line 1455 of yacc.c  */
#line 209 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 60:

/* Line 1455 of yacc.c  */
#line 212 "bison.y"
    { (yyval.intval) = (yyvsp[(3) - (3)].intval);;}
    break;

  case 61:

/* Line 1455 of yacc.c  */
#line 216 "bison.y"
    { (yyval.intval) = 1; emit_sel_name((yyvsp[(3) - (3)].strval));;}
    break;

  case 62:

/* Line 1455 of yacc.c  */
#line 217 "bison.y"
    { (yyval.intval) = (yyvsp[(1) - (5)].intval) + 1; emit_sel_name((yyvsp[(5) - (5)].strval));;}
    break;

  case 63:

/* Line 1455 of yacc.c  */
#line 218 "bison.y"
    { emit_sel_name("*");;}
    break;

  case 64:

/* Line 1455 of yacc.c  */
#line 222 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 65:

/* Line 1455 of yacc.c  */
#line 223 "bison.y"
    {(yyval.intval) = (yyvsp[(1) - (3)].intval) + 1; ;}
    break;

  case 66:

/* Line 1455 of yacc.c  */
#line 227 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 67:

/* Line 1455 of yacc.c  */
#line 228 "bison.y"
    { (yyval.intval) = 1 + (yyvsp[(3) - (3)].intval); ;}
    break;

  case 68:

/* Line 1455 of yacc.c  */
#line 231 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 70:

/* Line 1455 of yacc.c  */
#line 236 "bison.y"
    { emit("FILTER BY"); ;}
    break;

  case 71:

/* Line 1455 of yacc.c  */
#line 240 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (4)].strval), 'I');;}
    break;

  case 72:

/* Line 1455 of yacc.c  */
#line 241 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'L');;}
    break;

  case 73:

/* Line 1455 of yacc.c  */
#line 242 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'R');;}
    break;

  case 74:

/* Line 1455 of yacc.c  */
#line 243 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'O');;}
    break;

  case 75:

/* Line 1455 of yacc.c  */
#line 244 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (5)].strval), 'I'); ;}
    break;

  case 76:

/* Line 1455 of yacc.c  */
#line 245 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'L'); ;}
    break;

  case 77:

/* Line 1455 of yacc.c  */
#line 246 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'R'); ;}
    break;

  case 78:

/* Line 1455 of yacc.c  */
#line 247 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'O'); ;}
    break;

  case 79:

/* Line 1455 of yacc.c  */
#line 249 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 80:

/* Line 1455 of yacc.c  */
#line 252 "bison.y"
    { emit_limit((yyvsp[(2) - (2)].intval)); ;}
    break;

  case 81:

/* Line 1455 of yacc.c  */
#line 254 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 82:

/* Line 1455 of yacc.c  */
#line 257 "bison.y"
    { emit_sort((yyvsp[(4) - (4)].strval), 0); ;}
    break;

  case 83:

/* Line 1455 of yacc.c  */
#line 258 "bison.y"
    { emit_sort((yyvsp[(4) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 84:

/* Line 1455 of yacc.c  */
#line 259 "bison.y"
    { emit_presort((yyvsp[(3) - (3)].strval)); ;}
    break;



/* Line 1455 of yacc.c  */
#line 2281 "bison.cu"
      default: break;
    }
  YY_SYMBOL_PRINT ("-> $$ =", yyr1[yyn], &yyval, &yyloc);

  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);

  *++yyvsp = yyval;

  /* Now `shift' the result of the reduction.  Determine what state
     that goes to, based on the state we popped back to and the rule
     number reduced by.  */

  yyn = yyr1[yyn];

  yystate = yypgoto[yyn - YYNTOKENS] + *yyssp;
  if (0 <= yystate && yystate <= YYLAST && yycheck[yystate] == *yyssp)
    yystate = yytable[yystate];
  else
    yystate = yydefgoto[yyn - YYNTOKENS];

  goto yynewstate;


/*------------------------------------.
| yyerrlab -- here on detecting error |
`------------------------------------*/
yyerrlab:
  /* If not already recovering from an error, report this error.  */
  if (!yyerrstatus)
    {
      ++yynerrs;
#if ! YYERROR_VERBOSE
      yyerror (YY_("syntax error"));
#else
      {
	YYSIZE_T yysize = yysyntax_error (0, yystate, yychar);
	if (yymsg_alloc < yysize && yymsg_alloc < YYSTACK_ALLOC_MAXIMUM)
	  {
	    YYSIZE_T yyalloc = 2 * yysize;
	    if (! (yysize <= yyalloc && yyalloc <= YYSTACK_ALLOC_MAXIMUM))
	      yyalloc = YYSTACK_ALLOC_MAXIMUM;
	    if (yymsg != yymsgbuf)
	      YYSTACK_FREE (yymsg);
	    yymsg = (char *) YYSTACK_ALLOC (yyalloc);
	    if (yymsg)
	      yymsg_alloc = yyalloc;
	    else
	      {
		yymsg = yymsgbuf;
		yymsg_alloc = sizeof yymsgbuf;
	      }
	  }

	if (0 < yysize && yysize <= yymsg_alloc)
	  {
	    (void) yysyntax_error (yymsg, yystate, yychar);
	    yyerror (yymsg);
	  }
	else
	  {
	    yyerror (YY_("syntax error"));
	    if (yysize != 0)
	      goto yyexhaustedlab;
	  }
      }
#endif
    }



  if (yyerrstatus == 3)
    {
      /* If just tried and failed to reuse lookahead token after an
	 error, discard it.  */

      if (yychar <= YYEOF)
	{
	  /* Return failure if at end of input.  */
	  if (yychar == YYEOF)
	    YYABORT;
	}
      else
	{
	  yydestruct ("Error: discarding",
		      yytoken, &yylval);
	  yychar = YYEMPTY;
	}
    }

  /* Else will try to reuse lookahead token after shifting the error
     token.  */
  goto yyerrlab1;


/*---------------------------------------------------.
| yyerrorlab -- error raised explicitly by YYERROR.  |
`---------------------------------------------------*/
yyerrorlab:

  /* Pacify compilers like GCC when the user code never invokes
     YYERROR and the label yyerrorlab therefore never appears in user
     code.  */
  if (/*CONSTCOND*/ 0)
     goto yyerrorlab;

  /* Do not reclaim the symbols of the rule which action triggered
     this YYERROR.  */
  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);
  yystate = *yyssp;
  goto yyerrlab1;


/*-------------------------------------------------------------.
| yyerrlab1 -- common code for both syntax error and YYERROR.  |
`-------------------------------------------------------------*/
yyerrlab1:
  yyerrstatus = 3;	/* Each real token shifted decrements this.  */

  for (;;)
    {
      yyn = yypact[yystate];
      if (yyn != YYPACT_NINF)
	{
	  yyn += YYTERROR;
	  if (0 <= yyn && yyn <= YYLAST && yycheck[yyn] == YYTERROR)
	    {
	      yyn = yytable[yyn];
	      if (0 < yyn)
		break;
	    }
	}

      /* Pop the current state because it cannot handle the error token.  */
      if (yyssp == yyss)
	YYABORT;


      yydestruct ("Error: popping",
		  yystos[yystate], yyvsp);
      YYPOPSTACK (1);
      yystate = *yyssp;
      YY_STACK_PRINT (yyss, yyssp);
    }

  *++yyvsp = yylval;


  /* Shift the error token.  */
  YY_SYMBOL_PRINT ("Shifting", yystos[yyn], yyvsp, yylsp);

  yystate = yyn;
  goto yynewstate;


/*-------------------------------------.
| yyacceptlab -- YYACCEPT comes here.  |
`-------------------------------------*/
yyacceptlab:
  yyresult = 0;
  goto yyreturn;

/*-----------------------------------.
| yyabortlab -- YYABORT comes here.  |
`-----------------------------------*/
yyabortlab:
  yyresult = 1;
  goto yyreturn;

#if !defined(yyoverflow) || YYERROR_VERBOSE
/*-------------------------------------------------.
| yyexhaustedlab -- memory exhaustion comes here.  |
`-------------------------------------------------*/
yyexhaustedlab:
  yyerror (YY_("memory exhausted"));
  yyresult = 2;
  /* Fall through.  */
#endif

yyreturn:
  if (yychar != YYEMPTY)
     yydestruct ("Cleanup: discarding lookahead",
		 yytoken, &yylval);
  /* Do not reclaim the symbols of the rule which action triggered
     this YYABORT or YYACCEPT.  */
  YYPOPSTACK (yylen);
  YY_STACK_PRINT (yyss, yyssp);
  while (yyssp != yyss)
    {
      yydestruct ("Cleanup: popping",
		  yystos[*yyssp], yyvsp);
      YYPOPSTACK (1);
    }
#ifndef yyoverflow
  if (yyss != yyssa)
    YYSTACK_FREE (yyss);
#endif
#if YYERROR_VERBOSE
  if (yymsg != yymsgbuf)
    YYSTACK_FREE (yymsg);
#endif
  /* Make sure YYID is used.  */
  return YYID (yyresult);
}



/* Line 1675 of yacc.c  */
#line 261 "bison.y"


bool scan_state;
unsigned int statement_count;

int execute_file(int ac, char **av)
{
    bool just_once  = 0;
    string script;
    process_count = 1000000000; //1GB by default
    verbose = 0;
	ssd = 0;
	delta = 0;
    total_buffer_size = 0;
	hash_seed = 100;

    for (int i = 1; i < ac; i++) {
        if(strcmp(av[i],"-l") == 0) {
            process_count = 1000000*atoff(av[i+1]);
        }
        else if(strcmp(av[i],"-v") == 0) {
            verbose = 1;
        }
        else if(strcmp(av[i],"-delta") == 0) {
            delta = 1;
        }		
        else if(strcmp(av[i],"-ssd") == 0) {
            ssd = 1;
        }		
        else if(strcmp(av[i],"-i") == 0) {
            interactive = 1;
            break;
        }
        else if(strcmp(av[i],"-s") == 0) {
            just_once = 1;
            interactive = 1;
            script = av[i+1];
        };
    };

    load_col_data(data_dict, "data.dictionary");
	tot_disk = 0;

    if (!interactive) {
        if((yyin = fopen(av[ac-1], "r")) == nullptr) {
            perror(av[ac-1]);
            exit(1);
        };

        if(yyparse()) {
            printf("SQL scan parse failed\n");
            exit(1);
        };

        scan_state = 1;
        std::clock_t start1 = std::clock();

        load_vars();
		
        statement_count = 0;
        clean_queues();

        yyin = fopen(av[ac-1], "r");
        PROC_FLUSH_BUF ( yyin );
        statement_count = 0;

        extern FILE *yyin;
        context = CreateCudaDevice(0, nullptr, verbose);

        if(!yyparse()) {
            if(verbose)
                cout << "SQL scan parse worked " << endl;
        }
        else
            cout << "SQL scan parse failed" << endl;

        fclose(yyin);
        for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
            (*it).second->free();
        };

        if(verbose) {
            cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
			cout<< "disk time " << ( tot_disk / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
        };
    }
    else {
        context = CreateCudaDevice(0, nullptr, verbose);        
        if(!just_once)
            getline(cin, script);

        while (script != "exit" && script != "EXIT") {

            used_vars.clear();
            yy_scan_string(script.c_str());
            scan_state = 0;
            statement_count = 0;
            clean_queues();
            if(yyparse()) {
                printf("SQL scan parse failed \n");
                getline(cin, script);
                continue;
            };

            scan_state = 1;

            load_vars();

            statement_count = 0;
            clean_queues();
            yy_scan_string(script.c_str());
            std::clock_t start1 = std::clock();

            if(!yyparse()) {
                if(verbose)
                    cout << "SQL scan parse worked " <<  endl;
            };
            for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
                (*it).second->free();
            };
            varNames.clear();

            if(verbose) {
                cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << endl;
            };
            if(!just_once)
                getline(cin, script);
            else
                script = "exit";
        };

        while(!buffer_names.empty()) {
            //delete [] buffers[buffer_names.front()];
			hipHostFree(buffers[buffer_names.front()]);
            buffer_sizes.erase(buffer_names.front());
            buffers.erase(buffer_names.front());
            buffer_names.pop();
        };
		for(auto it = index_buffers.begin(); it != index_buffers.end();it++) {
			hipHostFree(it->second);
        };

    };
    if(save_dict) {
        save_col_data(data_dict,"data.dictionary");
	};	

    if(alloced_sz) {
        hipFree(alloced_tmp);
        alloced_sz = 0;
    };
	scratch.resize(0);
	scratch.shrink_to_fit();
    return 0;
}



//external c global to report errors
//char alenka_err[4048];


int alenkaExecute(char *s)
{
    YY_BUFFER_STATE bp;

    total_buffer_size = 0;
    scan_state = 0;
    load_col_data(data_dict, "data.dictionary");
    std::clock_t start;

    if(verbose)
        start = std::clock();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    int ret = yyparse();
    //printf("execute: returned [%d]\n", ret);
    if(!ret) {
        if(verbose)
            cout << "SQL scan parse worked" << endl;
    }

    scan_state = 1;
    load_vars();
    statement_count = 0;
    clean_queues();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    if(!yyparse()) {
        if(verbose)
            cout << "SQL scan parse worked " << endl;
    }
    else
        cout << "SQL scan parse failed" << endl;

    yy_delete_buffer(bp);

    // Clear Vars
    for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
        (*it).second->free();
    };
    varNames.clear();

    if(verbose)
        cout<< "statement time " <<  ( ( std::clock() - start ) / (double)CLOCKS_PER_SEC ) << endl;
    if(save_dict)
        save_col_data(data_dict,"data.dictionary");
    return ret;
}



