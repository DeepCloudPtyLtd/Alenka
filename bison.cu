#include "hip/hip_runtime.h"

/* A Bison parser, made by GNU Bison 2.4.1.  */

/* Skeleton implementation for Bison's Yacc-like parsers in C
   
      Copyright (C) 1984, 1989, 1990, 2000, 2001, 2002, 2003, 2004, 2005, 2006
   Free Software Foundation, Inc.
   
   This program is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   This program is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.  */

/* As a special exception, you may create a larger work that contains
   part or all of the Bison parser skeleton and distribute that work
   under terms of your choice, so long as that work isn't itself a
   parser generator using the skeleton or a modified version thereof
   as a parser skeleton.  Alternatively, if you modify or redistribute
   the parser skeleton itself, you may (at your option) remove this
   special exception, which will cause the skeleton and the resulting
   Bison output files to be licensed under the GNU General Public
   License without this special exception.
   
   This special exception was added by the Free Software Foundation in
   version 2.2 of Bison.  */

/* C LALR(1) parser skeleton written by Richard Stallman, by
   simplifying the original so-called "semantic" parser.  */

/* All symbols defined below should begin with yy or YY, to avoid
   infringing on user name space.  This should be done even for local
   variables, as they might otherwise be expanded by user macros.
   There are some unavoidable exceptions within include files to
   define necessary library symbols; they are noted "INFRINGES ON
   USER NAME SPACE" below.  */

/* Identify Bison output.  */
#define YYBISON 1

/* Bison version.  */
#define YYBISON_VERSION "2.4.1"

/* Skeleton name.  */
#define YYSKELETON_NAME "yacc.c"

/* Pure parsers.  */
#define YYPURE 0

/* Push parsers.  */
#define YYPUSH 0

/* Pull parsers.  */
#define YYPULL 1

/* Using locations.  */
#define YYLSP_NEEDED 0



/* Copy the first part of user declarations.  */

/* Line 189 of yacc.c  */
#line 15 "bison.y"



#include "lex.yy.c"
#include "cm.h"
#include "operators.h"




/* Line 189 of yacc.c  */
#line 84 "bison.cu"

/* Enabling traces.  */
#ifndef YYDEBUG
# define YYDEBUG 0
#endif

/* Enabling verbose error messages.  */
#ifdef YYERROR_VERBOSE
# undef YYERROR_VERBOSE
# define YYERROR_VERBOSE 1
#else
# define YYERROR_VERBOSE 0
#endif

/* Enabling the token table.  */
#ifndef YYTOKEN_TABLE
# define YYTOKEN_TABLE 0
#endif


/* Tokens.  */
#ifndef YYTOKENTYPE
# define YYTOKENTYPE
   /* Put the tokens into the symbol table, so that GDB and other debuggers
      know about them.  */
   enum yytokentype {
     FILENAME = 258,
     NAME = 259,
     STRING = 260,
     INTNUM = 261,
     DECIMAL1 = 262,
     BOOL1 = 263,
     APPROXNUM = 264,
     USERVAR = 265,
     ASSIGN = 266,
     EQUAL = 267,
     OR = 268,
     XOR = 269,
     AND = 270,
     DISTINCT = 271,
     REGEXP = 272,
     LIKE = 273,
     IS = 274,
     IN = 275,
     NOT = 276,
     BETWEEN = 277,
     COMPARISON = 278,
     SHIFT = 279,
     MOD = 280,
     FROM = 281,
     MULITE = 282,
     DELETE = 283,
     LOAD = 284,
     FILTER = 285,
     BY = 286,
     JOIN = 287,
     STORE = 288,
     INTO = 289,
     GROUP = 290,
     SELECT = 291,
     AS = 292,
     ORDER = 293,
     ASC = 294,
     DESC = 295,
     COUNT = 296,
     USING = 297,
     SUM = 298,
     AVG = 299,
     MIN = 300,
     MAX = 301,
     LIMIT = 302,
     ON = 303,
     BINARY = 304,
     LEFT = 305,
     RIGHT = 306,
     OUTER = 307,
     SORT = 308,
     SEGMENTS = 309,
     PRESORTED = 310,
     PARTITION = 311,
     INSERT = 312,
     WHERE = 313,
     DISPLAY = 314,
     CASE = 315,
     WHEN = 316,
     THEN = 317,
     ELSE = 318,
     END = 319,
     REFERENCES = 320,
     SHOW = 321,
     TABLES = 322,
     TABLE = 323,
     DESCRIBE = 324,
     DROP = 325,
     CREATE = 326,
     BITMAP = 327,
     INDEX = 328
   };
#endif



#if ! defined YYSTYPE && ! defined YYSTYPE_IS_DECLARED
typedef union YYSTYPE
{

/* Line 214 of yacc.c  */
#line 25 "bison.y"

    long long int intval;
    double floatval;
    char *strval;
    int subtok;



/* Line 214 of yacc.c  */
#line 202 "bison.cu"
} YYSTYPE;
# define YYSTYPE_IS_TRIVIAL 1
# define yystype YYSTYPE /* obsolescent; will be withdrawn */
# define YYSTYPE_IS_DECLARED 1
#endif


/* Copy the second part of user declarations.  */


/* Line 264 of yacc.c  */
#line 214 "bison.cu"

#ifdef short
# undef short
#endif

#ifdef YYTYPE_UINT8
typedef YYTYPE_UINT8 yytype_uint8;
#else
typedef unsigned char yytype_uint8;
#endif

#ifdef YYTYPE_INT8
typedef YYTYPE_INT8 yytype_int8;
#elif (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
typedef signed char yytype_int8;
#else
typedef short int yytype_int8;
#endif

#ifdef YYTYPE_UINT16
typedef YYTYPE_UINT16 yytype_uint16;
#else
typedef unsigned short int yytype_uint16;
#endif

#ifdef YYTYPE_INT16
typedef YYTYPE_INT16 yytype_int16;
#else
typedef short int yytype_int16;
#endif

#ifndef YYSIZE_T
# ifdef __SIZE_TYPE__
#  define YYSIZE_T __SIZE_TYPE__
# elif defined size_t
#  define YYSIZE_T size_t
# elif ! defined YYSIZE_T && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#  include <stddef.h> /* INFRINGES ON USER NAME SPACE */
#  define YYSIZE_T size_t
# else
#  define YYSIZE_T unsigned int
# endif
#endif

#define YYSIZE_MAXIMUM ((YYSIZE_T) -1)

#ifndef YY_
# if YYENABLE_NLS
#  if ENABLE_NLS
#   include <libintl.h> /* INFRINGES ON USER NAME SPACE */
#   define YY_(msgid) dgettext ("bison-runtime", msgid)
#  endif
# endif
# ifndef YY_
#  define YY_(msgid) msgid
# endif
#endif

/* Suppress unused-variable warnings by "using" E.  */
#if ! defined lint || defined __GNUC__
# define YYUSE(e) ((void) (e))
#else
# define YYUSE(e) /* empty */
#endif

/* Identity function, used to suppress warnings about constant conditions.  */
#ifndef lint
# define YYID(n) (n)
#else
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static int
YYID (int yyi)
#else
static int
YYID (yyi)
    int yyi;
#endif
{
  return yyi;
}
#endif

#if ! defined yyoverflow || YYERROR_VERBOSE

/* The parser invokes alloca or malloc; define the necessary symbols.  */

# ifdef YYSTACK_USE_ALLOCA
#  if YYSTACK_USE_ALLOCA
#   ifdef __GNUC__
#    define YYSTACK_ALLOC __builtin_alloca
#   elif defined __BUILTIN_VA_ARG_INCR
#    include <alloca.h> /* INFRINGES ON USER NAME SPACE */
#   elif defined _AIX
#    define YYSTACK_ALLOC __alloca
#   elif defined _MSC_VER
#    include <malloc.h> /* INFRINGES ON USER NAME SPACE */
#    define alloca _alloca
#   else
#    define YYSTACK_ALLOC alloca
#    if ! defined _ALLOCA_H && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
#     include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#     ifndef _STDLIB_H
#      define _STDLIB_H 1
#     endif
#    endif
#   endif
#  endif
# endif

# ifdef YYSTACK_ALLOC
   /* Pacify GCC's `empty if-body' warning.  */
#  define YYSTACK_FREE(Ptr) do { /* empty */; } while (YYID (0))
#  ifndef YYSTACK_ALLOC_MAXIMUM
    /* The OS might guarantee only one guard page at the bottom of the stack,
       and a page size can be as small as 4096 bytes.  So we cannot safely
       invoke alloca (N) if N exceeds 4096.  Use a slightly smaller number
       to allow for a few compiler-allocated temporary stack slots.  */
#   define YYSTACK_ALLOC_MAXIMUM 4032 /* reasonable circa 2006 */
#  endif
# else
#  define YYSTACK_ALLOC YYMALLOC
#  define YYSTACK_FREE YYFREE
#  ifndef YYSTACK_ALLOC_MAXIMUM
#   define YYSTACK_ALLOC_MAXIMUM YYSIZE_MAXIMUM
#  endif
#  if (defined __cplusplus && ! defined _STDLIB_H \
       && ! ((defined YYMALLOC || defined malloc) \
	     && (defined YYFREE || defined free)))
#   include <stdlib.h> /* INFRINGES ON USER NAME SPACE */
#   ifndef _STDLIB_H
#    define _STDLIB_H 1
#   endif
#  endif
#  ifndef YYMALLOC
#   define YYMALLOC malloc
#   if ! defined malloc && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void *malloc (YYSIZE_T); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
#  ifndef YYFREE
#   define YYFREE free
#   if ! defined free && ! defined _STDLIB_H && (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
void free (void *); /* INFRINGES ON USER NAME SPACE */
#   endif
#  endif
# endif
#endif /* ! defined yyoverflow || YYERROR_VERBOSE */


#if (! defined yyoverflow \
     && (! defined __cplusplus \
	 || (defined YYSTYPE_IS_TRIVIAL && YYSTYPE_IS_TRIVIAL)))

/* A type that is properly aligned for any stack member.  */
union yyalloc
{
  yytype_int16 yyss_alloc;
  YYSTYPE yyvs_alloc;
};

/* The size of the maximum gap between one aligned stack and the next.  */
# define YYSTACK_GAP_MAXIMUM (sizeof (union yyalloc) - 1)

/* The size of an array large to enough to hold all stacks, each with
   N elements.  */
# define YYSTACK_BYTES(N) \
     ((N) * (sizeof (yytype_int16) + sizeof (YYSTYPE)) \
      + YYSTACK_GAP_MAXIMUM)

/* Copy COUNT objects from FROM to TO.  The source and destination do
   not overlap.  */
# ifndef YYCOPY
#  if defined __GNUC__ && 1 < __GNUC__
#   define YYCOPY(To, From, Count) \
      __builtin_memcpy (To, From, (Count) * sizeof (*(From)))
#  else
#   define YYCOPY(To, From, Count)		\
      do					\
	{					\
	  YYSIZE_T yyi;				\
	  for (yyi = 0; yyi < (Count); yyi++)	\
	    (To)[yyi] = (From)[yyi];		\
	}					\
      while (YYID (0))
#  endif
# endif

/* Relocate STACK from its old location to the new one.  The
   local variables YYSIZE and YYSTACKSIZE give the old and new number of
   elements in the stack, and YYPTR gives the new location of the
   stack.  Advance YYPTR to a properly aligned location for the next
   stack.  */
# define YYSTACK_RELOCATE(Stack_alloc, Stack)				\
    do									\
      {									\
	YYSIZE_T yynewbytes;						\
	YYCOPY (&yyptr->Stack_alloc, Stack, yysize);			\
	Stack = &yyptr->Stack_alloc;					\
	yynewbytes = yystacksize * sizeof (*Stack) + YYSTACK_GAP_MAXIMUM; \
	yyptr += yynewbytes / sizeof (*yyptr);				\
      }									\
    while (YYID (0))

#endif

/* YYFINAL -- State number of the termination state.  */
#define YYFINAL  22
/* YYLAST -- Last index in YYTABLE.  */
#define YYLAST   643

/* YYNTOKENS -- Number of terminals.  */
#define YYNTOKENS  91
/* YYNNTS -- Number of nonterminals.  */
#define YYNNTS  14
/* YYNRULES -- Number of rules.  */
#define YYNRULES  84
/* YYNRULES -- Number of states.  */
#define YYNSTATES  250

/* YYTRANSLATE(YYLEX) -- Bison symbol number corresponding to YYLEX.  */
#define YYUNDEFTOK  2
#define YYMAXUTOK   328

#define YYTRANSLATE(YYX)						\
  ((unsigned int) (YYX) <= YYMAXUTOK ? yytranslate[YYX] : YYUNDEFTOK)

/* YYTRANSLATE[YYLEX] -- Bison symbol number corresponding to YYLEX.  */
static const yytype_uint8 yytranslate[] =
{
       0,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    21,     2,     2,     2,    32,    26,     2,
      84,    85,    30,    28,    87,    29,    86,    31,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,    90,    83,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,    34,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,    88,    25,    89,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     2,     2,     2,     2,
       2,     2,     2,     2,     2,     2,     1,     2,     3,     4,
       5,     6,     7,     8,     9,    10,    11,    12,    13,    14,
      15,    16,    17,    18,    19,    20,    22,    23,    24,    27,
      33,    35,    36,    37,    38,    39,    40,    41,    42,    43,
      44,    45,    46,    47,    48,    49,    50,    51,    52,    53,
      54,    55,    56,    57,    58,    59,    60,    61,    62,    63,
      64,    65,    66,    67,    68,    69,    70,    71,    72,    73,
      74,    75,    76,    77,    78,    79,    80,    81,    82
};

#if YYDEBUG
/* YYPRHS[YYN] -- Index of the first RHS symbol of rule number YYN in
   YYRHS.  */
static const yytype_uint16 yyprhs[] =
{
       0,     0,     3,     6,    10,    12,    20,    33,    39,    46,
      55,    65,    73,    76,    84,    90,    98,   101,   105,   129,
     131,   135,   137,   139,   141,   143,   145,   147,   162,   172,
     184,   191,   194,   197,   202,   207,   212,   217,   222,   225,
     229,   233,   237,   241,   245,   249,   253,   257,   261,   265,
     269,   272,   275,   279,   283,   289,   293,   302,   306,   311,
     312,   316,   320,   326,   328,   330,   334,   336,   340,   341,
     343,   346,   351,   357,   363,   369,   375,   382,   389,   396,
     397,   400,   401,   406,   414
};

/* YYRHS -- A `-1'-separated list of the rules' RHS.  */
static const yytype_int8 yyrhs[] =
{
      92,     0,    -1,    93,    83,    -1,    92,    93,    83,    -1,
      94,    -1,     4,    11,    45,    97,    35,     4,    96,    -1,
       4,    11,    38,     3,    51,    84,     3,    85,    46,    84,
      98,    85,    -1,     4,    11,    39,     4,   101,    -1,     4,
      11,    47,     4,    40,   100,    -1,     4,    11,    45,    97,
      35,     4,   102,    96,    -1,    42,     4,    43,     3,    51,
      84,     3,    85,   103,    -1,    42,     4,    43,     3,   103,
      58,   104,    -1,    78,     4,    -1,    66,    43,     4,    45,
      97,    35,     4,    -1,    37,    35,     4,    67,    95,    -1,
      68,     4,    51,    84,     3,    85,   103,    -1,    75,    76,
      -1,    79,    77,     4,    -1,    80,    81,    82,     4,    57,
       4,    84,     4,    86,     4,    85,    35,     4,    87,     4,
      67,     4,    86,     4,    12,     4,    86,     4,    -1,     4,
      -1,     4,    86,     4,    -1,    10,    -1,     5,    -1,     6,
      -1,     9,    -1,     7,    -1,     8,    -1,     4,    88,     6,
      89,    90,     4,    84,     6,    85,    74,     4,    84,     4,
      85,    -1,     4,    88,     6,    89,    90,     4,    84,     6,
      85,    -1,     4,    88,     6,    89,    90,     4,    74,     4,
      84,     4,    85,    -1,     4,    88,     6,    89,    90,     4,
      -1,     4,    48,    -1,     4,    49,    -1,    50,    84,    95,
      85,    -1,    52,    84,    95,    85,    -1,    53,    84,    95,
      85,    -1,    54,    84,    95,    85,    -1,    55,    84,    95,
      85,    -1,    16,    95,    -1,    95,    28,    95,    -1,    95,
      29,    95,    -1,    95,    30,    95,    -1,    95,    31,    95,
      -1,    95,    32,    95,    -1,    95,    33,    95,    -1,    95,
      15,    95,    -1,    95,    12,    95,    -1,    95,    13,    95,
      -1,    95,    14,    95,    -1,    95,    27,    95,    -1,    22,
      95,    -1,    21,    95,    -1,    95,    24,    95,    -1,    95,
      18,    95,    -1,    95,    24,    84,    94,    85,    -1,    84,
      95,    85,    -1,    69,    70,    95,    71,    95,    72,    95,
      73,    -1,    95,    19,     8,    -1,    95,    19,    22,     8,
      -1,    -1,    44,    40,    99,    -1,    95,    46,     4,    -1,
      97,    87,    95,    46,     4,    -1,    30,    -1,    95,    -1,
      98,    87,    95,    -1,    95,    -1,    95,    87,    99,    -1,
      -1,    99,    -1,    40,    95,    -1,    41,     4,    57,    95,
      -1,    59,    41,     4,    57,    95,    -1,    60,    41,     4,
      57,    95,    -1,    61,    41,     4,    57,    95,    -1,    41,
       4,    57,    95,   102,    -1,    59,    41,     4,    57,    95,
     102,    -1,    60,    41,     4,    57,    95,   102,    -1,    61,
      41,     4,    57,    95,   102,    -1,    -1,    56,     6,    -1,
      -1,    62,    63,    40,     4,    -1,    62,    63,    40,     4,
      65,    40,     6,    -1,    64,    40,     4,    -1
};

/* YYRLINE[YYN] -- source line where rule number YYN was defined.  */
static const yytype_uint16 yyrline[] =
{
       0,   120,   120,   121,   125,   128,   130,   132,   134,   136,
     138,   140,   142,   144,   146,   148,   150,   152,   154,   159,
     160,   161,   162,   163,   164,   165,   166,   167,   168,   169,
     170,   171,   172,   173,   174,   175,   176,   177,   178,   182,
     183,   184,   185,   186,   187,   189,   190,   191,   192,   193,
     194,   195,   196,   197,   199,   200,   201,   205,   206,   209,
     212,   216,   217,   218,   222,   223,   227,   228,   231,   233,
     236,   240,   241,   242,   243,   244,   245,   246,   247,   249,
     252,   254,   257,   258,   259
};
#endif

#if YYDEBUG || YYERROR_VERBOSE || YYTOKEN_TABLE
/* YYTNAME[SYMBOL-NUM] -- String name of the symbol SYMBOL-NUM.
   First, the terminals, then, starting at YYNTOKENS, nonterminals.  */
static const char *const yytname[] =
{
  "$end", "error", "$undefined", "FILENAME", "NAME", "STRING", "INTNUM",
  "DECIMAL1", "BOOL1", "APPROXNUM", "USERVAR", "ASSIGN", "EQUAL", "OR",
  "XOR", "AND", "DISTINCT", "REGEXP", "LIKE", "IS", "IN", "'!'", "NOT",
  "BETWEEN", "COMPARISON", "'|'", "'&'", "SHIFT", "'+'", "'-'", "'*'",
  "'/'", "'%'", "MOD", "'^'", "FROM", "MULITE", "DELETE", "LOAD", "FILTER",
  "BY", "JOIN", "STORE", "INTO", "GROUP", "SELECT", "AS", "ORDER", "ASC",
  "DESC", "COUNT", "USING", "SUM", "AVG", "MIN", "MAX", "LIMIT", "ON",
  "BINARY", "LEFT", "RIGHT", "OUTER", "SORT", "SEGMENTS", "PRESORTED",
  "PARTITION", "INSERT", "WHERE", "DISPLAY", "CASE", "WHEN", "THEN",
  "ELSE", "END", "REFERENCES", "SHOW", "TABLES", "TABLE", "DESCRIBE",
  "DROP", "CREATE", "BITMAP", "INDEX", "';'", "'('", "')'", "'.'", "','",
  "'{'", "'}'", "':'", "$accept", "stmt_list", "stmt", "select_stmt",
  "expr", "opt_group_list", "expr_list", "load_list", "val_list",
  "opt_val_list", "opt_where", "join_list", "opt_limit", "sort_def", 0
};
#endif

# ifdef YYPRINT
/* YYTOKNUM[YYLEX-NUM] -- Internal token number corresponding to
   token YYLEX-NUM.  */
static const yytype_uint16 yytoknum[] =
{
       0,   256,   257,   258,   259,   260,   261,   262,   263,   264,
     265,   266,   267,   268,   269,   270,   271,   272,   273,   274,
     275,    33,   276,   277,   278,   124,    38,   279,    43,    45,
      42,    47,    37,   280,    94,   281,   282,   283,   284,   285,
     286,   287,   288,   289,   290,   291,   292,   293,   294,   295,
     296,   297,   298,   299,   300,   301,   302,   303,   304,   305,
     306,   307,   308,   309,   310,   311,   312,   313,   314,   315,
     316,   317,   318,   319,   320,   321,   322,   323,   324,   325,
     326,   327,   328,    59,    40,    41,    46,    44,   123,   125,
      58
};
# endif

/* YYR1[YYN] -- Symbol number of symbol that rule YYN derives.  */
static const yytype_uint8 yyr1[] =
{
       0,    91,    92,    92,    93,    94,    94,    94,    94,    94,
      94,    94,    94,    94,    94,    94,    94,    94,    94,    95,
      95,    95,    95,    95,    95,    95,    95,    95,    95,    95,
      95,    95,    95,    95,    95,    95,    95,    95,    95,    95,
      95,    95,    95,    95,    95,    95,    95,    95,    95,    95,
      95,    95,    95,    95,    95,    95,    95,    95,    95,    96,
      96,    97,    97,    97,    98,    98,    99,    99,   100,   100,
     101,   102,   102,   102,   102,   102,   102,   102,   102,   103,
     103,   104,   104,   104,   104
};

/* YYR2[YYN] -- Number of symbols composing right hand side of rule YYN.  */
static const yytype_uint8 yyr2[] =
{
       0,     2,     2,     3,     1,     7,    12,     5,     6,     8,
       9,     7,     2,     7,     5,     7,     2,     3,    23,     1,
       3,     1,     1,     1,     1,     1,     1,    14,     9,    11,
       6,     2,     2,     4,     4,     4,     4,     4,     2,     3,
       3,     3,     3,     3,     3,     3,     3,     3,     3,     3,
       2,     2,     3,     3,     5,     3,     8,     3,     4,     0,
       3,     3,     5,     1,     1,     3,     1,     3,     0,     1,
       2,     4,     5,     5,     5,     5,     6,     6,     6,     0,
       2,     0,     4,     7,     3
};

/* YYDEFACT[STATE-NAME] -- Default rule to reduce with in state
   STATE-NUM when YYTABLE doesn't specify something else to do.  Zero
   means the default is an error.  */
static const yytype_uint8 yydefact[] =
{
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     4,     0,     0,     0,     0,     0,    16,    12,
       0,     0,     1,     0,     2,     0,     0,     0,     0,     0,
       0,     0,     0,    17,     0,     3,     0,     0,    19,    22,
      23,    25,    26,    24,    21,     0,     0,     0,    63,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
      79,     0,     0,     0,     0,     0,     7,    31,    32,     0,
       0,    38,    51,    50,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,     0,     0,
       0,     0,     0,     0,     0,     0,     0,     0,    68,    14,
       0,     0,     0,     0,     0,     0,     0,    70,    20,     0,
       0,     0,     0,     0,     0,     0,    55,    46,    47,    48,
      45,    53,    57,     0,     0,    52,    49,    39,    40,    41,
      42,    43,    44,    61,    59,     0,    66,    69,     8,     0,
      80,    81,     0,    79,     0,     0,     0,    33,    34,    35,
      36,    37,     0,    58,    19,     0,     0,     0,     0,     0,
       0,     5,    59,     0,     0,     0,     0,     0,    11,    13,
      15,     0,     0,     0,     0,    54,     0,     0,     0,     0,
       0,     9,    62,    67,    79,     0,     0,     0,     0,    30,
       0,     0,    60,     0,     0,     0,    10,     0,    84,     0,
       0,     0,     0,     0,    71,     0,     0,     0,    82,     0,
      64,     0,     0,     0,    56,    75,    72,    73,    74,     0,
       0,     6,     0,     0,    28,    76,    77,    78,     0,     0,
      65,     0,     0,    83,     0,    29,     0,     0,     0,     0,
       0,     0,    27,     0,     0,     0,     0,     0,     0,    18
};

/* YYDEFGOTO[NTERM-NUM].  */
static const yytype_int16 yydefgoto[] =
{
      -1,    10,    11,    12,   136,   161,    57,   211,   137,   138,
      66,   162,   102,   168
};

/* YYPACT[STATE-NUM] -- Index in YYTABLE of the portion describing
   STATE-NUM.  */
#define YYPACT_NINF -188
static const yytype_int16 yypact[] =
{
     163,     5,     1,    23,     2,    42,    -7,    64,     8,    22,
     184,     3,  -188,    61,   101,    75,   119,    73,  -188,  -188,
     130,    55,  -188,    56,  -188,   137,   144,   122,   145,    74,
     147,   115,    77,  -188,   158,  -188,   117,   126,   -44,  -188,
    -188,  -188,  -188,  -188,  -188,   149,   149,   149,  -188,    85,
      97,    98,    99,   102,   120,   149,   515,   -34,   152,   149,
      28,   122,   182,   132,   103,   149,  -188,  -188,  -188,   189,
     188,    63,   610,   610,   149,   149,   149,   149,   149,   149,
     283,   149,   149,   149,   149,   149,     0,   203,   149,   149,
     149,   149,   149,   149,   149,   192,   193,   149,   149,   564,
     114,   208,   157,   -28,   131,   213,   217,   564,  -188,   133,
     305,   327,   351,   373,   395,   469,  -188,   564,   584,    83,
      63,   600,  -188,   215,    67,   238,   216,   -18,   -18,  -188,
    -188,  -188,  -188,  -188,   -35,   541,   261,  -188,  -188,   224,
    -188,   -29,   226,   172,   148,   150,   146,  -188,  -188,  -188,
    -188,  -188,   149,  -188,    -8,   154,   230,   197,   199,   210,
     219,  -188,   233,   250,   149,   176,   218,   242,  -188,  -188,
    -188,   274,   237,   280,   441,  -188,   229,   149,   295,   296,
     299,  -188,  -188,  -188,   172,   264,   301,   220,   225,   -63,
     149,   149,  -188,   251,   265,   268,  -188,   317,  -188,   322,
     149,   323,   324,   419,   491,   149,   149,   149,   263,   246,
     564,   -48,   259,   262,  -188,  -188,   491,   491,   491,   304,
     314,  -188,   149,   346,   278,  -188,  -188,  -188,   347,   357,
     564,   277,   363,  -188,   284,  -188,   288,   369,   370,   309,
     292,   385,  -188,   307,   390,   383,   392,   312,   407,  -188
};

/* YYPGOTO[NTERM-NUM].  */
static const yytype_int16 yypgoto[] =
{
    -188,  -188,   389,   291,   -27,   254,   356,  -188,  -154,  -188,
    -188,  -187,  -141,  -188
};

/* YYTABLE[YYPACT[STATE-NUM]].  What to do in state STATE-NUM.  If
   positive, shift that token.  If negative, reduce the rule which
   number is the opposite.  If zero, do what YYDEFACT says.
   If YYTABLE_NINF, syntax error.  */
#define YYTABLE_NINF -1
static const yytype_int16 yytable[] =
{
      56,    96,   170,    13,    67,    68,   156,   142,   122,   157,
     183,   201,    91,    92,    93,    94,    13,   215,    71,    72,
      73,   202,   123,   192,   158,   159,   160,    15,    80,   225,
     226,   227,    99,   166,    56,   167,    14,   221,   107,   222,
      67,    68,    69,   196,    70,    16,    17,   110,   111,   112,
     113,   114,   115,    97,   117,   118,   119,   120,   121,    97,
     125,   126,   127,   128,   129,   130,   131,   132,    19,    18,
     135,   154,    39,    40,    41,    42,    43,    44,    69,   100,
      70,    85,    86,    45,   101,    20,    24,    87,    46,    47,
      88,    89,    90,    91,    92,    93,    94,    80,    84,    25,
      26,    85,    86,    21,     2,    29,    27,    87,    28,     3,
      88,    89,    90,    91,    92,    93,    94,    49,    30,    50,
      51,    52,    53,    31,    32,   174,    38,    39,    40,    41,
      42,    43,    44,     4,    33,     5,    54,    34,    45,    35,
      36,    59,     6,    46,    47,     7,     8,     9,    37,    58,
      60,    55,    48,    38,    39,    40,    41,    42,    43,    44,
      61,    62,    63,   203,   204,    45,    65,     1,    64,    74,
      46,    47,    49,   210,    50,    51,    52,    53,   216,   217,
     218,    75,    76,    77,    22,   104,    78,   106,     1,   105,
      79,    54,    98,   108,   109,   230,   133,   134,   139,    49,
       2,    50,    51,    52,    53,     3,    55,    38,    39,    40,
      41,    42,    43,    44,   140,   141,   143,   144,    54,    45,
     145,     2,   146,   153,    46,    47,     3,   165,   101,     4,
     169,     5,   171,    55,   176,   172,   173,   177,     6,   175,
     178,     7,     8,     9,    89,    90,    91,    92,    93,    94,
       4,   179,     5,    49,   182,    50,    51,    52,    53,     6,
     180,   184,     7,     8,     9,    88,    89,    90,    91,    92,
      93,    94,    54,    81,    82,    83,    84,   157,   187,    85,
      86,   185,   186,   188,   189,    87,   191,   124,    88,    89,
      90,    91,    92,    93,    94,    81,    82,    83,    84,   193,
     194,    85,    86,   195,   197,   198,   199,    87,   205,   200,
      88,    89,    90,    91,    92,    93,    94,    81,    82,    83,
      84,   208,   206,    85,    86,   207,   209,   212,   219,    87,
     213,   220,    88,    89,    90,    91,    92,    93,    94,    81,
      82,    83,    84,   223,   228,    85,    86,   224,   164,   229,
     231,    87,   232,   233,    88,    89,    90,    91,    92,    93,
      94,   234,   235,    81,    82,    83,    84,   236,   116,    85,
      86,   237,   238,   239,   240,    87,   241,   242,    88,    89,
      90,    91,    92,    93,    94,    81,    82,    83,    84,   243,
     147,    85,    86,   244,   245,   246,   247,    87,   248,    23,
      88,    89,    90,    91,    92,    93,    94,    81,    82,    83,
      84,   249,   148,    85,    86,   155,   181,   103,     0,    87,
       0,     0,    88,    89,    90,    91,    92,    93,    94,     0,
       0,    81,    82,    83,    84,     0,   149,    85,    86,     0,
       0,     0,     0,    87,     0,     0,    88,    89,    90,    91,
      92,    93,    94,    81,    82,    83,    84,     0,   150,    85,
      86,     0,     0,     0,     0,    87,     0,     0,    88,    89,
      90,    91,    92,    93,    94,     0,     0,     0,     0,     0,
     151,    81,    82,    83,    84,     0,     0,    85,    86,     0,
       0,     0,   214,    87,     0,     0,    88,    89,    90,    91,
      92,    93,    94,    81,    82,    83,    84,     0,     0,    85,
      86,     0,     0,   190,     0,    87,     0,     0,    88,    89,
      90,    91,    92,    93,    94,     0,     0,    81,    82,    83,
      84,     0,   156,    85,    86,     0,     0,     0,     0,    87,
     152,     0,    88,    89,    90,    91,    92,    93,    94,     0,
     158,   159,   160,    81,    82,    83,    84,     0,     0,    85,
      86,    95,     0,     0,     0,    87,     0,     0,    88,    89,
      90,    91,    92,    93,    94,     0,    81,    82,    83,    84,
       0,     0,    85,    86,     0,     0,     0,   163,    87,     0,
       0,    88,    89,    90,    91,    92,    93,    94,    83,    84,
       0,     0,    85,    86,     0,     0,     0,     0,    87,     0,
       0,    88,    89,    90,    91,    92,    93,    94,    -1,    -1,
       0,     0,     0,     0,    87,     0,     0,    88,    89,    90,
      91,    92,    93,    94,    87,     0,     0,    88,    89,    90,
      91,    92,    93,    94
};

static const yytype_int16 yycheck[] =
{
      27,    35,   143,    11,    48,    49,    41,    35,     8,    44,
     164,    74,    30,    31,    32,    33,    11,   204,    45,    46,
      47,    84,    22,   177,    59,    60,    61,     4,    55,   216,
     217,   218,    59,    62,    61,    64,    35,    85,    65,    87,
      48,    49,    86,   184,    88,    43,     4,    74,    75,    76,
      77,    78,    79,    87,    81,    82,    83,    84,    85,    87,
      87,    88,    89,    90,    91,    92,    93,    94,     4,    76,
      97,     4,     5,     6,     7,     8,     9,    10,    86,    51,
      88,    18,    19,    16,    56,    77,    83,    24,    21,    22,
      27,    28,    29,    30,    31,    32,    33,   124,    15,    38,
      39,    18,    19,    81,    37,     4,    45,    24,    47,    42,
      27,    28,    29,    30,    31,    32,    33,    50,    43,    52,
      53,    54,    55,     4,    51,   152,     4,     5,     6,     7,
       8,     9,    10,    66,     4,    68,    69,    82,    16,    83,
       3,    67,    75,    21,    22,    78,    79,    80,     4,     4,
       3,    84,    30,     4,     5,     6,     7,     8,     9,    10,
      45,    84,     4,   190,   191,    16,    40,     4,    51,    84,
      21,    22,    50,   200,    52,    53,    54,    55,   205,   206,
     207,    84,    84,    84,     0,     3,    84,    84,     4,    57,
      70,    69,    40,     4,     6,   222,     4,     4,    84,    50,
      37,    52,    53,    54,    55,    42,    84,     4,     5,     6,
       7,     8,     9,    10,     6,    58,    85,     4,    69,    16,
       3,    37,    89,     8,    21,    22,    42,     3,    56,    66,
       4,    68,    84,    84,     4,    85,    90,    40,    75,    85,
      41,    78,    79,    80,    28,    29,    30,    31,    32,    33,
      66,    41,    68,    50,     4,    52,    53,    54,    55,    75,
      41,    85,    78,    79,    80,    27,    28,    29,    30,    31,
      32,    33,    69,    12,    13,    14,    15,    44,     4,    18,
      19,    63,    40,    46,     4,    24,    57,    84,    27,    28,
      29,    30,    31,    32,    33,    12,    13,    14,    15,     4,
       4,    18,    19,     4,    40,     4,    86,    24,    57,    84,
      27,    28,    29,    30,    31,    32,    33,    12,    13,    14,
      15,     4,    57,    18,    19,    57,     4,     4,    65,    24,
       6,    85,    27,    28,    29,    30,    31,    32,    33,    12,
      13,    14,    15,    84,    40,    18,    19,    85,    87,    35,
       4,    24,    74,     6,    27,    28,    29,    30,    31,    32,
      33,     4,    85,    12,    13,    14,    15,     4,    85,    18,
      19,    87,    84,     4,     4,    24,    67,    85,    27,    28,
      29,    30,    31,    32,    33,    12,    13,    14,    15,     4,
      85,    18,    19,    86,     4,    12,     4,    24,    86,    10,
      27,    28,    29,    30,    31,    32,    33,    12,    13,    14,
      15,     4,    85,    18,    19,   124,   162,    61,    -1,    24,
      -1,    -1,    27,    28,    29,    30,    31,    32,    33,    -1,
      -1,    12,    13,    14,    15,    -1,    85,    18,    19,    -1,
      -1,    -1,    -1,    24,    -1,    -1,    27,    28,    29,    30,
      31,    32,    33,    12,    13,    14,    15,    -1,    85,    18,
      19,    -1,    -1,    -1,    -1,    24,    -1,    -1,    27,    28,
      29,    30,    31,    32,    33,    -1,    -1,    -1,    -1,    -1,
      85,    12,    13,    14,    15,    -1,    -1,    18,    19,    -1,
      -1,    -1,    73,    24,    -1,    -1,    27,    28,    29,    30,
      31,    32,    33,    12,    13,    14,    15,    -1,    -1,    18,
      19,    -1,    -1,    72,    -1,    24,    -1,    -1,    27,    28,
      29,    30,    31,    32,    33,    -1,    -1,    12,    13,    14,
      15,    -1,    41,    18,    19,    -1,    -1,    -1,    -1,    24,
      71,    -1,    27,    28,    29,    30,    31,    32,    33,    -1,
      59,    60,    61,    12,    13,    14,    15,    -1,    -1,    18,
      19,    46,    -1,    -1,    -1,    24,    -1,    -1,    27,    28,
      29,    30,    31,    32,    33,    -1,    12,    13,    14,    15,
      -1,    -1,    18,    19,    -1,    -1,    -1,    46,    24,    -1,
      -1,    27,    28,    29,    30,    31,    32,    33,    14,    15,
      -1,    -1,    18,    19,    -1,    -1,    -1,    -1,    24,    -1,
      -1,    27,    28,    29,    30,    31,    32,    33,    18,    19,
      -1,    -1,    -1,    -1,    24,    -1,    -1,    27,    28,    29,
      30,    31,    32,    33,    24,    -1,    -1,    27,    28,    29,
      30,    31,    32,    33
};

/* YYSTOS[STATE-NUM] -- The (internal number of the) accessing
   symbol of state STATE-NUM.  */
static const yytype_uint8 yystos[] =
{
       0,     4,    37,    42,    66,    68,    75,    78,    79,    80,
      92,    93,    94,    11,    35,     4,    43,     4,    76,     4,
      77,    81,     0,    93,    83,    38,    39,    45,    47,     4,
      43,     4,    51,     4,    82,    83,     3,     4,     4,     5,
       6,     7,     8,     9,    10,    16,    21,    22,    30,    50,
      52,    53,    54,    55,    69,    84,    95,    97,     4,    67,
       3,    45,    84,     4,    51,    40,   101,    48,    49,    86,
      88,    95,    95,    95,    84,    84,    84,    84,    84,    70,
      95,    12,    13,    14,    15,    18,    19,    24,    27,    28,
      29,    30,    31,    32,    33,    46,    35,    87,    40,    95,
      51,    56,   103,    97,     3,    57,    84,    95,     4,     6,
      95,    95,    95,    95,    95,    95,    85,    95,    95,    95,
      95,    95,     8,    22,    84,    95,    95,    95,    95,    95,
      95,    95,    95,     4,     4,    95,    95,    99,   100,    84,
       6,    58,    35,    85,     4,     3,    89,    85,    85,    85,
      85,    85,    71,     8,     4,    94,    41,    44,    59,    60,
      61,    96,   102,    46,    87,     3,    62,    64,   104,     4,
     103,    84,    85,    90,    95,    85,     4,    40,    41,    41,
      41,    96,     4,    99,    85,    63,    40,     4,    46,     4,
      72,    57,    99,     4,     4,     4,   103,    40,     4,    86,
      84,    74,    84,    95,    95,    57,    57,    57,     4,     4,
      95,    98,     4,     6,    73,   102,    95,    95,    95,    65,
      85,    85,    87,    84,    85,   102,   102,   102,    40,    35,
      95,     4,    74,     6,     4,    85,     4,    87,    84,     4,
       4,    67,    85,     4,    86,     4,    12,     4,    86,     4
};

#define yyerrok		(yyerrstatus = 0)
#define yyclearin	(yychar = YYEMPTY)
#define YYEMPTY		(-2)
#define YYEOF		0

#define YYACCEPT	goto yyacceptlab
#define YYABORT		goto yyabortlab
#define YYERROR		goto yyerrorlab


/* Like YYERROR except do call yyerror.  This remains here temporarily
   to ease the transition to the new meaning of YYERROR, for GCC.
   Once GCC version 2 has supplanted version 1, this can go.  */

#define YYFAIL		goto yyerrlab

#define YYRECOVERING()  (!!yyerrstatus)

#define YYBACKUP(Token, Value)					\
do								\
  if (yychar == YYEMPTY && yylen == 1)				\
    {								\
      yychar = (Token);						\
      yylval = (Value);						\
      yytoken = YYTRANSLATE (yychar);				\
      YYPOPSTACK (1);						\
      goto yybackup;						\
    }								\
  else								\
    {								\
      yyerror (YY_("syntax error: cannot back up")); \
      YYERROR;							\
    }								\
while (YYID (0))


#define YYTERROR	1
#define YYERRCODE	256


/* YYLLOC_DEFAULT -- Set CURRENT to span from RHS[1] to RHS[N].
   If N is 0, then set CURRENT to the empty location which ends
   the previous symbol: RHS[0] (always defined).  */

#define YYRHSLOC(Rhs, K) ((Rhs)[K])
#ifndef YYLLOC_DEFAULT
# define YYLLOC_DEFAULT(Current, Rhs, N)				\
    do									\
      if (YYID (N))                                                    \
	{								\
	  (Current).first_line   = YYRHSLOC (Rhs, 1).first_line;	\
	  (Current).first_column = YYRHSLOC (Rhs, 1).first_column;	\
	  (Current).last_line    = YYRHSLOC (Rhs, N).last_line;		\
	  (Current).last_column  = YYRHSLOC (Rhs, N).last_column;	\
	}								\
      else								\
	{								\
	  (Current).first_line   = (Current).last_line   =		\
	    YYRHSLOC (Rhs, 0).last_line;				\
	  (Current).first_column = (Current).last_column =		\
	    YYRHSLOC (Rhs, 0).last_column;				\
	}								\
    while (YYID (0))
#endif


/* YY_LOCATION_PRINT -- Print the location on the stream.
   This macro was not mandated originally: define only if we know
   we won't break user code: when these are the locations we know.  */

#ifndef YY_LOCATION_PRINT
# if YYLTYPE_IS_TRIVIAL
#  define YY_LOCATION_PRINT(File, Loc)			\
     fprintf (File, "%d.%d-%d.%d",			\
	      (Loc).first_line, (Loc).first_column,	\
	      (Loc).last_line,  (Loc).last_column)
# else
#  define YY_LOCATION_PRINT(File, Loc) ((void) 0)
# endif
#endif


/* YYLEX -- calling `yylex' with the right arguments.  */

#ifdef YYLEX_PARAM
# define YYLEX yylex (YYLEX_PARAM)
#else
# define YYLEX yylex ()
#endif

/* Enable debugging if requested.  */
#if YYDEBUG

# ifndef YYFPRINTF
#  include <stdio.h> /* INFRINGES ON USER NAME SPACE */
#  define YYFPRINTF fprintf
# endif

# define YYDPRINTF(Args)			\
do {						\
  if (yydebug)					\
    YYFPRINTF Args;				\
} while (YYID (0))

# define YY_SYMBOL_PRINT(Title, Type, Value, Location)			  \
do {									  \
  if (yydebug)								  \
    {									  \
      YYFPRINTF (stderr, "%s ", Title);					  \
      yy_symbol_print (stderr,						  \
		  Type, Value); \
      YYFPRINTF (stderr, "\n");						  \
    }									  \
} while (YYID (0))


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_value_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_value_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (!yyvaluep)
    return;
# ifdef YYPRINT
  if (yytype < YYNTOKENS)
    YYPRINT (yyoutput, yytoknum[yytype], *yyvaluep);
# else
  YYUSE (yyoutput);
# endif
  switch (yytype)
    {
      default:
	break;
    }
}


/*--------------------------------.
| Print this symbol on YYOUTPUT.  |
`--------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_symbol_print (FILE *yyoutput, int yytype, YYSTYPE const * const yyvaluep)
#else
static void
yy_symbol_print (yyoutput, yytype, yyvaluep)
    FILE *yyoutput;
    int yytype;
    YYSTYPE const * const yyvaluep;
#endif
{
  if (yytype < YYNTOKENS)
    YYFPRINTF (yyoutput, "token %s (", yytname[yytype]);
  else
    YYFPRINTF (yyoutput, "nterm %s (", yytname[yytype]);

  yy_symbol_value_print (yyoutput, yytype, yyvaluep);
  YYFPRINTF (yyoutput, ")");
}

/*------------------------------------------------------------------.
| yy_stack_print -- Print the state stack from its BOTTOM up to its |
| TOP (included).                                                   |
`------------------------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_stack_print (yytype_int16 *yybottom, yytype_int16 *yytop)
#else
static void
yy_stack_print (yybottom, yytop)
    yytype_int16 *yybottom;
    yytype_int16 *yytop;
#endif
{
  YYFPRINTF (stderr, "Stack now");
  for (; yybottom <= yytop; yybottom++)
    {
      int yybot = *yybottom;
      YYFPRINTF (stderr, " %d", yybot);
    }
  YYFPRINTF (stderr, "\n");
}

# define YY_STACK_PRINT(Bottom, Top)				\
do {								\
  if (yydebug)							\
    yy_stack_print ((Bottom), (Top));				\
} while (YYID (0))


/*------------------------------------------------.
| Report that the YYRULE is going to be reduced.  |
`------------------------------------------------*/

#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yy_reduce_print (YYSTYPE *yyvsp, int yyrule)
#else
static void
yy_reduce_print (yyvsp, yyrule)
    YYSTYPE *yyvsp;
    int yyrule;
#endif
{
  int yynrhs = yyr2[yyrule];
  int yyi;
  unsigned long int yylno = yyrline[yyrule];
  YYFPRINTF (stderr, "Reducing stack by rule %d (line %lu):\n",
	     yyrule - 1, yylno);
  /* The symbols being reduced.  */
  for (yyi = 0; yyi < yynrhs; yyi++)
    {
      YYFPRINTF (stderr, "   $%d = ", yyi + 1);
      yy_symbol_print (stderr, yyrhs[yyprhs[yyrule] + yyi],
		       &(yyvsp[(yyi + 1) - (yynrhs)])
		       		       );
      YYFPRINTF (stderr, "\n");
    }
}

# define YY_REDUCE_PRINT(Rule)		\
do {					\
  if (yydebug)				\
    yy_reduce_print (yyvsp, Rule); \
} while (YYID (0))

/* Nonzero means print parse trace.  It is left uninitialized so that
   multiple parsers can coexist.  */
int yydebug;
#else /* !YYDEBUG */
# define YYDPRINTF(Args)
# define YY_SYMBOL_PRINT(Title, Type, Value, Location)
# define YY_STACK_PRINT(Bottom, Top)
# define YY_REDUCE_PRINT(Rule)
#endif /* !YYDEBUG */


/* YYINITDEPTH -- initial size of the parser's stacks.  */
#ifndef	YYINITDEPTH
# define YYINITDEPTH 200
#endif

/* YYMAXDEPTH -- maximum size the stacks can grow to (effective only
   if the built-in stack extension method is used).

   Do not make this value too large; the results are undefined if
   YYSTACK_ALLOC_MAXIMUM < YYSTACK_BYTES (YYMAXDEPTH)
   evaluated with infinite-precision integer arithmetic.  */

#ifndef YYMAXDEPTH
# define YYMAXDEPTH 10000
#endif



#if YYERROR_VERBOSE

# ifndef yystrlen
#  if defined __GLIBC__ && defined _STRING_H
#   define yystrlen strlen
#  else
/* Return the length of YYSTR.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static YYSIZE_T
yystrlen (const char *yystr)
#else
static YYSIZE_T
yystrlen (yystr)
    const char *yystr;
#endif
{
  YYSIZE_T yylen;
  for (yylen = 0; yystr[yylen]; yylen++)
    continue;
  return yylen;
}
#  endif
# endif

# ifndef yystpcpy
#  if defined __GLIBC__ && defined _STRING_H && defined _GNU_SOURCE
#   define yystpcpy stpcpy
#  else
/* Copy YYSRC to YYDEST, returning the address of the terminating '\0' in
   YYDEST.  */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static char *
yystpcpy (char *yydest, const char *yysrc)
#else
static char *
yystpcpy (yydest, yysrc)
    char *yydest;
    const char *yysrc;
#endif
{
  char *yyd = yydest;
  const char *yys = yysrc;

  while ((*yyd++ = *yys++) != '\0')
    continue;

  return yyd - 1;
}
#  endif
# endif

# ifndef yytnamerr
/* Copy to YYRES the contents of YYSTR after stripping away unnecessary
   quotes and backslashes, so that it's suitable for yyerror.  The
   heuristic is that double-quoting is unnecessary unless the string
   contains an apostrophe, a comma, or backslash (other than
   backslash-backslash).  YYSTR is taken from yytname.  If YYRES is
   null, do not copy; instead, return the length of what the result
   would have been.  */
static YYSIZE_T
yytnamerr (char *yyres, const char *yystr)
{
  if (*yystr == '"')
    {
      YYSIZE_T yyn = 0;
      char const *yyp = yystr;

      for (;;)
	switch (*++yyp)
	  {
	  case '\'':
	  case ',':
	    goto do_not_strip_quotes;

	  case '\\':
	    if (*++yyp != '\\')
	      goto do_not_strip_quotes;
	    /* Fall through.  */
	  default:
	    if (yyres)
	      yyres[yyn] = *yyp;
	    yyn++;
	    break;

	  case '"':
	    if (yyres)
	      yyres[yyn] = '\0';
	    return yyn;
	  }
    do_not_strip_quotes: ;
    }

  if (! yyres)
    return yystrlen (yystr);

  return yystpcpy (yyres, yystr) - yyres;
}
# endif

/* Copy into YYRESULT an error message about the unexpected token
   YYCHAR while in state YYSTATE.  Return the number of bytes copied,
   including the terminating null byte.  If YYRESULT is null, do not
   copy anything; just return the number of bytes that would be
   copied.  As a special case, return 0 if an ordinary "syntax error"
   message will do.  Return YYSIZE_MAXIMUM if overflow occurs during
   size calculation.  */
static YYSIZE_T
yysyntax_error (char *yyresult, int yystate, int yychar)
{
  int yyn = yypact[yystate];

  if (! (YYPACT_NINF < yyn && yyn <= YYLAST))
    return 0;
  else
    {
      int yytype = YYTRANSLATE (yychar);
      YYSIZE_T yysize0 = yytnamerr (0, yytname[yytype]);
      YYSIZE_T yysize = yysize0;
      YYSIZE_T yysize1;
      int yysize_overflow = 0;
      enum { YYERROR_VERBOSE_ARGS_MAXIMUM = 5 };
      char const *yyarg[YYERROR_VERBOSE_ARGS_MAXIMUM];
      int yyx;

# if 0
      /* This is so xgettext sees the translatable formats that are
	 constructed on the fly.  */
      YY_("syntax error, unexpected %s");
      YY_("syntax error, unexpected %s, expecting %s");
      YY_("syntax error, unexpected %s, expecting %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s");
      YY_("syntax error, unexpected %s, expecting %s or %s or %s or %s");
# endif
      char *yyfmt;
      char const *yyf;
      static char const yyunexpected[] = "syntax error, unexpected %s";
      static char const yyexpecting[] = ", expecting %s";
      static char const yyor[] = " or %s";
      char yyformat[sizeof yyunexpected
		    + sizeof yyexpecting - 1
		    + ((YYERROR_VERBOSE_ARGS_MAXIMUM - 2)
		       * (sizeof yyor - 1))];
      char const *yyprefix = yyexpecting;

      /* Start YYX at -YYN if negative to avoid negative indexes in
	 YYCHECK.  */
      int yyxbegin = yyn < 0 ? -yyn : 0;

      /* Stay within bounds of both yycheck and yytname.  */
      int yychecklim = YYLAST - yyn + 1;
      int yyxend = yychecklim < YYNTOKENS ? yychecklim : YYNTOKENS;
      int yycount = 1;

      yyarg[0] = yytname[yytype];
      yyfmt = yystpcpy (yyformat, yyunexpected);

      for (yyx = yyxbegin; yyx < yyxend; ++yyx)
	if (yycheck[yyx + yyn] == yyx && yyx != YYTERROR)
	  {
	    if (yycount == YYERROR_VERBOSE_ARGS_MAXIMUM)
	      {
		yycount = 1;
		yysize = yysize0;
		yyformat[sizeof yyunexpected - 1] = '\0';
		break;
	      }
	    yyarg[yycount++] = yytname[yyx];
	    yysize1 = yysize + yytnamerr (0, yytname[yyx]);
	    yysize_overflow |= (yysize1 < yysize);
	    yysize = yysize1;
	    yyfmt = yystpcpy (yyfmt, yyprefix);
	    yyprefix = yyor;
	  }

      yyf = YY_(yyformat);
      yysize1 = yysize + yystrlen (yyf);
      yysize_overflow |= (yysize1 < yysize);
      yysize = yysize1;

      if (yysize_overflow)
	return YYSIZE_MAXIMUM;

      if (yyresult)
	{
	  /* Avoid sprintf, as that infringes on the user's name space.
	     Don't have undefined behavior even if the translation
	     produced a string with the wrong number of "%s"s.  */
	  char *yyp = yyresult;
	  int yyi = 0;
	  while ((*yyp = *yyf) != '\0')
	    {
	      if (*yyp == '%' && yyf[1] == 's' && yyi < yycount)
		{
		  yyp += yytnamerr (yyp, yyarg[yyi++]);
		  yyf += 2;
		}
	      else
		{
		  yyp++;
		  yyf++;
		}
	    }
	}
      return yysize;
    }
}
#endif /* YYERROR_VERBOSE */


/*-----------------------------------------------.
| Release the memory associated to this symbol.  |
`-----------------------------------------------*/

/*ARGSUSED*/
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
static void
yydestruct (const char *yymsg, int yytype, YYSTYPE *yyvaluep)
#else
static void
yydestruct (yymsg, yytype, yyvaluep)
    const char *yymsg;
    int yytype;
    YYSTYPE *yyvaluep;
#endif
{
  YYUSE (yyvaluep);

  if (!yymsg)
    yymsg = "Deleting";
  YY_SYMBOL_PRINT (yymsg, yytype, yyvaluep, yylocationp);

  switch (yytype)
    {

      default:
	break;
    }
}

/* Prevent warnings from -Wmissing-prototypes.  */
#ifdef YYPARSE_PARAM
#if defined __STDC__ || defined __cplusplus
int yyparse (void *YYPARSE_PARAM);
#else
int yyparse ();
#endif
#else /* ! YYPARSE_PARAM */
#if defined __STDC__ || defined __cplusplus
int yyparse (void);
#else
int yyparse ();
#endif
#endif /* ! YYPARSE_PARAM */


/* The lookahead symbol.  */
int yychar;

/* The semantic value of the lookahead symbol.  */
YYSTYPE yylval;

/* Number of syntax errors so far.  */
int yynerrs;



/*-------------------------.
| yyparse or yypush_parse.  |
`-------------------------*/

#ifdef YYPARSE_PARAM
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void *YYPARSE_PARAM)
#else
int
yyparse (YYPARSE_PARAM)
    void *YYPARSE_PARAM;
#endif
#else /* ! YYPARSE_PARAM */
#if (defined __STDC__ || defined __C99__FUNC__ \
     || defined __cplusplus || defined _MSC_VER)
int
yyparse (void)
#else
int
yyparse ()

#endif
#endif
{


    int yystate;
    /* Number of tokens to shift before error messages enabled.  */
    int yyerrstatus;

    /* The stacks and their tools:
       `yyss': related to states.
       `yyvs': related to semantic values.

       Refer to the stacks thru separate pointers, to allow yyoverflow
       to reallocate them elsewhere.  */

    /* The state stack.  */
    yytype_int16 yyssa[YYINITDEPTH];
    yytype_int16 *yyss;
    yytype_int16 *yyssp;

    /* The semantic value stack.  */
    YYSTYPE yyvsa[YYINITDEPTH];
    YYSTYPE *yyvs;
    YYSTYPE *yyvsp;

    YYSIZE_T yystacksize;

  int yyn;
  int yyresult;
  /* Lookahead token as an internal (translated) token number.  */
  int yytoken;
  /* The variables used to return semantic value and location from the
     action routines.  */
  YYSTYPE yyval;

#if YYERROR_VERBOSE
  /* Buffer for error messages, and its allocated size.  */
  char yymsgbuf[128];
  char *yymsg = yymsgbuf;
  YYSIZE_T yymsg_alloc = sizeof yymsgbuf;
#endif

#define YYPOPSTACK(N)   (yyvsp -= (N), yyssp -= (N))

  /* The number of symbols on the RHS of the reduced rule.
     Keep to zero when no symbol should be popped.  */
  int yylen = 0;

  yytoken = 0;
  yyss = yyssa;
  yyvs = yyvsa;
  yystacksize = YYINITDEPTH;

  YYDPRINTF ((stderr, "Starting parse\n"));

  yystate = 0;
  yyerrstatus = 0;
  yynerrs = 0;
  yychar = YYEMPTY; /* Cause a token to be read.  */

  /* Initialize stack pointers.
     Waste one element of value and location stack
     so that they stay on the same level as the state stack.
     The wasted elements are never initialized.  */
  yyssp = yyss;
  yyvsp = yyvs;

  goto yysetstate;

/*------------------------------------------------------------.
| yynewstate -- Push a new state, which is found in yystate.  |
`------------------------------------------------------------*/
 yynewstate:
  /* In all cases, when you get here, the value and location stacks
     have just been pushed.  So pushing a state here evens the stacks.  */
  yyssp++;

 yysetstate:
  *yyssp = yystate;

  if (yyss + yystacksize - 1 <= yyssp)
    {
      /* Get the current used size of the three stacks, in elements.  */
      YYSIZE_T yysize = yyssp - yyss + 1;

#ifdef yyoverflow
      {
	/* Give user a chance to reallocate the stack.  Use copies of
	   these so that the &'s don't force the real ones into
	   memory.  */
	YYSTYPE *yyvs1 = yyvs;
	yytype_int16 *yyss1 = yyss;

	/* Each stack pointer address is followed by the size of the
	   data in use in that stack, in bytes.  This used to be a
	   conditional around just the two extra args, but that might
	   be undefined if yyoverflow is a macro.  */
	yyoverflow (YY_("memory exhausted"),
		    &yyss1, yysize * sizeof (*yyssp),
		    &yyvs1, yysize * sizeof (*yyvsp),
		    &yystacksize);

	yyss = yyss1;
	yyvs = yyvs1;
      }
#else /* no yyoverflow */
# ifndef YYSTACK_RELOCATE
      goto yyexhaustedlab;
# else
      /* Extend the stack our own way.  */
      if (YYMAXDEPTH <= yystacksize)
	goto yyexhaustedlab;
      yystacksize *= 2;
      if (YYMAXDEPTH < yystacksize)
	yystacksize = YYMAXDEPTH;

      {
	yytype_int16 *yyss1 = yyss;
	union yyalloc *yyptr =
	  (union yyalloc *) YYSTACK_ALLOC (YYSTACK_BYTES (yystacksize));
	if (! yyptr)
	  goto yyexhaustedlab;
	YYSTACK_RELOCATE (yyss_alloc, yyss);
	YYSTACK_RELOCATE (yyvs_alloc, yyvs);
#  undef YYSTACK_RELOCATE
	if (yyss1 != yyssa)
	  YYSTACK_FREE (yyss1);
      }
# endif
#endif /* no yyoverflow */

      yyssp = yyss + yysize - 1;
      yyvsp = yyvs + yysize - 1;

      YYDPRINTF ((stderr, "Stack size increased to %lu\n",
		  (unsigned long int) yystacksize));

      if (yyss + yystacksize - 1 <= yyssp)
	YYABORT;
    }

  YYDPRINTF ((stderr, "Entering state %d\n", yystate));

  if (yystate == YYFINAL)
    YYACCEPT;

  goto yybackup;

/*-----------.
| yybackup.  |
`-----------*/
yybackup:

  /* Do appropriate processing given the current state.  Read a
     lookahead token if we need one and don't already have one.  */

  /* First try to decide what to do without reference to lookahead token.  */
  yyn = yypact[yystate];
  if (yyn == YYPACT_NINF)
    goto yydefault;

  /* Not known => get a lookahead token if don't already have one.  */

  /* YYCHAR is either YYEMPTY or YYEOF or a valid lookahead symbol.  */
  if (yychar == YYEMPTY)
    {
      YYDPRINTF ((stderr, "Reading a token: "));
      yychar = YYLEX;
    }

  if (yychar <= YYEOF)
    {
      yychar = yytoken = YYEOF;
      YYDPRINTF ((stderr, "Now at end of input.\n"));
    }
  else
    {
      yytoken = YYTRANSLATE (yychar);
      YY_SYMBOL_PRINT ("Next token is", yytoken, &yylval, &yylloc);
    }

  /* If the proper action on seeing token YYTOKEN is to reduce or to
     detect an error, take that action.  */
  yyn += yytoken;
  if (yyn < 0 || YYLAST < yyn || yycheck[yyn] != yytoken)
    goto yydefault;
  yyn = yytable[yyn];
  if (yyn <= 0)
    {
      if (yyn == 0 || yyn == YYTABLE_NINF)
	goto yyerrlab;
      yyn = -yyn;
      goto yyreduce;
    }

  /* Count tokens shifted since error; after three, turn off error
     status.  */
  if (yyerrstatus)
    yyerrstatus--;

  /* Shift the lookahead token.  */
  YY_SYMBOL_PRINT ("Shifting", yytoken, &yylval, &yylloc);

  /* Discard the shifted token.  */
  yychar = YYEMPTY;

  yystate = yyn;
  *++yyvsp = yylval;

  goto yynewstate;


/*-----------------------------------------------------------.
| yydefault -- do the default action for the current state.  |
`-----------------------------------------------------------*/
yydefault:
  yyn = yydefact[yystate];
  if (yyn == 0)
    goto yyerrlab;
  goto yyreduce;


/*-----------------------------.
| yyreduce -- Do a reduction.  |
`-----------------------------*/
yyreduce:
  /* yyn is the number of a rule to reduce with.  */
  yylen = yyr2[yyn];

  /* If YYLEN is nonzero, implement the default value of the action:
     `$$ = $1'.

     Otherwise, the following line sets YYVAL to garbage.
     This behavior is undocumented and Bison
     users should not rely upon it.  Assigning to YYVAL
     unconditionally makes the parser a bit smaller, and it avoids a
     GCC warning that YYVAL may be used uninitialized.  */
  yyval = yyvsp[1-yylen];


  YY_REDUCE_PRINT (yyn);
  switch (yyn)
    {
        case 4:

/* Line 1455 of yacc.c  */
#line 125 "bison.y"
    { emit("STMT"); ;}
    break;

  case 5:

/* Line 1455 of yacc.c  */
#line 129 "bison.y"
    { emit_select((yyvsp[(1) - (7)].strval), (yyvsp[(6) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 6:

/* Line 1455 of yacc.c  */
#line 131 "bison.y"
    {  emit_load((yyvsp[(1) - (12)].strval), (yyvsp[(4) - (12)].strval), (yyvsp[(11) - (12)].intval), (yyvsp[(7) - (12)].strval)); ;}
    break;

  case 7:

/* Line 1455 of yacc.c  */
#line 133 "bison.y"
    {  emit_filter((yyvsp[(1) - (5)].strval), (yyvsp[(4) - (5)].strval));;}
    break;

  case 8:

/* Line 1455 of yacc.c  */
#line 135 "bison.y"
    {  emit_order((yyvsp[(1) - (6)].strval), (yyvsp[(4) - (6)].strval), (yyvsp[(6) - (6)].intval));;}
    break;

  case 9:

/* Line 1455 of yacc.c  */
#line 137 "bison.y"
    {  emit_join((yyvsp[(1) - (8)].strval),(yyvsp[(6) - (8)].strval),(yyvsp[(7) - (8)].intval),0,-1); ;}
    break;

  case 10:

/* Line 1455 of yacc.c  */
#line 139 "bison.y"
    {  emit_store((yyvsp[(2) - (9)].strval),(yyvsp[(4) - (9)].strval),(yyvsp[(7) - (9)].strval)); ;}
    break;

  case 11:

/* Line 1455 of yacc.c  */
#line 141 "bison.y"
    {  emit_store_binary((yyvsp[(2) - (7)].strval),(yyvsp[(4) - (7)].strval)); ;}
    break;

  case 12:

/* Line 1455 of yacc.c  */
#line 143 "bison.y"
    {  emit_describe_table((yyvsp[(2) - (2)].strval));;}
    break;

  case 13:

/* Line 1455 of yacc.c  */
#line 145 "bison.y"
    {  emit_insert((yyvsp[(3) - (7)].strval), (yyvsp[(7) - (7)].strval));;}
    break;

  case 14:

/* Line 1455 of yacc.c  */
#line 147 "bison.y"
    {  emit_delete((yyvsp[(3) - (5)].strval));;}
    break;

  case 15:

/* Line 1455 of yacc.c  */
#line 149 "bison.y"
    {  emit_display((yyvsp[(2) - (7)].strval), (yyvsp[(5) - (7)].strval));;}
    break;

  case 16:

/* Line 1455 of yacc.c  */
#line 151 "bison.y"
    {  emit_show_tables();;}
    break;

  case 17:

/* Line 1455 of yacc.c  */
#line 153 "bison.y"
    {  emit_drop_table((yyvsp[(3) - (3)].strval));;}
    break;

  case 18:

/* Line 1455 of yacc.c  */
#line 155 "bison.y"
    {  emit_create_bitmap_index((yyvsp[(4) - (23)].strval), (yyvsp[(6) - (23)].strval), (yyvsp[(8) - (23)].strval), (yyvsp[(10) - (23)].strval), (yyvsp[(19) - (23)].strval), (yyvsp[(23) - (23)].strval));;}
    break;

  case 19:

/* Line 1455 of yacc.c  */
#line 159 "bison.y"
    { emit_name((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 20:

/* Line 1455 of yacc.c  */
#line 160 "bison.y"
    { emit_fieldname((yyvsp[(1) - (3)].strval), (yyvsp[(3) - (3)].strval)); ;}
    break;

  case 21:

/* Line 1455 of yacc.c  */
#line 161 "bison.y"
    { emit("USERVAR %s", (yyvsp[(1) - (1)].strval)); ;}
    break;

  case 22:

/* Line 1455 of yacc.c  */
#line 162 "bison.y"
    { emit_string((yyvsp[(1) - (1)].strval)); ;}
    break;

  case 23:

/* Line 1455 of yacc.c  */
#line 163 "bison.y"
    { emit_number((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 24:

/* Line 1455 of yacc.c  */
#line 164 "bison.y"
    { emit_float((yyvsp[(1) - (1)].floatval)); ;}
    break;

  case 25:

/* Line 1455 of yacc.c  */
#line 165 "bison.y"
    { emit_decimal((yyvsp[(1) - (1)].intval)); ;}
    break;

  case 26:

/* Line 1455 of yacc.c  */
#line 166 "bison.y"
    { emit("BOOL %d", (yyvsp[(1) - (1)].intval)); ;}
    break;

  case 27:

/* Line 1455 of yacc.c  */
#line 167 "bison.y"
    { emit_varchar((yyvsp[(1) - (14)].strval), (yyvsp[(3) - (14)].intval), (yyvsp[(6) - (14)].strval), (yyvsp[(8) - (14)].intval), (yyvsp[(11) - (14)].strval), (yyvsp[(13) - (14)].strval));;}
    break;

  case 28:

/* Line 1455 of yacc.c  */
#line 168 "bison.y"
    { emit_varchar((yyvsp[(1) - (9)].strval), (yyvsp[(3) - (9)].intval), (yyvsp[(6) - (9)].strval), (yyvsp[(8) - (9)].intval), "", "");;}
    break;

  case 29:

/* Line 1455 of yacc.c  */
#line 169 "bison.y"
    { emit_var((yyvsp[(1) - (11)].strval), (yyvsp[(3) - (11)].intval), (yyvsp[(6) - (11)].strval), (yyvsp[(8) - (11)].strval), (yyvsp[(10) - (11)].strval));;}
    break;

  case 30:

/* Line 1455 of yacc.c  */
#line 170 "bison.y"
    { emit_var((yyvsp[(1) - (6)].strval), (yyvsp[(3) - (6)].intval), (yyvsp[(6) - (6)].strval), "", "");;}
    break;

  case 31:

/* Line 1455 of yacc.c  */
#line 171 "bison.y"
    { emit_var_asc((yyvsp[(1) - (2)].strval));;}
    break;

  case 32:

/* Line 1455 of yacc.c  */
#line 172 "bison.y"
    { emit_var_desc((yyvsp[(1) - (2)].strval));;}
    break;

  case 33:

/* Line 1455 of yacc.c  */
#line 173 "bison.y"
    { emit_count(); ;}
    break;

  case 34:

/* Line 1455 of yacc.c  */
#line 174 "bison.y"
    { emit_sum(); ;}
    break;

  case 35:

/* Line 1455 of yacc.c  */
#line 175 "bison.y"
    { emit_average(); ;}
    break;

  case 36:

/* Line 1455 of yacc.c  */
#line 176 "bison.y"
    { emit_min(); ;}
    break;

  case 37:

/* Line 1455 of yacc.c  */
#line 177 "bison.y"
    { emit_max(); ;}
    break;

  case 38:

/* Line 1455 of yacc.c  */
#line 178 "bison.y"
    { emit_distinct(); ;}
    break;

  case 39:

/* Line 1455 of yacc.c  */
#line 182 "bison.y"
    { emit_add(); ;}
    break;

  case 40:

/* Line 1455 of yacc.c  */
#line 183 "bison.y"
    { emit_minus(); ;}
    break;

  case 41:

/* Line 1455 of yacc.c  */
#line 184 "bison.y"
    { emit_mul(); ;}
    break;

  case 42:

/* Line 1455 of yacc.c  */
#line 185 "bison.y"
    { emit_div(); ;}
    break;

  case 43:

/* Line 1455 of yacc.c  */
#line 186 "bison.y"
    { emit("MOD"); ;}
    break;

  case 44:

/* Line 1455 of yacc.c  */
#line 187 "bison.y"
    { emit("MOD"); ;}
    break;

  case 45:

/* Line 1455 of yacc.c  */
#line 189 "bison.y"
    { emit_and(); ;}
    break;

  case 46:

/* Line 1455 of yacc.c  */
#line 190 "bison.y"
    { emit_eq(); ;}
    break;

  case 47:

/* Line 1455 of yacc.c  */
#line 191 "bison.y"
    { emit_or(); ;}
    break;

  case 48:

/* Line 1455 of yacc.c  */
#line 192 "bison.y"
    { emit("XOR"); ;}
    break;

  case 49:

/* Line 1455 of yacc.c  */
#line 193 "bison.y"
    { emit("SHIFT %s", (yyvsp[(2) - (3)].subtok)==1?"left":"right"); ;}
    break;

  case 50:

/* Line 1455 of yacc.c  */
#line 194 "bison.y"
    { emit("NOT"); ;}
    break;

  case 51:

/* Line 1455 of yacc.c  */
#line 195 "bison.y"
    { emit("NOT"); ;}
    break;

  case 52:

/* Line 1455 of yacc.c  */
#line 196 "bison.y"
    { emit_cmp((yyvsp[(2) - (3)].subtok)); ;}
    break;

  case 53:

/* Line 1455 of yacc.c  */
#line 197 "bison.y"
    { emit_cmp(7); ;}
    break;

  case 54:

/* Line 1455 of yacc.c  */
#line 199 "bison.y"
    { emit("CMPSELECT %d", (yyvsp[(2) - (5)].subtok)); ;}
    break;

  case 55:

/* Line 1455 of yacc.c  */
#line 200 "bison.y"
    {emit("EXPR");;}
    break;

  case 56:

/* Line 1455 of yacc.c  */
#line 201 "bison.y"
    { emit_case(); ;}
    break;

  case 57:

/* Line 1455 of yacc.c  */
#line 205 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(3) - (3)].intval)); ;}
    break;

  case 58:

/* Line 1455 of yacc.c  */
#line 206 "bison.y"
    { emit("ISBOOL %d", (yyvsp[(4) - (4)].intval)); emit("NOT"); ;}
    break;

  case 59:

/* Line 1455 of yacc.c  */
#line 209 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 60:

/* Line 1455 of yacc.c  */
#line 212 "bison.y"
    { (yyval.intval) = (yyvsp[(3) - (3)].intval);;}
    break;

  case 61:

/* Line 1455 of yacc.c  */
#line 216 "bison.y"
    { (yyval.intval) = 1; emit_sel_name((yyvsp[(3) - (3)].strval));;}
    break;

  case 62:

/* Line 1455 of yacc.c  */
#line 217 "bison.y"
    { (yyval.intval) = (yyvsp[(1) - (5)].intval) + 1; emit_sel_name((yyvsp[(5) - (5)].strval));;}
    break;

  case 63:

/* Line 1455 of yacc.c  */
#line 218 "bison.y"
    { emit_sel_name("*");;}
    break;

  case 64:

/* Line 1455 of yacc.c  */
#line 222 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 65:

/* Line 1455 of yacc.c  */
#line 223 "bison.y"
    {(yyval.intval) = (yyvsp[(1) - (3)].intval) + 1; ;}
    break;

  case 66:

/* Line 1455 of yacc.c  */
#line 227 "bison.y"
    { (yyval.intval) = 1; ;}
    break;

  case 67:

/* Line 1455 of yacc.c  */
#line 228 "bison.y"
    { (yyval.intval) = 1 + (yyvsp[(3) - (3)].intval); ;}
    break;

  case 68:

/* Line 1455 of yacc.c  */
#line 231 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 70:

/* Line 1455 of yacc.c  */
#line 236 "bison.y"
    { emit("FILTER BY"); ;}
    break;

  case 71:

/* Line 1455 of yacc.c  */
#line 240 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (4)].strval), 'I');;}
    break;

  case 72:

/* Line 1455 of yacc.c  */
#line 241 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'L');;}
    break;

  case 73:

/* Line 1455 of yacc.c  */
#line 242 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'R');;}
    break;

  case 74:

/* Line 1455 of yacc.c  */
#line 243 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (5)].strval), 'O');;}
    break;

  case 75:

/* Line 1455 of yacc.c  */
#line 244 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(2) - (5)].strval), 'I'); ;}
    break;

  case 76:

/* Line 1455 of yacc.c  */
#line 245 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'L'); ;}
    break;

  case 77:

/* Line 1455 of yacc.c  */
#line 246 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'R'); ;}
    break;

  case 78:

/* Line 1455 of yacc.c  */
#line 247 "bison.y"
    { (yyval.intval) = 1; emit_join_tab((yyvsp[(3) - (6)].strval), 'O'); ;}
    break;

  case 79:

/* Line 1455 of yacc.c  */
#line 249 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 80:

/* Line 1455 of yacc.c  */
#line 252 "bison.y"
    { emit_limit((yyvsp[(2) - (2)].intval)); ;}
    break;

  case 81:

/* Line 1455 of yacc.c  */
#line 254 "bison.y"
    { /* nil */
    (yyval.intval) = 0;
;}
    break;

  case 82:

/* Line 1455 of yacc.c  */
#line 257 "bison.y"
    { emit_sort((yyvsp[(4) - (4)].strval), 0); ;}
    break;

  case 83:

/* Line 1455 of yacc.c  */
#line 258 "bison.y"
    { emit_sort((yyvsp[(4) - (7)].strval), (yyvsp[(7) - (7)].intval)); ;}
    break;

  case 84:

/* Line 1455 of yacc.c  */
#line 259 "bison.y"
    { emit_presort((yyvsp[(3) - (3)].strval)); ;}
    break;



/* Line 1455 of yacc.c  */
#line 2262 "bison.cu"
      default: break;
    }
  YY_SYMBOL_PRINT ("-> $$ =", yyr1[yyn], &yyval, &yyloc);

  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);

  *++yyvsp = yyval;

  /* Now `shift' the result of the reduction.  Determine what state
     that goes to, based on the state we popped back to and the rule
     number reduced by.  */

  yyn = yyr1[yyn];

  yystate = yypgoto[yyn - YYNTOKENS] + *yyssp;
  if (0 <= yystate && yystate <= YYLAST && yycheck[yystate] == *yyssp)
    yystate = yytable[yystate];
  else
    yystate = yydefgoto[yyn - YYNTOKENS];

  goto yynewstate;


/*------------------------------------.
| yyerrlab -- here on detecting error |
`------------------------------------*/
yyerrlab:
  /* If not already recovering from an error, report this error.  */
  if (!yyerrstatus)
    {
      ++yynerrs;
#if ! YYERROR_VERBOSE
      yyerror (YY_("syntax error"));
#else
      {
	YYSIZE_T yysize = yysyntax_error (0, yystate, yychar);
	if (yymsg_alloc < yysize && yymsg_alloc < YYSTACK_ALLOC_MAXIMUM)
	  {
	    YYSIZE_T yyalloc = 2 * yysize;
	    if (! (yysize <= yyalloc && yyalloc <= YYSTACK_ALLOC_MAXIMUM))
	      yyalloc = YYSTACK_ALLOC_MAXIMUM;
	    if (yymsg != yymsgbuf)
	      YYSTACK_FREE (yymsg);
	    yymsg = (char *) YYSTACK_ALLOC (yyalloc);
	    if (yymsg)
	      yymsg_alloc = yyalloc;
	    else
	      {
		yymsg = yymsgbuf;
		yymsg_alloc = sizeof yymsgbuf;
	      }
	  }

	if (0 < yysize && yysize <= yymsg_alloc)
	  {
	    (void) yysyntax_error (yymsg, yystate, yychar);
	    yyerror (yymsg);
	  }
	else
	  {
	    yyerror (YY_("syntax error"));
	    if (yysize != 0)
	      goto yyexhaustedlab;
	  }
      }
#endif
    }



  if (yyerrstatus == 3)
    {
      /* If just tried and failed to reuse lookahead token after an
	 error, discard it.  */

      if (yychar <= YYEOF)
	{
	  /* Return failure if at end of input.  */
	  if (yychar == YYEOF)
	    YYABORT;
	}
      else
	{
	  yydestruct ("Error: discarding",
		      yytoken, &yylval);
	  yychar = YYEMPTY;
	}
    }

  /* Else will try to reuse lookahead token after shifting the error
     token.  */
  goto yyerrlab1;


/*---------------------------------------------------.
| yyerrorlab -- error raised explicitly by YYERROR.  |
`---------------------------------------------------*/
yyerrorlab:

  /* Pacify compilers like GCC when the user code never invokes
     YYERROR and the label yyerrorlab therefore never appears in user
     code.  */
  if (/*CONSTCOND*/ 0)
     goto yyerrorlab;

  /* Do not reclaim the symbols of the rule which action triggered
     this YYERROR.  */
  YYPOPSTACK (yylen);
  yylen = 0;
  YY_STACK_PRINT (yyss, yyssp);
  yystate = *yyssp;
  goto yyerrlab1;


/*-------------------------------------------------------------.
| yyerrlab1 -- common code for both syntax error and YYERROR.  |
`-------------------------------------------------------------*/
yyerrlab1:
  yyerrstatus = 3;	/* Each real token shifted decrements this.  */

  for (;;)
    {
      yyn = yypact[yystate];
      if (yyn != YYPACT_NINF)
	{
	  yyn += YYTERROR;
	  if (0 <= yyn && yyn <= YYLAST && yycheck[yyn] == YYTERROR)
	    {
	      yyn = yytable[yyn];
	      if (0 < yyn)
		break;
	    }
	}

      /* Pop the current state because it cannot handle the error token.  */
      if (yyssp == yyss)
	YYABORT;


      yydestruct ("Error: popping",
		  yystos[yystate], yyvsp);
      YYPOPSTACK (1);
      yystate = *yyssp;
      YY_STACK_PRINT (yyss, yyssp);
    }

  *++yyvsp = yylval;


  /* Shift the error token.  */
  YY_SYMBOL_PRINT ("Shifting", yystos[yyn], yyvsp, yylsp);

  yystate = yyn;
  goto yynewstate;


/*-------------------------------------.
| yyacceptlab -- YYACCEPT comes here.  |
`-------------------------------------*/
yyacceptlab:
  yyresult = 0;
  goto yyreturn;

/*-----------------------------------.
| yyabortlab -- YYABORT comes here.  |
`-----------------------------------*/
yyabortlab:
  yyresult = 1;
  goto yyreturn;

#if !defined(yyoverflow) || YYERROR_VERBOSE
/*-------------------------------------------------.
| yyexhaustedlab -- memory exhaustion comes here.  |
`-------------------------------------------------*/
yyexhaustedlab:
  yyerror (YY_("memory exhausted"));
  yyresult = 2;
  /* Fall through.  */
#endif

yyreturn:
  if (yychar != YYEMPTY)
     yydestruct ("Cleanup: discarding lookahead",
		 yytoken, &yylval);
  /* Do not reclaim the symbols of the rule which action triggered
     this YYABORT or YYACCEPT.  */
  YYPOPSTACK (yylen);
  YY_STACK_PRINT (yyss, yyssp);
  while (yyssp != yyss)
    {
      yydestruct ("Cleanup: popping",
		  yystos[*yyssp], yyvsp);
      YYPOPSTACK (1);
    }
#ifndef yyoverflow
  if (yyss != yyssa)
    YYSTACK_FREE (yyss);
#endif
#if YYERROR_VERBOSE
  if (yymsg != yymsgbuf)
    YYSTACK_FREE (yymsg);
#endif
  /* Make sure YYID is used.  */
  return YYID (yyresult);
}



/* Line 1675 of yacc.c  */
#line 261 "bison.y"


bool scan_state;
unsigned int statement_count;

int execute_file(int ac, char **av)
{
    bool just_once  = 0;
    string script;

    process_count = 6200000;
    verbose = 0;
	ssd = 0;
	delta = 0;
    total_buffer_size = 0;
	hash_seed = 100;

    for (int i = 1; i < ac; i++) {
        if(strcmp(av[i],"-l") == 0) {
            process_count = atoff(av[i+1]);
        }
        else if(strcmp(av[i],"-v") == 0) {
            verbose = 1;
        }
        else if(strcmp(av[i],"-delta") == 0) {
            delta = 1;
        }		
        else if(strcmp(av[i],"-ssd") == 0) {
            ssd = 1;
        }		
        else if(strcmp(av[i],"-i") == 0) {
            interactive = 1;
            break;
        }
        else if(strcmp(av[i],"-s") == 0) {
            just_once = 1;
            interactive = 1;
            script = av[i+1];
        };
    };

    load_col_data(data_dict, "data.dictionary");
	tot_disk = 0;

    if (!interactive) {
        if((yyin = fopen(av[ac-1], "r")) == nullptr) {
            perror(av[ac-1]);
            exit(1);
        };

        if(yyparse()) {
            printf("SQL scan parse failed\n");
            exit(1);
        };

        scan_state = 1;
        std::clock_t start1 = std::clock();

        load_vars();
		
        statement_count = 0;
        clean_queues();

        yyin = fopen(av[ac-1], "r");
        PROC_FLUSH_BUF ( yyin );
        statement_count = 0;

        extern FILE *yyin;
        context = CreateCudaDevice(0, nullptr, verbose);

        if(!yyparse()) {
            if(verbose)
                cout << "SQL scan parse worked " << endl;
        }
        else
            cout << "SQL scan parse failed" << endl;

        fclose(yyin);
        for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
            (*it).second->free();
        };

        if(verbose) {
            cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
			cout<< "disk time " << ( tot_disk / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << endl;
        };
    }
    else {
        context = CreateCudaDevice(0, nullptr, verbose);        
        if(!just_once)
            getline(cin, script);

        while (script != "exit" && script != "EXIT") {

            used_vars.clear();
            yy_scan_string(script.c_str());
            scan_state = 0;
            statement_count = 0;
            clean_queues();
            if(yyparse()) {
                printf("SQL scan parse failed \n");
                getline(cin, script);
                continue;
            };

            scan_state = 1;

            load_vars();

            statement_count = 0;
            clean_queues();
            yy_scan_string(script.c_str());
            std::clock_t start1 = std::clock();

            if(!yyparse()) {
                if(verbose)
                    cout << "SQL scan parse worked " <<  endl;
            };
            for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
                (*it).second->free();
            };
            varNames.clear();

            if(verbose) {
                cout<< "cycle time " << ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << endl;
            };
            if(!just_once)
                getline(cin, script);
            else
                script = "exit";
        };

        while(!buffer_names.empty()) {
            delete [] buffers[buffer_names.front()];
            buffer_sizes.erase(buffer_names.front());
            buffers.erase(buffer_names.front());
            buffer_names.pop();
        };
		for(auto it = index_buffers.begin(); it != index_buffers.end();it++) {
			hipHostFree(it->second);
        };

    };
    if(save_dict)
        save_col_data(data_dict,"data.dictionary");

    if(alloced_sz) {
        hipFree(alloced_tmp);
        alloced_sz = 0;
    };
    if(raw_decomp_length) {
        hipFree(raw_decomp);
        raw_decomp_length = 0;
    };

    return 0;
}



//external c global to report errors
//char alenka_err[4048];


int alenkaExecute(char *s)
{
    YY_BUFFER_STATE bp;

    total_buffer_size = 0;
    scan_state = 0;
    load_col_data(data_dict, "data.dictionary");
    std::clock_t start;

    if(verbose)
        start = std::clock();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    int ret = yyparse();
    //printf("execute: returned [%d]\n", ret);
    if(!ret) {
        if(verbose)
            cout << "SQL scan parse worked" << endl;
    }

    scan_state = 1;
    load_vars();
    statement_count = 0;
    clean_queues();
    bp = yy_scan_string(s);
    yy_switch_to_buffer(bp);
    if(!yyparse()) {
        if(verbose)
            cout << "SQL scan parse worked " << endl;
    }
    else
        cout << "SQL scan parse failed" << endl;

    yy_delete_buffer(bp);

    // Clear Vars
    for (auto it=varNames.begin() ; it != varNames.end(); ++it ) {
        (*it).second->free();
    };
    varNames.clear();

    if(verbose)
        cout<< "statement time " <<  ( ( std::clock() - start ) / (double)CLOCKS_PER_SEC ) << endl;
    if(save_dict)
        save_col_data(data_dict,"data.dictionary");
    return ret;
}



