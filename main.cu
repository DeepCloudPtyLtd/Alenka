#include "hip/hip_runtime.h"
/*
*
*    This file is part of Alenka.
*
*    Alenka is free software: you can redistribute it and/or modify
*    it under the terms of the GNU General Public License as published by
*    the Free Software Foundation, either version 3 of the License, or
*    (at your option) any later version.
*
*    Alenka is distributed in the hope that it will be useful,
*    but WITHOUT ANY WARRANTY; without even the implied warranty of
*    MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
*    GNU General Public License for more details.
*
*    You should have received a copy of the GNU General Public License
*    along with Alenka.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <map>
#include <string>
#include <iostream>
#include <ctime>


using namespace std;
#include "alenka.h"

int main(int ac, char **av)
{
    std::clock_t start;
    int x;


    // test QPS via alenkaExecute	-- this section is the only C++ dependency
    if (string(av[1]) == "--QPS-test") {
        alenkaInit(NULL);
        start = std::clock();
        for (x=0; x< 1000; x++)  {
            alenkaExecute("A1 := SELECT  count(n_name) AS col1 FROM nation;\n DISPLAY A1 USING ('|');");
        }
        cout<< "Ave QPS is : " <<  ( 1000/ (( std::clock() - start ) / (double)CLOCKS_PER_SEC )) << endl;
        alenkaClose();
    }
    else {				// ordinary alenka file mode
        if (ac < 2) {
            cout << "Usage : alenka [--QPS-test] | [ [-l process_count] [-v] script.sql ]" << endl;
            exit(1);
        }
        else
            return execute_file( ac, av) ;
    }
}


