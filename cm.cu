#include "hip/hip_runtime.h"
/*
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <cctype>
#include <algorithm>
#include <functional>
#include <numeric>
#include "cm.h"
#include "atof.h"
#include "compress.cu"
#include "sorts.cu"


#ifdef _WIN64
#define atoll(S) _atoi64(S)
#endif


using namespace std;
using namespace thrust::placeholders;


std::clock_t tot;
unsigned long long int total_count = 0;
unsigned int total_segments = 0;
unsigned int total_max;
unsigned int process_count;
map <unsigned int, unsigned int> str_offset;
long long int totalRecs = 0;
bool fact_file_loaded = 1;
char map_check;
void* d_v = NULL;
void* s_v = NULL;
unsigned int oldCount;
queue<string> op_sort;
queue<string> op_type;
queue<string> op_value;
queue<int_type> op_nums;
queue<float_type> op_nums_f;
queue<string> col_aliases;

void* alloced_tmp;
unsigned int alloced_sz = 0;
bool alloced_switch = 0;

map<string,CudaSet*> varNames; //  STL map to manage CudaSet variables
map<string,string> setMap; //map to keep track of column names and set names


struct is_match
{
    __host__ __device__
    bool operator()(unsigned int x)
    {
        return x != 4294967295;
    }
};


struct f_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) < EPSILON) && ((x-y) > -EPSILON));
    }
};


struct f_less
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((y-x) > EPSILON);
    }
};

struct f_greater
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((x-y) > EPSILON);
    }
};

struct f_greater_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_less_equal
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((y-x) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_not_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return !(((x-y) < EPSILON) && ((x-y) > -EPSILON));
    }
};


struct long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (float_type)x;
    }
};


struct l_to_ui
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (unsigned int)x;
    }
};

struct float_to_decimal
{
    __host__ __device__
    float_type operator()(const float_type x)
    {
        return (int_type)(x*100);
    }
};


struct to_zero
{
    __host__ __device__
    bool operator()(const int_type x)
    {
        if(x == -1)
            return 0;
        else
            return 1;
    }
};



struct div_long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x, const float_type y)
    {
        return (float_type)x/y;
    }
};


struct long_to_float
{
    __host__ __device__
    float_type operator()(const long long int x)
    {
        return (((float_type)x)/100.0);
    }
};


// trim from start
static inline std::string &ltrim(std::string &s) {
    s.erase(s.begin(), std::find_if(s.begin(), s.end(), std::not1(std::ptr_fun<int, int>(std::isspace))));
    return s;
}

// trim from end
static inline std::string &rtrim(std::string &s) {
    s.erase(std::find_if(s.rbegin(), s.rend(), std::not1(std::ptr_fun<int, int>(std::isspace))).base(), s.end());
    return s;
}

// trim from both ends
static inline std::string &trim(std::string &s) {
    return ltrim(rtrim(s));
}

char *mystrtok(char **m,char *s,char c)
{
  char *p=s?s:*m;
  if( !*p )
    return 0;
  *m=strchr(p,c);
  if( *m )
    *(*m)++=0;
  else
    *m=p+strlen(p);
  return p;
}


void allocColumns(CudaSet* a, queue<string> fields);
void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, unsigned int& count);
void mygather(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, unsigned int count, unsigned int g_size);
void mycopy(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, unsigned int count, unsigned int g_size);
void write_compressed_char(string file_name, unsigned int index, unsigned int mCount);
unsigned long long int largest_prm(CudaSet* a);
unsigned int max_tmp(CudaSet* a);


unsigned int curr_segment = 10000000;

size_t getFreeMem();
char zone_map_check(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, CudaSet* a, unsigned int segment);

float total_time1 = 0;


CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs)
    : mColumnCount(0), mRecCount(0)
{
    initialize(nameRef, typeRef, sizeRef, colsRef, Recs);
    keep = false;
    partial_load = 0;
    source = 1;
    text_source = 1;
    grp = NULL;
};

CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs, char* file_name)
    : mColumnCount(0),  mRecCount(0)
{
    initialize(nameRef, typeRef, sizeRef, colsRef, Recs, file_name);
    keep = false;
    partial_load = 1;
    source = 1;
    text_source = 0;
    grp = NULL;
};

CudaSet::CudaSet(unsigned int RecordCount, unsigned int ColumnCount)
{
    initialize(RecordCount, ColumnCount);
    keep = false;
    partial_load = 0;
    source = 0;
    text_source = 0;
    grp = NULL;
};

CudaSet::CudaSet(queue<string> op_sel, queue<string> op_sel_as)
{
    initialize(op_sel, op_sel_as);
    keep = false;
    partial_load = 0;
    source = 0;
    text_source = 0;
    grp = NULL;
};

CudaSet::CudaSet(CudaSet* a, CudaSet* b, queue<string> op_sel, queue<string> op_sel_as)
{
    initialize(a,b, op_sel, op_sel_as);
    keep = false;
    partial_load = 0;
    source = 0;
    text_source = 0;
    grp = NULL;
};


CudaSet::~CudaSet()
{
    free();
};


void CudaSet::allocColumnOnDevice(unsigned int colIndex, unsigned long long int RecordCount)
{
    if (type[colIndex] == 0) {
		d_columns_int[type_index[colIndex]].resize(RecordCount);
    }
    else if (type[colIndex] == 1)
        d_columns_float[type_index[colIndex]].resize(RecordCount);
    else {
        void* d;
		unsigned long long int sz = (unsigned long long int)RecordCount*char_size[type_index[colIndex]];
        hipError_t cudaStatus = hipMalloc(&d, sz);
		if(cudaStatus != hipSuccess) {
			cout << "Could not allocate " << sz << " bytes of GPU memory for " << RecordCount << " records " << endl;
			exit(0);
		};
        d_columns_char[type_index[colIndex]] = (char*)d;
    };
};


void CudaSet::decompress_char_hash(unsigned int colIndex, unsigned int segment, unsigned int i_cnt)
{

    unsigned int bits_encoded, fit_count, sz, vals_count, real_count, old_count;
    const unsigned int len = char_size[type_index[colIndex]];

    char f1[100];
    strcpy(f1, load_file_name);
    strcat(f1,".");
    char col_pos[3];
    itoaa(cols[colIndex],col_pos);
    strcat(f1,col_pos);

    strcat(f1,".");
    itoaa(segment,col_pos);
    strcat(f1,col_pos);
    FILE* f;
    f = fopen (f1 , "rb" );
    fread(&sz, 4, 1, f);
    char* d_array = new char[sz*len];
    fread((void*)d_array, sz*len, 1, f);

    unsigned long long int* hashes  = new unsigned long long int[sz];

    for(unsigned int i = 0; i < sz ; i++) {
        hashes[i] = MurmurHash64A(&d_array[i*len], len, hash_seed); // divide by 2 so it will fit into a signed long long
    };

    void* d;
    hipMalloc((void **) &d, sz*int_size);
    hipMemcpy( d, (void *) hashes, sz*8, hipMemcpyHostToDevice);

    thrust::device_ptr<unsigned long long int> dd_int((unsigned long long int*)d);

    delete[] d_array;
    delete[] hashes;

    fread(&fit_count, 4, 1, f);
    fread(&bits_encoded, 4, 1, f);
    fread(&vals_count, 4, 1, f);
    fread(&real_count, 4, 1, f);

    unsigned long long int* int_array = new unsigned long long int[vals_count];
    fread((void*)int_array, 1, vals_count*8, f);
    fclose(f);

    void* d_val;
    hipMalloc((void **) &d_val, vals_count*8);
    hipMemcpy(d_val, (void *) int_array, vals_count*8, hipMemcpyHostToDevice);

    thrust::device_ptr<unsigned long long int> mval((unsigned long long int*)d_val);


    delete[] int_array;

    void* d_int;
    hipMalloc((void **) &d_int, real_count*4);

    // convert bits to ints and then do gather

    void* d_v;
    hipMalloc((void **) &d_v, 8);
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v);
    dd_v[1] = fit_count;
    dd_v[0] = bits_encoded;

    thrust::counting_iterator<unsigned int> begin(0);
    decompress_functor_str ff((unsigned long long int*)d_val,(unsigned int*)d_int, (unsigned int*)d_v);
    thrust::for_each(begin, begin + real_count, ff);

    //thrust::device_ptr<long long int> dd_int((long long int*)d);
    thrust::device_ptr<unsigned int> dd_val((unsigned int*)d_int);

    if(!prm.empty()) {
        if(prm_index[segment] == 'R') {
            thrust::device_ptr<int_type> d_tmp = thrust::device_malloc<int_type>(real_count);
            thrust::gather(dd_val, dd_val + real_count, dd_int, d_tmp);

            if(prm_d.size() == 0) // find the largest prm segment
                prm_d.resize(largest_prm(this));
            hipMemcpy((void**)(thrust::raw_pointer_cast(prm_d.data())), (void**)prm[segment],
                       4*prm_count[segment], hipMemcpyHostToDevice);

            old_count = d_columns_int[i_cnt].size();
            d_columns_int[i_cnt].resize(old_count + prm_count[segment]);
            thrust::gather(prm_d.begin(), prm_d.begin() + prm_count[segment], d_tmp, d_columns_int[i_cnt].begin() + old_count);
            thrust::device_free(d_tmp);

        }
        else if(prm_index[segment] == 'A') {
            old_count = d_columns_int[i_cnt].size();
            d_columns_int[i_cnt].resize(old_count + real_count);
            thrust::gather(dd_val, dd_val + real_count, dd_int, d_columns_int[i_cnt].begin() + old_count);
        }
    }
    else {

        old_count = d_columns_int[i_cnt].size();
        d_columns_int[i_cnt].resize(old_count + real_count);
        thrust::gather(dd_val, dd_val + real_count, dd_int, d_columns_int[i_cnt].begin() + old_count);

    };

    hipFree(d);
    hipFree(d_val);
    hipFree(d_v);
    hipFree(d_int);
};




// takes a char column , hashes strings, copies them to a gpu
void CudaSet::add_hashed_strings(string field, unsigned int segment, unsigned int i_cnt)
{
    unsigned int colInd2 = columnNames.find(field)->second;
    CudaSet *t = varNames[setMap[field]];

    if(not_compressed) { // decompressed strings on a host

        unsigned int old_count;
        unsigned long long int* hashes  = new unsigned long long int[t->mRecCount];

        for(unsigned int i = 0; i < t->mRecCount ; i++) {
            hashes[i] = MurmurHash64A(t->h_columns_char[t->type_index[colInd2]] + i*t->char_size[t->type_index[colInd2]] + segment*t->maxRecs*t->char_size[t->type_index[colInd2]], t->char_size[t->type_index[colInd2]], hash_seed);
		};	

        if(!prm.empty()) {
            if(prm_index[segment] == 'R') {

                thrust::device_ptr<unsigned long long int> d_tmp = thrust::device_malloc<unsigned long long int>(t->mRecCount);
                thrust::copy(hashes, hashes+mRecCount, d_tmp);

                if(prm_d.size() == 0) // find the largest prm segment
                    prm_d.resize(largest_prm(this));

                hipMemcpy((void**)(thrust::raw_pointer_cast(prm_d.data())), (void**)prm[segment],
                           4*prm_count[segment], hipMemcpyHostToDevice);

                old_count = d_columns_int[i_cnt].size();
                d_columns_int[i_cnt].resize(old_count + prm_count[segment]);
                thrust::gather(prm_d.begin(), prm_d.begin() + prm_count[segment], d_tmp, d_columns_int[i_cnt].begin() + old_count);
                thrust::device_free(d_tmp);

            }
            else if(prm_index[segment] == 'A') {
                old_count = d_columns_int[i_cnt].size();
                d_columns_int[i_cnt].resize(old_count + mRecCount);
                thrust::copy(hashes, hashes + mRecCount, d_columns_int[i_cnt].begin() + old_count);
            }
        }
        else {
            old_count = d_columns_int[i_cnt].size();
            d_columns_int[i_cnt].resize(old_count + mRecCount);
            thrust::copy(hashes, hashes + mRecCount, d_columns_int[i_cnt].begin() + old_count);
        }
		delete [] hashes;
    }
    else { // hash the dictionary
        decompress_char_hash(colInd2, segment, i_cnt);
    };
};


void CudaSet::resize_join(unsigned int addRecs)
{    
    mRecCount = mRecCount + addRecs;
	bool prealloc = 0;
    for(unsigned int i=0; i < mColumnCount; i++) {
        if(type[i] == 0) {
            h_columns_int[type_index[i]].resize(mRecCount);
        }
        else if(type[i] == 1) {
            h_columns_float[type_index[i]].resize(mRecCount);
        }
        else {
            if (h_columns_char[type_index[i]]) {			    
                if (mRecCount > prealloc_char_size) {                    
                    h_columns_char[type_index[i]] = (char*)realloc(h_columns_char[type_index[i]], (unsigned long long int)mRecCount*(unsigned long long int)char_size[type_index[i]]);
					prealloc = 1;
                };
            }
            else {
                h_columns_char[type_index[i]] = new char[(unsigned long long int)mRecCount*(unsigned long long int)char_size[type_index[i]]];
            };
        };

    };
	if(prealloc)
		prealloc_char_size = mRecCount;
};


void CudaSet::resize(unsigned int addRecs)
{    
    mRecCount = mRecCount + addRecs;
    for(unsigned int i=0; i <mColumnCount; i++) {
        if(type[i] == 0) {
            h_columns_int[type_index[i]].resize(mRecCount);
        }
        else if(type[i] == 1) {
            h_columns_float[type_index[i]].resize(mRecCount);
        }
        else {
            if (h_columns_char[type_index[i]]) {
                h_columns_char[type_index[i]] = (char*)realloc(h_columns_char[type_index[i]], (unsigned long long int)mRecCount*(unsigned long long int)char_size[type_index[i]]);
            }
            else {
                h_columns_char[type_index[i]] = new char[(unsigned long long int)mRecCount*(unsigned long long int)char_size[type_index[i]]];
            };
        };

    };
};

void CudaSet::reserve(unsigned int Recs)
{

    for(unsigned int i=0; i <mColumnCount; i++) {
        if(type[i] == 0)
            h_columns_int[type_index[i]].reserve(Recs);
        else if(type[i] == 1)
            h_columns_float[type_index[i]].reserve(Recs);
        else {
		    h_columns_char[type_index[i]] = new char[(unsigned long long int)Recs*(unsigned long long int)char_size[type_index[i]]];            
			if(h_columns_char[type_index[i]] == NULL) {
			    cout << "Could not allocate on a host " << Recs << " records of size " << char_size[type_index[i]] << endl;
			    exit(0);
			};
            prealloc_char_size = Recs;
        };

    };
};


void CudaSet::deAllocColumnOnDevice(unsigned int colIndex)
{
    if (type[colIndex] == 0 && !d_columns_int.empty()) {
        d_columns_int[type_index[colIndex]].resize(0);
        d_columns_int[type_index[colIndex]].shrink_to_fit();
    }
    else if (type[colIndex] == 1 && !d_columns_float.empty()) {
        d_columns_float[type_index[colIndex]].resize(0);
        d_columns_float[type_index[colIndex]].shrink_to_fit();
    }
    else if (type[colIndex] == 2 && d_columns_char[type_index[colIndex]] != NULL) {
        hipFree(d_columns_char[type_index[colIndex]]);
        d_columns_char[type_index[colIndex]] = NULL;
    };
};

void CudaSet::allocOnDevice(unsigned long long int RecordCount)
{
    for(unsigned int i=0; i < mColumnCount; i++)
        allocColumnOnDevice(i, RecordCount);
};

void CudaSet::deAllocOnDevice()
{
    for(unsigned int i=0; i <mColumnCount; i++)
        deAllocColumnOnDevice(i);

    if(!columnGroups.empty() && mRecCount !=0) {
        hipFree(grp);
        grp = NULL;
    };

    if(!prm.empty()) { // free the sources
        string some_field;
        map<string,int>::iterator it=columnNames.begin();
        some_field = (*it).first;

        if(setMap[some_field].compare(name)) {
            CudaSet* t = varNames[setMap[some_field]];
            t->deAllocOnDevice();
        };
    };
};

void CudaSet::resizeDeviceColumn(unsigned int RecCount, unsigned int colIndex)
{
    if (RecCount) {
        if (type[colIndex] == 0)
            d_columns_int[type_index[colIndex]].resize(mRecCount+RecCount);
        else if (type[colIndex] == 1)
            d_columns_float[type_index[colIndex]].resize(mRecCount+RecCount);
        else {
            if (d_columns_char[type_index[colIndex]] != NULL)
                hipFree(d_columns_char[type_index[colIndex]]);
            void *d;
            hipMalloc((void **) &d, (mRecCount+RecCount)*char_size[type_index[colIndex]]);
            d_columns_char[type_index[colIndex]] = (char*)d;
        };
    };
};



void CudaSet::resizeDevice(unsigned int RecCount)
{
    if (RecCount)
        for(unsigned int i=0; i < mColumnCount; i++)
            resizeDeviceColumn(RecCount, i);
};

bool CudaSet::onDevice(unsigned int i)
{
    unsigned j = type_index[i];

    if (type[i] == 0) {
        if (d_columns_int.empty())
            return 0;
        if (d_columns_int[j].size() == 0)
            return 0;
    }
    else if (type[i] == 1) {
        if (d_columns_float.empty())
            return 0;
        if(d_columns_float[j].size() == 0)
            return 0;
    }
    else if  (type[i] == 2) {
        if(d_columns_char.empty())
            return 0;
        if(d_columns_char[j] == NULL)
            return 0;
    };
    return 1;
}



CudaSet* CudaSet::copyDeviceStruct()
{

    CudaSet* a = new CudaSet(mRecCount, mColumnCount);
    a->not_compressed = not_compressed;
    a->segCount = segCount;
    a->maxRecs = maxRecs;

    for ( map<string,int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
        a->columnNames[(*it).first] = (*it).second;

    for(unsigned int i=0; i < mColumnCount; i++) {
        a->cols[i] = cols[i];
        a->type[i] = type[i];

        if(a->type[i] == 0) {
            a->d_columns_int.push_back(thrust::device_vector<int_type>());
            a->h_columns_int.push_back(thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >());
            a->type_index[i] = a->d_columns_int.size()-1;
        }
        else if(a->type[i] == 1) {
            a->d_columns_float.push_back(thrust::device_vector<float_type>());
            a->h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >());
            a->type_index[i] = a->d_columns_float.size()-1;
            a->decimal[i] = decimal[i];
        }
        else {
            a->h_columns_char.push_back(NULL);
            a->d_columns_char.push_back(NULL);
            a->type_index[i] = a->d_columns_char.size()-1;
			a->char_size.push_back(char_size[type_index[i]]);
        };
    };
    a->load_file_name = load_file_name;

    a->mRecCount = 0;
    return a;
}


unsigned long long int CudaSet::readSegmentsFromFile(unsigned int segNum, unsigned int colIndex)
{
    
    char f1[100];
    strcpy(f1, load_file_name);
    strcat(f1,".");
    char col_pos[3];
    itoaa(cols[colIndex],col_pos);
    strcat(f1,col_pos);
    unsigned int cnt;

    strcat(f1,".");
    itoaa(segNum,col_pos);
    strcat(f1,col_pos);

	std::clock_t start1 = std::clock();
	
	
    FILE* f;

    f = fopen(f1, "rb" );
	if(f == NULL) {
		cout << "Error opening " << f1 << " file " << endl;
		exit(0);
	};
	size_t rr;
	

    if(type[colIndex] == 0) {	    
        fread(h_columns_int[type_index[colIndex]].data(), 4, 1, f);		
        cnt = ((unsigned int*)(h_columns_int[type_index[colIndex]].data()))[0];
    	//cout << "start fread " << f1 << " " << (cnt+8)*8 - 4 << endl;
        rr = fread((unsigned int*)(h_columns_int[type_index[colIndex]].data()) + 1, 1, (cnt+8)*8 - 4, f);
		if(rr != (cnt+8)*8 - 4) {
			cout << "Couldn't read  " << (cnt+8)*8 - 4 << " bytes from " << f1  << endl;
			exit(0);
		};
		//cout << "end fread " << rr << endl;
    }
    else if(type[colIndex] == 1) {	    
        fread(h_columns_float[type_index[colIndex]].data(), 4, 1, f);		
        cnt = ((unsigned int*)(h_columns_float[type_index[colIndex]].data()))[0];
		//cout << "start fread " << f1 << " " << (cnt+8)*8 - 4 << endl;
        rr = fread((unsigned int*)(h_columns_float[type_index[colIndex]].data()) + 1, 1, (cnt+8)*8 - 4, f);
		if(rr != (cnt+8)*8 - 4) {
			cout << "Couldn't read  " << (cnt+8)*8 - 4 << " bytes from " << f1  << endl;
			exit(0);
		};		
		//cout << "end fread " << rr << endl;
    }
    else {
        decompress_char(f, colIndex, segNum);
    };
	
	tot = tot +    ( std::clock() - start1 );

    fclose(f);
	
    return 0;
};


void CudaSet::decompress_char(FILE* f, unsigned int colIndex, unsigned int segNum)
{


    unsigned int bits_encoded, fit_count, sz, vals_count, real_count;
    const unsigned int len = char_size[type_index[colIndex]];
		
    fread(&sz, 4, 1, f);
    char* d_array = new char[sz*len];
    fread((void*)d_array, sz*len, 1, f);
    void* d;
    hipMalloc((void **) &d, sz*len);
    hipMemcpy( d, (void *) d_array, sz*len, hipMemcpyHostToDevice);
    delete[] d_array;
	

    fread(&fit_count, 4, 1, f);
    fread(&bits_encoded, 4, 1, f);
    fread(&vals_count, 4, 1, f);
    fread(&real_count, 4, 1, f);
	
	thrust::device_ptr<unsigned int> param = thrust::device_malloc<unsigned int>(2);
    param[1] = fit_count;
    param[0] = bits_encoded;
	
    unsigned long long int* int_array = new unsigned long long int[vals_count];
    fread((void*)int_array, 1, vals_count*8, f);
    //fclose(f);
	
    void* d_val;
    hipMalloc((void **) &d_val, vals_count*8);
    hipMemcpy(d_val, (void *) int_array, vals_count*8, hipMemcpyHostToDevice);
    delete[] int_array;	

    void* d_int;
    hipMalloc((void **) &d_int, real_count*4);

    // convert bits to ints and then do gather
	

    thrust::counting_iterator<unsigned int> begin(0);
    decompress_functor_str ff((unsigned long long int*)d_val,(unsigned int*)d_int, (unsigned int*)thrust::raw_pointer_cast(param));
    thrust::for_each(begin, begin + real_count, ff);
	
    //thrust::device_ptr<unsigned int> dd_r((unsigned int*)d_int);
    //for(int z = 0 ; z < 3; z++)
    //cout << "DD " << dd_r[z] << endl;

    //void* d_char;
    //hipMalloc((void **) &d_char, real_count*len);
    //hipMemset(d_char, 0, real_count*len);
    //str_gather(d_int, real_count, d, d_char, len);
    if(str_offset.count(colIndex) == 0)
        str_offset[colIndex] = 0;
    //cout << "str off " << str_offset[colIndex] << endl;
	//cout << "prm cnt of seg " << segNum << " is " << prm.empty() << endl; 
    if(!alloced_switch)
        str_gather(d_int, real_count, d, d_columns_char[type_index[colIndex]] + str_offset[colIndex]*len, len);
    else
        str_gather(d_int, real_count, d, alloced_tmp, len);
		
		
    if(!prm.empty()) {
        str_offset[colIndex] = str_offset[colIndex] + prm_count[segNum];
    }
    else {
        str_offset[colIndex] = str_offset[colIndex] + real_count;
    };

    //if(d_columns_char[type_index[colIndex]])
    //    hipFree(d_columns_char[type_index[colIndex]]);
    //d_columns_char[type_index[colIndex]] = (char*)d_char;

    mRecCount = real_count;

    hipFree(d);
    hipFree(d_val);
    thrust::device_free(param);
    hipFree(d_int);
}



void CudaSet::CopyColumnToGpu(unsigned int colIndex,  unsigned int segment)
{    
    if(not_compressed) 	{
	    // calculate how many records we need to copy
		if(segment < segCount-1) {
			mRecCount = maxRecs;
		}
        else {
			mRecCount = oldRecCount - maxRecs*(segCount-1);
        };		
	
        switch(type[colIndex]) {
        case 0 :
            if(!alloced_switch)
                thrust::copy(h_columns_int[type_index[colIndex]].begin() + maxRecs*segment, h_columns_int[type_index[colIndex]].begin() + maxRecs*segment + mRecCount, d_columns_int[type_index[colIndex]].begin());
            else {
                thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
                thrust::copy(h_columns_int[type_index[colIndex]].begin() + maxRecs*segment, h_columns_int[type_index[colIndex]].begin() + maxRecs*segment + mRecCount, d_col);
            };
            break;
        case 1 :
            if(!alloced_switch)
                thrust::copy(h_columns_float[type_index[colIndex]].begin() + maxRecs*segment, h_columns_float[type_index[colIndex]].begin() + maxRecs*segment + mRecCount, d_columns_float[type_index[colIndex]].begin());
            else {
                thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
                thrust::copy(h_columns_float[type_index[colIndex]].begin() + maxRecs*segment, h_columns_float[type_index[colIndex]].begin() + maxRecs*segment + mRecCount, d_col);
            };
            break;
        default :			
            if(!alloced_switch)
                hipMemcpy(d_columns_char[type_index[colIndex]], h_columns_char[type_index[colIndex]] + maxRecs*segment*char_size[type_index[colIndex]], char_size[type_index[colIndex]]*mRecCount, hipMemcpyHostToDevice);
            else
                hipMemcpy(alloced_tmp, h_columns_char[type_index[colIndex]] + maxRecs*segment*char_size[type_index[colIndex]], char_size[type_index[colIndex]]*mRecCount, hipMemcpyHostToDevice);
        };
    }
    else {
        unsigned long long int data_offset;
		
        if (partial_load) 
            data_offset = readSegmentsFromFile(segment,colIndex);

        if(type[colIndex] != 2) {
            if(d_v == NULL)
                CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
            if(s_v == NULL);
            CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));
        };

        if(type[colIndex] == 0) {
            if(!alloced_switch) {
                mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data()), h_columns_int[type_index[colIndex]].data() + data_offset, d_v, s_v);
            }
            else {
                mRecCount = pfor_decompress(alloced_tmp, h_columns_int[type_index[colIndex]].data() + data_offset, d_v, s_v);
            };
			
        }
        else if(type[colIndex] == 1) {
            if(decimal[colIndex]) {
                if(!alloced_switch) {
                    mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data()) , h_columns_float[type_index[colIndex]].data() + data_offset, d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data()));
                    thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[type_index[colIndex]].begin(), long_to_float());
                }
                else {
                    mRecCount = pfor_decompress(alloced_tmp, h_columns_float[type_index[colIndex]].data() + data_offset, d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)alloced_tmp);
                    thrust::device_ptr<float_type> d_col_float((float_type*)alloced_tmp);
                    thrust::transform(d_col_int,d_col_int+mRecCount, d_col_float, long_to_float());
                };
            }
            //else // uncompressed float
            //hipMemcpy( d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
            // will have to fix it later so uncompressed data will be written by segments too
        }
    };
}



void CudaSet::CopyColumnToGpu(unsigned int colIndex) // copy all segments
{
    if(not_compressed) {
        switch(type[colIndex]) {
        case 0 :
            thrust::copy(h_columns_int[type_index[colIndex]].begin(), h_columns_int[type_index[colIndex]].begin() + mRecCount, d_columns_int[type_index[colIndex]].begin());
            break;
        case 1 :
            thrust::copy(h_columns_float[type_index[colIndex]].begin(), h_columns_float[type_index[colIndex]].begin() + mRecCount, d_columns_float[type_index[colIndex]].begin());
            break;
        default :
            hipMemcpy(d_columns_char[type_index[colIndex]], h_columns_char[type_index[colIndex]], char_size[type_index[colIndex]]*mRecCount, hipMemcpyHostToDevice);
        };
    }
    else {
        long long int data_offset;
        unsigned long long int totalRecs = 0;
        if(d_v == NULL)
            CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
        if(s_v == NULL);
        CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));

        str_offset[colIndex] = 0;
        for(unsigned int i = 0; i < segCount; i++) {

            if (partial_load)
                data_offset = readSegmentsFromFile(i,colIndex);


            if(type[colIndex] == 0) {
                mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data() + totalRecs), h_columns_int[type_index[colIndex]].data() + data_offset, d_v, s_v);
            }
            else if(type[colIndex] == 1) {
                if(decimal[colIndex]) {
                    mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() + totalRecs) , h_columns_float[type_index[colIndex]].data() + data_offset, d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() + totalRecs));
                    thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[type_index[colIndex]].begin() + totalRecs, long_to_float());
                }
                // else  uncompressed float
                //hipMemcpy( d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
                // will have to fix it later so uncompressed data will be written by segments too
            };

            totalRecs = totalRecs + mRecCount;
        };

        mRecCount = totalRecs;
    };
}



void CudaSet::CopyColumnToHost(int colIndex, unsigned int offset, unsigned int RecCount)
{

    switch(type[colIndex]) {
    case 0 :
        thrust::copy(d_columns_int[type_index[colIndex]].begin(), d_columns_int[type_index[colIndex]].begin() + RecCount, h_columns_int[type_index[colIndex]].begin() + offset);
        break;
    case 1 :
        thrust::copy(d_columns_float[type_index[colIndex]].begin(), d_columns_float[type_index[colIndex]].begin() + RecCount, h_columns_float[type_index[colIndex]].begin() + offset);
        break;
    default :
        hipMemcpy(h_columns_char[type_index[colIndex]] + offset*char_size[type_index[colIndex]], d_columns_char[type_index[colIndex]], char_size[type_index[colIndex]]*RecCount, hipMemcpyDeviceToHost);
    }
}



void CudaSet::CopyColumnToHost(int colIndex)
{
    CopyColumnToHost(colIndex, 0, mRecCount);
}

void CudaSet::CopyToHost(unsigned int offset, unsigned int count)
{
    for(unsigned int i = 0; i < mColumnCount; i++) {
        CopyColumnToHost(i, offset, count);
    };
}

float_type* CudaSet::get_float_type_by_name(string name)
{
    unsigned int colIndex = columnNames.find(name)->second;
    return thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data());
}

int_type* CudaSet::get_int_by_name(string name)
{
    unsigned int colIndex = columnNames.find(name)->second;
    return thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data());
}

float_type* CudaSet::get_host_float_by_name(string name)
{
    unsigned int colIndex = columnNames.find(name)->second;
    return thrust::raw_pointer_cast(h_columns_float[type_index[colIndex]].data());
}

int_type* CudaSet::get_host_int_by_name(string name)
{
    unsigned int colIndex = columnNames.find(name)->second;
    return thrust::raw_pointer_cast(h_columns_int[type_index[colIndex]].data());
}



void CudaSet::GroupBy(stack<string> columnRef, unsigned int int_col_count)
{
    int grpInd, colIndex;

    if(grp)
        hipFree(grp);

    CUDA_SAFE_CALL(hipMalloc((void **) &grp, mRecCount * sizeof(bool)));
    thrust::device_ptr<bool> d_grp(grp);

    thrust::sequence(d_grp, d_grp+mRecCount, 0, 0);

    thrust::device_ptr<bool> d_group = thrust::device_malloc<bool>(mRecCount);

    d_group[mRecCount-1] = 1;
    unsigned int i_count = 0;

    for(int i = 0; i < columnRef.size(); columnRef.pop()) {

        columnGroups.push(columnRef.top()); // save for future references
        colIndex = columnNames[columnRef.top()];

        if(!onDevice(colIndex)) {
            allocColumnOnDevice(colIndex,mRecCount);
            CopyColumnToGpu(colIndex,  mRecCount);
            grpInd = 1;
        }
        else
            grpInd = 0;

        if (type[colIndex] == 0) {  // int_type
            thrust::transform(d_columns_int[type_index[colIndex]].begin(), d_columns_int[type_index[colIndex]].begin() + mRecCount - 1,
                              d_columns_int[type_index[colIndex]].begin()+1, d_group, thrust::not_equal_to<int_type>());
        }
        else if (type[colIndex] == 1) {  // float_type
            thrust::transform(d_columns_float[type_index[colIndex]].begin(), d_columns_float[type_index[colIndex]].begin() + mRecCount - 1,
                              d_columns_float[type_index[colIndex]].begin()+1, d_group, f_not_equal_to());
        }
        else  {  // Char
            //str_grp(d_columns_char[type_index[colIndex]], mRecCount, d_group, char_size[type_index[colIndex]]);
            //use int_type

            thrust::transform(d_columns_int[int_col_count+i_count].begin(), d_columns_int[int_col_count+i_count].begin() + mRecCount - 1,
                              d_columns_int[int_col_count+i_count].begin()+1, d_group, thrust::not_equal_to<int_type>());
            i_count++;

        };
        thrust::transform(d_group, d_group+mRecCount, d_grp, d_grp, thrust::logical_or<bool>());

        if (grpInd == 1)
            deAllocColumnOnDevice(colIndex);
    };

    thrust::device_free(d_group);
    grp_count = thrust::count(d_grp, d_grp+mRecCount,1);
};


void CudaSet::addDeviceColumn(int_type* col, int colIndex, string colName, unsigned int recCount)
{
    if (columnNames.find(colName) == columnNames.end()) {
        columnNames[colName] = colIndex;
        type[colIndex] = 0;
        d_columns_int.push_back(thrust::device_vector<int_type>(recCount));
        h_columns_int.push_back(thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >());
        type_index[colIndex] = d_columns_int.size()-1;
    }
    else {  // already exists, my need to resize it
        if(d_columns_int[type_index[colIndex]].size() < recCount) {
            d_columns_int[type_index[colIndex]].resize(recCount);
        };
    };
    // copy data to d columns
    thrust::device_ptr<int_type> d_col((int_type*)col);
    thrust::copy(d_col, d_col+recCount, d_columns_int[type_index[colIndex]].begin());
};

void CudaSet::addDeviceColumn(float_type* col, int colIndex, string colName, unsigned int recCount)
{
    if (columnNames.find(colName) == columnNames.end()) {
        columnNames[colName] = colIndex;
        type[colIndex] = 1;
        d_columns_float.push_back(thrust::device_vector<float_type>(recCount));
        h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >());
        type_index[colIndex] = d_columns_float.size()-1;
    }
    else {  // already exists, my need to resize it
        if(d_columns_float[type_index[colIndex]].size() < recCount)
            d_columns_float[type_index[colIndex]].resize(recCount);
    };

    thrust::device_ptr<float_type> d_col((float_type*)col);
    thrust::copy(d_col, d_col+recCount, d_columns_float[type_index[colIndex]].begin());
};

void CudaSet::compress(char* file_name, unsigned int offset, unsigned int check_type, unsigned int check_val, void* d, unsigned int mCount)
{
    char str[100];
    char col_pos[3];
	thrust::device_vector<unsigned int> permutation;
	
	total_count = total_count + mCount;
	total_segments = total_segments + 1;
	if (mCount > total_max)
		total_max = mCount;
	
	if(!op_sort.empty()) { //sort the segment
		//copy the key columns to device		
		queue<string> sf(op_sort);				
		
		permutation.resize(mRecCount);
		thrust::sequence(permutation.begin(), permutation.begin() + mRecCount,0,1);
		unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation.data());
		void* temp;	
        cout << "sorting " << getFreeMem() << endl; 
		
		unsigned int max_c = max_char(this, sf);				

		if(max_c > float_size)
			CUDA_SAFE_CALL(hipMalloc((void **) &temp, mRecCount*max_c));
		else
			CUDA_SAFE_CALL(hipMalloc((void **) &temp, mRecCount*float_size));
		
        string sort_type = "ASC";		

        while(!sf.empty()) {
            int colInd = columnNames[sf.front()];
			
			allocColumnOnDevice(colInd, maxRecs);
			CopyColumnToGpu(colInd);			
			
            if (type[colInd] == 0)
                update_permutation(d_columns_int[type_index[colInd]], raw_ptr, mRecCount, sort_type, (int_type*)temp);
            else if (type[colInd] == 1)
                update_permutation(d_columns_float[type_index[colInd]], raw_ptr, mRecCount, sort_type, (float_type*)temp);
            else {
                update_permutation_char(d_columns_char[type_index[colInd]], raw_ptr, mRecCount, sort_type, (char*)temp, char_size[type_index[colInd]]);
            };
			deAllocColumnOnDevice(colInd);
			sf.pop();
        };	
        hipFree(temp);		
	};
	
	
	for(unsigned int i = 0; i< mColumnCount; i++) {			    
		strcpy(str, file_name);
		strcat(str,".");
		itoaa(cols[i],col_pos);
		strcat(str,col_pos);
		curr_file = str;
		strcat(str,".");
		itoaa(total_segments-1,col_pos);
		strcat(str,col_pos);	
		
		if(!op_sort.empty()) {
			allocColumnOnDevice(i, maxRecs);
			CopyColumnToGpu(i);			
		};	
	
		if(type[i] == 0) {
			thrust::device_ptr<int_type> d_col((int_type*)d);					
			if(!op_sort.empty()) {
				thrust::gather(permutation.begin(), permutation.end(), d_columns_int[type_index[i]].begin(), d_col);
			}
			else {
				thrust::copy(h_columns_int[type_index[i]].begin() + offset, h_columns_int[type_index[i]].begin() + offset + mCount, d_col);
			};	
			pfor_compress( d, mCount*int_size, str, h_columns_int[type_index[i]], 0, 0);
		}
		else if(type[i] == 1) {
			if(decimal[i]) {
				thrust::device_ptr<float_type> d_col((float_type*)d);				
				if(!op_sort.empty()) {
					thrust::gather(permutation.begin(), permutation.end(), d_columns_float[type_index[i]].begin(), d_col);
				}
				else {
					thrust::copy(h_columns_float[type_index[i]].begin() + offset, h_columns_float[type_index[i]].begin() + offset + mCount, d_col);
				};	
				thrust::device_ptr<long long int> d_col_dec((long long int*)d);
				thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());
				pfor_compress( d, mCount*float_size, str, h_columns_float[type_index[i]], 1, 0);
			}
			else { // do not compress -- float
				thrust::device_ptr<float_type> d_col((float_type*)d);				
				if(!op_sort.empty()) {
					thrust::gather(permutation.begin(), permutation.end(), d_columns_float[type_index[i]].begin(), d_col);
					thrust::copy(d_col, d_col+mRecCount, h_columns_float[type_index[i]].begin());
				};
				fstream binary_file(str,ios::out|ios::binary|fstream::app);
				binary_file.write((char *)&mCount, 4);
				binary_file.write((char *)(h_columns_float[type_index[i]].data() + offset),mCount*float_size);
				unsigned int comp_type = 3;
				binary_file.write((char *)&comp_type, 4);
				binary_file.close();
			};
		}
		else { //char
			if(!op_sort.empty()) {				
				unsigned int*  h_permutation = new unsigned int[mRecCount];
				thrust::copy(permutation.begin(), permutation.end(), h_permutation);
				char* t = new char[char_size[type_index[i]]*mRecCount];
				apply_permutation_char_host(h_columns_char[type_index[i]], h_permutation, mRecCount, t, char_size[type_index[i]]);
				thrust::copy(t, t+ char_size[type_index[i]]*mRecCount, h_columns_char[type_index[i]]);
				delete [] t;
			};	
			compress_char(str, i, mCount, offset);
		};	   

        if(check_type == 1) {
			if(fact_file_loaded) {
                writeHeader(file_name, cols[i]);
            }
		}
		else {
			if(check_val == 0) {
				writeHeader(file_name, cols[i]);
			};
		};
		
	}; 
	permutation.resize(0);
	permutation.shrink_to_fit();
}	


void CudaSet::writeHeader(char* file_name, unsigned int col) {
    char str[100];
    char col_pos[3];

    strcpy(str, file_name);
    strcat(str,".");
    itoaa(col,col_pos);
    strcat(str,col_pos);
    string ff = str;
    strcat(str,".header");

    fstream binary_file(str,ios::out|ios::binary|ios::app);
    binary_file.write((char *)&total_count, 8);
    binary_file.write((char *)&total_segments, 4);
    binary_file.write((char *)&total_max, 4);
    binary_file.write((char *)&cnt_counts[ff], 4);
    binary_file.close();
};


void CudaSet::writeSortHeader(char* file_name)
{
    char str[100];
	unsigned int idx;

    strcpy(str, file_name);
    strcat(str,".sort");

    fstream binary_file(str,ios::out|ios::binary|ios::app);
	idx = op_sort.size();
	binary_file.write((char *)&idx, 4);
	queue<string> os(op_sort);
	while(!os.empty()) {
	    idx = columnNames[os.front()];
		binary_file.write((char *)&idx, 4);
		os.pop();
	};	
    binary_file.close();

}



void CudaSet::Store(char* file_name, char* sep, unsigned int limit, bool binary )
{
    if (mRecCount == 0 && binary == 1) { // write tails
        for(unsigned int i = 0; i< mColumnCount; i++) {
            writeHeader(file_name, cols[i]);
        };
        return;
    };

    unsigned int mCount, cnt;

    if(limit != 0 && limit < mRecCount)
        mCount = limit;
    else
        mCount = mRecCount;

    if(binary == 0) {

        char buffer [33];
        queue<string> op_vx;
        for ( map<string,int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
            op_vx.push((*it).first);
        curr_segment = 1000000;
        FILE *file_pr = fopen(file_name, "w");
        if (file_pr  == NULL)
            cout << "Could not open file " << file_name << endl;
			
        if(prm.size() || source)
            allocColumns(this, op_vx);
        unsigned int curr_seg = 0, cnt = 0;
        unsigned curr_count, sum_printed = 0;
        while(sum_printed < mCount) {

            if(prm.size() || source)  {
                copyColumns(this, op_vx, curr_seg, cnt);
                // if host arrays are empty
                unsigned int olRecs = mRecCount;
                resize(mRecCount);
                mRecCount = olRecs;
                CopyToHost(0,mRecCount);
                if(sum_printed + mRecCount <= mCount)
                    curr_count = mRecCount;
                else {
                    curr_count = mCount - sum_printed;
                };
            }
            else
                curr_count = mCount;

            sum_printed = sum_printed + mRecCount;
            string ss;			

            for(unsigned int i=0; i < curr_count; i++) {
                for(unsigned int j=0; j < mColumnCount; j++) {
                    if (type[j] == 0) {
                        sprintf(buffer, "%lld", (h_columns_int[type_index[j]])[i] );
                        fputs(buffer,file_pr);
                        fputs(sep, file_pr);
                    }
                    else if (type[j] == 1) {
                        sprintf(buffer, "%.2f", (h_columns_float[type_index[j]])[i] );
                        fputs(buffer,file_pr);
                        fputs(sep, file_pr);
                    }
                    else {
                        ss.assign(h_columns_char[type_index[j]] + (i*char_size[type_index[j]]), char_size[type_index[j]]);
                        trim(ss);
                        fputs(ss.c_str(), file_pr);
                        fputs(sep, file_pr);
                    };
                };
                if (i != mCount -1)
                    fputs("\n",file_pr);
            };
            curr_seg++;
        };
        fclose(file_pr);
    }
    else if(text_source) {  //writing a binary file using a text file as a source

        //char str[100];
        //char col_pos[3];

        void* d;
        CUDA_SAFE_CALL(hipMalloc((void **) &d, mCount*float_size));

		compress(file_name, 0, 1, 0, d, mCount);
		writeSortHeader(file_name);
        /*for(unsigned int i = 0; i< mColumnCount; i++) {
            strcpy(str, file_name);
            strcat(str,".");
            itoaa(cols[i],col_pos);
            strcat(str,col_pos);
            curr_file = str;

            strcat(str,".");
            itoaa(total_segments-1,col_pos);
            strcat(str,col_pos);			

            if(type[i] == 0) {
                thrust::device_ptr<int_type> d_col((int_type*)d);
                thrust::copy(h_columns_int[type_index[i]].begin(), h_columns_int[type_index[i]].begin() + mCount, d_col);
                pfor_compress( d, mCount*int_size, str, h_columns_int[type_index[i]], 0, 0);
            }
            else if(type[i] == 1) {
                if(decimal[i]) {
                    thrust::device_ptr<float_type> d_col((float_type*)d);				
                    thrust::copy(h_columns_float[type_index[i]].begin(), h_columns_float[type_index[i]].begin() + mCount, d_col);
                    thrust::device_ptr<long long int> d_col_dec((long long int*)d);
                    thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());
                    pfor_compress( d, mCount*float_size, str, h_columns_float[type_index[i]], 1, 0);
                }
                else { // do not compress -- float
                    fstream binary_file(str,ios::out|ios::binary|fstream::app);
                    binary_file.write((char *)&mCount, 4);
                    binary_file.write((char *)(h_columns_float[type_index[i]].data()),mCount*float_size);
                    unsigned int comp_type = 3;
                    binary_file.write((char *)&comp_type, 4);
                    binary_file.close();
                };
            }
            else { //char
                compress_char(str, i, mCount, 0);
            };

            if(fact_file_loaded) {
                writeHeader(file_name, cols[i]);
            };

        };
		*/


        for(unsigned int i = 0; i< mColumnCount; i++)
            if(type[i] == 2)
                deAllocColumnOnDevice(i);

        hipFree(d);

    }
    else { //writing a binary file using a binary file as a source
        fact_file_loaded = 1;
		unsigned int offset = 0;
		
		void* d;
		if(mRecCount < process_count) {
			CUDA_SAFE_CALL(hipMalloc((void **) &d, mRecCount*float_size));
		}
		else {
			CUDA_SAFE_CALL(hipMalloc((void **) &d, process_count*float_size));
		};  
		
		
		if(!not_compressed) { // records are compressed, for example after filter op.
		//decompress to host
		    queue<string> op_vx;
			for ( map<string,int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it ) {
				op_vx.push((*it).first);
			};	
			
			allocColumns(this, op_vx);
			unsigned int oldCnt = mRecCount;
			mRecCount = 0;
			resize(oldCnt);
			mRecCount = oldCnt;
			for(unsigned int i = 0; i < segCount; i++) {   
				cnt = 0;
				copyColumns(this, op_vx, i, cnt);		
				reset_offsets();
				CopyToHost(0, mRecCount);
				offset = offset + mRecCount;		
				compress(file_name, 0, 0, i - (segCount-1), d, mRecCount);		
			};			
			//mRecCount = offset;			
		}
		else {
		
		
		// now we have decompressed records on the host
		//call setSegments and compress columns in every segment		
		
			segCount = mRecCount/process_count + 1;	
			offset = 0;	
		
			for(unsigned int z = 0; z < segCount; z++) {   
		
				if(z < segCount-1) {
					if(mRecCount < process_count) {
						mCount = mRecCount;
					}
					else {
						mCount = process_count;
					} 				
				}	
				else
					mCount = mRecCount - (segCount-1)*process_count;

				compress(file_name, offset, 0, z - (segCount-1), d, mCount);
				offset = offset + mCount;		
			};	
			hipFree(d);        
		};	
    };
}


void CudaSet::compress_char(string file_name, unsigned int index, unsigned int mCount, unsigned int offset)
{
    std::map<string,unsigned int> dict;
    std::vector<string> dict_ordered;
    std::vector<unsigned int> dict_val;
    map<string,unsigned int>::iterator iter;
    unsigned int bits_encoded;
    char* field;
    unsigned int len = char_size[type_index[index]];

    field = new char[len];

    for (unsigned int i = 0 ; i < mCount; i++) {

        strncpy(field, h_columns_char[type_index[index]] + (i+offset)*len, char_size[type_index[index]]);

        if((iter = dict.find(field)) != dict.end()) {
            dict_val.push_back(iter->second);
        }
        else {
            string f = field;
            dict[f] = dict.size();
            dict_val.push_back(dict.size()-1);
            dict_ordered.push_back(f);
        };
    };
	delete [] field;

    bits_encoded = (unsigned int)ceil(log2(double(dict.size()+1)));

    char *cc = new char[len+1];
    unsigned int sz = dict_ordered.size();
    // write to a file
    fstream binary_file(file_name.c_str(),ios::out|ios::binary);
    binary_file.write((char *)&sz, 4);
    for(unsigned int i = 0; i < dict_ordered.size(); i++) {
        memset(&cc[0], 0, len);
        strcpy(cc,dict_ordered[i].c_str());
        binary_file.write(cc, len);
    };

    delete [] cc;
    unsigned int fit_count = 64/bits_encoded;
    unsigned long long int val = 0;
    binary_file.write((char *)&fit_count, 4);
    binary_file.write((char *)&bits_encoded, 4);
    unsigned int curr_cnt = 1;
    unsigned int vals_count = dict_val.size()/fit_count;
    if(!vals_count || dict_val.size()%fit_count)
        vals_count++;
    binary_file.write((char *)&vals_count, 4);
    unsigned int real_count = dict_val.size();
    binary_file.write((char *)&real_count, 4);

    for(unsigned int i = 0; i < dict_val.size(); i++) {

        val = val | dict_val[i];

        if(curr_cnt < fit_count)
            val = val << bits_encoded;

        if( (curr_cnt == fit_count) || (i == (dict_val.size() - 1)) ) {
            if (curr_cnt < fit_count) {
                val = val << ((fit_count-curr_cnt)-1)*bits_encoded;
            };
            curr_cnt = 1;
            binary_file.write((char *)&val, 8);
            val = 0;
        }
        else
            curr_cnt = curr_cnt + 1;
    };
    binary_file.close();
};




int CudaSet::LoadBigFile(const char* file_name, const char* sep )
{
    char line[1000];
    unsigned int current_column, count = 0, index;
	char *p,*t;

    if (file_p == NULL)
        file_p = fopen(file_name, "r");
    if (file_p  == NULL) {
        cout << "Could not open file " << file_name << endl;
		exit(0);
	};	

	map<unsigned int,unsigned int> col_map;
	for(unsigned int i = 0; i < mColumnCount; i++) {
		col_map[cols[i]] = i;
	};		

    while (count < process_count && fgets(line, 1000, file_p) != NULL) {
        strtok(line, "\n");
        current_column = 0;
		
        for(t=mystrtok(&p,line,'|');t;t=mystrtok(&p,0,'|')) {
			current_column++;
			if(col_map.find(current_column) == col_map.end()) {
				continue;					  
			};	
          
			index = col_map[current_column];
            if (type[index] == 0) {
                if (strchr(t,'-') == NULL) {
					(h_columns_int[type_index[index]])[count] = atoll(t);
                }
                else {   // handling possible dates
                    strncpy(t+4,t+5,2);
                    strncpy(t+6,t+8,2);
                    t[8] = '\0';
                    (h_columns_int[type_index[index]])[count] = atoll(t);
                };
            }
            else if (type[index] == 1) {
				(h_columns_float[type_index[index]])[count] = atoff(t);
			}	
            else  {//char
                strcpy(h_columns_char[type_index[index]] + count*char_size[type_index[index]], t);
            }
        };
        count++;
    };

    mRecCount = count;
	
    if(count < process_count)  {
        fclose(file_p);
        return 1;
    }
    else
        return 0;
};


void CudaSet::free()  {

    if (!seq)
        delete seq;

    for(unsigned int i = 0; i < mColumnCount; i++ ) {
        if(type[i] == 2 && h_columns_char[type_index[i]] && prm.empty()) {
            delete [] h_columns_char[type_index[i]];
            h_columns_char[type_index[i]] = NULL;
        }
		else {
			if(type[i] == 0 ) {			
				h_columns_int[type_index[i]].resize(0);
				h_columns_int[type_index[i]].shrink_to_fit();
			}	
			else if(type[i] == 1) {			
		        h_columns_float[type_index[i]].resize(0);
				h_columns_float[type_index[i]].shrink_to_fit();
			};			
        }
    }
    
			
    if(!prm.empty()) { // free the sources
        string some_field;
        map<string,int>::iterator it=columnNames.begin();
        some_field = (*it).first;
        CudaSet* t = varNames[setMap[some_field]];
        t->deAllocOnDevice();

    };

    delete type;
    delete cols;

    if(!columnGroups.empty() && mRecCount !=0 && grp != NULL)
        hipFree(grp);

    for(unsigned int i = 0; i < prm.size(); i++)
        delete [] prm[i];
		
};


bool* CudaSet::logical_and(bool* column1, bool* column2)
{
    thrust::device_ptr<bool> dev_ptr1(column1);
    thrust::device_ptr<bool> dev_ptr2(column2);

    thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_and<bool>());

    thrust::device_free(dev_ptr2);
    return column1;
}


bool* CudaSet::logical_or(bool* column1, bool* column2)
{

    thrust::device_ptr<bool> dev_ptr1(column1);
    thrust::device_ptr<bool> dev_ptr2(column2);

    thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_or<bool>());
    thrust::device_free(dev_ptr2);
    return column1;
}



bool* CudaSet::compare(int_type s, int_type d, int_type op_type)
{
    bool res;

    if (op_type == 2) // >
        if(d>s) res = 1;
        else res = 0;
    else if (op_type == 1)  // <
        if(d<s) res = 1;
        else res = 0;
    else if (op_type == 6) // >=
        if(d>=s) res = 1;
        else res = 0;
    else if (op_type == 5)  // <=
        if(d<=s) res = 1;
        else res = 0;
    else if (op_type == 4)// =
        if(d==s) res = 1;
        else res = 0;
    else // !=
        if(d!=s) res = 1;
        else res = 0;

    thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
    thrust::sequence(p, p+mRecCount,res,(bool)0);

    return thrust::raw_pointer_cast(p);
};


bool* CudaSet::compare(float_type s, float_type d, int_type op_type)
{
    bool res;

    if (op_type == 2) // >
        if ((d-s) > EPSILON) res = 1;
        else res = 0;
    else if (op_type == 1)  // <
        if ((s-d) > EPSILON) res = 1;
        else res = 0;
    else if (op_type == 6) // >=
        if (((d-s) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
    else if (op_type == 5)  // <=
        if (((s-d) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
    else if (op_type == 4)// =
        if (((d-s) < EPSILON) && ((d-s) > -EPSILON)) res = 1;
        else res = 0;
    else // !=
        if (!(((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;

    thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
    thrust::sequence(p, p+mRecCount,res,(bool)0);

    return thrust::raw_pointer_cast(p);
}


bool* CudaSet::compare(int_type* column1, int_type d, int_type op_type)
{
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr(column1);


    if (op_type == 2) // >
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater<int_type>());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less<int_type>());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater_equal<int_type>());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less_equal<int_type>());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::equal_to<int_type>());
    else // !=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::not_equal_to<int_type>());

    return thrust::raw_pointer_cast(temp);

}

bool* CudaSet::compare(float_type* column1, float_type d, int_type op_type)
{
    thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr(column1);

    if (op_type == 2) // >
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_equal_to());
    else // !=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_not_equal_to());

    return thrust::raw_pointer_cast(res);
}


bool* CudaSet::compare(int_type* column1, int_type* column2, int_type op_type)
{
    thrust::device_ptr<int_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr2(column2);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater<int_type>());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less<int_type>());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater_equal<int_type>());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less_equal<int_type>());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::equal_to<int_type>());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::not_equal_to<int_type>());

    return thrust::raw_pointer_cast(temp);
}

bool* CudaSet::compare(float_type* column1, float_type* column2, int_type op_type)
{
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<float_type> dev_ptr2(column2);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

    return thrust::raw_pointer_cast(temp);

}


bool* CudaSet::compare(float_type* column1, int_type* column2, int_type op_type)
{
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr(column2);
    thrust::device_ptr<float_type> dev_ptr2 = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr2, long_to_float_type());

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

    thrust::device_free(dev_ptr2);
    return thrust::raw_pointer_cast(temp);
}


float_type* CudaSet::op(int_type* column1, float_type* column2, string op_type, int reverse)
{

    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr(column1);

    thrust::transform(dev_ptr, dev_ptr + mRecCount, temp, long_to_float_type()); // in-place transformation

    thrust::device_ptr<float_type> dev_ptr1(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };

    return thrust::raw_pointer_cast(temp);

}




int_type* CudaSet::op(int_type* column1, int_type* column2, string op_type, int reverse)
{

    thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr2(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<int_type>());
    }
    else  {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
    }

    return thrust::raw_pointer_cast(temp);

}

float_type* CudaSet::op(float_type* column1, float_type* column2, string op_type, int reverse)
{

    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<float_type> dev_ptr2(column2);
	
    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0) 
			thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<float_type>());			
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<float_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
    };
    return thrust::raw_pointer_cast(temp);
}

int_type* CudaSet::op(int_type* column1, int_type d, string op_type, int reverse)
{
    thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
    thrust::fill(temp, temp+mRecCount, d);

    thrust::device_ptr<int_type> dev_ptr1(column1);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<int_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
    };
    return thrust::raw_pointer_cast(temp);
}

float_type* CudaSet::op(int_type* column1, float_type d, string op_type, int reverse)
{
    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::fill(temp, temp+mRecCount, d);

    thrust::device_ptr<int_type> dev_ptr(column1);
    thrust::device_ptr<float_type> dev_ptr1 = thrust::device_malloc<float_type>(mRecCount);
    thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr1, long_to_float_type());

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
    }
    else  {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };
    thrust::device_free(dev_ptr1);
    return thrust::raw_pointer_cast(temp);
}


float_type* CudaSet::op(float_type* column1, float_type d, string op_type,int reverse)
{
    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr1(column1);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::divides<float_type>());
    }
    else	{
        if (op_type.compare("MUL") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };

    return thrust::raw_pointer_cast(temp);

}


void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs, char* file_name) // compressed data for DIM tables
{
    mColumnCount = nameRef.size();
    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];
    unsigned int cnt;
    file_p = NULL;
    FILE* f;
    char f1[100];
	
    prealloc_char_size = 0;
    not_compressed = 0;
    mRecCount = Recs;
	oldRecCount = Recs;
    load_file_name = file_name;
	
	strcpy(f1, file_name);
    strcat(f1, ".sort");
	cout << "opening " << f1 << endl;
    f = fopen (f1 , "rb" );
	if(f != NULL) {
		unsigned int sz, idx;
		fread((char *)&sz, 4, 1, f);
		for(unsigned int j = 0; j < sz; j++) {
			fread((char *)&idx, 4, 1, f);
			sorted_fields.push(idx);
			//cout << "presorted on " << idx << endl;
		};	
		fclose(f);
	};	
	
	tmp_table = 0;

    for(unsigned int i=0; i < mColumnCount; i++) {

        columnNames[nameRef.front()] = i;
        cols[i] = colsRef.front();
        seq = 0;

        strcpy(f1, file_name);
        strcat(f1,".");
        char col_pos[3];
        itoaa(colsRef.front(),col_pos);
        strcat(f1,col_pos); // read the size of a segment

        strcat(f1, ".header");
        f = fopen (f1 , "rb" );
        for(unsigned int j = 0; j < 5; j++)
            fread((char *)&cnt, 4, 1, f);
        fclose(f);
        //cout << "creating " << f1 << " " << cnt << endl;

        if ((typeRef.front()).compare("int") == 0) {
            type[i] = 0;
            decimal[i] = 0;
            h_columns_int.push_back(thrust::host_vector<int_type, pinned_allocator<int_type> >(cnt + 9));
            d_columns_int.push_back(thrust::device_vector<int_type>());
            type_index[i] = h_columns_int.size()-1;
        }
        else if ((typeRef.front()).compare("float") == 0) {
            type[i] = 1;
            decimal[i] = 0;
            h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >(cnt + 9));
            d_columns_float.push_back(thrust::device_vector<float_type >());
            type_index[i] = h_columns_float.size()-1;
        }
        else if ((typeRef.front()).compare("decimal") == 0) {
            type[i] = 1;
            decimal[i] = 1;
            h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >(cnt + 9));
            d_columns_float.push_back(thrust::device_vector<float_type>());
            type_index[i] = h_columns_float.size()-1;
        }
        else {
            type[i] = 2;
            decimal[i] = 0;
            h_columns_char.push_back(NULL);
            d_columns_char.push_back(NULL);
            char_size.push_back(sizeRef.front());
            type_index[i] = h_columns_char.size()-1;
        };

        nameRef.pop();
        typeRef.pop();
        sizeRef.pop();
        colsRef.pop();
    };
};



void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, int_type Recs)
{
    mColumnCount = nameRef.size();
    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];
	prealloc_char_size = 0;

    file_p = NULL;
	tmp_table = 0;

    mRecCount = Recs;
	oldRecCount = Recs;
    segCount = 1;

    for(unsigned int i=0; i < mColumnCount; i++) {

        columnNames[nameRef.front()] = i;
        cols[i] = colsRef.front();
        seq = 0;

        if ((typeRef.front()).compare("int") == 0) {
            type[i] = 0;
            decimal[i] = 0;
            h_columns_int.push_back(thrust::host_vector<int_type, pinned_allocator<int_type> >());
            d_columns_int.push_back(thrust::device_vector<int_type>());
            type_index[i] = h_columns_int.size()-1;
        }
        else if ((typeRef.front()).compare("float") == 0) {
            type[i] = 1;
            decimal[i] = 0;
            h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >());
            d_columns_float.push_back(thrust::device_vector<float_type>());
            type_index[i] = h_columns_float.size()-1;
        }
        else if ((typeRef.front()).compare("decimal") == 0) {
            type[i] = 1;
            decimal[i] = 1;
            h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >());
            d_columns_float.push_back(thrust::device_vector<float_type>());
            type_index[i] = h_columns_float.size()-1;
        }

        else {
            type[i] = 2;
            decimal[i] = 0;
            h_columns_char.push_back(NULL);
            d_columns_char.push_back(NULL);
            char_size.push_back(sizeRef.front());
            type_index[i] = h_columns_char.size()-1;
        };
        nameRef.pop();
        typeRef.pop();
        sizeRef.pop();
        colsRef.pop();
    };
};

void CudaSet::initialize(unsigned int RecordCount, unsigned int ColumnCount)
{
    mRecCount = RecordCount;
	oldRecCount = RecordCount;
    mColumnCount = ColumnCount;
	prealloc_char_size = 0;

    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];
    seq = 0;

    for(unsigned int i =0; i < mColumnCount; i++) {
        cols[i] = i;
    };


};


void CudaSet::initialize(queue<string> op_sel, queue<string> op_sel_as)
{
    mRecCount = 0;	
	mColumnCount = op_sel.size();

    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];
    
    seq = 0;    
    segCount = 1;
    not_compressed = 1;
    col_aliases = op_sel_as;
	prealloc_char_size = 0;

    unsigned int index;
	unsigned int i = 0;
    while(!op_sel.empty()) {
	
	    if(!setMap.count(op_sel.front())) {
			cout << "coudn't find column " << op_sel.front() << endl;
		    exit(0);
		};
		
		
        CudaSet* a = varNames[setMap[op_sel.front()]]; 
		
		if(i == 0)
		    maxRecs = a->maxRecs;

        index = a->columnNames[op_sel.front()];
        cols[i] = i;
        decimal[i] = a->decimal[i];
    	columnNames[op_sel.front()] = i;

        if (a->type[index] == 0)  {
            d_columns_int.push_back(thrust::device_vector<int_type>());
            h_columns_int.push_back(thrust::host_vector<int_type, pinned_allocator<int_type> >());
            type[i] = 0;
            type_index[i] = h_columns_int.size()-1;
        }
        else if ((a->type)[index] == 1) {
            d_columns_float.push_back(thrust::device_vector<float_type>());
            h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >());
            type[i] = 1;
            type_index[i] = h_columns_float.size()-1;
        }
        else {
            h_columns_char.push_back(NULL);
            d_columns_char.push_back(NULL);
            type[i] = 2;
            type_index[i] = h_columns_char.size()-1;
            char_size.push_back(a->char_size[a->type_index[index]]);
        };
	    i++;
        op_sel.pop();
	 };

}


void CudaSet::initialize(CudaSet* a, CudaSet* b, queue<string> op_sel, queue<string> op_sel_as)
{
    mRecCount = 0;
	
	mColumnCount = 0;
	queue<string> q_cnt(op_sel);
	unsigned int i = 0;
	set<string> field_names;
	while(!q_cnt.empty()) {
        if(a->columnNames.find(q_cnt.front()) !=  a->columnNames.end() || b->columnNames.find(q_cnt.front()) !=  b->columnNames.end())  {
			field_names.insert(q_cnt.front());
		};
        q_cnt.pop();
    }	
	mColumnCount = field_names.size();
	
    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];
    maxRecs = b->maxRecs;

    map<string,int>::iterator it;
    seq = 0;
    
    segCount = 1;
    not_compressed = 1;

    col_aliases = op_sel_as;
	prealloc_char_size = 0;

    unsigned int index;
	i = 0;
    while(!op_sel.empty() && (columnNames.find(op_sel.front()) ==  columnNames.end())) {

        if((it = a->columnNames.find(op_sel.front())) !=  a->columnNames.end()) {
            index = it->second;
            cols[i] = i;
            decimal[i] = a->decimal[i];
			columnNames[op_sel.front()] = i;
			
            if (a->type[index] == 0)  {
                d_columns_int.push_back(thrust::device_vector<int_type>());
                h_columns_int.push_back(thrust::host_vector<int_type, pinned_allocator<int_type> >());
                type[i] = 0;
                type_index[i] = h_columns_int.size()-1;
            }
            else if ((a->type)[index] == 1) {
                d_columns_float.push_back(thrust::device_vector<float_type>());
                h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >());				
                type[i] = 1;
                type_index[i] = h_columns_float.size()-1;
            }
            else {
                h_columns_char.push_back(NULL);
                d_columns_char.push_back(NULL);
                type[i] = 2;
                type_index[i] = h_columns_char.size()-1;
                char_size.push_back(a->char_size[a->type_index[index]]);                
            };
			i++;
        }
        else if((it = b->columnNames.find(op_sel.front())) !=  b->columnNames.end()) {
            index = it->second;
			columnNames[op_sel.front()] = i;			
            cols[i] = i;
            decimal[i] = b->decimal[index];

            if ((b->type)[index] == 0) {
                d_columns_int.push_back(thrust::device_vector<int_type>());
				h_columns_int.push_back(thrust::host_vector<int_type, pinned_allocator<int_type> >());
				type[i] = 0;
                type_index[i] = h_columns_int.size()-1;
            }
            else if ((b->type)[index] == 1) {
                d_columns_float.push_back(thrust::device_vector<float_type>());
				h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >());				
                type[i] = 1;
                type_index[i] = h_columns_float.size()-1;
            }
            else {
                h_columns_char.push_back(NULL);
                d_columns_char.push_back(NULL);
                type[i] = 2;
                type_index[i] = h_columns_char.size()-1;
                char_size.push_back(b->char_size[b->type_index[index]]);
            };
			i++;
        }		
        op_sel.pop();
	 };
};



int_type reverse_op(int_type op_type)
{
    if (op_type == 2) // >
        return 5;
    else if (op_type == 1)  // <
        return 6;
    else if (op_type == 6) // >=
        return 1;
    else if (op_type == 5)  // <=
        return 2;
    else return op_type;
}


size_t getFreeMem()
{
    size_t available, total;
    hipMemGetInfo(&available, &total);
    return available;
} ;



void allocColumns(CudaSet* a, queue<string> fields)
{
    if(!a->prm.empty()) {
        unsigned int max_sz = max_tmp(a) ;		
        CudaSet* t = varNames[setMap[fields.front()]];
        if(max_sz*t->maxRecs > alloced_sz) {
            if(alloced_sz) {
                hipFree(alloced_tmp);
            };
            hipMalloc((void **) &alloced_tmp, max_sz*t->maxRecs);
            alloced_sz = max_sz*t->maxRecs;
        }
    }
    else {

        while(!fields.empty()) {
            if(setMap.count(fields.front()) > 0) {

                unsigned int idx = a->columnNames[fields.front()];
                bool onDevice = 0;

                if(a->type[idx] == 0) {
                    if(a->d_columns_int[a->type_index[idx]].size() > 0) {
                        onDevice = 1;
                    }
                }
                else if(a->type[idx] == 1) {
                    if(a->d_columns_float[a->type_index[idx]].size() > 0) {
                        onDevice = 1;
                    };
                }
                else {
                    if((a->d_columns_char[a->type_index[idx]]) != NULL) {
                        onDevice = 1;
                    };
                };

                if (!onDevice) {
                    if(a->prm.empty()) {
                        a->allocColumnOnDevice(idx, a->maxRecs);
                    }
                    else {
                        a->allocColumnOnDevice(idx, largest_prm(a));
                    };
                }
            }
            fields.pop();
        };
    };
}


unsigned long long int largest_prm(CudaSet* a)
{
    unsigned long long int maxx = 0;

    for(unsigned int i = 0; i < a->prm_count.size(); i++)
        if(maxx < a->prm_count[i])
            maxx = a->prm_count[i];
    if(maxx == 0)
        maxx = a->maxRecs;
    return maxx;
};


void gatherColumns(CudaSet* a, CudaSet* t, string field, unsigned int segment, unsigned int& count)
{

    unsigned int tindex = t->columnNames[field];
    unsigned int idx = a->columnNames[field];

    //find the largest possible size of a gathered segment
    if(!a->onDevice(idx)) {
        unsigned int max_count = 0;

        for(unsigned int i = 0; i < a->prm.size(); i++)
            if (a->prm_count[i] > max_count)
                max_count = a->prm_count[i];
        a->allocColumnOnDevice(idx, max_count);
    };



    unsigned int g_size = a->prm_count[segment];

    if(a->prm_index[segment] == 'R') {

        if(a->prm_d.size() == 0) // find the largest prm segment
            a->prm_d.resize(largest_prm(a));

        if(curr_segment != segment) {
            hipMemcpy((void**)(thrust::raw_pointer_cast(a->prm_d.data())), (void**)a->prm[segment],
                       4*g_size, hipMemcpyHostToDevice);
            curr_segment = segment;
        };

        mygather(tindex, idx, a, t, count, g_size);
    }
    else {
        mycopy(tindex, idx, a, t, count, g_size);
    };

    a->mRecCount = g_size;
}

unsigned int getSegmentRecCount(CudaSet* a, unsigned int segment) {
    if (segment == a->segCount-1) {
        return oldCount - a->maxRecs*segment;
    }
    else
        return 	a->maxRecs;
}



void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, unsigned int& count)
{
    set<string> uniques;
    CudaSet *t;

    while(!fields.empty()) {
        if (uniques.count(fields.front()) == 0 && setMap.count(fields.front()) > 0)	{
            if(!a->prm.empty()) {
                t = varNames[setMap[fields.front()]];
                if(a->prm_count[segment]) {
                    alloced_switch = 1;
                    t->CopyColumnToGpu(t->columnNames[fields.front()], segment); 
                    gatherColumns(a, t, fields.front(), segment, count);
                    alloced_switch = 0;
                }
                else
                    a->mRecCount = 0;
            }
            else {
                a->CopyColumnToGpu(a->columnNames[fields.front()], segment); 
            };
            uniques.insert(fields.front());
        };
        fields.pop();
    };
}



void setPrm(CudaSet* a, CudaSet* b, char val, unsigned int segment) {

    b->prm.push_back(NULL);
    b->prm_index.push_back(val);

    if (val == 'A') {
        b->mRecCount = b->mRecCount + getSegmentRecCount(a,segment);
        b->prm_count.push_back(getSegmentRecCount(a, segment));
    }
    else {
        b->prm_count.push_back(0);
    };
}



void mygather(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, unsigned int offset, unsigned int g_size)
{
    if(t->type[tindex] == 0) {
        if(!alloced_switch) {
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           t->d_columns_int[t->type_index[tindex]].begin(), a->d_columns_int[a->type_index[idx]].begin() + offset);
        }
        else {
            thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           d_col, a->d_columns_int[a->type_index[idx]].begin() + offset);
        };
    }
    else if(t->type[tindex] == 1) {
        if(!alloced_switch) {
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           t->d_columns_float[t->type_index[tindex]].begin(), a->d_columns_float[a->type_index[idx]].begin() + offset);
        }
        else {
            thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           d_col, a->d_columns_float[a->type_index[idx]].begin() + offset);
        };
    }
    else {
        if(!alloced_switch) {	
		
            str_gather((void*)thrust::raw_pointer_cast(a->prm_d.data()), g_size,
                       (void*)t->d_columns_char[t->type_index[tindex]], (void*)(a->d_columns_char[a->type_index[idx]] + offset*a->char_size[a->type_index[idx]]), a->char_size[a->type_index[idx]] );
					   
        }
        else {
            str_gather((void*)thrust::raw_pointer_cast(a->prm_d.data()), g_size,
                       alloced_tmp, (void*)(a->d_columns_char[a->type_index[idx]] + offset*a->char_size[a->type_index[idx]]), a->char_size[a->type_index[idx]] );
        };
    }
};

void mycopy(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, unsigned int offset, unsigned int g_size)
{
    if(t->type[tindex] == 0) {
        if(!alloced_switch) {
            thrust::copy(t->d_columns_int[t->type_index[tindex]].begin(), t->d_columns_int[t->type_index[tindex]].begin() + g_size,
                         a->d_columns_int[a->type_index[idx]].begin() + offset);
        }
        else {
            thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
            thrust::copy(d_col, d_col + g_size, a->d_columns_int[a->type_index[idx]].begin() + offset);

        };
    }
    else if(t->type[tindex] == 1) {
        if(!alloced_switch) {
            thrust::copy(t->d_columns_float[t->type_index[tindex]].begin(), t->d_columns_float[t->type_index[tindex]].begin() + g_size,
                         a->d_columns_float[a->type_index[idx]].begin() + offset);
        }
        else {
            thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
            thrust::copy(d_col, d_col + g_size,	a->d_columns_float[a->type_index[idx]].begin() + offset);
        };
    }
    else {
        if(!alloced_switch) {
            hipMemcpy((void**)(a->d_columns_char[a->type_index[idx]] + offset*a->char_size[a->type_index[idx]]), (void**)t->d_columns_char[t->type_index[tindex]],
                       g_size*t->char_size[t->type_index[tindex]], hipMemcpyDeviceToDevice);
        }
        else {
            hipMemcpy((void**)(a->d_columns_char[a->type_index[idx]] + offset*a->char_size[a->type_index[idx]]), alloced_tmp,
                       g_size*t->char_size[t->type_index[tindex]], hipMemcpyDeviceToDevice);
        };
    };
};



unsigned int load_queue(queue<string> c1, CudaSet* right, bool str_join, string f2, unsigned int &rcount)
{
    queue<string> cc;
    while(!c1.empty()) {
        if(right->columnNames.find(c1.front()) !=  right->columnNames.end()) {
            if(f2 != c1.front() || str_join) {
                cc.push(c1.front());
            };
        };
        c1.pop();
    };
    if(!str_join && right->columnNames.find(f2) !=  right->columnNames.end()) {
        cc.push(f2);
    };

    unsigned int cnt_r = 0;
    if(!right->prm.empty()) {	
        allocColumns(right, cc);
        rcount = std::accumulate(right->prm_count.begin(), right->prm_count.end(), 0 );
    }
    else
        rcount = right->mRecCount;

    queue<string> ct(cc);
    reset_offsets();

    while(!ct.empty()) {
        right->allocColumnOnDevice(right->columnNames[ct.front()], rcount);
        ct.pop();
    };


    ct = cc;
    if(right->prm.empty()) {
        //copy all records
        while(!ct.empty()) {
            right->CopyColumnToGpu(right->columnNames[ct.front()]);
            ct.pop();
        };
        cnt_r = right->mRecCount;
    }
    else {
        //copy and gather all records
        for(unsigned int i = 0; i < right->segCount; i++) {
			reset_offsets();
            copyColumns(right, cc, i, cnt_r);
            cnt_r = cnt_r + right->prm_count[i];
        };
    };
    return cnt_r;

}

unsigned int max_char(CudaSet* a)
{
    unsigned int max_char = 0;
    for(unsigned int i = 0; i < a->char_size.size(); i++)
        if (a->char_size[i] > max_char)
            max_char = a->char_size[i];

    return max_char;
};

unsigned int max_char(CudaSet* a, set<string> field_names)
{
    unsigned int max_char = 0, i;
    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        i = a->columnNames[*it];	
		if (a->type[i] == 2) {
			if (a->char_size[a->type_index[i]] > max_char)
				max_char = a->char_size[a->type_index[i]];
		};
	};	
    return max_char;
};

unsigned int max_char(CudaSet* a, queue<string> field_names)
{
    unsigned int max_char = 0, i;
    while (!field_names.empty()) {
        i = a->columnNames[field_names.front()];	
		if (a->type[i] == 2) {
			if (a->char_size[a->type_index[i]] > max_char)
				max_char = a->char_size[a->type_index[i]];
		};
		field_names.pop();
	};	
    return max_char;
};



unsigned int max_tmp(CudaSet* a)
{
    unsigned int max_sz = 0;
    for(unsigned int i = 0; i < a->mColumnCount; i++) {
        if(a->type[i] == 0) {
            if(int_size > max_sz)
                max_sz = int_size;
        }
        else if(a->type[i] == 1) {
            if(float_size > max_sz)
                max_sz = float_size;
        };
    };
    unsigned int m_char = max_char(a);
    if(m_char > max_sz)
        return m_char;
    else
        return max_sz;

};


void reset_offsets() {
    map<unsigned int, unsigned int>::iterator iter;

    for (iter = str_offset.begin(); iter != str_offset.end(); ++iter) {
        iter->second = 0;
    };

};

void setSegments(CudaSet* a, queue<string> cols)
{
	size_t mem_available = getFreeMem();
	unsigned int tot_sz = 0, idx;
	while(!cols.empty()) {
	    idx = a->columnNames[cols.front()];
	    if(a->type[idx] != 2)
			tot_sz = tot_sz + int_size;
		else
            tot_sz = tot_sz + a->char_size[a->type_index[idx]];
        cols.pop();		
	};
	if(a->mRecCount*tot_sz > mem_available/3) { //default is 3
	    a->segCount = (a->mRecCount*tot_sz)/(mem_available/5) + 1;	
		a->maxRecs = (a->mRecCount/a->segCount)+1;
	};

};

void update_permutation_char(char* key, unsigned int* permutation, unsigned int RecCount, string SortType, char* tmp, unsigned int len)
{

    str_gather((void*)permutation, RecCount, (void*)key, (void*)tmp, len);

    // stable_sort the permuted keys and update the permutation
    if (SortType.compare("DESC") == 0 )
        str_sort(tmp, RecCount, permutation, 1, len);
    else
        str_sort(tmp, RecCount, permutation, 0, len);
}

void update_permutation_char_host(char* key, unsigned int* permutation, unsigned int RecCount, string SortType, char* tmp, unsigned int len)
{
    str_gather_host(permutation, RecCount, (void*)key, (void*)tmp, len);

    if (SortType.compare("DESC") == 0 )
        str_sort_host(tmp, RecCount, permutation, 1, len);
    else
        str_sort_host(tmp, RecCount, permutation, 0, len);

}



void apply_permutation_char(char* key, unsigned int* permutation, unsigned int RecCount, char* tmp, unsigned int len)
{
    // copy keys to temporary vector
    hipMemcpy( (void*)tmp, (void*) key, RecCount*len, hipMemcpyDeviceToDevice);
    // permute the keys
    str_gather((void*)permutation, RecCount, (void*)tmp, (void*)key, len);
}


void apply_permutation_char_host(char* key, unsigned int* permutation, unsigned int RecCount, char* res, unsigned int len)
{
    str_gather_host(permutation, RecCount, (void*)key, (void*)res, len);
}



