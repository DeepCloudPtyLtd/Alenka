#include "hip/hip_runtime.h"
/*
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */
 
 

#include <cctype>
#include <algorithm>
#include <functional>
#include <numeric>
#include "cm.h"
#include "atof.h"
#include "compress.cu"
#include "sorts.cu"
#include "filter.h"
#include "row.h"


#ifdef _WIN64
#define atoll(S) _atoi64(S)
#endif


using namespace std;
using namespace thrust::placeholders;


size_t total_count = 0, total_max;
std::clock_t tot;
unsigned int total_segments = 0;
unsigned int process_count;
size_t alloced_sz = 0;
bool fact_file_loaded = 1;
bool verbose;
void* d_v = NULL;
void* s_v = NULL;
queue<string> op_sort;
queue<string> op_presort;
queue<string> op_type;
bool op_case = 0;
queue<string> op_value;
queue<int_type> op_nums;
queue<float_type> op_nums_f;
queue<string> col_aliases;
map<string, map<string, col_data> > data_dict;

void* alloced_tmp;
bool alloced_switch = 0;

map<string,CudaSet*> varNames; //  STL map to manage CudaSet variables
map<string,string> setMap; //map to keep track of column names and set names


struct is_match
{
    __host__ __device__
    bool operator()(unsigned int x)
    {
        return x != 4294967295;
    }
};


struct f_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) < EPSILON) && ((x-y) > -EPSILON));
    }
};


struct f_less
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((y-x) > EPSILON);
    }
};

struct f_greater
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((x-y) > EPSILON);
    }
};

struct f_greater_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((x-y) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_less_equal
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return (((y-x) > EPSILON) || (((x-y) < EPSILON) && ((x-y) > -EPSILON)));
    }
};

struct f_not_equal_to
{
    __host__ __device__
    bool operator()(const float_type x, const float_type y)
    {
        return ((x-y) > EPSILON) || ((x-y) < -EPSILON);
    }
};


struct long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (float_type)x;
    }
};


struct l_to_ui
{
    __host__ __device__
    float_type operator()(const int_type x)
    {
        return (unsigned int)x;
    }
};


struct to_zero
{
    __host__ __device__
    bool operator()(const int_type x)
    {
        if(x == -1)
            return 0;
        else
            return 1;
    }
};



struct div_long_to_float_type
{
    __host__ __device__
    float_type operator()(const int_type x, const float_type y)
    {
        return (float_type)x/y;
    }
};



// trim from start
static inline std::string &ltrim(std::string &s) {
    s.erase(s.begin(), std::find_if(s.begin(), s.end(), std::not1(std::ptr_fun<int, int>(std::isspace))));
    return s;
}

// trim from end
static inline std::string &rtrim(std::string &s) {
    s.erase(std::find_if(s.rbegin(), s.rend(), std::not1(std::ptr_fun<int, int>(std::isspace))).base(), s.end());
    return s;
}

// trim from both ends
static inline std::string &trim(std::string &s) {
    return ltrim(rtrim(s));
}

char *mystrtok(char **m,char *s,const char c)
{
    char *p=s?s:*m;
    if( !*p )
        return 0;
    *m=strchr(p,c);
    if( *m )
        *(*m)++=0;
    else
        *m=p+strlen(p);
    return p;
}


void allocColumns(CudaSet* a, queue<string> fields);
void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, size_t& count, bool rsz, bool flt);
void mygather(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, size_t count, size_t g_size);
void mycopy(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, size_t count, size_t g_size);
void write_compressed_char(string file_name, unsigned int index, size_t mCount);
size_t max_tmp(CudaSet* a);
size_t getFreeMem();
char zone_map_check(queue<string> op_type, queue<string> op_value, queue<int_type> op_nums,queue<float_type> op_nums_f, CudaSet* a, unsigned int segment);
void filter_op(char *s, char *f, unsigned int segment);


CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, queue<string> &references, queue<string> &references_names)
    : mColumnCount(0), mRecCount(0)
{
    initialize(nameRef, typeRef, sizeRef, colsRef, Recs, references, references_names);
    keep = false;
    source = 1;
    text_source = 1;
    grp = NULL;
	fil_f = NULL;
	fil_s = NULL;
	grp_type = NULL;
};

CudaSet::CudaSet(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, string file_name, unsigned int max)
    : mColumnCount(0),  mRecCount(0)
{
	maxRecs = max;
    initialize(nameRef, typeRef, sizeRef, colsRef, Recs, file_name);
    keep = false;
    source = 1;
    text_source = 0;
    grp = NULL;
	fil_f = NULL;
	fil_s = NULL;
	grp_type = NULL;

};

CudaSet::CudaSet(size_t RecordCount, unsigned int ColumnCount)
{
    initialize(RecordCount, ColumnCount);
    keep = false;
    source = 0;
    text_source = 0;
    grp = NULL;
	fil_f = NULL;
	fil_s = NULL;
	grp_type = NULL;	
};

CudaSet::CudaSet(queue<string> op_sel, queue<string> op_sel_as)
{
    initialize(op_sel, op_sel_as);
    keep = false;
    source = 0;
    text_source = 0;
    grp = NULL;
	fil_f = NULL;
	fil_s = NULL;
	grp_type = NULL;	
};

CudaSet::CudaSet(CudaSet* a, CudaSet* b, queue<string> op_sel, queue<string> op_sel_as)
{
    initialize(a,b, op_sel, op_sel_as);
    keep = false;
    source = 0;
    text_source = 0;
    grp = NULL;
	fil_f = NULL;
	fil_s = NULL;
	grp_type = NULL;	
};


CudaSet::~CudaSet()
{
    free();
};


void CudaSet::allocColumnOnDevice(unsigned int colIndex, size_t RecordCount)
{
    if (type[colIndex] == 0) {
        d_columns_int[type_index[colIndex]].resize(RecordCount);
    }
    else if (type[colIndex] == 1)
        d_columns_float[type_index[colIndex]].resize(RecordCount);
    else {
        void* d;
        size_t sz = RecordCount*char_size[type_index[colIndex]];
        hipError_t cudaStatus = hipMalloc(&d, sz);
        if(cudaStatus != hipSuccess) {
            cout << "Could not allocate " << sz << " bytes of GPU memory for " << RecordCount << " records " << endl;
            exit(0);
        };
        d_columns_char[type_index[colIndex]] = (char*)d;
    };
};


void CudaSet::decompress_char_hash(string colname, unsigned int segment, size_t i_cnt)
{
    unsigned int bits_encoded, fit_count, sz, vals_count, real_count;
	unsigned int colIndex = columnNames[colname];
    size_t old_count;
    const unsigned int len = char_size[type_index[colIndex]];

    string f1 = load_file_name + "." + colname + "." + int_to_string(segment);

    FILE* f;
    f = fopen (f1.c_str() , "rb" );
    fread(&sz, 4, 1, f);
    char* d_array = new char[sz*len];
    fread((void*)d_array, sz*len, 1, f);

    unsigned long long int* hashes  = new unsigned long long int[sz];

    for(unsigned int i = 0; i < sz ; i++) {
        hashes[i] = MurmurHash64A(&d_array[i*len], len, hash_seed); // divide by 2 so it will fit into a signed long long
    };

    void* d;
    hipMalloc((void **) &d, sz*int_size);
    hipMemcpy( d, (void *) hashes, sz*8, hipMemcpyHostToDevice);

    thrust::device_ptr<unsigned long long int> dd_int((unsigned long long int*)d);
	
    delete[] d_array;
    delete[] hashes;

    fread(&fit_count, 4, 1, f);
    fread(&bits_encoded, 4, 1, f);
    fread(&vals_count, 4, 1, f);
    fread(&real_count, 4, 1, f);

    unsigned long long int* int_array = new unsigned long long int[vals_count];
    fread((void*)int_array, 1, vals_count*8, f);
    fclose(f);

    void* d_val;
    hipMalloc((void **) &d_val, vals_count*8);
    hipMemcpy(d_val, (void *) int_array, vals_count*8, hipMemcpyHostToDevice);

    thrust::device_ptr<unsigned long long int> mval((unsigned long long int*)d_val);
    delete[] int_array;
    void* d_int;
    hipMalloc((void **) &d_int, real_count*4);
	
    // convert bits to ints and then do gather

    void* d_v1;
    hipMalloc((void **) &d_v1, 8);
    thrust::device_ptr<unsigned int> dd_v((unsigned int*)d_v1);
	
    dd_v[1] = fit_count;
    dd_v[0] = bits_encoded;

    thrust::counting_iterator<unsigned int> begin(0);
    decompress_functor_str ff((unsigned long long int*)d_val,(unsigned int*)d_int, (unsigned int*)d_v1);
    thrust::for_each(begin, begin + real_count, ff);

    thrust::device_ptr<unsigned int> dd_val((unsigned int*)d_int);

    if(filtered) {
        if(prm_index == 'R') {
            thrust::device_ptr<int_type> d_tmp = thrust::device_malloc<int_type>(real_count);
            thrust::gather(dd_val, dd_val + real_count, dd_int, d_tmp);
            old_count = d_columns_int[i_cnt].size();
            d_columns_int[i_cnt].resize(old_count + mRecCount);
            thrust::gather(prm_d.begin(), prm_d.begin() + mRecCount, d_tmp, d_columns_int[i_cnt].begin() + old_count);
            thrust::device_free(d_tmp);

        }
        else if(prm_index == 'A') {
            old_count = d_columns_int[i_cnt].size();
            d_columns_int[i_cnt].resize(old_count + real_count);
            thrust::gather(dd_val, dd_val + real_count, dd_int, d_columns_int[i_cnt].begin() + old_count);
        }
    }
    else {
        old_count = d_columns_int[i_cnt].size();
        d_columns_int[i_cnt].resize(old_count + real_count);
        thrust::gather(dd_val, dd_val + real_count, dd_int, d_columns_int[i_cnt].begin() + old_count);
    };

    hipFree(d);
    hipFree(d_val);
    hipFree(d_v1);
    hipFree(d_int);	
};




// takes a char column , hashes strings, copies them to a gpu
void CudaSet::add_hashed_strings(string field, unsigned int segment, size_t i_cnt)
{
    unsigned int colInd2 = columnNames.find(field)->second;
    CudaSet *t = varNames[setMap[field]];

    if(not_compressed) { // decompressed strings on a host

        size_t old_count;
        unsigned long long int* hashes  = new unsigned long long int[t->mRecCount];

        for(unsigned int i = 0; i < t->mRecCount ; i++) {
            hashes[i] = MurmurHash64A(t->h_columns_char[t->type_index[colInd2]] + i*t->char_size[t->type_index[colInd2]] + segment*t->maxRecs*t->char_size[t->type_index[colInd2]], t->char_size[t->type_index[colInd2]], hash_seed);
        };

        if(filtered) {

            if(prm_index == 'R') {

                thrust::device_ptr<unsigned long long int> d_tmp = thrust::device_malloc<unsigned long long int>(t->mRecCount);
                thrust::copy(hashes, hashes+mRecCount, d_tmp);
                old_count = d_columns_int[i_cnt].size();
                d_columns_int[i_cnt].resize(old_count + mRecCount);
                thrust::gather(prm_d.begin(), prm_d.begin() + mRecCount, d_tmp, d_columns_int[i_cnt].begin() + old_count);
                thrust::device_free(d_tmp);

            }
            else if(prm_index == 'A') {
                old_count = d_columns_int[i_cnt].size();
                d_columns_int[i_cnt].resize(old_count + mRecCount);
                thrust::copy(hashes, hashes + mRecCount, d_columns_int[i_cnt].begin() + old_count);
            }
        }
        else {

            old_count = d_columns_int[i_cnt].size();
            d_columns_int[i_cnt].resize(old_count + mRecCount);
            thrust::copy(hashes, hashes + mRecCount, d_columns_int[i_cnt].begin() + old_count);
        }
        delete [] hashes;
    }
    else { // hash the dictionary
        decompress_char_hash(field, segment, i_cnt);
    };
};


void CudaSet::resize_join(size_t addRecs)
{
    mRecCount = mRecCount + addRecs;
    bool prealloc = 0;
    for(unsigned int i=0; i < mColumnCount; i++) {
        if(type[i] == 0) {
            h_columns_int[type_index[i]].resize(mRecCount);
        }
        else if(type[i] == 1) {
            h_columns_float[type_index[i]].resize(mRecCount);
        }
        else {
            if (h_columns_char[type_index[i]]) {
                if (mRecCount > prealloc_char_size) {
                    h_columns_char[type_index[i]] = (char*)realloc(h_columns_char[type_index[i]], mRecCount*char_size[type_index[i]]);
                    prealloc = 1;
                };
            }
            else {
                h_columns_char[type_index[i]] = new char[mRecCount*char_size[type_index[i]]];
            };
        };

    };
    if(prealloc)
        prealloc_char_size = mRecCount;
};


void CudaSet::resize(size_t addRecs)
{
    mRecCount = mRecCount + addRecs;
	for(unsigned int i=0; i <mColumnCount; i++) {
        if(type[i] == 0) {
            h_columns_int[type_index[i]].resize(mRecCount);
        }
        else if(type[i] == 1) {
            h_columns_float[type_index[i]].resize(mRecCount);
        }
        else {
            if (h_columns_char[type_index[i]]) {
                h_columns_char[type_index[i]] = (char*)realloc(h_columns_char[type_index[i]], mRecCount*char_size[type_index[i]]);
            }
            else {
                h_columns_char[type_index[i]] = new char[mRecCount*char_size[type_index[i]]];
            };
        };

    };
};

void CudaSet::reserve(size_t Recs)
{

    for(unsigned int i=0; i <mColumnCount; i++) {
        if(type[i] == 0)
            h_columns_int[type_index[i]].reserve(Recs);
        else if(type[i] == 1)
            h_columns_float[type_index[i]].reserve(Recs);
        else {
            h_columns_char[type_index[i]] = new char[Recs*char_size[type_index[i]]];
            if(h_columns_char[type_index[i]] == NULL) {
                cout << "Could not allocate on a host " << Recs << " records of size " << char_size[type_index[i]] << endl;
                exit(0);
            };
            prealloc_char_size = Recs;
        };

    };
};


void CudaSet::deAllocColumnOnDevice(unsigned int colIndex)
{
    if (type[colIndex] == 0 && !d_columns_int.empty()) {
		if(d_columns_int[type_index[colIndex]].size() > 0) {
			d_columns_int[type_index[colIndex]].resize(0);
			d_columns_int[type_index[colIndex]].shrink_to_fit();
		};	
    }
    else if (type[colIndex] == 1 && !d_columns_float.empty()) {
		if (d_columns_float[type_index[colIndex]].size() > 0) {
			d_columns_float[type_index[colIndex]].resize(0);
			d_columns_float[type_index[colIndex]].shrink_to_fit();
		};	
    }
    else if (type[colIndex] == 2 && d_columns_char[type_index[colIndex]] != NULL) {
        hipFree(d_columns_char[type_index[colIndex]]);
        d_columns_char[type_index[colIndex]] = NULL;
    };
};

void CudaSet::allocOnDevice(size_t RecordCount)
{
    for(unsigned int i=0; i < mColumnCount; i++)
        allocColumnOnDevice(i, RecordCount);
};

void CudaSet::deAllocOnDevice()
{
    for(unsigned int i=0; i < mColumnCount; i++)
        deAllocColumnOnDevice(i);
		
	for(unsigned int i=0; i < d_columns_int.size(); i++)	{
		if(d_columns_int[i].size() > 0) {
			d_columns_int[i].resize(0);
			d_columns_int[i].shrink_to_fit();
		};	
	};	

	for(unsigned int i=0; i < d_columns_float.size(); i++)	{
		if(d_columns_float[i].size() > 0) {
			d_columns_float[i].resize(0);
			d_columns_float[i].shrink_to_fit();
		};	
	};	
	

    if(grp) {
        hipFree(grp);
        grp = NULL;
    };

	
    if(filtered) { // free the sources
        string some_field;
        map<string,unsigned int>::iterator it=columnNames.begin();
        some_field = (*it).first;

        if(setMap[some_field].compare(name)) {
			if(varNames.find(setMap[some_field]) != varNames.end()) {
				CudaSet* t = varNames[setMap[some_field]];
				t->deAllocOnDevice();
			};	
        };
    };
};

void CudaSet::resizeDeviceColumn(size_t RecCount, unsigned int colIndex)
{
//   if (RecCount) {
    if (type[colIndex] == 0) {
        d_columns_int[type_index[colIndex]].resize(mRecCount+RecCount);
    }
    else if (type[colIndex] == 1)
        d_columns_float[type_index[colIndex]].resize(mRecCount+RecCount);
    else {
        if (d_columns_char[type_index[colIndex]] != NULL)
            hipFree(d_columns_char[type_index[colIndex]]);
        void *d;
        hipMalloc((void **) &d, (mRecCount+RecCount)*char_size[type_index[colIndex]]);
        d_columns_char[type_index[colIndex]] = (char*)d;
    };
//    };
};



void CudaSet::resizeDevice(size_t RecCount)
{
    //  if (RecCount)
    for(unsigned int i=0; i < mColumnCount; i++)
        resizeDeviceColumn(RecCount, i);
};

bool CudaSet::onDevice(unsigned int i)
{
    size_t j = type_index[i];

    if (type[i] == 0) {
        if (d_columns_int.empty())
            return 0;
        if (d_columns_int[j].size() == 0)
            return 0;
    }
    else if (type[i] == 1) {
        if (d_columns_float.empty())
            return 0;
        if(d_columns_float[j].size() == 0)
            return 0;
    }
    else if  (type[i] == 2) {
        if(d_columns_char.empty())
            return 0;
        if(d_columns_char[j] == NULL)
            return 0;
    };
    return 1;
}



CudaSet* CudaSet::copyDeviceStruct()
{

    CudaSet* a = new CudaSet(mRecCount, mColumnCount);
    a->not_compressed = not_compressed;
    a->segCount = segCount;
    a->maxRecs = maxRecs;
	a->ref_joins = ref_joins;
	a->ref_sets = ref_sets;
	a->ref_cols = ref_cols;

    for ( map<string,unsigned int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
        a->columnNames[(*it).first] = (*it).second;

    for(unsigned int i=0; i < mColumnCount; i++) {
        a->cols[i] = cols[i];
        a->type[i] = type[i];

        if(a->type[i] == 0) {
            a->d_columns_int.push_back(thrust::device_vector<int_type>());
            a->h_columns_int.push_back(thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >());
            a->type_index[i] = a->d_columns_int.size()-1;
        }
        else if(a->type[i] == 1) {
            a->d_columns_float.push_back(thrust::device_vector<float_type>());
            a->h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >());
            a->type_index[i] = a->d_columns_float.size()-1;
            a->decimal[i] = decimal[i];
        }
        else {
            a->h_columns_char.push_back(NULL);
            a->d_columns_char.push_back(NULL);
            a->type_index[i] = a->d_columns_char.size()-1;
            a->char_size.push_back(char_size[type_index[i]]);
        };
    };
    a->load_file_name = load_file_name;

    a->mRecCount = 0;
    return a;
}


void CudaSet::readSegmentsFromFile(unsigned int segNum, string colname, size_t offset)
{
	std::clock_t start1 = std::clock();	
    string f1(load_file_name);
	unsigned int colIndex = columnNames[colname];
    f1 += "." + colname + "." + int_to_string(segNum);
    unsigned int cnt;

    FILE* f;
    f = fopen(f1.c_str(), "rb" );
    if(f == NULL) {
        cout << "Error opening " << f1 << " file " << endl;
        exit(0);
    };
    size_t rr;	

    if(type[colIndex] == 0) {	    
		if(1 > h_columns_int[type_index[colIndex]].size())
			h_columns_int[type_index[colIndex]].resize(1);		
        fread(h_columns_int[type_index[colIndex]].data(), 4, 1, f);
        cnt = ((unsigned int*)(h_columns_int[type_index[colIndex]].data()))[0];		
		if(cnt > h_columns_int[type_index[colIndex]].size()/8 + 10)
			h_columns_int[type_index[colIndex]].resize(cnt/8 + 10);			
        rr = fread((unsigned int*)(h_columns_int[type_index[colIndex]].data()) + 1, 1, cnt+52, f);
        if(rr != cnt+52) {
            cout << "Couldn't read  " << cnt+52 << " bytes from " << f1  << " ,read only " << rr << endl;
            exit(0);
        };
    }
    else if(type[colIndex] == 1) {		
		if(1 > h_columns_float[type_index[colIndex]].size())
			h_columns_float[type_index[colIndex]].resize(1);		
        fread(h_columns_float[type_index[colIndex]].data(), 4, 1, f);
        cnt = ((unsigned int*)(h_columns_float[type_index[colIndex]].data()))[0];
		if(cnt > h_columns_float[type_index[colIndex]].size()/8 + 10)
			h_columns_float[type_index[colIndex]].resize(cnt/8 + 10);				
        rr = fread((unsigned int*)(h_columns_float[type_index[colIndex]].data()) + 1, 1, cnt+52, f);
        if(rr != cnt+52) {
            cout << "Couldn't read  " << cnt+52 << " bytes from " << f1  << endl;
            exit(0);
        };		
    }
    else {
        decompress_char(f, colIndex, segNum, offset);
    };
    fclose(f);
	tot = tot + (std::clock() - start1);
	//if(verbose)
	//	std::cout<< "read from file time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';	
};



void CudaSet::decompress_char(FILE* f, unsigned int colIndex, unsigned int segNum, size_t offset)
{
    unsigned int bits_encoded, fit_count, sz, vals_count, real_count;
    const unsigned int len = char_size[type_index[colIndex]];
	std::clock_t start1 = std::clock();	
	
    fread(&sz, 4, 1, f);
    char* d_array = new char[sz*len];
    fread((void*)d_array, sz*len, 1, f);
	tot = tot + (std::clock() - start1);
    void* d;
    hipMalloc((void **) &d, sz*len);
    hipMemcpy( d, (void *) d_array, sz*len, hipMemcpyHostToDevice);
    delete[] d_array;

	start1 = std::clock();	
    fread(&fit_count, 4, 1, f);
    fread(&bits_encoded, 4, 1, f);
    fread(&vals_count, 4, 1, f);
    fread(&real_count, 4, 1, f);
	tot = tot + (std::clock() - start1);

    thrust::device_ptr<unsigned int> param = thrust::device_malloc<unsigned int>(2);
    param[1] = fit_count;
    param[0] = bits_encoded;

    unsigned long long int* int_array = new unsigned long long int[vals_count];
    fread((void*)int_array, 1, vals_count*8, f);
	
    //fclose(f);

    void* d_val;
    hipMalloc((void **) &d_val, vals_count*8);
    hipMemcpy(d_val, (void *) int_array, vals_count*8, hipMemcpyHostToDevice);
    delete[] int_array;

    void* d_int;
    hipMalloc((void **) &d_int, real_count*4);

    thrust::counting_iterator<unsigned int> begin(0);
    decompress_functor_str ff((unsigned long long int*)d_val,(unsigned int*)d_int, (unsigned int*)thrust::raw_pointer_cast(param));
    thrust::for_each(begin, begin + real_count, ff);

    if(!alloced_switch)
		str_gather(d_int, real_count, d, d_columns_char[type_index[colIndex]] + offset*len, len);
    else
        str_gather(d_int, real_count, d, alloced_tmp, len);
    mRecCount = real_count;

    hipFree(d);
    hipFree(d_val);
    thrust::device_free(param);
    hipFree(d_int);	
}



void CudaSet::CopyColumnToGpu(string colname,  unsigned int segment, size_t offset)
{

	unsigned int colIndex = columnNames[colname];
    if(not_compressed) 	{
        // calculate how many records we need to copy
        if(segment < segCount-1) {
            mRecCount = maxRecs;
        }
        else {
            mRecCount = hostRecCount - maxRecs*(segCount-1);
        };

        switch(type[colIndex]) {
        case 0 :
            if(!alloced_switch)
                thrust::copy(h_columns_int[type_index[colIndex]].begin() + maxRecs*segment, h_columns_int[type_index[colIndex]].begin() + maxRecs*segment + mRecCount, d_columns_int[type_index[colIndex]].begin() + offset);
            else {
                thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
                thrust::copy(h_columns_int[type_index[colIndex]].begin() + maxRecs*segment, h_columns_int[type_index[colIndex]].begin() + maxRecs*segment + mRecCount, d_col);
            };
            break;
        case 1 :
            if(!alloced_switch) {
                thrust::copy(h_columns_float[type_index[colIndex]].begin() + maxRecs*segment, h_columns_float[type_index[colIndex]].begin() + maxRecs*segment + mRecCount, d_columns_float[type_index[colIndex]].begin() + offset);
            }
            else {
                thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
                thrust::copy(h_columns_float[type_index[colIndex]].begin() + maxRecs*segment, h_columns_float[type_index[colIndex]].begin() + maxRecs*segment + mRecCount, d_col);
            };
            break;
        default :
            if(!alloced_switch) {
                hipMemcpy(d_columns_char[type_index[colIndex]] + char_size[type_index[colIndex]]*offset, h_columns_char[type_index[colIndex]] + maxRecs*segment*char_size[type_index[colIndex]], char_size[type_index[colIndex]]*mRecCount, hipMemcpyHostToDevice);
            }
            else
                hipMemcpy(alloced_tmp , h_columns_char[type_index[colIndex]] + maxRecs*segment*char_size[type_index[colIndex]], char_size[type_index[colIndex]]*mRecCount, hipMemcpyHostToDevice);
        };
    }
    else {
	
        readSegmentsFromFile(segment,colname, offset);

        if(type[colIndex] != 2) {
            if(d_v == NULL)
                CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
            if(s_v == NULL)
                CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));
        };
		

        if(type[colIndex] == 0) {
            if(!alloced_switch) {
                mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data() + offset), h_columns_int[type_index[colIndex]].data(), d_v, s_v);
            }
            else {
                mRecCount = pfor_decompress(alloced_tmp, h_columns_int[type_index[colIndex]].data(), d_v, s_v);
            };
        }
        else if(type[colIndex] == 1) {
            if(decimal[colIndex]) {
                if(!alloced_switch) {
                    mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() + offset) , h_columns_float[type_index[colIndex]].data(), d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() + offset));
                    thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[type_index[colIndex]].begin(), long_to_float());
                }
                else {
                    mRecCount = pfor_decompress(alloced_tmp, h_columns_float[type_index[colIndex]].data(), d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)alloced_tmp);
                    thrust::device_ptr<float_type> d_col_float((float_type*)alloced_tmp);
                    thrust::transform(d_col_int,d_col_int+mRecCount, d_col_float, long_to_float());
                };
            }
            //else // uncompressed float
            //hipMemcpy( d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
            // will have to fix it later so uncompressed data will be written by segments too
        }
		
    };
}



void CudaSet::CopyColumnToGpu(string colname) // copy all segments
{
	unsigned int colIndex = columnNames[colname];
    if(not_compressed) {
        switch(type[colIndex]) {
        case 0 :
            thrust::copy(h_columns_int[type_index[colIndex]].begin(), h_columns_int[type_index[colIndex]].begin() + mRecCount, d_columns_int[type_index[colIndex]].begin());
            break;
        case 1 :
            thrust::copy(h_columns_float[type_index[colIndex]].begin(), h_columns_float[type_index[colIndex]].begin() + mRecCount, d_columns_float[type_index[colIndex]].begin());
            break;
        default :
            hipMemcpy(d_columns_char[type_index[colIndex]], h_columns_char[type_index[colIndex]], char_size[type_index[colIndex]]*mRecCount, hipMemcpyHostToDevice);
        };
    }
    else {
        size_t totals = 0;
        if(d_v == NULL)
            CUDA_SAFE_CALL(hipMalloc((void **) &d_v, 12));
        if(s_v == NULL)
            CUDA_SAFE_CALL(hipMalloc((void **) &s_v, 8));

		size_t cnt = 0;
        for(unsigned int i = 0; i < segCount; i++) {

            readSegmentsFromFile(i,colname, cnt);

            if(type[colIndex] == 0) {
                mRecCount = pfor_decompress(thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data() + totals), h_columns_int[type_index[colIndex]].data(), d_v, s_v);
            }
            else if(type[colIndex] == 1) {
                if(decimal[colIndex]) {
                    mRecCount = pfor_decompress( thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() + totals) , h_columns_float[type_index[colIndex]].data(), d_v, s_v);
                    thrust::device_ptr<long long int> d_col_int((long long int*)thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data() + totals));
                    thrust::transform(d_col_int,d_col_int+mRecCount,d_columns_float[type_index[colIndex]].begin() + totals, long_to_float());
                }
                // else  uncompressed float
                //hipMemcpy( d_columns[colIndex], (void *) ((float_type*)h_columns[colIndex] + offset), count*float_size, hipMemcpyHostToDevice);
                // will have to fix it later so uncompressed data will be written by segments too
            };
			cnt = cnt + mRecCount;

            //totalRecs = totals + mRecCount;
        };

        mRecCount = totals;
    };
}



void CudaSet::CopyColumnToHost(int colIndex, size_t offset, size_t RecCount)
{

    switch(type[colIndex]) {
    case 0 :
        thrust::copy(d_columns_int[type_index[colIndex]].begin(), d_columns_int[type_index[colIndex]].begin() + RecCount, h_columns_int[type_index[colIndex]].begin() + offset);
        break;
    case 1 :
        thrust::copy(d_columns_float[type_index[colIndex]].begin(), d_columns_float[type_index[colIndex]].begin() + RecCount, h_columns_float[type_index[colIndex]].begin() + offset);
        break;
    default :
        hipMemcpy(h_columns_char[type_index[colIndex]] + offset*char_size[type_index[colIndex]], d_columns_char[type_index[colIndex]], char_size[type_index[colIndex]]*RecCount, hipMemcpyDeviceToHost);
    }
}



void CudaSet::CopyColumnToHost(int colIndex)
{
    CopyColumnToHost(colIndex, 0, mRecCount);
}

void CudaSet::CopyToHost(size_t offset, size_t count)
{
    for(unsigned int i = 0; i < mColumnCount; i++) {
        CopyColumnToHost(i, offset, count);
    };
}

float_type* CudaSet::get_float_type_by_name(string name)
{
    unsigned int colIndex = columnNames.find(name)->second;
    return thrust::raw_pointer_cast(d_columns_float[type_index[colIndex]].data());
}

int_type* CudaSet::get_int_by_name(string name)
{
    unsigned int colIndex = columnNames.find(name)->second;
    return thrust::raw_pointer_cast(d_columns_int[type_index[colIndex]].data());
}

float_type* CudaSet::get_host_float_by_name(string name)
{
    unsigned int colIndex = columnNames.find(name)->second;
    return thrust::raw_pointer_cast(h_columns_float[type_index[colIndex]].data());
}

int_type* CudaSet::get_host_int_by_name(string name)
{
    unsigned int colIndex = columnNames.find(name)->second;
    return thrust::raw_pointer_cast(h_columns_int[type_index[colIndex]].data());
}



void CudaSet::GroupBy(stack<string> columnRef, unsigned int int_col_count)
{
    unsigned int colIndex;

    if(grp)
        hipFree(grp);

    CUDA_SAFE_CALL(hipMalloc((void **) &grp, mRecCount * sizeof(bool)));
    thrust::device_ptr<bool> d_grp(grp);

    thrust::sequence(d_grp, d_grp+mRecCount, 0, 0);

    thrust::device_ptr<bool> d_group = thrust::device_malloc<bool>(mRecCount);

    d_group[mRecCount-1] = 1;
    unsigned int i_count = 0;

    for(int i = 0; i < columnRef.size(); columnRef.pop()) {

        columnGroups.push(columnRef.top()); // save for future references
        colIndex = columnNames[columnRef.top()];


        if (type[colIndex] == 0) {  // int_type
            thrust::transform(d_columns_int[type_index[colIndex]].begin(), d_columns_int[type_index[colIndex]].begin() + mRecCount - 1,
                              d_columns_int[type_index[colIndex]].begin()+1, d_group, thrust::not_equal_to<int_type>());
        }
        else if (type[colIndex] == 1) {  // float_type
            thrust::transform(d_columns_float[type_index[colIndex]].begin(), d_columns_float[type_index[colIndex]].begin() + mRecCount - 1,
                              d_columns_float[type_index[colIndex]].begin()+1, d_group, f_not_equal_to());
        }
        else  {  // Char
            //str_grp(d_columns_char[type_index[colIndex]], mRecCount, d_group, char_size[type_index[colIndex]]);
            //use int_type

            thrust::transform(d_columns_int[int_col_count+i_count].begin(), d_columns_int[int_col_count+i_count].begin() + mRecCount - 1,
                              d_columns_int[int_col_count+i_count].begin()+1, d_group, thrust::not_equal_to<int_type>());
            i_count++;

        };
        thrust::transform(d_group, d_group+mRecCount, d_grp, d_grp, thrust::logical_or<bool>());

    };

    thrust::device_free(d_group);
    grp_count = thrust::count(d_grp, d_grp+mRecCount,1);
};


void CudaSet::addDeviceColumn(int_type* col, int colIndex, string colName, size_t recCount)
{
    if (columnNames.find(colName) == columnNames.end()) {
        columnNames[colName] = colIndex;
        type[colIndex] = 0;
        d_columns_int.push_back(thrust::device_vector<int_type>(recCount));
        h_columns_int.push_back(thrust::host_vector<int_type, uninitialized_host_allocator<int_type> >());
        type_index[colIndex] = d_columns_int.size()-1;
    }
    else {  // already exists, my need to resize it
        if(d_columns_int[type_index[colIndex]].size() < recCount) {
            d_columns_int[type_index[colIndex]].resize(recCount);
        };
    };
    // copy data to d columns
    thrust::device_ptr<int_type> d_col((int_type*)col);
    thrust::copy(d_col, d_col+recCount, d_columns_int[type_index[colIndex]].begin());
};

void CudaSet::addDeviceColumn(float_type* col, int colIndex, string colName, size_t recCount, bool is_decimal)
{
    if (columnNames.find(colName) == columnNames.end()) {
        columnNames[colName] = colIndex;
        type[colIndex] = 1;
        d_columns_float.push_back(thrust::device_vector<float_type>(recCount));
        h_columns_float.push_back(thrust::host_vector<float_type, uninitialized_host_allocator<float_type> >());
        type_index[colIndex] = d_columns_float.size()-1;
    }
    else {  // already exists, my need to resize it
        if(d_columns_float[type_index[colIndex]].size() < recCount)
            d_columns_float[type_index[colIndex]].resize(recCount);
    };

	decimal[colIndex] = is_decimal;
    thrust::device_ptr<float_type> d_col((float_type*)col);
    thrust::copy(d_col, d_col+recCount, d_columns_float[type_index[colIndex]].begin());
};

void CudaSet::compress(string file_name, size_t offset, unsigned int check_type, unsigned int check_val, size_t mCount)
{
    string str(file_name);
    thrust::device_vector<unsigned int> permutation;
	
	void* d;
	CUDA_SAFE_CALL(hipMalloc((void **) &d, mCount*float_size));	

    total_count = total_count + mCount;
    if (mCount > total_max && op_sort.empty()) {
        total_max = mCount;
	};	

    if(!op_sort.empty()) { //sort the segment
        //copy the key columns to device
        queue<string> sf(op_sort);

        permutation.resize(mRecCount);
        thrust::sequence(permutation.begin(), permutation.begin() + mRecCount,0,1);
        unsigned int* raw_ptr = thrust::raw_pointer_cast(permutation.data());
        void* temp;

        CUDA_SAFE_CALL(hipMalloc((void **) &temp, mRecCount*max_char(this, sf)));

        string sort_type = "ASC";

        while(!sf.empty()) {
            int colInd = columnNames[sf.front()];

            allocColumnOnDevice(colInd, maxRecs);
            CopyColumnToGpu(sf.front());

            if (type[colInd] == 0)
                update_permutation(d_columns_int[type_index[colInd]], raw_ptr, mRecCount, sort_type, (int_type*)temp);
            else if (type[colInd] == 1)
                update_permutation(d_columns_float[type_index[colInd]], raw_ptr, mRecCount, sort_type, (float_type*)temp);
            else {
                update_permutation_char(d_columns_char[type_index[colInd]], raw_ptr, mRecCount, sort_type, (char*)temp, char_size[type_index[colInd]]);
            };
            deAllocColumnOnDevice(colInd);
            sf.pop();
        };
        hipFree(temp);
    };

	// here we need to check for partitions and if partition_count > 0 -> create partitions
	if(mCount < partition_count || partition_count == 0)
		partition_count = 1;
	unsigned int partition_recs = mCount/partition_count;
	
	if(!op_sort.empty()) {
	    if(total_max < partition_recs)
			total_max = partition_recs;
	};	
	
	total_segments++;
	unsigned int old_segments = total_segments;
	size_t new_offset;
	//for(unsigned int i = 0; i< mColumnCount; i++) {
	unsigned int i;
    for (map<string,unsigned int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it ) {

		i = it->second;
		string colname = it->first;
		
		str = file_name + "." + colname;
		curr_file = str;
		str += "." + int_to_string(total_segments-1);
		new_offset = 0;

		if(!op_sort.empty()) {
			allocColumnOnDevice(i, maxRecs);
			CopyColumnToGpu(it->first);
		};		

		if(type[i] == 0) {
			thrust::device_ptr<int_type> d_col((int_type*)d);
			if(!op_sort.empty()) {
				thrust::gather(permutation.begin(), permutation.end(), d_columns_int[type_index[i]].begin(), d_col);
				
				for(unsigned int p = 0; p < partition_count; p++) {
					str = file_name + "." + colname;
					curr_file = str;
					str += "." + int_to_string(total_segments-1);
					if (p < partition_count - 1) {
						pfor_compress( (int_type*)d + new_offset, partition_recs*int_size, str, h_columns_int[type_index[i]], 0);
					}	
					else {	
						pfor_compress( (int_type*)d + new_offset, (mCount - partition_recs*p)*int_size, str, h_columns_int[type_index[i]], 0);
					};	
					new_offset = new_offset + partition_recs;
					total_segments++;	
				};
			}
			else {
				thrust::copy(h_columns_int[type_index[i]].begin() + offset, h_columns_int[type_index[i]].begin() + offset + mCount, d_col);
				pfor_compress( d, mCount*int_size, str, h_columns_int[type_index[i]], 0);
			};			
		}
		else if(type[i] == 1) {
			if(decimal[i]) {
				thrust::device_ptr<float_type> d_col((float_type*)d);
				if(!op_sort.empty()) {
					thrust::gather(permutation.begin(), permutation.end(), d_columns_float[type_index[i]].begin(), d_col);
					thrust::device_ptr<long long int> d_col_dec((long long int*)d);
					thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());
					
					for(unsigned int p = 0; p < partition_count; p++) {
						str = file_name + "." + colname;
						curr_file = str;
						str += "." + int_to_string(total_segments-1);
						if (p < partition_count - 1)
							pfor_compress( (int_type*)d + new_offset, partition_recs*float_size, str, h_columns_float[type_index[i]], 1);
						else	
							pfor_compress( (int_type*)d + new_offset, (mCount - partition_recs*p)*float_size, str, h_columns_float[type_index[i]], 1);
						new_offset = new_offset + partition_recs;
						total_segments++;	
					};					
				}
				else {
					thrust::copy(h_columns_float[type_index[i]].begin() + offset, h_columns_float[type_index[i]].begin() + offset + mCount, d_col);
					thrust::device_ptr<long long int> d_col_dec((long long int*)d);
					thrust::transform(d_col,d_col+mCount,d_col_dec, float_to_long());
					pfor_compress( d, mCount*float_size, str, h_columns_float[type_index[i]], 1);					
				};
			}
			else { // do not compress -- float
				thrust::device_ptr<float_type> d_col((float_type*)d);
				if(!op_sort.empty()) {
					thrust::gather(permutation.begin(), permutation.end(), d_columns_float[type_index[i]].begin(), d_col);
					thrust::copy(d_col, d_col+mRecCount, h_columns_float[type_index[i]].begin());
					for(unsigned int p = 0; p < partition_count; p++) {
						str = file_name + "." + colname;
						curr_file = str;
						str += "." + int_to_string(total_segments-1);
						unsigned int curr_cnt;
						if (p < partition_count - 1)
							curr_cnt = partition_recs;
						else
							curr_cnt = mCount - partition_recs*p;
					
						fstream binary_file(str.c_str(),ios::out|ios::binary|fstream::app);
						binary_file.write((char *)&curr_cnt, 4);
						binary_file.write((char *)(h_columns_float[type_index[i]].data() + new_offset),curr_cnt*float_size);
						new_offset = new_offset + partition_recs;
						unsigned int comp_type = 3;
						binary_file.write((char *)&comp_type, 4);
						binary_file.close();
					};					
				}
				else {				
					fstream binary_file(str.c_str(),ios::out|ios::binary|fstream::app);
					binary_file.write((char *)&mCount, 4);
					binary_file.write((char *)(h_columns_float[type_index[i]].data() + offset),mCount*float_size);
					unsigned int comp_type = 3;
					binary_file.write((char *)&comp_type, 4);
					binary_file.close();				
				};
			};
		}
		else { //char
			if(!op_sort.empty()) {
				unsigned int*  h_permutation = new unsigned int[mRecCount];
				thrust::copy(permutation.begin(), permutation.end(), h_permutation);
				char* t = new char[char_size[type_index[i]]*mRecCount];
				apply_permutation_char_host(h_columns_char[type_index[i]], h_permutation, mRecCount, t, char_size[type_index[i]]);
				delete [] h_permutation;
				thrust::copy(t, t+ char_size[type_index[i]]*mRecCount, h_columns_char[type_index[i]]);
				delete [] t;
				for(unsigned int p = 0; p < partition_count; p++) {		
					str = file_name + "." + colname;
					curr_file = str;
					str += "." + int_to_string(total_segments-1);
				
					if (p < partition_count - 1)
						compress_char(str, i, partition_recs, new_offset);
					else	
						compress_char(str, i, mCount - partition_recs*p, new_offset);
					new_offset = new_offset + partition_recs;
					total_segments++;	
				};	
			}
			else
				compress_char(str, i, mCount, offset);
		};
		
		
		if((check_type == 1 && fact_file_loaded) || (check_type == 1 && check_val == 0)) {
			if(!op_sort.empty())
				writeHeader(file_name, it->first, total_segments-1);
			else {
				writeHeader(file_name, it->first, total_segments);
			};
		};
		
		total_segments = old_segments;
    };
	hipFree(d);

	if(!op_sort.empty()) {
		total_segments = (old_segments-1)+partition_count;
	};	
    permutation.resize(0);
    permutation.shrink_to_fit();	
}


void CudaSet::writeHeader(string file_name, string colname, unsigned int tot_segs) {
    string str = file_name + "." + colname;
    string ff = str;
    str += ".header";
	
    fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
    binary_file.write((char *)&total_count, 8);
    binary_file.write((char *)&tot_segs, 4);
    binary_file.write((char *)&total_max, 4);
    binary_file.write((char *)&cnt_counts[ff], 4);
    binary_file.close();
};

void CudaSet::reWriteHeader(string file_name, string colname, unsigned int tot_segs, size_t newRecs, size_t maxRecs1) {
    string str = file_name + "." + colname;
    string ff = str;
    str += ".header";	
    fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
	binary_file.write((char *)&newRecs, 8);
    binary_file.write((char *)&tot_segs, 4);
	binary_file.write((char *)&maxRecs1, 4);
    binary_file.close();
};



void CudaSet::writeSortHeader(string file_name)
{
    string str(file_name);
    unsigned int idx;

    if(!op_sort.empty()) {
        str += ".sort";
        fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
        idx = (unsigned int)op_sort.size();
        binary_file.write((char *)&idx, 4);
        queue<string> os(op_sort);
        while(!os.empty()) {
            //idx = cols[columnNames[os.front()]];
			if(verbose)
				cout << "sorted on " << idx << endl;
			idx = os.front().size();	
            binary_file.write((char *)&idx, 4);
			binary_file.write(os.front().data(), idx);
            os.pop();
        };
        binary_file.close();
    }
    else if(!op_presort.empty()) {
        str += ".presort";
        fstream binary_file(str.c_str(),ios::out|ios::binary|ios::trunc);
        idx = (unsigned int)op_presort.size();
        binary_file.write((char *)&idx, 4);
        queue<string> os(op_presort);
        while(!os.empty()) {
            //idx = cols[columnNames[os.front()]];
			idx = os.front().size();	
            binary_file.write((char *)&idx, 4);
			binary_file.write(os.front().data(), idx);
            os.pop();
        };
        binary_file.close();
    };
}

using namespace mgpu;

void CudaSet::Display(unsigned int limit, bool binary, bool term)
{
        #define MAXCOLS 128
        #define MAXFIELDSIZE 128        

        //-- This should/will be converted to an array holding pointers of malloced sized structures--
        char    bigbuf[MAXCOLS * MAXFIELDSIZE];
        char    *fields[MAXCOLS];
        const   char *dcolumns[MAXCOLS];
        size_t  mCount;         // num records in play
        bool    print_all = 0;
        string  ss;
	int rows = 0;

        if(limit != 0 && limit < mRecCount)
            mCount = limit;
        else {
            mCount = mRecCount;
            print_all = 1;
        };

        cout << "mRecCount=" << mRecCount << " mcount = " << mCount << " term " << term <<  " limit=" << limit << " print_all=" << print_all << endl;

        map<unsigned int, string> ordered_columnNames;
        for (map<string,unsigned int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
                ordered_columnNames[it->second] = it->first;

        unsigned int cc =0;
        for (map<unsigned int, string>::iterator it=ordered_columnNames.begin() ; it != ordered_columnNames.end(); ++it )
        {
                fields[cc] = &(bigbuf[cc*MAXFIELDSIZE]);                        // a hack to avoid malloc overheads     - refine later
                dcolumns[cc++] = it->second.c_str();
        }

     // The goal here is to loop fast and avoid any double handling of outgoing data - pointers are good.
        if(not_compressed && prm_d.size() == 0) {
            for(unsigned int i=0; i < mCount; i++) {                            // for each record
                  for(unsigned int j=0; j < mColumnCount; j++) {                // for each col
                    if (type[j] == 0)
                        sprintf(fields[j], "%lld", (h_columns_int[type_index[j]])[i] );
                    else if (type[j] == 1)
                        sprintf(fields[j], "%.2f", (h_columns_float[type_index[j]])[i] );
                    else {
						strncpy(fields[j], h_columns_char[type_index[j]] + (i*char_size[type_index[j]]), char_size[type_index[j]]);
                        //ss.assign(h_columns_char[type_index[j]] + (i*char_size[type_index[j]]), char_size[type_index[j]]);
                        //fields[j] = (char *) ss.c_str();
                    };
                  };
                  row_cb(NULL, mColumnCount, (char **)fields, (char **)dcolumns);
                  rows++;
            };
        }
        else {
                        queue<string> op_vx;
                        for (map<string,unsigned int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
                                op_vx.push((*it).first);
                        //curr_segment = 1000000;

                        if(prm_d.size() || source) {
                                allocColumns(this, op_vx);
                        };
                        unsigned int curr_seg = 0;
                        size_t cnt = 0;
                        size_t curr_count, sum_printed = 0;
                        resize(maxRecs);
                        while(sum_printed < mCount || print_all) {

                                if(prm_d.size() || source)  {                            // if host arrays are empty
                                        copyColumns(this, op_vx, curr_seg, cnt);
                                        size_t olRecs = mRecCount;
                                        mRecCount = olRecs;
                                        CopyToHost(0,mRecCount);
                                        if(sum_printed + mRecCount <= mCount || print_all)
                                                curr_count = mRecCount;
                                        else
                                                curr_count = mCount - sum_printed;
                                }
                                else
                                        curr_count = mCount;

                                sum_printed = sum_printed + mRecCount;
                                for(unsigned int i=0; i < curr_count; i++) {
                                        for(unsigned int j=0; j < mColumnCount; j++) {
                                                if (type[j] == 0)
                                                        sprintf(fields[j], "%lld", (h_columns_int[type_index[j]])[i] );
                                                else if (type[j] == 1)
                                                        sprintf(fields[j], "%.2f", (h_columns_float[type_index[j]])[i] );
                                                else {
                                                        ss.assign(h_columns_char[type_index[j]] + (i*char_size[type_index[j]]), char_size[type_index[j]]);
                                                        fields[j] = (char *) ss.c_str();
                                                };
                                        };
                                        row_cb(NULL, mColumnCount, (char **)fields, (char**)dcolumns);
                                        rows++;
                                };
                                curr_seg++;
                                if(curr_seg == segCount)
                                        print_all = 0;
                        };
        };      // end else
}
 
void CudaSet::Store(string file_name, char* sep, unsigned int limit, bool binary, bool term)
{
    if (mRecCount == 0 && binary == 1 && !term) { // write tails
        //for(unsigned int i = 0; i< mColumnCount; i++) {
		for (map<string,unsigned int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it ) {
            writeHeader(file_name, it->first, total_segments);
        };
        return;
    };
	
    size_t mCount;
	bool print_all = 0;
	
    if(limit != 0 && limit < mRecCount)
        mCount = limit;
    else {
        mCount = mRecCount;
		print_all = 1;
	};	
	//cout << "mCount " << mCount << " " << mRecCount << endl;

    if(binary == 0) {
	
		FILE *file_pr;
		if(!term) {
			file_pr = fopen(file_name.c_str(), "w");
			if (file_pr  == NULL)
				cout << "Could not open file " << file_name << endl;
		}
		else
			file_pr = stdout;
			
		char buffer [33];			
		string ss;
		
		if(not_compressed && prm_d.size() == 0) {
            for(unsigned int i=0; i < mCount; i++) {
                for(unsigned int j=0; j < mColumnCount; j++) {
                    if (type[j] == 0) {
                        sprintf(buffer, "%lld", (h_columns_int[type_index[j]])[i] );
                        fputs(buffer,file_pr);
                        fputs(sep, file_pr);
                    }
                    else if (type[j] == 1) {
                        sprintf(buffer, "%.2f", (h_columns_float[type_index[j]])[i] );
                        fputs(buffer,file_pr);
                        fputs(sep, file_pr);
                    }
                    else {
                        ss.assign(h_columns_char[type_index[j]] + (i*char_size[type_index[j]]), char_size[type_index[j]]);
                        trim(ss);
                        fputs(ss.c_str(), file_pr);
                        fputs(sep, file_pr);
                    };
                };
                if (i != mCount -1 )
                    fputs("\n",file_pr);
            };		
			if(!term)
				fclose(file_pr);
		}
		else {

			queue<string> op_vx;
			for (map<string,unsigned int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it )
				op_vx.push((*it).first);

			if(prm_d.size() || source) {
				allocColumns(this, op_vx);
			};	
			
			unsigned int curr_seg = 0;
			size_t cnt = 0;
			size_t curr_count, sum_printed = 0;
			mRecCount = 0;
			resize(maxRecs);	
			
			while(sum_printed < mCount || print_all) {

				if(prm_d.size() || source)  {
					copyColumns(this, op_vx, curr_seg, cnt);
					if(curr_seg == 0) {
						if(limit != 0 && limit < mRecCount) {
							mCount = limit;
							print_all = 0;
						}	
						else {
							mCount = mRecCount;
							print_all = 1;
						};	
					
					};
					
                // if host arrays are empty
					size_t olRecs = mRecCount;                
					mRecCount = olRecs;
					CopyToHost(0,mRecCount);
					//cout << "start " << sum_printed << " " <<  mRecCount << " " <<  mCount << endl;
					if(sum_printed + mRecCount <= mCount || print_all) {
						curr_count = mRecCount;
					}	
					else {
						curr_count = mCount - sum_printed;
					};
				}
				else {
					curr_count = mCount;
				};	

				sum_printed = sum_printed + mRecCount;            
				//cout << "sum printed " << sum_printed << " " << curr_count << " " << curr_seg << endl;
			
				for(unsigned int i=0; i < curr_count; i++) {				
					for(unsigned int j=0; j < mColumnCount; j++) {
						if (type[j] == 0) {
							sprintf(buffer, "%lld", (h_columns_int[type_index[j]])[i] );
							fputs(buffer,file_pr);
							fputs(sep, file_pr);
						}
						else if (type[j] == 1) {
							sprintf(buffer, "%.2f", (h_columns_float[type_index[j]])[i] );
							fputs(buffer,file_pr);
							fputs(sep, file_pr);
						}
						else {
							ss.assign(h_columns_char[type_index[j]] + (i*char_size[type_index[j]]), char_size[type_index[j]]);
							trim(ss);
							fputs(ss.c_str(), file_pr);
							fputs(sep, file_pr);
						};
					};
					if (i != mCount -1 && (curr_seg != segCount || i < curr_count))
						fputs("\n",file_pr);
				};
				curr_seg++;
				if(curr_seg == segCount)
					print_all = 0;
			};
			if(!term) {
				fclose(file_pr);
			};	
		};		
    }
    else {
	
		//lets update the data dictionary
		for (map<string,unsigned int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it ) {
	
			if(decimal[(*it).second] == 1)
				data_dict[file_name][(*it).first].col_type = 3;
			else	
				data_dict[file_name][(*it).first].col_type = type[(*it).second]; 
			if(type[(*it).second] != 2)
				data_dict[file_name][(*it).first].col_length = 0;
			else	
				data_dict[file_name][(*it).first].col_length = char_size[type_index[(*it).second]];
		};		

	
		if(text_source) {  //writing a binary file using a text file as a source

			// time to perform join checks on REFERENCES dataset segments	
			//for(unsigned int i = 0; i< mColumnCount; i++) {
			for (map<string,unsigned int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it ) {
				unsigned int i = columnNames[it->first];
				if(ref_sets.find(i) != ref_sets.end()) {
				
					string f1 = file_name + "." + it->first + ".refs";
					fstream f_file;
					if(total_segments == 0) {
						f_file.open(f1.c_str(), ios::out|ios::trunc|ios::binary);
						unsigned int len = ref_sets[i].size();
						f_file.write((char *)&len, 4);
						f_file.write(ref_sets[i].c_str(), len);
						len = ref_cols[i].size();
						f_file.write((char *)&len, 4);
						f_file.write(ref_cols[i].c_str(), len);
					}	
					else {	
						f_file.open(f1.c_str(), ios::out|ios::app|ios::binary);
					};					
				
					f1 = ref_sets[i] + "." + ref_cols[i] + ".header";
					FILE* ff = fopen(f1.c_str(), "rb");
					if(ff == NULL) {
						cout << "Couldn't open file " << f1 << endl;
						exit(0);
					};
					unsigned int ref_segCount, ref_maxRecs;
					fread((char *)&ref_segCount, 4, 1, ff);
					fread((char *)&ref_segCount, 4, 1, ff);
					fread((char *)&ref_segCount, 4, 1, ff);
					fread((char *)&ref_maxRecs, 4, 1, ff);
					fclose(ff);				
					//cout << "CALC " << i << " " << ref_sets[i] << " " << ref_cols[i] << " " << ref_segCount << endl;
				
					CudaSet* a = new CudaSet(maxRecs, 1);
					a->h_columns_int.push_back(thrust::host_vector<int_type, pinned_allocator<int_type> >());
					a->d_columns_int.push_back(thrust::device_vector<int_type>(ref_maxRecs));
					a->type[0] = 0;
					a->type_index[0] = 0;
					a->not_compressed = 0;
					a->load_file_name = ref_sets[i];
					a->cols[0] = 1;
					a->columnNames[ref_cols[i]] = 0;
					MGPU_MEM(int) aIndicesDevice, bIndicesDevice;
					size_t res_count;
				
					if(!onDevice(i)) {
						allocColumnOnDevice(i, maxRecs);					
					};	
					CopyColumnToGpu(it->first);
					thrust::sort(d_columns_int[type_index[i]].begin(), d_columns_int[type_index[i]].begin() + mRecCount);				

					f_file.write((char *)&total_segments, 4);					
					f_file.write((char *)&ref_segCount, 4);				
					for(unsigned int z = 0; z < ref_segCount; z++) {

						a->CopyColumnToGpu(ref_cols[i], z, 0);
						thrust::sort(a->d_columns_int[0].begin(), a->d_columns_int[0].begin() + a->mRecCount);
						// check if there is a join result
						//cout << "join " << mRecCount << " " << a->mRecCount << endl;					
				
						res_count = RelationalJoin<MgpuJoinKindInner>(thrust::raw_pointer_cast(d_columns_int[type_index[i]].data()), mRecCount,
									thrust::raw_pointer_cast(a->d_columns_int[0].data()), a->mRecCount,
									&aIndicesDevice, &bIndicesDevice,
									mgpu::less<int_type>(), *context);
					//cout << "RES " << i << " " << total_segments << ":" << z << " " << res_count << endl;			
						f_file.write((char *)&z, 4);
						f_file.write((char *)&res_count, 8);
					};
					f_file.close();
					a->deAllocColumnOnDevice(0);
					a->free();				
				};
			};	
			compress(file_name, 0, 1, 0, mCount);		
			for(unsigned int i = 0; i< mColumnCount; i++)
				if(type[i] == 2)
					deAllocColumnOnDevice(i);
		}
		else { //writing a binary file using a binary file as a source
			fact_file_loaded = 1;
			size_t offset = 0;

			if(!not_compressed) { // records are compressed, for example after filter op.
				//decompress to host
				queue<string> op_vx;
				for (map<string,unsigned int>::iterator it=columnNames.begin() ; it != columnNames.end(); ++it ) {
					op_vx.push((*it).first);
				};

				allocColumns(this, op_vx);
				size_t oldCnt = mRecCount;
				mRecCount = 0;
				resize(oldCnt);
				mRecCount = oldCnt;
				for(unsigned int i = 0; i < segCount; i++) {
					size_t cnt = 0;
					copyColumns(this, op_vx, i, cnt);
					CopyToHost(0, mRecCount);
					offset = offset + mRecCount;
					compress(file_name, 0, 0, i - (segCount-1), mRecCount);
				};
			}
			else {
				// now we have decompressed records on the host
				//call setSegments and compress columns in every segment

				segCount = (mRecCount/process_count + 1);
				offset = 0;

				for(unsigned int z = 0; z < segCount; z++) {

					if(z < segCount-1) {
						if(mRecCount < process_count) {
							mCount = mRecCount;
						}
						else {
							mCount = process_count;
						}
					}
					else {
						mCount = mRecCount - (segCount-1)*process_count;
					};	
					compress(file_name, offset, 0, z - (segCount-1), mCount);
					offset = offset + mCount;
				};
			};
		};
	};
}


void CudaSet::compress_char(string file_name, unsigned int index, size_t mCount, size_t offset)
{
    std::map<string,unsigned int> dict;
    std::vector<string> dict_ordered;
    std::vector<unsigned int> dict_val;
    map<string,unsigned int>::iterator iter;
    unsigned int bits_encoded, ss;    
    unsigned int len = char_size[type_index[index]];

    for (unsigned int i = 0 ; i < mCount; i++) {

		string f(h_columns_char[type_index[index]] + (i+offset)*len, len);
		
        if((iter = dict.find(f)) != dict.end()) {
            dict_val.push_back(iter->second);
        }
        else {            
			ss = (unsigned int)dict.size();
            dict[f] = ss;
            dict_val.push_back(ss);
            dict_ordered.push_back(f);
        };
    };

    bits_encoded = (unsigned int)ceil(log2(double(dict.size()+1)));

    char *cc = new char[len+1];
    unsigned int sz = (unsigned int)dict_ordered.size();
    // write to a file
    fstream binary_file(file_name.c_str(),ios::out|ios::binary);
    binary_file.write((char *)&sz, 4);
    for(unsigned int i = 0; i < dict_ordered.size(); i++) {
        memset(&cc[0], 0, len);
        strcpy(cc,dict_ordered[i].c_str());
        binary_file.write(cc, len);
    };

    delete [] cc;
    unsigned int fit_count = 64/bits_encoded;
    unsigned long long int val = 0;
    binary_file.write((char *)&fit_count, 4);
    binary_file.write((char *)&bits_encoded, 4);
    unsigned int curr_cnt = 1;
    unsigned int vals_count = (unsigned int)dict_val.size()/fit_count;
    if(!vals_count || dict_val.size()%fit_count)
        vals_count++;
    binary_file.write((char *)&vals_count, 4);
    unsigned int real_count = (unsigned int)dict_val.size();
    binary_file.write((char *)&real_count, 4);

    for(unsigned int i = 0; i < dict_val.size(); i++) {

        val = val | dict_val[i];
		
        if(curr_cnt < fit_count)
            val = val << bits_encoded;

        if( (curr_cnt == fit_count) || (i == (dict_val.size() - 1)) ) {
            if (curr_cnt < fit_count) {
                val = val << ((fit_count-curr_cnt)-1)*bits_encoded;
            };
            curr_cnt = 1;
            binary_file.write((char *)&val, 8);
            val = 0;
        }
        else
            curr_cnt = curr_cnt + 1;
    };
    binary_file.close();
};



bool CudaSet::LoadBigFile(FILE* file_p)
{
    char line[1000];
    unsigned int current_column, count = 0, index;
    char *p,*t;
	const char* sep = separator.c_str();

    map<unsigned int,unsigned int> col_map;
    for(unsigned int i = 0; i < mColumnCount; i++) {
        col_map[cols[i]] = i;
    };	

    while (count < process_count && fgets(line, 1000, file_p) != NULL) {
        strtok(line, "\n");
        current_column = 0;

        for(t=mystrtok(&p,line,*sep); t; t=mystrtok(&p,0,*sep)) {
            current_column++;
            if(col_map.find(current_column) == col_map.end()) {
                continue;
            };

            index = col_map[current_column];
            if (type[index] == 0) {
                if (strchr(t,'-') == NULL) {
                    (h_columns_int[type_index[index]])[count] = atoll(t);
                }
                else {   // handling possible dates
                    strncpy(t+4,t+5,2);
                    strncpy(t+6,t+8,2);
                    t[8] = '\0';
                    (h_columns_int[type_index[index]])[count] = atoll(t);
                };
            }
            else if (type[index] == 1) {
                (h_columns_float[type_index[index]])[count] = atoff(t);
            }
            else  {//char
                strcpy(h_columns_char[type_index[index]] + count*char_size[type_index[index]], t);
            }
        };
        count++;
    };

    mRecCount = count;

    if(count < process_count)  {
        fclose(file_p);
        return 1;
    }
    else
        return 0;
};


void CudaSet::free()  {
	
    for(unsigned int i = 0; i < mColumnCount; i++ ) {
		if(type[i] == 2 && h_columns_char[type_index[i]]) {
            delete [] h_columns_char[type_index[i]];
            h_columns_char[type_index[i]] = NULL;
        }
        else {
            if(type[i] == 0 ) {
                h_columns_int[type_index[i]].resize(0);
                h_columns_int[type_index[i]].shrink_to_fit();
            }
            else if(type[i] == 1) {
                h_columns_float[type_index[i]].resize(0);
                h_columns_float[type_index[i]].shrink_to_fit();
            };			
        }
    };
	
	prm_d.resize(0);
	prm_d.shrink_to_fit();
	deAllocOnDevice();
	
    delete type;
	delete decimal;
	if(grp_type)
		delete grp_type;
    delete cols;
	if(fil_s)
		delete fil_s;
	if(fil_f)	
		delete fil_f;

};


bool* CudaSet::logical_and(bool* column1, bool* column2)
{
    thrust::device_ptr<bool> dev_ptr1(column1);
    thrust::device_ptr<bool> dev_ptr2(column2);

    thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_and<bool>());

    thrust::device_free(dev_ptr2);
    return column1;
}


bool* CudaSet::logical_or(bool* column1, bool* column2)
{

    thrust::device_ptr<bool> dev_ptr1(column1);
    thrust::device_ptr<bool> dev_ptr2(column2);

    thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, dev_ptr1, thrust::logical_or<bool>());
    thrust::device_free(dev_ptr2);
    return column1;
}



bool* CudaSet::compare(int_type s, int_type d, int_type op_type)
{
    bool res;

    if (op_type == 2) // >
        if(d>s) res = 1;
        else res = 0;
    else if (op_type == 1)  // <
        if(d<s) res = 1;
        else res = 0;
    else if (op_type == 6) // >=
        if(d>=s) res = 1;
        else res = 0;
    else if (op_type == 5)  // <=
        if(d<=s) res = 1;
        else res = 0;
    else if (op_type == 4)// =
        if(d==s) res = 1;
        else res = 0;
    else // !=
        if(d!=s) res = 1;
        else res = 0;

    thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
    thrust::sequence(p, p+mRecCount,res,(bool)0);

    return thrust::raw_pointer_cast(p);
};


bool* CudaSet::compare(float_type s, float_type d, int_type op_type)
{
    bool res;

    if (op_type == 2) // >
        if ((d-s) > EPSILON) res = 1;
        else res = 0;
    else if (op_type == 1)  // <
        if ((s-d) > EPSILON) res = 1;
        else res = 0;
    else if (op_type == 6) // >=
        if (((d-s) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
    else if (op_type == 5)  // <=
        if (((s-d) > EPSILON) || (((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;
    else if (op_type == 4)// =
        if (((d-s) < EPSILON) && ((d-s) > -EPSILON)) res = 1;
        else res = 0;
    else // !=
        if (!(((d-s) < EPSILON) && ((d-s) > -EPSILON))) res = 1;
        else res = 0;

    thrust::device_ptr<bool> p = thrust::device_malloc<bool>(mRecCount);
    thrust::sequence(p, p+mRecCount,res,(bool)0);

    return thrust::raw_pointer_cast(p);
}


bool* CudaSet::compare(int_type* column1, int_type d, int_type op_type)
{
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr(column1);


    if (op_type == 2) // >
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater<int_type>());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less<int_type>());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::greater_equal<int_type>());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::less_equal<int_type>());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::equal_to<int_type>());
    else // !=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), temp, thrust::not_equal_to<int_type>());

    return thrust::raw_pointer_cast(temp);

}

bool* CudaSet::compare(float_type* column1, float_type d, int_type op_type)
{
    thrust::device_ptr<bool> res = thrust::device_malloc<bool>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr(column1);
	
    if (op_type == 2) // >
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_equal_to());
    else  // != 
        thrust::transform(dev_ptr, dev_ptr+mRecCount, thrust::make_constant_iterator(d), res, f_not_equal_to());

    return thrust::raw_pointer_cast(res);	
}


bool* CudaSet::compare(int_type* column1, int_type* column2, int_type op_type)
{
    thrust::device_ptr<int_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr2(column2);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater<int_type>());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less<int_type>());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::greater_equal<int_type>());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::less_equal<int_type>());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::equal_to<int_type>());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::not_equal_to<int_type>());

    return thrust::raw_pointer_cast(temp);
}

bool* CudaSet::compare(float_type* column1, float_type* column2, int_type op_type)
{
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<float_type> dev_ptr2(column2);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

    return thrust::raw_pointer_cast(temp);

}


bool* CudaSet::compare(float_type* column1, int_type* column2, int_type op_type)
{
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr(column2);
    thrust::device_ptr<float_type> dev_ptr2 = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<bool> temp = thrust::device_malloc<bool>(mRecCount);

    thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr2, long_to_float_type());

    if (op_type == 2) // >
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater());
    else if (op_type == 1)  // <
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less());
    else if (op_type == 6) // >=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_greater_equal_to());
    else if (op_type == 5)  // <=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_less_equal());
    else if (op_type == 4)// =
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_equal_to());
    else // !=
        thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, f_not_equal_to());

    thrust::device_free(dev_ptr2);
    return thrust::raw_pointer_cast(temp);
}


float_type* CudaSet::op(int_type* column1, float_type* column2, string op_type, int reverse)
{

    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr(column1);

    thrust::transform(dev_ptr, dev_ptr + mRecCount, temp, long_to_float_type()); // in-place transformation

    thrust::device_ptr<float_type> dev_ptr1(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };

    return thrust::raw_pointer_cast(temp);

}




int_type* CudaSet::op(int_type* column1, int_type* column2, string op_type, int reverse)
{

    thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
    thrust::device_ptr<int_type> dev_ptr1(column1);
    thrust::device_ptr<int_type> dev_ptr2(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<int_type>());
    }
    else  {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
    }

    return thrust::raw_pointer_cast(temp);

}

float_type* CudaSet::op(float_type* column1, float_type* column2, string op_type, int reverse)
{

    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr1(column1);
    thrust::device_ptr<float_type> dev_ptr2(column2);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, dev_ptr2, temp, thrust::divides<float_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr2, dev_ptr2+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());
    };
    return thrust::raw_pointer_cast(temp);
}

int_type* CudaSet::op(int_type* column1, int_type d, string op_type, int reverse)
{
    thrust::device_ptr<int_type> temp = thrust::device_malloc<int_type>(mRecCount);
    thrust::fill(temp, temp+mRecCount, d);

    thrust::device_ptr<int_type> dev_ptr1(column1);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<int_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<int_type>());
    }
    else {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<int_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<int_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<int_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<int_type>());
    };
    return thrust::raw_pointer_cast(temp);
}

float_type* CudaSet::op(int_type* column1, float_type d, string op_type, int reverse)
{
    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::fill(temp, temp+mRecCount, d);

    thrust::device_ptr<int_type> dev_ptr(column1);
    thrust::device_ptr<float_type> dev_ptr1 = thrust::device_malloc<float_type>(mRecCount);
    thrust::transform(dev_ptr, dev_ptr + mRecCount, dev_ptr1, long_to_float_type());

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, temp, temp, thrust::divides<float_type>());
    }
    else  {
        if (op_type.compare("MUL") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(temp, temp+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };
    thrust::device_free(dev_ptr1);
    return thrust::raw_pointer_cast(temp);
}


float_type* CudaSet::op(float_type* column1, float_type d, string op_type,int reverse)
{
    thrust::device_ptr<float_type> temp = thrust::device_malloc<float_type>(mRecCount);
    thrust::device_ptr<float_type> dev_ptr1(column1);

    if(reverse == 0) {
        if (op_type.compare("MUL") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::minus<float_type>());
        else
            thrust::transform(dev_ptr1, dev_ptr1+mRecCount, thrust::make_constant_iterator(d), temp, thrust::divides<float_type>());
    }
    else	{
        if (op_type.compare("MUL") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::multiplies<float_type>());
        else if (op_type.compare("ADD") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::plus<float_type>());
        else if (op_type.compare("MINUS") == 0)
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::minus<float_type>());
        else
            thrust::transform(thrust::make_constant_iterator(d), thrust::make_constant_iterator(d)+mRecCount, dev_ptr1, temp, thrust::divides<float_type>());

    };

    return thrust::raw_pointer_cast(temp);

}





void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, string file_name) // compressed data for DIM tables
{
    mColumnCount = (unsigned int)nameRef.size();
    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];
    FILE* f;
    string f1;
	unsigned int cnt;
	char buffer[4000];
	string str;

    prealloc_char_size = 0;
    not_compressed = 0;
    mRecCount = Recs;
    hostRecCount = Recs;
	totalRecs = Recs;
    load_file_name = file_name;

    f1 = file_name + ".sort";
    f = fopen (f1.c_str() , "rb" );
    if(f != NULL) {
        unsigned int sz, idx;
        fread((char *)&sz, 4, 1, f);
        for(unsigned int j = 0; j < sz; j++) {
            fread((char *)&idx, 4, 1, f);
			fread(buffer, idx, 1, f);
			str.assign(buffer, idx);
            sorted_fields.push(str);
			if(verbose)
				cout << "segment sorted on " << idx << endl;
        };
        fclose(f);
    };

    f1 = file_name + ".presort";
    f = fopen (f1.c_str() , "rb" );
    if(f != NULL) {
        unsigned int sz, idx;
        fread((char *)&sz, 4, 1, f);
        for(unsigned int j = 0; j < sz; j++) {
            fread((char *)&idx, 4, 1, f);
			fread(buffer, idx, 1, f);
			str.assign(buffer, idx);			
            presorted_fields.push(str);
			if(verbose)
				cout << "presorted on " << idx << endl;
        };
        fclose(f);
    };

    tmp_table = 0;
    filtered = 0;	

    for(unsigned int i=0; i < mColumnCount; i++) {

		//f1 = file_name + "." + nameRef.front() + ".0";
		//f = fopen (f1.c_str() , "rb" );
		//fread((char *)&bytes, 4, 1, f); //need to read metadata such as type and length
		//fclose(f);
		
        columnNames[nameRef.front()] = i;
        cols[i] = colsRef.front();

		if (((typeRef.front()).compare("decimal") == 0) || ((typeRef.front()).compare("int") == 0)) {
			f1 = file_name + "." + nameRef.front() + ".0";		
			f = fopen (f1.c_str() , "rb" );
			for(unsigned int j = 0; j < 6; j++)
				fread((char *)&cnt, 4, 1, f);
			fclose(f);
			compTypes[nameRef.front()] = cnt;
		};		
		
		//check the references
		f1 = file_name + "." + nameRef.front() + ".refs";
		f = fopen (f1.c_str() , "rb" );
		if(f != NULL) {					
			unsigned int len;
		    fread(&len, 4, 1, f);
		    char* array = new char[len];
		    fread((void*)array, len, 1, f);
			ref_sets[i] = array;
			delete [] array;	
			unsigned int segs, seg_num, curr_seg;
			size_t res_count;
			fread(&len, 4, 1, f);
			char* array1 = new char[len];
		    fread((void*)array1, len, 1, f);
			ref_cols[i] = array1;
			delete [] array1;	
			
			unsigned int bytes_read = fread((void*)&curr_seg, 4, 1, f);			
						
			while(bytes_read == 1) {				
				fread((void*)&segs, 4, 1, f); //ref seg count
				//cout << "for " << i << " read " << array << " and " << z << " " << segs << endl;
			
				for(unsigned int j = 0; j < segs; j++) {
					fread((void*)&seg_num, 4, 1, f);			
					fread((void*)&res_count, 8, 1, f);		
					//cout << "curr_seg " << curr_seg << " " << seg_num << " " << res_count << endl;	
					if(res_count)
						ref_joins[i][curr_seg].insert(seg_num);
					else	
						ref_joins[i][curr_seg].insert(std::numeric_limits<unsigned int>::max());
				};
				bytes_read = fread((void*)&curr_seg, 4, 1, f);					
			};				
			fclose(f);
		};		
		
		
        
        if ((typeRef.front()).compare("int") == 0) {
            type[i] = 0;
            decimal[i] = 0;
			h_columns_int.push_back(thrust::host_vector<int_type, pinned_allocator<int_type> >());
            d_columns_int.push_back(thrust::device_vector<int_type>());
            type_index[i] = h_columns_int.size()-1;
        }
        else if ((typeRef.front()).compare("float") == 0) {
            type[i] = 1;
            decimal[i] = 0;
			h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >());
            d_columns_float.push_back(thrust::device_vector<float_type >());
            type_index[i] = h_columns_float.size()-1;
        }
        else if ((typeRef.front()).compare("decimal") == 0) {
            type[i] = 1;
            decimal[i] = 1;
			h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >());
            d_columns_float.push_back(thrust::device_vector<float_type>());
            type_index[i] = h_columns_float.size()-1;
        }
        else {
            type[i] = 2;
            decimal[i] = 0;
            h_columns_char.push_back(NULL);
            d_columns_char.push_back(NULL);
            char_size.push_back(sizeRef.front());
            type_index[i] = h_columns_char.size()-1;
        };

        nameRef.pop();
        typeRef.pop();
        sizeRef.pop();
        colsRef.pop();
    };
};



void CudaSet::initialize(queue<string> &nameRef, queue<string> &typeRef, queue<int> &sizeRef, queue<int> &colsRef, size_t Recs, queue<string> &references, queue<string> &references_names)
{
    mColumnCount = (unsigned int)nameRef.size();
    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];
    prealloc_char_size = 0;

    tmp_table = 0;
    filtered = 0;

    mRecCount = Recs;
    hostRecCount = Recs;
    segCount = 1;

    for(unsigned int i=0; i < mColumnCount; i++) {

        columnNames[nameRef.front()] = i;
        cols[i] = colsRef.front();

        if ((typeRef.front()).compare("int") == 0) {
            type[i] = 0;
            decimal[i] = 0;
            h_columns_int.push_back(thrust::host_vector<int_type, pinned_allocator<int_type> >());
            d_columns_int.push_back(thrust::device_vector<int_type>());
            type_index[i] = h_columns_int.size()-1;
        }
        else if ((typeRef.front()).compare("float") == 0) {
            type[i] = 1;
            decimal[i] = 0;
            h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >());
            d_columns_float.push_back(thrust::device_vector<float_type>());
            type_index[i] = h_columns_float.size()-1;
        }
        else if ((typeRef.front()).compare("decimal") == 0) {
            type[i] = 1;
            decimal[i] = 1;
            h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >());
            d_columns_float.push_back(thrust::device_vector<float_type>());
            type_index[i] = h_columns_float.size()-1;
        }

        else {
            type[i] = 2;
            decimal[i] = 0;
            h_columns_char.push_back(NULL);
            d_columns_char.push_back(NULL);
            char_size.push_back(sizeRef.front());
            type_index[i] = h_columns_char.size()-1;
        };
		
		if(!references.front().empty()) {
			ref_sets[i] = references.front();
			ref_cols[i] = references_names.front();
		};
        nameRef.pop();
        typeRef.pop();
        sizeRef.pop();
        colsRef.pop();
		references.pop();
		references_names.pop();
    };
};

void CudaSet::initialize(size_t RecordCount, unsigned int ColumnCount)
{
    mRecCount = RecordCount;
    hostRecCount = RecordCount;
    mColumnCount = ColumnCount;
    prealloc_char_size = 0;

    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];
    filtered = 0;

    for(unsigned int i =0; i < mColumnCount; i++) {
        cols[i] = i;
    };


};


void CudaSet::initialize(queue<string> op_sel, queue<string> op_sel_as)
{
    mRecCount = 0;
    mColumnCount = (unsigned int)op_sel.size();

    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];

    segCount = 1;
    not_compressed = 1;
    filtered = 0;
    col_aliases = op_sel_as;
    prealloc_char_size = 0;

    unsigned int index;
    unsigned int i = 0;
    while(!op_sel.empty()) {

        if(!setMap.count(op_sel.front())) {
            cout << "coudn't find column " << op_sel.front() << endl;
            exit(0);
        };


        CudaSet* a = varNames[setMap[op_sel.front()]];

        if(i == 0)
            maxRecs = a->maxRecs;

        index = a->columnNames[op_sel.front()];
        cols[i] = i;
        decimal[i] = a->decimal[i];
        columnNames[op_sel.front()] = i;

        if (a->type[index] == 0)  {
            d_columns_int.push_back(thrust::device_vector<int_type>());
            h_columns_int.push_back(thrust::host_vector<int_type, pinned_allocator<int_type> >());
            type[i] = 0;
            type_index[i] = h_columns_int.size()-1;
        }
        else if ((a->type)[index] == 1) {
            d_columns_float.push_back(thrust::device_vector<float_type>());
            h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >());
            type[i] = 1;
            type_index[i] = h_columns_float.size()-1;
        }
        else {
            h_columns_char.push_back(NULL);
            d_columns_char.push_back(NULL);
            type[i] = 2;
            type_index[i] = h_columns_char.size()-1;
            char_size.push_back(a->char_size[a->type_index[index]]);
        };
        i++;
        op_sel.pop();
    };

}


void CudaSet::initialize(CudaSet* a, CudaSet* b, queue<string> op_sel, queue<string> op_sel_as)
{
    mRecCount = 0;
    mColumnCount = 0;
    queue<string> q_cnt(op_sel);
    unsigned int i = 0;
    set<string> field_names;
    while(!q_cnt.empty()) {
        if(a->columnNames.find(q_cnt.front()) !=  a->columnNames.end() || b->columnNames.find(q_cnt.front()) !=  b->columnNames.end())  {
            field_names.insert(q_cnt.front());
        };
        q_cnt.pop();
    }
    mColumnCount = (unsigned int)field_names.size();

    type = new unsigned int[mColumnCount];
    cols = new unsigned int[mColumnCount];
    decimal = new bool[mColumnCount];
    maxRecs = b->maxRecs;

    map<string,unsigned int>::iterator it;

    segCount = 1;
    filtered = 0;
    not_compressed = 1;

    col_aliases = op_sel_as;
    prealloc_char_size = 0;

    unsigned int index;
    i = 0;
    while(!op_sel.empty() && (columnNames.find(op_sel.front()) ==  columnNames.end())) {

        if((it = a->columnNames.find(op_sel.front())) !=  a->columnNames.end()) {
            index = it->second;
            cols[i] = i;
            decimal[i] = a->decimal[i];
            columnNames[op_sel.front()] = i;

            if (a->type[index] == 0)  {
                d_columns_int.push_back(thrust::device_vector<int_type>());
                h_columns_int.push_back(thrust::host_vector<int_type, pinned_allocator<int_type> >());
                type[i] = 0;
                type_index[i] = h_columns_int.size()-1;
            }
            else if ((a->type)[index] == 1) {
                d_columns_float.push_back(thrust::device_vector<float_type>());
                h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >());
                type[i] = 1;
                type_index[i] = h_columns_float.size()-1;
            }
            else {
                h_columns_char.push_back(NULL);
                d_columns_char.push_back(NULL);
                type[i] = 2;
                type_index[i] = h_columns_char.size()-1;
                char_size.push_back(a->char_size[a->type_index[index]]);
            };
            i++;
        }
        else if((it = b->columnNames.find(op_sel.front())) !=  b->columnNames.end()) {
            index = it->second;
            columnNames[op_sel.front()] = i;
            cols[i] = i;
            decimal[i] = b->decimal[index];

            if ((b->type)[index] == 0) {
                d_columns_int.push_back(thrust::device_vector<int_type>());
                h_columns_int.push_back(thrust::host_vector<int_type, pinned_allocator<int_type> >());
                type[i] = 0;
                type_index[i] = h_columns_int.size()-1;
            }
            else if ((b->type)[index] == 1) {
                d_columns_float.push_back(thrust::device_vector<float_type>());
                h_columns_float.push_back(thrust::host_vector<float_type, pinned_allocator<float_type> >());
                type[i] = 1;
                type_index[i] = h_columns_float.size()-1;
            }
            else {
                h_columns_char.push_back(NULL);
                d_columns_char.push_back(NULL);
                type[i] = 2;
                type_index[i] = h_columns_char.size()-1;
                char_size.push_back(b->char_size[b->type_index[index]]);
            };
            i++;
        }
        op_sel.pop();
    };
};



int_type reverse_op(int_type op_type)
{
    if (op_type == 2) // >
        return 5;
    else if (op_type == 1)  // <
        return 6;
    else if (op_type == 6) // >=
        return 1;
    else if (op_type == 5)  // <=
        return 2;
    else return op_type;
}


size_t getFreeMem()
{
    size_t available, total;
    hipMemGetInfo(&available, &total);
    return available;
} ;



void allocColumns(CudaSet* a, queue<string> fields)
{
    if(a->filtered) {
        size_t max_sz = max_tmp(a) ;
        CudaSet* t = varNames[setMap[fields.front()]];
        if(max_sz*t->maxRecs > alloced_sz) {
            if(alloced_sz) {
                hipFree(alloced_tmp);
            };
            hipMalloc((void **) &alloced_tmp, max_sz*t->maxRecs);
            alloced_sz = max_sz*t->maxRecs;
        }
    }
    else {

        while(!fields.empty()) {
            if(setMap.count(fields.front()) > 0) {

                unsigned int idx = a->columnNames[fields.front()];
                bool onDevice = 0;

                if(a->type[idx] == 0) {
                    if(a->d_columns_int[a->type_index[idx]].size() > 0) {
                        onDevice = 1;
                    }
                }
                else if(a->type[idx] == 1) {
                    if(a->d_columns_float[a->type_index[idx]].size() > 0) {
                        onDevice = 1;
                    };
                }
                else {
                    if((a->d_columns_char[a->type_index[idx]]) != NULL) {
                        onDevice = 1;
                    };
                };

                if (!onDevice) {
                    a->allocColumnOnDevice(idx, a->maxRecs);
                }
            }
            fields.pop();
        };
    };
}



void gatherColumns(CudaSet* a, CudaSet* t, string field, unsigned int segment, size_t& count)
{

    unsigned int tindex = t->columnNames[field];
    unsigned int idx = a->columnNames[field];

    if(!a->onDevice(idx)) {
        a->allocColumnOnDevice(idx, a->maxRecs);
    };

    if(a->prm_index == 'R') {
        mygather(tindex, idx, a, t, count, a->mRecCount);
    }
    else {
        mycopy(tindex, idx, a, t, count, t->mRecCount);
		a->mRecCount = t->mRecCount;
    };
}


size_t getSegmentRecCount(CudaSet* a, unsigned int segment) {
    if (segment == a->segCount-1) {
        return a->hostRecCount - a->maxRecs*segment;
    }
    else
        return 	a->maxRecs;
}



void copyColumns(CudaSet* a, queue<string> fields, unsigned int segment, size_t& count, bool rsz, bool flt)
{
    set<string> uniques;
	
    if(a->filtered) { //filter the segment
        if(flt) {
            filter_op(a->fil_s, a->fil_f, segment);
        };
        if(rsz) {		    
            a->resizeDevice(count);
            a->devRecCount = count+a->mRecCount;
        };
    };


	while(!fields.empty()) {
        if (uniques.count(fields.front()) == 0 && setMap.count(fields.front()) > 0)	{
            if(a->filtered) {
                if(a->mRecCount) {
                    CudaSet *t = varNames[setMap[fields.front()]];
                    alloced_switch = 1;
                    t->CopyColumnToGpu(fields.front(), segment);
                    gatherColumns(a, t, fields.front(), segment, count);
                    alloced_switch = 0;
					a->orig_segs[t->load_file_name].insert(segment);
                };
            }
            else {
				if(a->mRecCount) {
					a->CopyColumnToGpu(fields.front(), segment, count);
				};	
            };
            uniques.insert(fields.front());
        };
        fields.pop();
    };	
}



void setPrm(CudaSet* a, CudaSet* b, char val, unsigned int segment) {

    b->prm_index = val;
    if (val == 'A') {
        b->mRecCount = getSegmentRecCount(a,segment);
    }
	else if (val == 'N') {
        b->mRecCount = 0;
    }
}



void mygather(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, size_t offset, size_t g_size)
{
    if(t->type[tindex] == 0) {
        if(!alloced_switch) {
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           t->d_columns_int[t->type_index[tindex]].begin(), a->d_columns_int[a->type_index[idx]].begin() + offset);
        }
        else {
            thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           d_col, a->d_columns_int[a->type_index[idx]].begin() + offset);
        };
    }
    else if(t->type[tindex] == 1) {
        if(!alloced_switch) {
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           t->d_columns_float[t->type_index[tindex]].begin(), a->d_columns_float[a->type_index[idx]].begin() + offset);
        }
        else {
            thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
            thrust::gather(a->prm_d.begin(), a->prm_d.begin() + g_size,
                           d_col, a->d_columns_float[a->type_index[idx]].begin() + offset);
        };
    }
    else {
        if(!alloced_switch) {
            str_gather((void*)thrust::raw_pointer_cast(a->prm_d.data()), g_size,
                       (void*)t->d_columns_char[t->type_index[tindex]], (void*)(a->d_columns_char[a->type_index[idx]] + offset*a->char_size[a->type_index[idx]]), (unsigned int)a->char_size[a->type_index[idx]] );
        }
        else {
            str_gather((void*)thrust::raw_pointer_cast(a->prm_d.data()), g_size,
                       alloced_tmp, (void*)(a->d_columns_char[a->type_index[idx]] + offset*a->char_size[a->type_index[idx]]), (unsigned int)a->char_size[a->type_index[idx]] );
        };
    }
};

void mycopy(unsigned int tindex, unsigned int idx, CudaSet* a, CudaSet* t, size_t offset, size_t g_size)
{
    if(t->type[tindex] == 0) {
        if(!alloced_switch) {
            thrust::copy(t->d_columns_int[t->type_index[tindex]].begin(), t->d_columns_int[t->type_index[tindex]].begin() + g_size,
                         a->d_columns_int[a->type_index[idx]].begin() + offset);
        }
        else {
            thrust::device_ptr<int_type> d_col((int_type*)alloced_tmp);
            thrust::copy(d_col, d_col + g_size, a->d_columns_int[a->type_index[idx]].begin() + offset);

        };
    }
    else if(t->type[tindex] == 1) {
        if(!alloced_switch) {
            thrust::copy(t->d_columns_float[t->type_index[tindex]].begin(), t->d_columns_float[t->type_index[tindex]].begin() + g_size,
                         a->d_columns_float[a->type_index[idx]].begin() + offset);
        }
        else {
            thrust::device_ptr<float_type> d_col((float_type*)alloced_tmp);
            thrust::copy(d_col, d_col + g_size,	a->d_columns_float[a->type_index[idx]].begin() + offset);
        };
    }
    else {
        if(!alloced_switch) {
            hipMemcpy((void**)(a->d_columns_char[a->type_index[idx]] + offset*a->char_size[a->type_index[idx]]), (void**)t->d_columns_char[t->type_index[tindex]],
                       g_size*t->char_size[t->type_index[tindex]], hipMemcpyDeviceToDevice);
        }
        else {
            hipMemcpy((void**)(a->d_columns_char[a->type_index[idx]] + offset*a->char_size[a->type_index[idx]]), alloced_tmp,
                       g_size*t->char_size[t->type_index[tindex]], hipMemcpyDeviceToDevice);
        };
    };
};



size_t load_queue(queue<string> c1, CudaSet* right, bool str_join, string f2, size_t &rcount,
                  unsigned int start_segment, unsigned int end_segment, bool rsz, bool flt)
{
    queue<string> cc;
    while(!c1.empty()) {
        if(right->columnNames.find(c1.front()) !=  right->columnNames.end()) {
            if(f2 != c1.front() || str_join) {
                cc.push(c1.front());
            };
        };
        c1.pop();
    };
    if(!str_join && right->columnNames.find(f2) !=  right->columnNames.end()) {
        cc.push(f2);
    };

    if(right->filtered) {
        allocColumns(right, cc);
        rcount = right->maxRecs;
    }
    else
        rcount = right->mRecCount;

    queue<string> ct(cc);

    while(!ct.empty()) {
        if(right->filtered && rsz) {
            right->mRecCount = 0;
        }
        else {
            right->allocColumnOnDevice(right->columnNames[ct.front()], rcount);
		};	
        ct.pop();
    };


    size_t cnt_r = 0;
    for(unsigned int i = start_segment; i < end_segment; i++) {
        if(!right->filtered)
            copyColumns(right, cc, i, cnt_r, rsz, 0);
        else
            copyColumns(right, cc, i, cnt_r, rsz, flt);
        cnt_r = cnt_r + right->mRecCount;
		//cout << "RIGHT SEG " <<  i << " " << cnt_r << " " << right->d_columns_int[1][0] << "-" << right->d_columns_int[1][cnt_r-1] << endl;			
    };
    right->mRecCount = cnt_r;
    return cnt_r;

}

size_t max_char(CudaSet* a)
{
    size_t max_char1 = 8;
    for(unsigned int i = 0; i < a->char_size.size(); i++)
        if (a->char_size[i] > max_char1)
            max_char1 = a->char_size[i];

    return max_char1;
};

size_t max_char(CudaSet* a, set<string> field_names)
{
    size_t max_char1 = 8, i;
    for (set<string>::iterator it=field_names.begin(); it!=field_names.end(); ++it) {
        i = a->columnNames[*it];
        if (a->type[i] == 2) {
            if (a->char_size[a->type_index[i]] > max_char1)
                max_char1 = a->char_size[a->type_index[i]];
        };
    };
    return max_char1;
};

size_t max_char(CudaSet* a, queue<string> field_names)
{
    size_t max_char = 8, i;
    while (!field_names.empty()) {
        i = a->columnNames[field_names.front()];
        if (a->type[i] == 2) {
            if (a->char_size[a->type_index[i]] > max_char)
                max_char = a->char_size[a->type_index[i]];
        };
        field_names.pop();
    };
    return max_char;
};



size_t max_tmp(CudaSet* a)
{
    size_t max_sz = 0;
    for(unsigned int i = 0; i < a->mColumnCount; i++) {
        if(a->type[i] == 0) {
            if(int_size > max_sz)
                max_sz = int_size;
        }
        else if(a->type[i] == 1) {
            if(float_size > max_sz)
                max_sz = float_size;
        };
    };
    size_t m_char = max_char(a);
    if(m_char > max_sz)
        return m_char;
    else
        return max_sz;

};


void setSegments(CudaSet* a, queue<string> cols)
{
    size_t mem_available = getFreeMem();
    size_t tot_sz = 0, idx;
    while(!cols.empty()) {
        idx = a->columnNames[cols.front()];
        if(a->type[idx] != 2)
            tot_sz = tot_sz + int_size;
        else
            tot_sz = tot_sz + a->char_size[a->type_index[idx]];
        cols.pop();
    };
    if(a->mRecCount*tot_sz > mem_available/3) { //default is 3
        a->segCount = (a->mRecCount*tot_sz)/(mem_available/5) + 1;
        a->maxRecs = (a->mRecCount/a->segCount)+1;
    };

};

void update_permutation_char(char* key, unsigned int* permutation, size_t RecCount, string SortType, char* tmp, unsigned int len)
{

    str_gather((void*)permutation, RecCount, (void*)key, (void*)tmp, len);

    // stable_sort the permuted keys and update the permutation
    if (SortType.compare("DESC") == 0 )
        str_sort(tmp, RecCount, permutation, 1, len);
    else
        str_sort(tmp, RecCount, permutation, 0, len);
}

void update_permutation_char_host(char* key, unsigned int* permutation, size_t RecCount, string SortType, char* tmp, unsigned int len)
{
    str_gather_host(permutation, RecCount, (void*)key, (void*)tmp, len);

    if (SortType.compare("DESC") == 0 )
        str_sort_host(tmp, RecCount, permutation, 1, len);
    else
        str_sort_host(tmp, RecCount, permutation, 0, len);

}



void apply_permutation_char(char* key, unsigned int* permutation, size_t RecCount, char* tmp, unsigned int len)
{
    // copy keys to temporary vector
    hipMemcpy( (void*)tmp, (void*) key, RecCount*len, hipMemcpyDeviceToDevice);
    // permute the keys
    str_gather((void*)permutation, RecCount, (void*)tmp, (void*)key, len);
}


void apply_permutation_char_host(char* key, unsigned int* permutation, size_t RecCount, char* res, unsigned int len)
{
    str_gather_host(permutation, RecCount, (void*)key, (void*)res, len);
}



void filter_op(char *s, char *f, unsigned int segment)
{
    CudaSet *a, *b;

    a = varNames.find(f)->second;
    a->name = f;
	//std::clock_t start1 = std::clock();	
	
    if(a->mRecCount == 0) {
        b = new CudaSet(0,1);
    }
    else {
		if(verbose)
			cout << "FILTER " << s << " " << f << " " << getFreeMem() << '\xd';

		
        b = varNames[s];
		b->name = s;
        size_t cnt = 0;
        allocColumns(a, b->fil_value);

        if (b->prm_d.size() == 0)
            b->prm_d.resize(a->maxRecs);

		//cout << "MAP CHECK start " << segment <<  endl;	
		char map_check = zone_map_check(b->fil_type,b->fil_value,b->fil_nums, b->fil_nums_f, a, segment);
		//cout << "MAP CHECK segment " << segment << " " << map_check <<  endl;
		
        if(map_check == 'R') {
            copyColumns(a, b->fil_value, segment, cnt);	
            bool* res = filter(b->fil_type,b->fil_value,b->fil_nums, b->fil_nums_f, a, segment);
		    thrust::device_ptr<bool> bp((bool*)res);    
			b->prm_index = 'R';
			b->mRecCount = thrust::count(bp, bp + (unsigned int)a->mRecCount, 1);
			thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)a->mRecCount),
							bp, b->prm_d.begin(), thrust::identity<bool>());
			if(segment == a->segCount-1)
				b->type_index = a->type_index;
			hipFree(res);
        }
        else  {
            setPrm(a,b,map_check,segment);
        };
        if(segment == a->segCount-1)
            a->deAllocOnDevice();
    }
	//cout << endl << "filter res " << b->mRecCount << endl;		
    //std::cout<< "filter time " <<  ( ( std::clock() - start1 ) / (double)CLOCKS_PER_SEC ) << " " << getFreeMem() << '\n';	
}


void sort_right(CudaSet* right, unsigned int colInd2, string f2, queue<string> op_g, queue<string> op_sel,
                bool decimal_join, bool& str_join, size_t& rcount) {
		
	size_t cnt_r = 0;		
    right->hostRecCount = right->mRecCount;
   
    if (right->type[colInd2]  == 2) {
        str_join = 1;
        right->d_columns_int.push_back(thrust::device_vector<int_type>());
        for(unsigned int i = 0; i < right->segCount; i++) {
            right->add_hashed_strings(f2, i, right->d_columns_int.size()-1);
        };
        cnt_r = right->d_columns_int[right->d_columns_int.size()-1].size();
    };

	
	//sort the segments and merge them on a host
	
    // need to allocate all right columns
		queue<string> op_alt1;
		op_alt1.push(f2);
		cnt_r = load_queue(op_alt1, right, str_join, "", rcount, 0, right->segCount);

		if(str_join) {
			colInd2 = right->mColumnCount+1;
			right->type_index[colInd2] = right->d_columns_int.size()-1;
		};	

		//here we need to make sure that right column is ordered. If not then we order it and keep the permutation
		bool sorted;

		if(str_join || !decimal_join) {
			sorted = thrust::is_sorted(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r);
		}
		else
			sorted = thrust::is_sorted(right->d_columns_float[right->type_index[colInd2]].begin(), right->d_columns_float[right->type_index[colInd2]].begin() + cnt_r);


		if(!sorted) {
		
			thrust::device_ptr<unsigned int> v = thrust::device_malloc<unsigned int>(cnt_r);
			thrust::sequence(v, v + cnt_r, 0, 1);
			thrust::sort_by_key(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r, v);
			thrust::copy(right->d_columns_int[right->type_index[colInd2]].begin(), right->d_columns_int[right->type_index[colInd2]].begin() + cnt_r, right->h_columns_int[right->type_index[colInd2]].begin());			

			right->resize(cnt_r);
			
			right->deAllocColumnOnDevice(colInd2);		
			
			void* d;
			CUDA_SAFE_CALL(hipMalloc((void **) &d, cnt_r*max_char(right)));
			
			unsigned int i;
			while(!op_sel.empty()) {		
				if (right->columnNames.find(op_sel.front()) != right->columnNames.end()) {
					i = right->columnNames[op_sel.front()];

					if(i != colInd2) {

						queue<string> op_alt2;
						op_alt2.push(op_sel.front());
						cnt_r = load_queue(op_alt2, right, str_join, "", rcount, 0, right->segCount, 0, 0);
						cout << "next load " << cnt_r << endl;

						if(right->type[i] == 0) {
							thrust::device_ptr<int_type> d_tmp((int_type*)d);
							thrust::gather(v, v+cnt_r, right->d_columns_int[right->type_index[i]].begin(), d_tmp);
							thrust::copy(d_tmp, d_tmp + cnt_r, right->h_columns_int[right->type_index[i]].begin());
						}
						else if(right->type[i] == 1) {
							thrust::device_ptr<float_type> d_tmp((float_type*)d);
							thrust::gather(v, v+cnt_r, right->d_columns_float[right->type_index[i]].begin(), d_tmp);
							thrust::copy(d_tmp, d_tmp + cnt_r, right->h_columns_float[right->type_index[i]].begin());
						}
						else {
							thrust::device_ptr<char> d_tmp((char*)d);
							str_gather(thrust::raw_pointer_cast(v), cnt_r, (void*)right->d_columns_char[right->type_index[i]], (void*) thrust::raw_pointer_cast(d_tmp), right->char_size[right->type_index[i]]);			
							hipMemcpy( (void*)right->h_columns_char[right->type_index[i]], (void*) thrust::raw_pointer_cast(d_tmp), cnt_r*right->char_size[right->type_index[i]], hipMemcpyDeviceToHost);
						};
						right->deAllocColumnOnDevice(i);
					};
				};
				op_sel.pop();
			};
			thrust::device_free(v);
			hipFree(d);
			right->not_compressed = 1;
		}								
}						


size_t load_right(CudaSet* right, unsigned int colInd2, string f2, queue<string> op_g, queue<string> op_sel,
                        queue<string> op_alt, bool decimal_join, bool& str_join,
                        size_t& rcount, unsigned int start_seg, unsigned int end_seg, bool rsz) {

    size_t cnt_r = 0;
    right->hostRecCount = right->mRecCount;
    //if join is on strings then add integer columns to left and right tables and modify colInd1 and colInd2

    if (right->type[colInd2]  == 2) {
        str_join = 1;
        right->d_columns_int.push_back(thrust::device_vector<int_type>());
        for(unsigned int i = start_seg; i < end_seg; i++) {
            right->add_hashed_strings(f2, i, right->d_columns_int.size()-1);
        };
        cnt_r = right->d_columns_int[right->d_columns_int.size()-1].size();
    };

    // need to allocate all right columns    
    if(right->not_compressed) {
        queue<string> op_alt1;
        op_alt1.push(f2);
        cnt_r = load_queue(op_alt1, right, str_join, "", rcount, start_seg, end_seg, rsz, 1);
    }
    else {
        cnt_r = load_queue(op_alt, right, str_join, f2, rcount, start_seg, end_seg, rsz, 1);
    };

    if(str_join) {
        colInd2 = right->mColumnCount+1;
        right->type_index[colInd2] = right->d_columns_int.size()-1;
    };	

    if(right->not_compressed) {
        queue<string> op_alt1;
        while(!op_alt.empty()) {
            if(f2.compare(op_alt.front())) {
                if (right->columnNames.find(op_alt.front()) != right->columnNames.end()) {
                    op_alt1.push(op_alt.front());
                };
            };
            op_alt.pop();
        };
		if(!op_alt1.empty())
			cnt_r = load_queue(op_alt1, right, str_join, "", rcount, start_seg, end_seg, 0, 0);
    };

    return cnt_r;
};

unsigned int calc_right_partition(CudaSet* left, CudaSet* right, queue<string> op_sel) {
	unsigned int tot_size = left->maxRecs*8;
	
	while(!op_sel.empty()) {
		if (right->columnNames.find(op_sel.front()) != right->columnNames.end()) {
					
		    if(right->type[right->columnNames[op_sel.front()]] <= 1) {
				tot_size = tot_size + right->maxRecs*8*right->segCount;
            }
            else {
				tot_size = tot_size + right->maxRecs*
									  right->char_size[right->type_index[right->columnNames[op_sel.front()]]]*
									  right->segCount;
			};
        };		
		op_sel.pop();			
	};		
	
	if(tot_size + 300000000 < getFreeMem())
		return right->segCount;
	else {	
		if(right->segCount == 1) { //need to partition it. Not compressed.
			right->segCount = ((tot_size + 300000000)/getFreeMem())+1;
			right->maxRecs = (right->mRecCount/right->segCount)+1;
			return 1;
		}
		else { //compressed
			return right->segCount / ((tot_size+300000000)/getFreeMem());
		};				
	};	
		
};


string int_to_string(int number){
    string number_string = "";
    char ones_char;
    int ones = 0;
    while(true){
        ones = number % 10;
        switch(ones){
            case 0: ones_char = '0'; break;
            case 1: ones_char = '1'; break;
            case 2: ones_char = '2'; break;
            case 3: ones_char = '3'; break;
            case 4: ones_char = '4'; break;
            case 5: ones_char = '5'; break;
            case 6: ones_char = '6'; break;
            case 7: ones_char = '7'; break;
            case 8: ones_char = '8'; break;
            case 9: ones_char = '9'; break;
            default : cout << ("Trouble converting number to string.");
        }
        number -= ones;
        number_string = ones_char + number_string;
        if(number == 0){
            break;
        }
        number = number/10;
    }
    return number_string;
}


void insert_records(char* f, char* s) {
	char buf[4096];
    size_t size, maxRecs;
	string str_s, str_d;	

	if(varNames.find(s) == varNames.end()) {
		cout << "couldn't find " << s << endl;
		exit(0);
	};	
	CudaSet *a;
    a = varNames.find(s)->second;
    a->name = s;	
	
	if(varNames.find(f) == varNames.end()) {
		cout << "couldn't find " << f << endl;
		exit(0);
	};	
	
	CudaSet *b;
    b = varNames.find(f)->second;
    b->name = f;	
	
	// if both source and destination are on disk
	if(a->source && b->source) {
		for(unsigned int i = 0; i < a->segCount; i++) {          	
			//for(unsigned int z = 0; z< a->mColumnCount; z++) {							
			for (map<string,unsigned int>::iterator it=a->columnNames.begin() ; it != a->columnNames.end(); ++it ) {
				str_s = a->load_file_name + "." + it->first + "." + int_to_string(i);		
				str_d = b->load_file_name + "." + it->first + "." + int_to_string(b->segCount + i);
				FILE* source = fopen(str_s.c_str(), "rb");
				FILE* dest = fopen(str_d.c_str(), "wb");
				while (size = fread(buf, 1, BUFSIZ, source)) {
					fwrite(buf, 1, size, dest);
				}
				fclose(source);
				fclose(dest);
			};		
		};
	
		if(a->maxRecs > b->maxRecs)
			maxRecs = a->maxRecs;
		else	
			maxRecs = b->maxRecs;
		
		for (map<string,unsigned int>::iterator it=b->columnNames.begin() ; it != b->columnNames.end(); ++it ) {
			b->reWriteHeader(b->load_file_name, it->first, a->segCount + b->segCount, a->totalRecs + b->totalRecs, maxRecs);				
		};		
	}
 	else if(!a->source && !b->source) { //if both source and destination are in memory
		size_t oldCount = b->mRecCount;
		b->resize(a->mRecCount);		
		for(unsigned int z = 0; z< b->mColumnCount; z++) {	
			if(b->type[z] == 0) {
				thrust::copy(a->h_columns_int[a->type_index[z]].begin(), a->h_columns_int[a->type_index[z]].begin() + a->mRecCount, b->h_columns_int[b->type_index[z]].begin() + oldCount);
			}
			else if(b->type[z] == 1) {
				thrust::copy(a->h_columns_float[a->type_index[z]].begin(), a->h_columns_float[a->type_index[z]].begin() + a->mRecCount, b->h_columns_float[b->type_index[z]].begin() + oldCount);			
			}
			else {
				hipMemcpy(b->h_columns_char[b->type_index[z]] + b->char_size[b->type_index[z]]*oldCount, a->h_columns_char[a->type_index[z]], a->char_size[a->type_index[z]]*a->mRecCount, hipMemcpyHostToHost);			
			};		
		};	
	}
	else if(!a->source && b->source) {
		
		total_segments = b->segCount;
		total_count = a->mRecCount;
		total_max = process_count;
		unsigned int segCount = (a->mRecCount/process_count + 1);
        size_t offset = 0, mCount;

        for(unsigned int z = 0; z < segCount; z++) {
            if(z < segCount-1) {
                if(a->mRecCount < process_count) {
                    mCount = a->mRecCount;
                }
                else {
                    mCount = process_count;
                }
            }
			else {
				mCount = a->mRecCount - (segCount-1)*process_count;			
			};				
			a->compress(b->load_file_name, offset, 0, z - (segCount-1), mCount);
            offset = offset + mCount;
        };
		//update headers
		total_count = a->mRecCount + b->mRecCount;
		//cout << "and now lets write " << a->mRecCount << " " <<  b->mRecCount << endl;
		for (map<string,unsigned int>::iterator it=b->columnNames.begin() ; it != b->columnNames.end(); ++it ) {
			b->writeHeader(b->load_file_name, it->first, total_segments);
		};	
	};	
};



void delete_records(char* f) {

    CudaSet *a;
    a = varNames.find(f)->second;
    a->name = f;	
	size_t totalRemoved = 0;
	size_t maxRecs = 0;

    if(!a->keep) { // temporary variable
		cout << "Delete operator is only applicable to disk based sets" << endl;
		cout << "for deleting records from derived sets please use filter operator " << endl;
		exit(0);
    }
    else {  // read matching segments, delete, compress and write on a disk replacing the original segments

		string str, str_old;
	    queue<string> op_vx;
		size_t cnt;
		for (map<string,unsigned int>::iterator it=a->columnNames.begin() ; it != a->columnNames.end(); ++it ) {
            op_vx.push((*it).first);        		
		};	
		allocColumns(a, op_vx);
		a->prm_d.resize(a->maxRecs);
		a->resize(a->maxRecs);
		size_t cc = a->mRecCount;
		size_t tmp;
		
        void* d;
        CUDA_SAFE_CALL(hipMalloc((void **) &d, a->maxRecs*float_size));	
        unsigned int new_seg_count = 0;
		char map_check;
		
		for(unsigned int i = 0; i < a->segCount; i++) {          

			map_check = zone_map_check(op_type,op_value,op_nums, op_nums_f, a, i);
			if(verbose)
				cout << "MAP CHECK segment " << i << " " << map_check <<  endl;
			if(map_check != 'N') {			
			
			    cnt = 0;
				copyColumns(a, op_vx, i, cnt);
				tmp = a->mRecCount;				
		
				if(a->mRecCount) {						
					bool* res = filter(op_type,op_value,op_nums, op_nums_f, a, i);
					thrust::device_ptr<bool> bp((bool*)res);
					thrust::copy_if(thrust::make_counting_iterator((unsigned int)0), thrust::make_counting_iterator((unsigned int)a->mRecCount),
									bp, a->prm_d.begin(), not_identity<bool>());						
					a->mRecCount = thrust::count(bp, bp + (unsigned int)a->mRecCount, 0);						
					hipFree(res);
					
					//cout << "Remained recs count " << a->mRecCount << endl;
					if(a->mRecCount > maxRecs)
						maxRecs = a->mRecCount;
										
					if (a->mRecCount) {
					
					    totalRemoved = totalRemoved + (tmp - a->mRecCount);
					    if (a->mRecCount == tmp) { //none deleted
							//cout << "rename " << i << " to " << new_seg_count << endl;
							if(new_seg_count != i) {
								//for(unsigned int z = 0; z< a->mColumnCount; z++) {
								for (map<string,unsigned int>::iterator it=a->columnNames.begin() ; it != a->columnNames.end(); ++it ) {
							
									str_old = a->load_file_name + "." + it->first;
									str_old += "." + int_to_string(i);
									str = a->load_file_name + "." + it->first;
									str += "." + int_to_string(new_seg_count);								
								
									remove(str.c_str());
									rename(str_old.c_str(), str.c_str());
								};	
							};   
							new_seg_count++;
							
						}
						else { //some deleted
					        //cout << "writing segment " << new_seg_count << endl;
							//for(unsigned int z = 0; z< a->mColumnCount; z++) {
							for (map<string,unsigned int>::iterator it=a->columnNames.begin() ; it != a->columnNames.end(); ++it ) {
								unsigned int z = it->second;
								str = a->load_file_name + "." + it->first;
								str += "." + int_to_string(new_seg_count);

								if(a->type[z] == 0) {
									thrust::device_ptr<int_type> d_col((int_type*)d);
									thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_int[a->type_index[z]].begin(), d_col);				
									pfor_compress( d, a->mRecCount*int_size, str, a->h_columns_int[a->type_index[z]], 0);
								}
								else if(a->type[z] == 1){
									thrust::device_ptr<float_type> d_col((float_type*)d);
									if(a->decimal[z]) {
										thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_float[a->type_index[z]].begin(), d_col);
										thrust::device_ptr<long long int> d_col_dec((long long int*)d);
										thrust::transform(d_col,d_col+a->mRecCount, d_col_dec, float_to_long());
										pfor_compress( d, a->mRecCount*float_size, str, a->h_columns_float[a->type_index[z]], 1);					
									}
									else {
										thrust::gather(a->prm_d.begin(), a->prm_d.begin() + a->mRecCount, a->d_columns_float[a->type_index[z]].begin(), d_col);
										thrust::copy(d_col, d_col + a->mRecCount, a->h_columns_float[a->type_index[z]].begin());	
										fstream binary_file(str.c_str(),ios::out|ios::binary);
										binary_file.write((char *)&a->mRecCount, 4);
										binary_file.write((char *)(a->h_columns_float[a->type_index[z]].data()),a->mRecCount*float_size);
										unsigned int comp_type = 3;
										binary_file.write((char *)&comp_type, 4);
										binary_file.close();													
									
									};
								}
								else {								
							        void* t;
									CUDA_SAFE_CALL(hipMalloc((void **) &t, tmp*a->char_size[a->type_index[z]]));
									apply_permutation_char(a->d_columns_char[a->type_index[z]], (unsigned int*)thrust::raw_pointer_cast(a->prm_d.data()), tmp, (char*)t, a->char_size[a->type_index[z]]);
									hipMemcpy(a->h_columns_char[a->type_index[z]], t, a->char_size[a->type_index[z]]*a->mRecCount, hipMemcpyDeviceToHost);
									hipFree(t);
									a->compress_char(str, z, a->mRecCount, 0);
								};	
							};															
							new_seg_count++;				
						};	
					}
					else {
						totalRemoved = totalRemoved + tmp;
					};
				}					
			}
            else {				
				if(new_seg_count != i) {
					//cout << "rename " << i << " to " << new_seg_count << endl;
					//for(unsigned int z = 0; z< a->mColumnCount; z++) {
					for (map<string,unsigned int>::iterator it=a->columnNames.begin() ; it != a->columnNames.end(); ++it ) {
							
						str_old = a->load_file_name + "." + it->first;
						str_old += "." + int_to_string(i);
						str = a->load_file_name + "." + it->first;
						str += "." + int_to_string(new_seg_count);								
								
						remove(str.c_str());														
						rename(str_old.c_str(), str.c_str());												
					};						
				};		
				new_seg_count++;	
				maxRecs	= a->maxRecs;			
			};	
			//cout << "TOTAL REM " << totalRemoved << endl;
		};	
		
		if (new_seg_count < a->segCount) {
			for(unsigned int i = new_seg_count; i < a->segCount; i++) {
				//cout << "delete segment " << i << endl;
				//for(unsigned int z = 0; z< a->mColumnCount; z++) {							
				for (map<string,unsigned int>::iterator it=a->columnNames.begin() ; it != a->columnNames.end(); ++it ) {
					str = a->load_file_name + "." + it->first;
					str += "." + int_to_string(i);								
					remove(str.c_str());	
				};	
			};					
		};
		
		for (map<string,unsigned int>::iterator it=a->columnNames.begin() ; it != a->columnNames.end(); ++it ) {
			a->reWriteHeader(a->load_file_name, it->first, new_seg_count, a->totalRecs-totalRemoved, maxRecs);				
		};				

		
		a->mRecCount = cc;
		a->prm_d.resize(0);
		a->segCount = new_seg_count;
        a->deAllocOnDevice();
		hipFree(d);
    };	
	

};


void save_col_data(map<string, map<string, col_data> >& data_dict, string file_name)
{
	size_t str_len;
	fstream binary_file(file_name.c_str(),ios::out|ios::binary|ios::trunc);
	size_t len = data_dict.size();
	binary_file.write((char *)&len, 8);
	for ( map<string, map<string, col_data> >::iterator it=data_dict.begin() ; it != data_dict.end(); ++it ) {
		str_len = (*it).first.size();
		binary_file.write((char *)&str_len, 8);
		binary_file.write((char *)(*it).first.data(), str_len);
		map<string, col_data> s = (*it).second;
		size_t len1 = s.size();
		binary_file.write((char *)&len1, 8);
		
		for ( map<string, col_data>::iterator sit=s.begin() ; sit != s.end(); ++sit ) {
			str_len = (*sit).first.size();
			binary_file.write((char *)&str_len, 8);
			binary_file.write((char *)(*sit).first.data(), str_len);
			binary_file.write((char *)&(*sit).second.col_type, 4);
			binary_file.write((char *)&(*sit).second.col_length, 4);
		};		
	};
	binary_file.close();
}

void load_col_data(map<string, map<string, col_data> >& data_dict, string file_name)
{
	size_t str_len, recs, len1;
	string str1, str2;
	char buffer[4000];
	unsigned int col_type, col_length;
	fstream binary_file;
	binary_file.open(file_name.c_str(),ios::in|ios::binary);
	if(binary_file.is_open()) {
		binary_file.read((char*)&recs, 8);
		for(unsigned int i = 0; i < recs; i++) {
			binary_file.read((char*)&str_len, 8);
			binary_file.read(buffer, str_len);
			str1.assign(buffer, str_len);
			binary_file.read((char*)&len1, 8);
		
			for(unsigned int j = 0; j < len1; j++) {
				binary_file.read((char*)&str_len, 8);
				binary_file.read(buffer, str_len);
				str2.assign(buffer, str_len);				
				binary_file.read((char*)&col_type, 4);
				binary_file.read((char*)&col_length, 4);
				data_dict[str1][str2].col_type = col_type;
				data_dict[str1][str2].col_length = col_length;
				//cout << "data DICT " << str1 << " " << str2 << " " << col_type << " " << col_length << endl;
			};
		};	
		binary_file.close();	
	}
	else {
		cout << "Coudn't open data dictionary" << endl;
	};	
	
	
}
